#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   USER NAME SPACE" below.  */

/** All of the code in bison is related to the tokens, and token definations for the Flex parser to collect upon input of the syntax defined.
In the Alenka program  Bison uses a function called excute_files as the main controlling of the program execution. Like all programs that use bison and flex 
the yyparse function is used as the scanner. yyparse is defind in this file.
**/
/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 17 "bison.y"



/** C/C++ Declarations of functions
**/
#include "lex.yy.c"
#include "cm.h"

    void clean_queues();
    void order_inplace(CudaSet* a, stack<string> exe_type, bool update_int);
    void yyerror(char *s, ...);
    void emit(char *s, ...);
    void emit_mul();
    void emit_add();
    void emit_minus();
    void emit_distinct();
    void emit_div();
    void emit_and();
    void emit_eq();
    void emit_or();
    void emit_cmp(int val);
    void emit_var(char *s, int c, char *f, char* ref, char* ref_name);
    void emit_var_asc(char *s);
    void emit_var_desc(char *s);
    void emit_name(char *name);
    void emit_count();
    void emit_sum();
    void emit_average();
    void emit_min();
    void emit_max();
    void emit_string(char *str);
    void emit_number(int_type val);
    void emit_float(float_type val);
    void emit_decimal(float_type val);
    void emit_sel_name(char* name);
    void emit_limit(int val);
    void emit_union(char *s, char *f1, char *f2);
    void emit_varchar(char *s, int c, char *f, int d, char *ref, char* ref_name);
    void emit_load(char *s, char *f, int d, char* sep);
    void emit_load_binary(const char *s, const char *f, int d);
    void emit_store(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f);
    void emit_filter(char *s, char *f);
    void emit_delete(char *f);
    void emit_insert(char *f, char* s);
    void emit_order(char *s, char *f, int e, int ll = 0);
    void emit_group(char *s, char *f, int e);
    void emit_select(char *s, char *f, int ll);
    void emit_join(char *s, char *j1, int grp);
    void emit_join_tab(char *s, char tp);
    void emit_distinct();
    void emit_join();
    void emit_sort(char* s, int p);
    void emit_presort(char* s);
    void emit_display(char *s, char* sep);
    void emit_case();
    void emit_show_tables();
    void emit_describe_table(char* table_name);
    void emit_drop_table(char* table_name);
    void process_error(int severity, string err);



/* Line 189 of yacc.c  */
#line 136 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 1
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     OR = 268,
     XOR = 269,
     AND = 270,
     DISTINCT = 271,
     REGEXP = 272,
     LIKE = 273,
     IS = 274,
     IN = 275,
     NOT = 276,
     BETWEEN = 277,
     COMPARISON = 278,
     SHIFT = 279,
     MOD = 280,
     FROM = 281,
     MULITE = 282,
     DELETE = 283,
     LOAD = 284,
     FILTER = 285,
     BY = 286,
     JOIN = 287,
     STORE = 288,
     INTO = 289,
     GROUP = 290,
     SELECT = 291,
     AS = 292,
     ORDER = 293,
     ASC = 294,
     DESC = 295,
     COUNT = 296,
     USING = 297,
     SUM = 298,
     AVG = 299,
     MIN = 300,
     MAX = 301,
     LIMIT = 302,
     ON = 303,
     BINARY = 304,
     LEFT = 305,
     RIGHT = 306,
     OUTER = 307,
     SORT = 308,
     SEGMENTS = 309,
     PRESORTED = 310,
     PARTITION = 311,
     INSERT = 312,
     WHERE = 313,
     DISPLAY = 314,
     CASE = 315,
     WHEN = 316,
     THEN = 317,
     ELSE = 318,
     END = 319,
     REFERENCES = 320,
     SHOW = 321,
     TABLES = 322,
     TABLE = 323,
     DESCRIBE = 324,
     DROP = 325
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 79 "bison.y"

    long long int intval;
    double floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 251 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 263 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  20
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   679

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  88
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  14
/* YYNRULES -- Number of rules.  */
#define YYNRULES  84
/* YYNRULES -- Number of states.  */
#define YYNSTATES  228

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   325

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    21,     2,     2,     2,    32,    26,     2,
      81,    82,    30,    28,    87,    29,    83,    31,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    86,    80,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    34,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    84,    25,    85,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    22,    23,    24,    27,
      33,    35,    36,    37,    38,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59,    60,    61,    62,    63,
      64,    65,    66,    67,    68,    69,    70,    71,    72,    73,
      74,    75,    76,    77,    78,    79
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    39,    46,
      55,    65,    73,    76,    84,    90,    98,   101,   105,   107,
     111,   113,   115,   117,   119,   121,   123,   138,   148,   160,
     167,   170,   173,   178,   183,   188,   193,   198,   201,   203,
     207,   211,   215,   219,   223,   227,   231,   235,   239,   243,
     247,   250,   253,   257,   261,   267,   271,   280,   284,   289,
     290,   294,   298,   304,   306,   308,   312,   314,   318,   319,
     321,   324,   329,   335,   341,   347,   353,   360,   367,   374,
     375,   378,   379,   384,   392
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      89,     0,    -1,    90,    80,    -1,    89,    90,    80,    -1,
      91,    -1,     4,    11,    45,    94,    35,     4,    93,    -1,
       4,    11,    38,     3,    51,    81,     3,    82,    46,    81,
      95,    82,    -1,     4,    11,    39,     4,    98,    -1,     4,
      11,    47,     4,    40,    97,    -1,     4,    11,    45,    94,
      35,     4,    99,    93,    -1,    42,     4,    43,     3,    51,
      81,     3,    82,   100,    -1,    42,     4,    43,     3,   100,
      58,   101,    -1,    78,     4,    -1,    66,    43,     4,    45,
      94,    35,     4,    -1,    37,    35,     4,    67,    92,    -1,
      68,     4,    51,    81,     3,    82,   100,    -1,    75,    76,
      -1,    79,    77,     4,    -1,     4,    -1,     4,    83,     4,
      -1,    10,    -1,     5,    -1,     6,    -1,     9,    -1,     7,
      -1,     8,    -1,     4,    84,     6,    85,    86,     4,    81,
       6,    82,    74,     4,    81,     4,    82,    -1,     4,    84,
       6,    85,    86,     4,    81,     6,    82,    -1,     4,    84,
       6,    85,    86,     4,    74,     4,    81,     4,    82,    -1,
       4,    84,     6,    85,    86,     4,    -1,     4,    48,    -1,
       4,    49,    -1,    50,    81,    92,    82,    -1,    52,    81,
      92,    82,    -1,    53,    81,    92,    82,    -1,    54,    81,
      92,    82,    -1,    55,    81,    92,    82,    -1,    16,    92,
      -1,    41,    -1,    92,    28,    92,    -1,    92,    29,    92,
      -1,    92,    30,    92,    -1,    92,    31,    92,    -1,    92,
      32,    92,    -1,    92,    33,    92,    -1,    92,    15,    92,
      -1,    92,    12,    92,    -1,    92,    13,    92,    -1,    92,
      14,    92,    -1,    92,    27,    92,    -1,    22,    92,    -1,
      21,    92,    -1,    92,    24,    92,    -1,    92,    18,    92,
      -1,    92,    24,    81,    91,    82,    -1,    81,    92,    82,
      -1,    69,    70,    92,    71,    92,    72,    92,    73,    -1,
      92,    19,     8,    -1,    92,    19,    22,     8,    -1,    -1,
      44,    40,    96,    -1,    92,    46,     4,    -1,    94,    87,
      92,    46,     4,    -1,    30,    -1,    92,    -1,    95,    87,
      92,    -1,    92,    -1,    92,    87,    96,    -1,    -1,    96,
      -1,    40,    92,    -1,    41,     4,    57,    92,    -1,    59,
      41,     4,    57,    92,    -1,    60,    41,     4,    57,    92,
      -1,    61,    41,     4,    57,    92,    -1,    41,     4,    57,
      92,    99,    -1,    59,    41,     4,    57,    92,    99,    -1,
      60,    41,     4,    57,    92,    99,    -1,    61,    41,     4,
      57,    92,    99,    -1,    -1,    56,     6,    -1,    -1,    62,
      63,    40,     4,    -1,    62,    63,    40,     4,    65,    40,
       6,    -1,    64,    40,     4,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint16 yyrline[] =
{
       0,   171,   171,   172,   176,   179,   181,   183,   185,   187,
     189,   191,   193,   195,   197,   199,   201,   203,   208,   209,
     210,   211,   212,   213,   214,   215,   216,   217,   218,   219,
     220,   221,   222,   223,   224,   225,   226,   227,   228,   232,
     233,   234,   235,   236,   237,   239,   240,   241,   242,   243,
     244,   245,   246,   247,   249,   250,   251,   255,   256,   259,
     262,   266,   267,   268,   272,   273,   277,   278,   281,   283,
     286,   290,   291,   292,   293,   294,   295,   296,   297,   299,
     302,   304,   307,   308,   309
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL", "OR",
  "XOR", "AND", "DISTINCT", "REGEXP", "LIKE", "IS", "IN", "'!'", "NOT",
  "BETWEEN", "COMPARISON", "'|'", "'&'", "SHIFT", "'+'", "'-'", "'*'",
  "'/'", "'%'", "MOD", "'^'", "FROM", "MULITE", "DELETE", "LOAD", "FILTER",
  "BY", "JOIN", "STORE", "INTO", "GROUP", "SELECT", "AS", "ORDER", "ASC",
  "DESC", "COUNT", "USING", "SUM", "AVG", "MIN", "MAX", "LIMIT", "ON",
  "BINARY", "LEFT", "RIGHT", "OUTER", "SORT", "SEGMENTS", "PRESORTED",
  "PARTITION", "INSERT", "WHERE", "DISPLAY", "CASE", "WHEN", "THEN",
  "ELSE", "END", "REFERENCES", "SHOW", "TABLES", "TABLE", "DESCRIBE",
  "DROP", "';'", "'('", "')'", "'.'", "'{'", "'}'", "':'", "','",
  "$accept", "stmt_list", "stmt", "select_stmt", "expr", "opt_group_list",
  "expr_list", "load_list", "val_list", "opt_val_list", "opt_where",
  "join_list", "opt_limit", "sort_def", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,    33,   276,   277,   278,   124,    38,   279,    43,    45,
      42,    47,    37,   280,    94,   281,   282,   283,   284,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
     306,   307,   308,   309,   310,   311,   312,   313,   314,   315,
     316,   317,   318,   319,   320,   321,   322,   323,   324,   325,
      59,    40,    41,    46,   123,   125,    58,    44
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    88,    89,    89,    90,    91,    91,    91,    91,    91,
      91,    91,    91,    91,    91,    91,    91,    91,    92,    92,
      92,    92,    92,    92,    92,    92,    92,    92,    92,    92,
      92,    92,    92,    92,    92,    92,    92,    92,    92,    92,
      92,    92,    92,    92,    92,    92,    92,    92,    92,    92,
      92,    92,    92,    92,    92,    92,    92,    92,    92,    93,
      93,    94,    94,    94,    95,    95,    96,    96,    97,    97,
      98,    99,    99,    99,    99,    99,    99,    99,    99,   100,
     100,   101,   101,   101,   101
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     5,     6,     8,
       9,     7,     2,     7,     5,     7,     2,     3,     1,     3,
       1,     1,     1,     1,     1,     1,    14,     9,    11,     6,
       2,     2,     4,     4,     4,     4,     4,     2,     1,     3,
       3,     3,     3,     3,     3,     3,     3,     3,     3,     3,
       2,     2,     3,     3,     5,     3,     8,     3,     4,     0,
       3,     3,     5,     1,     1,     3,     1,     3,     0,     1,
       2,     4,     5,     5,     5,     5,     6,     6,     6,     0,
       2,     0,     4,     7,     3
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     4,     0,     0,     0,     0,     0,    16,    12,     0,
       1,     0,     2,     0,     0,     0,     0,     0,     0,     0,
       0,    17,     3,     0,     0,    18,    21,    22,    24,    25,
      23,    20,     0,     0,     0,    63,    38,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,    79,     0,
       0,     0,     0,     7,    30,    31,     0,     0,    37,    51,
      50,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,    68,    14,     0,     0,     0,
       0,     0,     0,    70,    19,     0,     0,     0,     0,     0,
       0,     0,    55,    46,    47,    48,    45,    53,    57,     0,
       0,    52,    49,    39,    40,    41,    42,    43,    44,    61,
      59,     0,    66,    69,     8,     0,    80,    81,     0,    79,
       0,     0,    32,    33,    34,    35,    36,     0,    58,    18,
       0,     0,     0,     0,     0,     0,     5,    59,     0,     0,
       0,     0,     0,    11,    13,    15,     0,     0,     0,    54,
       0,     0,     0,     0,     0,     9,    62,    67,    79,     0,
       0,     0,    29,     0,     0,    60,     0,     0,     0,    10,
       0,    84,     0,     0,     0,     0,    71,     0,     0,     0,
      82,    64,     0,     0,     0,    56,    75,    72,    73,    74,
       0,     6,     0,     0,    27,    76,    77,    78,     0,    65,
       0,     0,    83,    28,     0,     0,     0,    26
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,     9,    10,    11,   132,   156,    55,   202,   133,   134,
      63,   157,    99,   163
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -184
static const yytype_int16 yypact[] =
{
      34,    -6,   -23,    16,   -22,    23,   -43,    41,   -19,     2,
     -28,  -184,   117,    71,    36,    81,    42,  -184,  -184,    90,
    -184,    17,  -184,    93,    95,   172,    97,    38,   103,    65,
      30,  -184,  -184,    67,    80,   -41,  -184,  -184,  -184,  -184,
    -184,  -184,   224,   224,   224,  -184,  -184,    40,    52,    58,
      59,    68,    83,   224,   565,   -20,   112,   224,   -15,   172,
     151,    85,   224,  -184,  -184,  -184,   161,   162,   630,   114,
     114,   224,   224,   224,   224,   224,   224,   320,   224,   224,
     224,   224,   224,     1,   276,   224,   224,   224,   224,   224,
     224,   224,   165,   166,   224,   224,   614,    94,   178,   113,
     -13,   101,   182,   614,  -184,   115,   346,   368,   391,   424,
     447,   519,  -184,   614,   292,   239,   630,   646,  -184,   189,
      82,    98,   186,    84,    84,  -184,  -184,  -184,  -184,  -184,
     -30,   591,   177,  -184,  -184,   183,  -184,    20,   194,   143,
     121,   125,  -184,  -184,  -184,  -184,  -184,   224,  -184,    24,
     130,   216,   181,   195,   196,   197,  -184,   179,   231,   224,
     157,   180,   202,  -184,  -184,  -184,   198,   243,   495,  -184,
     191,   224,   245,   246,   247,  -184,  -184,  -184,   143,   212,
     251,   175,   -71,   224,   224,  -184,   203,   204,   205,  -184,
     255,  -184,   224,   269,   281,   469,   541,   224,   224,   224,
     210,   614,    -4,   207,   208,  -184,  -184,   541,   541,   541,
     249,  -184,   224,   287,   220,  -184,  -184,  -184,   289,   614,
     214,   295,  -184,  -184,   219,   297,   221,  -184
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -184,  -184,   293,   184,   -25,   152,   253,  -184,  -155,  -184,
    -184,  -183,  -138,  -184
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_int16 yytable[] =
{
      54,   165,    20,   193,   177,    12,     1,    64,    65,   118,
     194,   151,    13,   206,   152,    93,   185,    68,    69,    70,
      14,    15,   138,   119,   215,   216,   217,    16,    77,   153,
     154,   155,    96,    17,    54,    12,    97,   103,     1,     2,
     189,    98,    66,    67,     3,    18,   106,   107,   108,   109,
     110,   111,    22,   113,   114,   115,   116,   117,    19,   121,
     122,   123,   124,   125,   126,   127,   128,    94,     4,   131,
       5,     2,    64,    65,    94,    27,     3,     6,   211,    28,
       7,     8,   161,   212,   162,    29,   149,    36,    37,    38,
      39,    40,    41,    30,    31,    77,    33,    32,    42,    34,
       4,    56,     5,    43,    44,    57,    58,    66,    67,     6,
      59,    60,     7,     8,    88,    89,    90,    91,    61,     2,
      62,    71,   168,    46,     3,    85,    86,    87,    88,    89,
      90,    91,    47,    72,    48,    49,    50,    51,    84,    73,
      74,    85,    86,    87,    88,    89,    90,    91,     4,    75,
       5,    52,    95,    76,   101,    23,    24,     6,   195,   196,
       7,     8,    25,    53,    26,   104,   102,   201,   105,   129,
     130,   137,   207,   208,   209,   135,    35,    36,    37,    38,
      39,    40,    41,   139,   136,   140,   160,   219,    42,    78,
      79,    80,    81,    43,    44,    82,    83,   148,   164,    98,
     141,    84,    45,   166,    85,    86,    87,    88,    89,    90,
      91,   167,   169,    46,    86,    87,    88,    89,    90,    91,
     170,   171,    47,   152,    48,    49,    50,    51,    35,    36,
      37,    38,    39,    40,    41,   176,   172,   173,   174,   178,
      42,    52,   180,   179,   181,    43,    44,   182,   184,   186,
     187,   188,   190,    53,    81,   191,   192,    82,    83,   200,
     197,   198,   199,    84,   159,    46,    85,    86,    87,    88,
      89,    90,    91,   203,    47,   210,    48,    49,    50,    51,
      35,    36,    37,    38,    39,    40,    41,   204,   213,   218,
     214,   220,    42,    52,   221,   222,   223,    43,    44,   224,
     225,   226,    21,   227,   150,    53,    80,    81,     0,   175,
      82,    83,   100,     0,     0,     0,    84,    46,     0,    85,
      86,    87,    88,    89,    90,    91,    47,     0,    48,    49,
      50,    51,    78,    79,    80,    81,     0,     0,    82,    83,
       0,     0,     0,     0,    84,    52,     0,    85,    86,    87,
      88,    89,    90,    91,     0,     0,     0,   120,    78,    79,
      80,    81,     0,     0,    82,    83,     0,     0,     0,     0,
      84,     0,     0,    85,    86,    87,    88,    89,    90,    91,
      78,    79,    80,    81,     0,     0,    82,    83,     0,     0,
       0,     0,    84,     0,     0,    85,    86,    87,    88,    89,
      90,    91,   112,    78,    79,    80,    81,     0,     0,    82,
      83,     0,     0,     0,     0,    84,     0,     0,    85,    86,
      87,    88,    89,    90,    91,     0,     0,     0,   142,     0,
       0,     0,     0,     0,     0,     0,    78,    79,    80,    81,
       0,     0,    82,    83,     0,     0,     0,     0,    84,     0,
     143,    85,    86,    87,    88,    89,    90,    91,     0,    78,
      79,    80,    81,     0,     0,    82,    83,     0,     0,     0,
       0,    84,     0,   144,    85,    86,    87,    88,    89,    90,
      91,    78,    79,    80,    81,     0,     0,    82,    83,     0,
       0,     0,     0,    84,     0,     0,    85,    86,    87,    88,
      89,    90,    91,     0,     0,     0,   145,    78,    79,    80,
      81,     0,     0,    82,    83,     0,     0,     0,     0,    84,
       0,     0,    85,    86,    87,    88,    89,    90,    91,   146,
       0,    78,    79,    80,    81,     0,     0,    82,    83,     0,
       0,     0,   205,    84,     0,     0,    85,    86,    87,    88,
      89,    90,    91,    78,    79,    80,    81,     0,     0,    82,
      83,     0,     0,     0,     0,    84,     0,   183,    85,    86,
      87,    88,    89,    90,    91,     0,     0,    78,    79,    80,
      81,     0,   151,    82,    83,     0,     0,     0,     0,    84,
     147,     0,    85,    86,    87,    88,    89,    90,    91,     0,
     153,   154,   155,    78,    79,    80,    81,     0,     0,    82,
      83,    92,     0,     0,     0,    84,     0,     0,    85,    86,
      87,    88,    89,    90,    91,     0,    78,    79,    80,    81,
       0,     0,    82,    83,     0,     0,     0,   158,    84,     0,
       0,    85,    86,    87,    88,    89,    90,    91,    82,    83,
       0,     0,     0,     0,    84,     0,     0,    85,    86,    87,
      88,    89,    90,    91,    -1,    -1,     0,     0,     0,     0,
      84,     0,     0,    85,    86,    87,    88,    89,    90,    91
};

static const yytype_int16 yycheck[] =
{
      25,   139,     0,    74,   159,    11,     4,    48,    49,     8,
      81,    41,    35,   196,    44,    35,   171,    42,    43,    44,
       4,    43,    35,    22,   207,   208,   209,     4,    53,    59,
      60,    61,    57,    76,    59,    11,    51,    62,     4,    37,
     178,    56,    83,    84,    42,     4,    71,    72,    73,    74,
      75,    76,    80,    78,    79,    80,    81,    82,    77,    84,
      85,    86,    87,    88,    89,    90,    91,    87,    66,    94,
      68,    37,    48,    49,    87,     4,    42,    75,    82,    43,
      78,    79,    62,    87,    64,     4,     4,     5,     6,     7,
       8,     9,    10,    51,     4,   120,     3,    80,    16,     4,
      66,     4,    68,    21,    22,    67,     3,    83,    84,    75,
      45,    81,    78,    79,    30,    31,    32,    33,    51,    37,
      40,    81,   147,    41,    42,    27,    28,    29,    30,    31,
      32,    33,    50,    81,    52,    53,    54,    55,    24,    81,
      81,    27,    28,    29,    30,    31,    32,    33,    66,    81,
      68,    69,    40,    70,     3,    38,    39,    75,   183,   184,
      78,    79,    45,    81,    47,     4,    81,   192,     6,     4,
       4,    58,   197,   198,   199,    81,     4,     5,     6,     7,
       8,     9,    10,    82,     6,     3,     3,   212,    16,    12,
      13,    14,    15,    21,    22,    18,    19,     8,     4,    56,
      85,    24,    30,    82,    27,    28,    29,    30,    31,    32,
      33,    86,    82,    41,    28,    29,    30,    31,    32,    33,
       4,    40,    50,    44,    52,    53,    54,    55,     4,     5,
       6,     7,     8,     9,    10,     4,    41,    41,    41,    82,
      16,    69,    40,    63,    46,    21,    22,     4,    57,     4,
       4,     4,    40,    81,    15,     4,    81,    18,    19,     4,
      57,    57,    57,    24,    87,    41,    27,    28,    29,    30,
      31,    32,    33,     4,    50,    65,    52,    53,    54,    55,
       4,     5,     6,     7,     8,     9,    10,     6,    81,    40,
      82,     4,    16,    69,    74,     6,    82,    21,    22,     4,
      81,     4,     9,    82,   120,    81,    14,    15,    -1,   157,
      18,    19,    59,    -1,    -1,    -1,    24,    41,    -1,    27,
      28,    29,    30,    31,    32,    33,    50,    -1,    52,    53,
      54,    55,    12,    13,    14,    15,    -1,    -1,    18,    19,
      -1,    -1,    -1,    -1,    24,    69,    -1,    27,    28,    29,
      30,    31,    32,    33,    -1,    -1,    -1,    81,    12,    13,
      14,    15,    -1,    -1,    18,    19,    -1,    -1,    -1,    -1,
      24,    -1,    -1,    27,    28,    29,    30,    31,    32,    33,
      12,    13,    14,    15,    -1,    -1,    18,    19,    -1,    -1,
      -1,    -1,    24,    -1,    -1,    27,    28,    29,    30,    31,
      32,    33,    82,    12,    13,    14,    15,    -1,    -1,    18,
      19,    -1,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    -1,    -1,    -1,    82,    -1,
      -1,    -1,    -1,    -1,    -1,    -1,    12,    13,    14,    15,
      -1,    -1,    18,    19,    -1,    -1,    -1,    -1,    24,    -1,
      82,    27,    28,    29,    30,    31,    32,    33,    -1,    12,
      13,    14,    15,    -1,    -1,    18,    19,    -1,    -1,    -1,
      -1,    24,    -1,    82,    27,    28,    29,    30,    31,    32,
      33,    12,    13,    14,    15,    -1,    -1,    18,    19,    -1,
      -1,    -1,    -1,    24,    -1,    -1,    27,    28,    29,    30,
      31,    32,    33,    -1,    -1,    -1,    82,    12,    13,    14,
      15,    -1,    -1,    18,    19,    -1,    -1,    -1,    -1,    24,
      -1,    -1,    27,    28,    29,    30,    31,    32,    33,    82,
      -1,    12,    13,    14,    15,    -1,    -1,    18,    19,    -1,
      -1,    -1,    73,    24,    -1,    -1,    27,    28,    29,    30,
      31,    32,    33,    12,    13,    14,    15,    -1,    -1,    18,
      19,    -1,    -1,    -1,    -1,    24,    -1,    72,    27,    28,
      29,    30,    31,    32,    33,    -1,    -1,    12,    13,    14,
      15,    -1,    41,    18,    19,    -1,    -1,    -1,    -1,    24,
      71,    -1,    27,    28,    29,    30,    31,    32,    33,    -1,
      59,    60,    61,    12,    13,    14,    15,    -1,    -1,    18,
      19,    46,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    -1,    12,    13,    14,    15,
      -1,    -1,    18,    19,    -1,    -1,    -1,    46,    24,    -1,
      -1,    27,    28,    29,    30,    31,    32,    33,    18,    19,
      -1,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,    29,
      30,    31,    32,    33,    18,    19,    -1,    -1,    -1,    -1,
      24,    -1,    -1,    27,    28,    29,    30,    31,    32,    33
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    37,    42,    66,    68,    75,    78,    79,    89,
      90,    91,    11,    35,     4,    43,     4,    76,     4,    77,
       0,    90,    80,    38,    39,    45,    47,     4,    43,     4,
      51,     4,    80,     3,     4,     4,     5,     6,     7,     8,
       9,    10,    16,    21,    22,    30,    41,    50,    52,    53,
      54,    55,    69,    81,    92,    94,     4,    67,     3,    45,
      81,    51,    40,    98,    48,    49,    83,    84,    92,    92,
      92,    81,    81,    81,    81,    81,    70,    92,    12,    13,
      14,    15,    18,    19,    24,    27,    28,    29,    30,    31,
      32,    33,    46,    35,    87,    40,    92,    51,    56,   100,
      94,     3,    81,    92,     4,     6,    92,    92,    92,    92,
      92,    92,    82,    92,    92,    92,    92,    92,     8,    22,
      81,    92,    92,    92,    92,    92,    92,    92,    92,     4,
       4,    92,    92,    96,    97,    81,     6,    58,    35,    82,
       3,    85,    82,    82,    82,    82,    82,    71,     8,     4,
      91,    41,    44,    59,    60,    61,    93,    99,    46,    87,
       3,    62,    64,   101,     4,   100,    82,    86,    92,    82,
       4,    40,    41,    41,    41,    93,     4,    96,    82,    63,
      40,    46,     4,    72,    57,    96,     4,     4,     4,   100,
      40,     4,    81,    74,    81,    92,    92,    57,    57,    57,
       4,    92,    95,     4,     6,    73,    99,    92,    92,    92,
      65,    82,    87,    81,    82,    99,    99,    99,    40,    92,
       4,    74,     6,    82,     4,    81,     4,    82
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 176 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 180 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 182 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval));;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 186 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 188 "bison.y"
    {  emit_join((yyvsp[(1) - (8)].strval),(yyvsp[(6) - (8)].strval),(yyvsp[(7) - (8)].intval)); ;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 190 "bison.y"
    {  emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 192 "bison.y"
    {  emit_store_binary((yyvsp[(2) - (7)].strval),(yyvsp[(4) - (7)].strval)); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 194 "bison.y"
    {  emit_describe_table((yyvsp[(2) - (2)].strval));;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 196 "bison.y"
    {  emit_insert((yyvsp[(3) - (7)].strval), (yyvsp[(7) - (7)].strval));;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 198 "bison.y"
    {  emit_delete((yyvsp[(3) - (5)].strval));;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 200 "bison.y"
    {  emit_display((yyvsp[(2) - (7)].strval), (yyvsp[(5) - (7)].strval));;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 202 "bison.y"
    {  emit_show_tables();;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 204 "bison.y"
    {  emit_drop_table((yyvsp[(3) - (3)].strval));;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 208 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 209 "bison.y"
    { emit("FIELDNAME %s.%s", (yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 210 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 211 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 212 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 213 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 214 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 215 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 216 "bison.y"
    { emit_varchar((yyvsp[(1) - (14)].strval), (yyvsp[(3) - (14)].intval), (yyvsp[(6) - (14)].strval), (yyvsp[(8) - (14)].intval), (yyvsp[(11) - (14)].strval), (yyvsp[(13) - (14)].strval));;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 217 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval), "", "");;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 218 "bison.y"
    { emit_var((yyvsp[(1) - (11)].strval), (yyvsp[(3) - (11)].intval), (yyvsp[(6) - (11)].strval), (yyvsp[(8) - (11)].strval), (yyvsp[(10) - (11)].strval));;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 219 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval), "", "");;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 220 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 221 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 222 "bison.y"
    { emit_count(); ;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 223 "bison.y"
    { emit_sum(); ;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 224 "bison.y"
    { emit_average(); ;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 225 "bison.y"
    { emit_min(); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 226 "bison.y"
    { emit_max(); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 227 "bison.y"
    { emit_distinct(); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 228 "bison.y"
    { emit_join(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 232 "bison.y"
    { emit_add(); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 233 "bison.y"
    { emit_minus(); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 234 "bison.y"
    { emit_mul(); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 235 "bison.y"
    { emit_div(); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 236 "bison.y"
    { emit("MOD"); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 237 "bison.y"
    { emit("MOD"); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 239 "bison.y"
    { emit_and(); ;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 240 "bison.y"
    { emit_eq(); ;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 241 "bison.y"
    { emit_or(); ;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 242 "bison.y"
    { emit("XOR"); ;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 243 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 244 "bison.y"
    { emit("NOT"); ;}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 245 "bison.y"
    { emit("NOT"); ;}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 246 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 247 "bison.y"
    { emit_cmp(7); ;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 249 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 250 "bison.y"
    {emit("EXPR");;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 251 "bison.y"
    { emit_case(); ;}
    break;

  case 57:

/* Line 1455 of yacc.c  */
#line 255 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 58:

/* Line 1455 of yacc.c  */
#line 256 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 59:

/* Line 1455 of yacc.c  */
#line 259 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 262 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 266 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 267 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 63:

/* Line 1455 of yacc.c  */
#line 268 "bison.y"
    { emit_sel_name("*");;}
    break;

  case 64:

/* Line 1455 of yacc.c  */
#line 272 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 65:

/* Line 1455 of yacc.c  */
#line 273 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 66:

/* Line 1455 of yacc.c  */
#line 277 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 67:

/* Line 1455 of yacc.c  */
#line 278 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 68:

/* Line 1455 of yacc.c  */
#line 281 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 70:

/* Line 1455 of yacc.c  */
#line 286 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 71:

/* Line 1455 of yacc.c  */
#line 290 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval), 'I');;}
    break;

  case 72:

/* Line 1455 of yacc.c  */
#line 291 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'L');;}
    break;

  case 73:

/* Line 1455 of yacc.c  */
#line 292 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'R');;}
    break;

  case 74:

/* Line 1455 of yacc.c  */
#line 293 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'O');;}
    break;

  case 75:

/* Line 1455 of yacc.c  */
#line 294 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval), 'I'); ;}
    break;

  case 76:

/* Line 1455 of yacc.c  */
#line 295 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'L'); ;}
    break;

  case 77:

/* Line 1455 of yacc.c  */
#line 296 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'R'); ;}
    break;

  case 78:

/* Line 1455 of yacc.c  */
#line 297 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'O'); ;}
    break;

  case 79:

/* Line 1455 of yacc.c  */
#line 299 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 80:

/* Line 1455 of yacc.c  */
#line 302 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;

  case 81:

/* Line 1455 of yacc.c  */
#line 304 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 82:

/* Line 1455 of yacc.c  */
#line 307 "bison.y"
    { emit_sort((yyvsp[(4) - (4)].strval), 0); ;}
    break;

  case 83:

/* Line 1455 of yacc.c  */
#line 308 "bison.y"
    { emit_sort((yyvsp[(4) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 84:

/* Line 1455 of yacc.c  */
#line 309 "bison.y"
    { emit_presort((yyvsp[(3) - (3)].strval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2308 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 311 "bison.y"


#include "filter.h"
#include "select.h"
#include "merge.h"
#include "zone_map.h"
#include "atof.h"
#include "sstream"
#include "sorts.cu"
#include "callbacks.h"

using namespace mgpu;
using namespace thrust::placeholders;
using namespace std;

size_t int_size = sizeof(int_type);
size_t float_size = sizeof(float_type);

queue<string> namevars;
queue<string> typevars;
queue<int> sizevars;
queue<int> cols;
queue<string> references;
queue<string> references_names;

queue<unsigned int> j_col_count;
unsigned int sel_count = 0;
unsigned int join_cnt = 0;
unsigned int distinct_cnt = 0;
unsigned int join_col_cnt = 0;
unsigned int join_tab_cnt = 0;
unsigned int tab_cnt = 0;
queue<string> op_join;
queue<char> join_type;
unsigned int partition_count;
unsigned int statement_count = 0;
map<string,unsigned int> stat;
map<unsigned int, unsigned int> join_and_cnt;
bool scan_state = 0;
map<string, map<string, bool> > used_vars;
bool save_dict = 0;
ContextPtr context;

void emit_multijoin(string s, string j1, string j2, unsigned int tab, char* res_name);
void filter_op(char *s, char *f, unsigned int segment);

/** The check_used_vars() function allows Alenka to check and see whether the input the user entered was actually vaild alenka variables ie table names & columns within
data.dictionary.It iterates through data.dict map structure to figure out if any variables match table and/or column names. This has empty parameter list
**/
void check_used_vars()
{
    for (map<string, map<string, col_data> >::iterator it=data_dict.begin() ; it != data_dict.end(); ++it ) {

        map<string, col_data> s = (*it).second;
        queue<string> vars(op_value);
        while(!vars.empty()) {
            if(s.count(vars.front()) != 0) {
                used_vars[(*it).first][vars.front()] = 1;
            };
            vars.pop();
        }
    };
}


/** 
 This function pushses NAME on to op_type queue and its assocaited value on to op_value queue
**/
void emit_name(char *name)
{
    op_type.push("NAME");
    op_value.push(name);
}

/** 
 This function is called when the keyword 'LIMIT'is parsed over. This function pushses LIMIT onto op_type queue.back() 
 The LIMIT keyword is used along with others to limit the amout of data or records stroed/displayed
 **/
void emit_limit(int val)
{
    op_nums.push(val);
}


/** 
 This function pushses STRING on to op_type queue and its assocaited value on to op_value queue
**/
void emit_string(char *str)
{   // remove the float_type quotes
    string sss(str,1, strlen(str)-2);
    op_type.push("STRING");
    op_value.push(sss);
}


/** 
 This function pushses 'NUMBER' on to op_type queue and its assocaited value on to op_value queue
**/
void emit_number(int_type val)
{
    op_type.push("NUMBER");
    op_nums.push(val);
}

/** 
 This function pushses 'FLOAT' on to op_type queue and its assocaited value on to op_value queue
**/
void emit_float(float_type val)
{
    op_type.push("FLOAT");
    op_nums_f.push(val);
}

/** 
 This function pushses 'DECIMAL' on to op_type queue and its assocaited value on to op_value queue
**/
void emit_decimal(float_type val)
{
    op_type.push("DECIMAL");
    op_nums_f.push(val);
}



/** 
 This function is called when the keyword symbol '*'is parsed over in bewteen expressions. This function pushses MUL onto op_type queue 
 **/
void emit_mul()
{
    op_type.push("MUL");
}

/** 
 This function is called when the keyword symbol '+'is parsed over in bewteen expressions. This function pushses ADD onto op_type queue 
 **/ 
void emit_add()
{
    op_type.push("ADD");
}

/** 
emit_div() function is called when the keyword  symbol'/' is parsed over in bewteen expressions.  
 **/
void emit_div()
{
    op_type.push("DIV");
}

unsigned int misses = 0;

/**
This function is called when the keyword 'AND'is parsed over. emit_and pushses AND onto op_type queue and increments join_col_cnt
 by 1.
**/
void emit_and()
{
    op_type.push("AND");
    join_col_cnt++;
}

/**
This function is called when the keyword 'EQUAL'is parsed over. emit_and pushses JOIN onto op_type queue.back() If variable misses is equal to 0 join_and_cnt[tab_cnt] 
to be join_col_cnt and increments tab_cnt by 1. Otherwise missess is decremented by 1
 by 1.
**/
void emit_eq()
{
    op_type.push("JOIN");
    if(misses == 0) {
        join_and_cnt[tab_cnt] = join_col_cnt;
        misses = join_col_cnt;
        join_col_cnt = 0;
        tab_cnt++;
    }
    else {
        misses--;
    }
}

/**
This function is called when the keyword 'DISTINCT'is parsed over. emit_and pushses DISTINCT onto op_type queue and increments distinct_cnt
 by 1.
**/
void emit_distinct()
{
    op_type.push("DISTINCT");
    distinct_cnt++;
}

/**
This function is called when the keyword 'JOIN' is parsed over.
*/
void emit_join()
{

}


/** 
emit_or() function is called when the keyword 'OR' is parsed over. This function pushses OR onto op_type queue 
 **/
void emit_or()
{
    op_type.push("OR");
}


/** 
 This function is called when the keyword '-'is parsed over in bewteen expressions. This function pushses MINUS onto op_type queue 
 **/
void emit_minus()
{
    op_type.push("MINUS");
}

/** 
emit_cmp is called when the keyword 'COMPARISON'is parsed over.This function pushses 'CMP' on to op_type queue and its assocaited value on to op_value queue.
**/
void emit_cmp(int val)
{
    op_type.push("CMP");
    op_nums.push(val);
}

/** 
emit function is used in multiple cases and instances of keywords.
**/
void emit(char *s, ...)
{
}
/**
emit_var is called when the parser encounters a column list dealing with colmns that are not of type char. Usually in Load command. This function
pushes the apporiate variables onto the various queue structures 
**/

void emit_var(char *s, int c, char *f, char* ref, char* ref_name)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(0);
    cols.push(c);
    references.push(ref);
    references_names.push(ref_name);
}

/** 
emit_var_asc is called when the keyword 'ASC'is parsed over.This function pushses 'ASC' on to op_value queue and  the value of s to op_type.
**/
void emit_var_asc(char *s)
{
    op_type.push(s);
    op_value.push("ASC");
}

/** 
emit_var_desc is called when the keyword 'DESC'is parsed over.This function pushses 'DESC' on to op_value queue and  the value of s to op_type.
**/
void emit_var_desc(char *s)
{
    op_type.push(s);
    op_value.push("DESC");
}

void emit_sort(char *s, int p)
{
    op_sort.push(s);
    partition_count = p;
}

void emit_presort(char *s)
{
    op_presort.push(s);
}


/**
emit_varchar is called when the parser encounters a column list dealing with columns that are of type char. Usually in Load command. This function
pushes the apporiate variables onto the various queue structures 
**/
void emit_varchar(char *s, int c, char *f, int d, char *ref, char* ref_name)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(d);
    cols.push(c);
    references.push(ref);
    references_names.push(ref_name);
}

/** 
emit_sel_name is called when a '*', ',', and 'AS' keyword/symbols are used in an Alenka command.
**/
void emit_sel_name(char *s)
{
    op_type.push("emit sel_name");
    op_value.push(s);
    sel_count++;
}

/**
Pushes the keyword "COUNT" on the op_type queue to represent finding the count
**/
void emit_count()
{
    op_type.push("COUNT");
}

/**
Pushes the keyword "SUM" on the op_type queue to represent finding the sum
**/
void emit_sum()
{
    op_type.push("SUM");
}


/**
Pushes the keyword "AVG" on the op_type queue to represent finding he average
**/
void emit_average()
{
    op_type.push("AVG");
}

/**
Pushes the keyword "MIN" on the op_type queue to represent finding the minimum
**/
void emit_min()
{
    op_type.push("MIN");
}

/**
Pushes the keyword "MAX" on the op_type queue to represent find the maximum
**/
void emit_max()
{
    op_type.push("MAX");
}

/**
 emit_join_tab is called when a specific join is wanted like RIGHT OR LEFT join. This function pushes type of join specified onto join_type queue.
**/
void emit_join_tab(char *s, char tp)
{
    op_join.push(s);
    join_tab_cnt++;
    join_type.push(tp);
};

/**
order_inplace changes the order of columns on device. This function accpets CudaSet pointer, stack of strings, set of strings and a boolean variable.
**/
void order_inplace_host(CudaSet* a, stack<string> exe_type, set<string> field_names, bool update_str)
{
    unsigned int* permutation = new unsigned int[a->mRecCount];
    thrust::sequence(permutation, permutation + a->mRecCount);

    size_t maxSize =  a->mRecCount;
    char* temp;
    temp = new char[maxSize*max_char(a)];
    stack<string> exe_type1(exe_type);
    stack<string> exe_value;

    while(!exe_type1.empty()) {
        exe_value.push("ASC");
        exe_type1.pop();
    };


    // sort on host

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {

        if (a->type[exe_type.top()] == 0)
            update_permutation_host(a->h_columns_int[exe_type.top()].data(), permutation, a->mRecCount, exe_value.top(), (int_type*)temp);
        else if (a->type[exe_type.top()] == 1)
            update_permutation_host(a->h_columns_float[exe_type.top()].data(), permutation, a->mRecCount,exe_value.top(), (float_type*)temp);
        else {
            update_permutation_char_host(a->h_columns_char[exe_type.top()], permutation, a->mRecCount, exe_value.top(), temp, a->char_size[exe_type.top()]);
        };
    };

    for (unsigned int i = 0; i < a->mColumnCount; i++) {
        if (a->type[a->columnNames[i]] == 0) {
            thrust::gather(permutation, permutation + a->mRecCount, a->h_columns_int[a->columnNames[i]].data(), (int_type*)temp);
            thrust::copy((int_type*)temp, (int_type*)temp + a->mRecCount, a->h_columns_int[a->columnNames[i]].data());
        }
        else if (a->type[a->columnNames[i]] == 1) {
            thrust::gather(permutation, permutation + a->mRecCount, a->h_columns_float[a->columnNames[i]].data(), (float_type*)temp);
            thrust::copy((float_type*)temp, (float_type*)temp + a->mRecCount, a->h_columns_float[a->columnNames[i]].data());
        }
        else {
            apply_permutation_char_host(a->h_columns_char[a->columnNames[i]], permutation, a->mRecCount, temp, a->char_size[a->columnNames[i]]);
            thrust::copy(temp, temp + a->mRecCount*a->char_size[a->columnNames[i]], a->h_columns_float[a->columnNames[i]].data());
        };
    };

    delete [] temp;
    delete [] permutation;
}


void order_inplace1(CudaSet* a, stack<string> exe_type, set<string> field_names, bool update_str)
{
    unsigned int sz = a->mRecCount;
    cout << "order_inplace1->Attempting to device_malloc for " << sz << " records" << endl;
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(sz);
    cout <<"order_inplace1->device_malloc successful, now sequence" << endl;
    thrust::sequence(permutation, permutation+sz,0,1);

    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
    void* temp;
    cout << "order_inplace1->getting ready to hipMalloc" << endl;
    CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*max_char(a, field_names)));

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {
    	cout << "order_inplace1->for loop [" << i << "]" << endl;
        if (a->type[exe_type.top()] == 0 ) {
            a->d_columns_int[exe_type.top()].resize(sz);
            thrust::copy(a->h_columns_int[exe_type.top()].begin(), a->h_columns_int[exe_type.top()].begin() + sz, a->d_columns_int[exe_type.top()].begin());
            update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, sz, "ASC", (int_type*)temp);
            a->d_columns_int[exe_type.top()].resize(0);
            a->d_columns_int[exe_type.top()].shrink_to_fit();
        }
        else if (a->type[exe_type.top()] == 1) {
            a->d_columns_float[exe_type.top()].resize(sz);
            thrust::copy(a->h_columns_float[exe_type.top()].begin(), a->h_columns_float[exe_type.top()].begin() + sz, a->d_columns_float[exe_type.top()].begin());
            update_permutation(a->d_columns_float[exe_type.top()], raw_ptr, sz,"ASC", (float_type*)temp);
            a->d_columns_float[exe_type.top()].resize(0);
            a->d_columns_float[exe_type.top()].shrink_to_fit();
        }
        else {
            // use int col int_col_count
            a->d_columns_int[exe_type.top()].resize(sz);
            thrust::copy(a->h_columns_int[exe_type.top()].begin(), a->h_columns_int[exe_type.top()].begin() + sz, a->d_columns_int[exe_type.top()].begin());
            update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, sz, "ASC", (int_type*)temp);
            a->d_columns_int[exe_type.top()].resize(0);
            a->d_columns_int[exe_type.top()].shrink_to_fit();
        };
    };


    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        if (a->type[*it] == 0) {
            a->d_columns_int[*it].resize(sz);
            thrust::copy(a->h_columns_int[*it].begin(), a->h_columns_int[*it].begin() + sz, a->d_columns_int[*it].begin());
            apply_permutation(a->d_columns_int[*it], raw_ptr, sz, (int_type*)temp);
            thrust::copy(a->d_columns_int[*it].begin(), a->d_columns_int[*it].begin() + sz, a->h_columns_int[*it].begin());
            a->d_columns_int[*it].resize(0);
            a->d_columns_int[*it].shrink_to_fit();
        }
        else if (a->type[*it] == 1) {
            a->d_columns_float[*it].resize(sz);
            thrust::copy(a->h_columns_float[*it].begin(), a->h_columns_float[*it].begin() + sz, a->d_columns_float[*it].begin());
            apply_permutation(a->d_columns_float[*it], raw_ptr, sz, (float_type*)temp);
            thrust::copy(a->d_columns_float[*it].begin(), a->d_columns_float[*it].begin() + sz, a->h_columns_float[*it].begin());
            a->d_columns_float[*it].resize(0);
            a->d_columns_float[*it].shrink_to_fit();
        }
        else {
            a->allocColumnOnDevice(*it, sz);
            hipMemcpy( a->d_columns_char[*it], (void *)a->h_columns_char[*it], sz*a->char_size[*it], hipMemcpyHostToDevice);
            apply_permutation_char(a->d_columns_char[*it], raw_ptr, sz, (char*)temp, a->char_size[*it]);
            hipMemcpy( a->h_columns_char[*it], a->d_columns_char[*it], sz*a->char_size[*it], hipMemcpyDeviceToHost);
            a->deAllocColumnOnDevice(*it);
            if(update_str) {
                a->d_columns_int[*it].resize(sz);
                hipMemcpy( a->d_columns_char[*it], (void *)a->h_columns_char[*it], sz*a->char_size[*it], hipMemcpyHostToDevice);
                apply_permutation(a->d_columns_int[*it], raw_ptr, sz, (int_type*)temp);
            }

        };
    };
    hipFree(temp);
    thrust::device_free(permutation);

    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        if (a->type[*it] == 0) {
            a->d_columns_int[*it].resize(sz);
            thrust::copy(a->h_columns_int[*it].begin(), a->h_columns_int[*it].begin() + sz, a->d_columns_int[*it].begin());
        }
        else if (a->type[*it] == 1) {
            a->d_columns_float[*it].resize(sz);
            thrust::copy(a->h_columns_float[*it].begin(), a->h_columns_float[*it].begin() + sz, a->d_columns_float[*it].begin());
        }
        else {
            a->allocColumnOnDevice(*it, sz);
            hipMemcpy( a->d_columns_char[*it], (void *)a->h_columns_char[*it], sz*a->char_size[*it], hipMemcpyHostToDevice);
        };

    };

}




void order_inplace(CudaSet* a, stack<string> exe_type, set<string> field_names, bool update_str)
{

    unsigned int sz = a->mRecCount;
    cout << "order_inplace->Attempting to device_malloc for " << sz << " records: " << getFreeMem() << endl;
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(sz);
    cout <<"order_inplace->device_malloc successful, now sequence: " << getFreeMem() << endl;
    thrust::sequence(permutation, permutation+sz,0,1);

    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
    void* temp;
    cout << "order_inplace->getting ready to hipMalloc" << endl;
    CUDA_SAFE_CALL(hipMalloc((void **) &temp, sz*max_char(a, field_names)));
    cout << "order_inplace->hipMalloc worked" << endl;

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {
    	cout << "order_inplace->first for loop[" << i << "]" << endl;
        if (a->type[exe_type.top()] == 0)
            update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, sz, "ASC", (int_type*)temp);
        else if (a->type[exe_type.top()] == 1)
            update_permutation(a->d_columns_float[exe_type.top()], raw_ptr, sz,"ASC", (float_type*)temp);
        else {
            // use int col int_col_count
            update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, sz, "ASC", (int_type*)temp);
        };
    };

    cout << "done with first for loop" << endl;


    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
    	cout << "order_inplace->second for loop" << endl;
        if (a->type[*it] == 0) {
            apply_permutation(a->d_columns_int[*it], raw_ptr, sz, (int_type*)temp);
        }
        else if (a->type[*it] == 1)
            apply_permutation(a->d_columns_float[*it], raw_ptr, sz, (float_type*)temp);
        else {
            apply_permutation_char(a->d_columns_char[*it], raw_ptr, sz, (char*)temp, a->char_size[*it]);
            if(update_str) {
                if(a->d_columns_int[*it].size() > 0) {
                    apply_permutation(a->d_columns_int[*it], raw_ptr, sz, (int_type*)temp);
                };
            };
        };
    };
    hipFree(temp);
    thrust::device_free(permutation);
}

///
/** 
This function checks for star joins and pops them from op_vals queues when it finds them 
**/
bool check_star_join(string j1)
{
    queue<string> op_vals(op_value);

    for(unsigned int i=0; i < sel_count; i++) {
        op_vals.pop();
        op_vals.pop();
    };

    if(join_tab_cnt > 1) {

        while(op_vals.size()) {
            if (std::find(varNames[j1]->columnNames.begin(), varNames[j1]->columnNames.end(), op_vals.front()) != varNames[j1]->columnNames.end()) {
                op_vals.pop();
                op_vals.pop();
            }
            else {
                return 0;
            };
        };
        return 1;
    }
    else
        return 0;
}

std::ostream &operator<<(std::ostream &os, const uint2 &x)
{
    os << x.x << ", " << x.y;
    return os;
}


/**
 emit_join is called when a join_list is parsed over in a command. Function used to join. This version of emit_join has 3 parameters two char arrays and interger.
 Function calls emit_multijoin when more then one join is needed
**/  
void emit_join(char *s, char *j1, int grp)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(j1) == stat.end() && data_dict.count(j1) == 0) {
            process_error(2, "Join : couldn't find variable " + string(j1) );
        };
        if (stat.find(op_join.front()) == stat.end() && data_dict.count(op_join.front()) == 0) {
            process_error(2, "Join : couldn't find variable " + op_join.front() );
        };
        stat[s] = statement_count;
        stat[j1] = statement_count;
        check_used_vars();
        while(!op_join.empty()) {
            stat[op_join.front()] = statement_count;
            op_join.pop();
        };
        return;
    };

    queue<string> op_m(op_value);
    queue<string> op_m1(op_value);

    if(join_tab_cnt > 1) {
        string tab_name;
        for(unsigned int i = 1; i <= join_tab_cnt; i++) {

            if(i == join_tab_cnt)
                tab_name = s;
            else
                tab_name = s + int_to_string(i);

            string j, j2;
            if(i == 1) {
                j2 = op_join.front();
                op_join.pop();
                j = op_join.front();
                op_join.pop();
            }
            else {
                if(!op_join.empty()) {
                    j = op_join.front();
                    op_join.pop();
                }
                else
                    j = j1;
                j2 = s + int_to_string(i-1);
            };
            emit_multijoin(tab_name, j, j2, i, s);
            op_value = op_m;
        };
    }
    else {
        string j2 = op_join.front();
        op_join.pop();
        emit_multijoin(s, j1, j2, 1, s);
    };


    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_m.front());
        op_m.pop();
        op_sel_as.push(op_m.front());
        op_m.pop();
    };
    while(!op_sel_as.empty()) {
        //cout << "alias " << op_sel.front() << " : " << op_sel_as.front() << endl;
        if(op_sel.front() != op_sel_as.front()) {
            if(varNames[s]->type[op_sel.front()] == 0) {
                varNames[s]->h_columns_int[op_sel_as.front()] = varNames[s]->h_columns_int[op_sel.front()];
                varNames[s]->h_columns_int.erase(op_sel.front());
                varNames[s]->d_columns_int[op_sel_as.front()] = varNames[s]->d_columns_int[op_sel.front()];
                varNames[s]->d_columns_int.erase(op_sel.front());
                varNames[s]->type[op_sel_as.front()] = 0;
                varNames[s]->type.erase(op_sel.front());
            }
            else if(varNames[s]->type[op_sel.front()] == 1) {
                varNames[s]->h_columns_float[op_sel_as.front()] = varNames[s]->h_columns_float[op_sel.front()];
                varNames[s]->h_columns_float.erase(op_sel.front());
                varNames[s]->d_columns_float[op_sel_as.front()] = varNames[s]->d_columns_float[op_sel.front()];
                varNames[s]->d_columns_float.erase(op_sel.front());
                varNames[s]->type[op_sel_as.front()] = 1;
                varNames[s]->type.erase(op_sel.front());
                varNames[s]->decimal.erase(op_sel.front());
            }
            else {
                varNames[s]->h_columns_char[op_sel_as.front()] = varNames[s]->h_columns_char[op_sel.front()];
                varNames[s]->h_columns_char.erase(op_sel.front());
                varNames[s]->d_columns_char[op_sel_as.front()] = varNames[s]->d_columns_char[op_sel.front()];
                varNames[s]->d_columns_char.erase(op_sel.front());
                varNames[s]->type[op_sel_as.front()] = 2;
                varNames[s]->type.erase(op_sel.front());
                varNames[s]->char_size[op_sel_as.front()] = varNames[s]->char_size[op_sel.front()];
                varNames[s]->char_size.erase(op_sel.front());
            };
            varNames[s]->decimal[op_sel_as.front()] = varNames[s]->decimal[op_sel.front()];
            std::vector<string>::iterator it;
            it = std::find(varNames[s]->columnNames.begin(), varNames[s]->columnNames.end(), op_sel.front());
            *it = op_sel_as.front();
        };
        op_sel_as.pop();
        op_sel.pop();
    };



    clean_queues();

    if(stat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };

    if(op_join.size()) {
        if(stat[op_join.front()] == statement_count && op_join.front().compare(j1) != 0) {
            varNames[op_join.front()]->free();
            varNames.erase(op_join.front());
        };
    };

}



///
/** 
emit_multijoin is callled by emit_join when join_tab_cnt > 1. This function is use to join multiple tables.
**/
void emit_multijoin(string s, string j1, string j2, unsigned int tab, char* res_name)
{

    if(varNames.find(j1) == varNames.end() || varNames.find(j2) == varNames.end()) {
        clean_queues();
        if(varNames.find(j1) == varNames.end())
            cout << "Couldn't find j1 " << j1 << endl;
        if(varNames.find(j2) == varNames.end())
            cout << "Couldn't find j2 " << j2 << endl;

        return;
    };

    CudaSet* left = varNames.find(j1)->second;
    CudaSet* right = varNames.find(j2)->second;

    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_value.front());
        op_value.pop();
        op_sel_as.push(op_value.front());
        op_value.pop();
    };

    queue<string> op_sel_s(op_sel);
    queue<string> op_sel_s_as(op_sel_as);
    queue<string> op_g(op_value);

    if(tab > 0) {
        for(unsigned int z = 0; z < join_tab_cnt - tab; z++) {
            for(unsigned int j = 0; j < join_and_cnt[z]*2 + 2; j++) {
                op_sel_s.push(op_g.front());
                op_sel_s_as.push(op_g.front());
                op_g.pop();
            };
        };
    };

    string f1 = op_g.front();
    op_g.pop();
    string f2 = op_g.front();
    op_g.pop();

    if (verbose)
        cout << "JOIN " << s <<  " " <<  f1 << " " << f2 << " " << getFreeMem() <<  endl;

    std::clock_t start1 = std::clock();
    CudaSet* c = new CudaSet(right, left, op_sel_s, op_sel_s_as);

    if ((left->mRecCount == 0 && !left->filtered) || (right->mRecCount == 0 && !right->filtered)) {
    	cout << "emit_multijoin->Created new CudaSet and returning" << endl;
        c = new CudaSet(left, right, op_sel_s, op_sel_s_as);
        varNames[res_name] = c;
        clean_queues();
        return;
    };

    if(join_tab_cnt > 1 && tab < join_tab_cnt)
        c->tmp_table = 1;
    else
        c->tmp_table = 0;

    string colname1, colname2;
    string tmpstr;
    cout << "emit_multijoin->looking for column names" << endl;
    if (std::find(left->columnNames.begin(), left->columnNames.end(), f1) != left->columnNames.end()) {
        colname1 = f1;
        if (std::find(right->columnNames.begin(), right->columnNames.end(), f2) != right->columnNames.end()) {
            colname2 = f2;
        }
        else {
            process_error(2, "Couldn't find column " + f2 );
        };
    }
    else if (std::find(right->columnNames.begin(), right->columnNames.end(), f1) != right->columnNames.end()) {
        colname2 = f1;
        tmpstr = f1;
        f1 = f2;
        if (std::find(left->columnNames.begin(), left->columnNames.end(), f2) != left->columnNames.end()) {
            colname1 = f2;
            f2 = tmpstr;
        }
        else {
            process_error(2, "Couldn't find column " +f2 );
        };
    }
    else {
        process_error(2, "Couldn't find column " + f1);
    };


    if (!((left->type[colname1] == 0 && right->type[colname2]  == 0) || (left->type[colname1] == 2 && right->type[colname2]  == 2)
            || (left->type[colname1] == 1 && right->type[colname2]  == 1 && left->decimal[colname1] && right->decimal[colname2]))) {
        process_error(2, "Joins on floats are not supported ");
    };


    bool decimal_join = 0;
    if (left->type[colname1] == 1 && right->type[colname2]  == 1)
        decimal_join = 1;

    queue<string> op_vd(op_g);
    queue<string> op_g1(op_g);
    queue<string> op_alt(op_sel);
    unsigned int jc = join_and_cnt[join_tab_cnt - tab];
    while(jc) {
        jc--;
        op_vd.pop();
        op_alt.push(op_vd.front());
        op_vd.pop();
    };

    bool str_join = 0;
    size_t rcount = 0, cnt_r;
    //need to sort the entire dataset by a key before loading segment by segment
    unsigned int r_parts = calc_right_partition(left, right, op_sel);
    //cout << "partitioned to " << r_parts << endl;
    unsigned int start_part = 0;
    queue<string> cc;

    if (left->type[colname1]  == 2) {
        left->d_columns_int[colname1] = thrust::device_vector<int_type>();
    }
    else {
        cc.push(f1);
        allocColumns(left, cc);
    };

    left->hostRecCount = left->mRecCount;

    size_t cnt_l, res_count, tot_count = 0, offset = 0, k = 0;
    queue<string> lc(cc);
    thrust::device_vector<int> p_tmp;
    thrust::device_vector<unsigned int> v_l(left->maxRecs);
    MGPU_MEM(int) aIndicesDevice, bIndicesDevice;
    std::vector<int> j_data;

    stack<string> exe_type;
    set<string> field_names;
    exe_type.push(f2);
    /*
	for(string *it=right->columnNames.begin(); it!=right->columnNames.end();it++) {
        if (std::find(c->columnNames.begin(), c->columnNames.end(), *it) != c->columnNames.end() || *it == f2) {
            field_names.insert(*it);
        };
    };
    */
    cout << "emit_multijoin->changed code, inserting field_names" << endl;
    for(unsigned int i = 0; i < right->columnNames.size(); i++) {
    	if(std::find(c->columnNames.begin(), c->columnNames.end(), right->columnNames[i]) != c->columnNames.end() || right->columnNames[i] == f2) {
    		field_names.insert(right->columnNames[i]);
    	}
    }

    right->hostRecCount = right->mRecCount;
    while(start_part < right->segCount) {

        bool rsz = 1;
        right->deAllocOnDevice();

        cout << "emit_multijoin->loading " << start_part << " " << r_parts << " " << getFreeMem() << endl;
        cout << "emit_multijoin->Tot segs " << right->segCount << endl;
        //if(right->not_compressed)
        //order_inplace_host(right, exe_type, field_names, 0);
        cout << "ordered " << endl;

        if(start_part + r_parts >= right->segCount ) {
            cnt_r = load_right(right, colname2, f2, op_g1, op_sel, op_alt, decimal_join, str_join, rcount, start_part, right->segCount, rsz);
            start_part = right->segCount;
        }
        else {
            cnt_r = load_right(right, colname2, f2, op_g1, op_sel, op_alt, decimal_join, str_join, rcount, start_part, start_part+r_parts, rsz);
            start_part = start_part+r_parts;
        };

        cout << "emit_multijoin->loaded " << cnt_r << " " << getFreeMem() << endl;
        right->mRecCount = cnt_r;

        cout << "emit_multijoin->sort on GPU or CPU.  FreeMem: " << getFreeMem() << ", Needed Mem: " << right->mRecCount*max_char(right) << endl;
        cout << "emit_multijoin->max_row=" << row_size(right) << ", max_row*mRecCount=" << right->mRecCount*row_size(right) << endl;
        //if(right->not_compressed && getFreeMem() < right->mRecCount*max_char(right)*2) {
        //Is there enough free memory on the GPU to sort on the GPU or do we need to do it on the CPU?
        if(right->not_compressed && getFreeMem() < right->mRecCount*row_size(right)) {
        	cout << "emit_multijoin->CopyToHost" << endl;
            right->CopyToHost(0, right->mRecCount);
            right->deAllocOnDevice();
            if (left->type[colname1]  != 2)
                order_inplace1(right, exe_type, field_names, 0);
            else
                order_inplace1(right, exe_type, field_names, 1);
        }
        else {
        	cout << "emit_multijoin->order_inplace" << endl;
            if (left->type[colname1]  != 2)
                order_inplace(right, exe_type, field_names, 0);
            else {
                order_inplace(right, exe_type, field_names, 1);
            };
        };

        for (unsigned int i = 0; i < left->segCount; i++) {

            if(verbose)
                //cout << "segment " << i <<  '\xd';
                cout << "segment " << i <<  endl;
            j_data.clear();
            std::clock_t start2 = std::clock();

            //for (set<unsigned int>::iterator it = left->ref_joins[colInd1][i].begin(); it != left->ref_joins[colInd1][i].end(); it++) {
            //	cout << "seg match " << *it << endl;
            //};

            //for (set<unsigned int>::iterator it = right->orig_segs[left->ref_sets[colInd1]].begin(); it != right->orig_segs[left->ref_sets[colInd1]].end(); it++) {
            //	cout << "right segs " << *it << endl;
            //};


            if(left->ref_joins[colname1][i].size() && right->orig_segs[left->ref_sets[colname1]].size()) {
                set_intersection(left->ref_joins[colname1][i].begin(),left->ref_joins[colname1][i].end(),
                                 right->orig_segs[left->ref_sets[colname1]].begin(), right->orig_segs[left->ref_sets[colname1]].end(),
                                 std::back_inserter(j_data));
                if(j_data.empty()) {
                    cout << "skipping a segment " << endl;
                    continue;
                };

            };


            cnt_l = 0;
            if (left->type[colname1]  != 2) {
                copyColumns(left, lc, i, cnt_l);
            }
            else {
                left->add_hashed_strings(f1, i);
            };


            if(!left->filtered) {
                if (left->type[colname1]  != 2)
                    cnt_l = left->mRecCount;
                else
                    cnt_l = left->d_columns_int[colname1].size();
            }
            else {
                cnt_l = left->mRecCount;
            };


            if (cnt_l) {

                // sort the left index column, save the permutation vector, it might be needed later

                thrust::device_ptr<int_type> d_col((int_type*)thrust::raw_pointer_cast(left->d_columns_int[colname1].data()));
                thrust::sequence(v_l.begin(), v_l.begin() + cnt_l,0,1);

                bool do_sort = 1;
                if(!left->sorted_fields.empty()) {
                    if(left->sorted_fields.front() == f1) {
                        do_sort = 0;
                    };
                }
                else if(!left->presorted_fields.empty()) {
                    if(left->presorted_fields.front() == f1) {
                        do_sort = 0;
                    };
                };

                if(do_sort)
                    thrust::sort_by_key(d_col, d_col + cnt_l, v_l.begin());
                else if(verbose)
                    cout << "No need of sorting " << endl;

                cout << "emit_multijoin->join " << cnt_l << ":" << cnt_r << " " << join_type.front() << endl;
                cout << "emit_multijoin->SZ " << left->d_columns_int[colname1].size() << endl;


                if (left->d_columns_int[colname1][0] > right->d_columns_int[colname2][cnt_r-1] ||
                        left->d_columns_int[colname1][cnt_l-1] < right->d_columns_int[colname2][0]) {
                    cout << endl << "skipping after copying " << endl;
                    continue;
                };
                //else
                //    cout << "JOINING " << left->d_columns_int[colname1][0] << ":" << left->d_columns_int[colname1][cnt_l-1] << " AND " << right->d_columns_int[colname2][0] << ":" << right->d_columns_int[colname2][cnt_r-1] << endl;

                //cout << "joining " << left->d_columns_int[colname1][0] << " : " << left->d_columns_int[colname1][cnt_l-1] << " and " << right->d_columns_int[colname2][0] << " : " << right->d_columns_int[colname2][cnt_r-1] << endl;


                char join_kind = join_type.front();

                if (left->type[colname1] == 2) {
                    thrust::device_ptr<int_type> d_col_r((int_type*)thrust::raw_pointer_cast(right->d_columns_int[colname2].data()));

                    //for(int z = 0; z < cnt_r ; z++)
                    //	cout << " R " << right->d_columns_int[colname2][z] << endl;

                    //for(int z = 0; z < cnt_l ; z++)
                    //	cout << " L " << left->d_columns_int[colname1][z] << endl;


                    res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(d_col), cnt_l,
                                thrust::raw_pointer_cast(d_col_r), cnt_r,
                                &aIndicesDevice, &bIndicesDevice,
                                mgpu::less<unsigned long long int>(), *context);

                }
                else {

                    if (join_kind == 'I')
                        res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
                                    thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
                                    &aIndicesDevice, &bIndicesDevice,
                                    mgpu::less<int_type>(), *context);
                    else if(join_kind == 'L')
                        res_count = RelationalJoin<MgpuJoinKindLeft>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
                                    thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
                                    &aIndicesDevice, &bIndicesDevice,
                                    mgpu::less<int_type>(), *context);
                    else if(join_kind == 'R')
                        res_count = RelationalJoin<MgpuJoinKindRight>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
                                    thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
                                    &aIndicesDevice, &bIndicesDevice,
                                    mgpu::less<int_type>(), *context);
                    else if(join_kind == 'O')
                        res_count = RelationalJoin<MgpuJoinKindOuter>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
                                    thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
                                    &aIndicesDevice, &bIndicesDevice,
                                    mgpu::less<int_type>(), *context);
                };

                cout << "emit_multijoin->RES " << res_count << " seg " << i << endl;

                int* r1 = aIndicesDevice->get();
                thrust::device_ptr<int> d_res1((int*)r1);
                int* r2 = bIndicesDevice->get();
                thrust::device_ptr<int> d_res2((int*)r2);


                if(res_count) {
                    p_tmp.resize(res_count);
                    thrust::sequence(p_tmp.begin(), p_tmp.end(),-1);
                    thrust::gather_if(d_res1, d_res1+res_count, d_res1, v_l.begin(), p_tmp.begin(), is_positive<int>());
                };


                // check if the join is a multicolumn join
                unsigned int mul_cnt = join_and_cnt[join_tab_cnt - tab];

                while(mul_cnt) {

                    mul_cnt--;
                    string f3 = op_g.front();
                    op_g.pop();
                    string f4 = op_g.front();
                    op_g.pop();

                    cout << "emit_multijoin->ADDITIONAL COL JOIN " << f3 << " " << f4 << " " << getFreeMem() << endl;

                    queue<string> rc;
                    rc.push(f3);

                    allocColumns(left, rc);
                    left->hostRecCount = left->mRecCount;
                    size_t offset = 0;
                    copyColumns(left, rc, i, offset, 0, 0);
                    rc.pop();

                    void* temp;
                    CUDA_SAFE_CALL(hipMalloc((void **) &temp, res_count*float_size));
                    void* temp1;
                    CUDA_SAFE_CALL(hipMalloc((void **) &temp1, res_count*float_size));
                    hipMemset(temp,0,res_count*float_size);
                    hipMemset(temp1,0,res_count*float_size);



                    if (res_count) {
                        thrust::device_ptr<bool> d_add = thrust::device_malloc<bool>(res_count);

                        if (left->type[f3] == 1 && right->type[f4]  == 1) {

                            if(right->d_columns_float[f4].size() == 0)
                                load_queue(rc, right, 0, f4, rcount, 0, right->segCount, 0, 0);

                            thrust::device_ptr<float_type> d_tmp((float_type*)temp);
                            thrust::device_ptr<float_type> d_tmp1((float_type*)temp1);
                            thrust::gather_if(p_tmp.begin(), p_tmp.end(), p_tmp.begin(), left->d_columns_float[f3].begin(), d_tmp, is_positive<int>());
                            thrust::gather_if(d_res2, d_res2+res_count, d_res2, right->d_columns_float[f4].begin(), d_tmp1, is_positive<int>());
                            thrust::transform(d_tmp, d_tmp+res_count, d_tmp1, d_add, float_equal_to());
                        }
                        else {
                            if(right->d_columns_int[f4].size() == 0) {
                                load_queue(rc, right, 0, f4, rcount, 0, right->segCount, 0, 0);
                            };
                            thrust::device_ptr<int_type> d_tmp((int_type*)temp);
                            thrust::device_ptr<int_type> d_tmp1((int_type*)temp1);
                            thrust::gather_if(p_tmp.begin(), p_tmp.end(), p_tmp.begin(), left->d_columns_int[f3].begin(), d_tmp, is_positive<int>());
                            thrust::gather_if(d_res2, d_res2+res_count, d_res2, right->d_columns_int[f4].begin(), d_tmp1, is_positive<int>());
                            thrust::transform(d_tmp, d_tmp+res_count, d_tmp1, d_add, thrust::equal_to<int_type>());
                        };

                        if (join_kind == 'I') {  // result count changes only in case of an inner join

                            unsigned int new_cnt = thrust::count(d_add, d_add+res_count, 1);
                            thrust::stable_partition(d_res2, d_res2 + res_count, d_add, thrust::identity<unsigned int>());
                            thrust::stable_partition(p_tmp.begin(), p_tmp.end(), d_add, thrust::identity<unsigned int>());
                            thrust::device_free(d_add);
                            res_count = new_cnt;
                        }
                        else { //otherwise we consider it a valid left join result with non-nulls on the left side and nulls on the right side
                            thrust::transform(d_res2, d_res2 + res_count, d_add , d_res2, set_minus());
                        };
                    };
                    hipFree(temp);
                    hipFree(temp1);
                };

                //string sss = s;
                //cout << " " << tot_count << " " << res_count << endl;
                //if(sss != "OLC") {
                //	cout << "RUN " << tot_count << " " << res_count << endl;

                tot_count = tot_count + res_count;

                if(res_count) {

                    for (map<string, set<unsigned int> >::iterator itr = left->orig_segs.begin(); itr != left->orig_segs.end(); itr++) {
                        for (set<unsigned int>::iterator it = itr->second.begin(); it != itr->second.end(); it++) {
                            cout << "emit_multijoin->LEFT SEGS " << itr->first << " : " << *it << endl;
                            c->orig_segs[itr->first].insert(*it);
                        };
                    };

                    for (map<string, set<unsigned int> >::iterator itr = right->orig_segs.begin(); itr != right->orig_segs.end(); itr++) {
                        for (set<unsigned int>::iterator it = itr->second.begin(); it != itr->second.end(); it++) {
                            cout << "emit_multijoin->RIGHT SEGS " << itr->first << " : " << *it << endl;
                            c->orig_segs[itr->first].insert(*it);
                        };
                    };

                    offset = c->mRecCount;
                    queue<string> op_sel1(op_sel_s);
                    c->resize_join(res_count);
                    void* temp;
                    CUDA_SAFE_CALL(hipMalloc((void **) &temp, res_count*max_char(c)));

                    bool copied = 0;
                    thrust::host_vector<unsigned int> prm_vh;
                    std::map<string,bool> processed;
                    bool cmp_type;


                    //std::clock_t start1 = std::clock();
                    while(!op_sel1.empty()) {

                        if (processed.find(op_sel1.front()) != processed.end()) {
                            op_sel1.pop();
                            continue;
                        }
                        else
                            processed[op_sel1.front()] = 1;

                        while(!cc.empty())
                            cc.pop();

                        cc.push(op_sel1.front());

                        if(std::find(left->columnNames.begin(), left->columnNames.end(), op_sel1.front()) !=  left->columnNames.end()) {
                            // copy field's segment to device, gather it and copy to the host

                            if(left->filtered)
                                cmp_type = varNames[left->source_name]->compTypes[op_sel1.front()];
                            else
                                cmp_type = left->compTypes[op_sel1.front()];

                            if ((((left->type[op_sel1.front()] == 0) || ((left->type[op_sel1.front()] == 1) &&
                                    left->decimal[op_sel1.front()])) && cmp_type == 0) && (op_sel1.front() != colname1) && left->not_compressed == 0) { // do the processing on host

                                void* h;
                                unsigned int cnt, bits;
                                int_type lower_val;


                                if(verbose)
                                    cout << "emit_multijoin->processing " << op_sel1.front() << " " << i << " " << cmp_type << endl;

                                if(!copied) {
                                    if(left->filtered && left->prm_index == 'R') {
                                        thrust::device_vector<unsigned int> prm_v(res_count);
                                        thrust::gather(p_tmp.begin(), p_tmp.begin() + res_count, left->prm_d.begin(), prm_v.begin());
                                        prm_vh = prm_v;
                                    }
                                    else {
                                        prm_vh = p_tmp;
                                    };
                                    copied = 1;
                                };

                                CudaSet *t;
                                if(left->filtered)
                                    t = varNames[left->source_name];
                                else
                                    t = left;

                                t->readSegmentsFromFile(i, op_sel1.front(), 0);

                                if(t->type[op_sel1.front()] == 0) {
                                    h = t->h_columns_int[op_sel1.front()].data();
                                }
                                else {
                                    h = t->h_columns_float[op_sel1.front()].data();
                                };

                                cnt = ((unsigned int*)h)[0];
                                lower_val = ((int_type*)(((unsigned int*)h)+1))[0];
                                bits = ((unsigned int*)((char*)h + cnt))[8];
                                cout << "emit_multijoin->" << cnt << " " << lower_val << " " << bits << endl;

                                if(bits == 8) {
                                    if(left->type[op_sel1.front()] == 0) {
                                        thrust::gather(prm_vh.begin(), prm_vh.end(), (char*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
                                    }
                                    else {
                                        int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                                        thrust::gather(prm_vh.begin(), prm_vh.end(), (char*)((unsigned int*)h + 6), ptr + offset);
                                    };
                                }
                                else if(bits == 16) {
                                    if(left->type[op_sel1.front()] == 0) {
                                        thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned short int*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
                                    }
                                    else {
                                        int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                                        thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned short int*)((unsigned int*)h + 6), ptr + offset);
                                    };
                                }
                                else if(bits == 32) {
                                    if(left->type[op_sel1.front()] == 0) {
                                        thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned int*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
                                    }
                                    else {
                                        int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                                        thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned int*)((unsigned int*)h + 6), ptr + offset);
                                    };
                                }
                                else if(bits == 64) {
                                    if(left->type[op_sel1.front()] == 0) {
                                        thrust::gather(prm_vh.begin(), prm_vh.end(),  (int_type*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
                                    }
                                    else {
                                        int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                                        thrust::gather(prm_vh.begin(), prm_vh.end(), (int_type*)((unsigned int*)h + 6), ptr + offset);
                                    };
                                };

                                if(left->type[op_sel1.front()] == 0) {
                                    thrust::transform(c->h_columns_int[op_sel1.front()].begin() + offset, c->h_columns_int[op_sel1.front()].begin() + offset + res_count,
                                                      thrust::make_constant_iterator(lower_val), c->h_columns_int[op_sel1.front()].begin() + offset, thrust::plus<int_type>());
                                }
                                else {
                                    int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
                                    thrust::transform(ptr + offset, ptr + offset + res_count,
                                                      thrust::make_constant_iterator(lower_val), ptr + offset, thrust::plus<int_type>());
                                    thrust::transform(ptr + offset, ptr + offset + res_count, c->h_columns_float[op_sel1.front()].begin() + offset, long_to_float());
                                };

                            }
                            else {

                                allocColumns(left, cc);
                                copyColumns(left, cc, i, k, 0, 0);

                                //gather
                                if(left->type[op_sel1.front()] == 0) {
                                    thrust::device_ptr<int_type> d_tmp((int_type*)temp);
                                    thrust::sequence(d_tmp, d_tmp+res_count,0,0);
                                    thrust::gather_if(p_tmp.begin(), p_tmp.begin() + res_count, p_tmp.begin(), left->d_columns_int[op_sel1.front()].begin(), d_tmp, is_positive<int>());
                                    thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[op_sel1.front()].begin() + offset);
                                }
                                else if(left->type[op_sel1.front()] == 1) {
                                    thrust::device_ptr<float_type> d_tmp((float_type*)temp);
                                    thrust::sequence(d_tmp, d_tmp+res_count,0,0);
                                    thrust::gather_if(p_tmp.begin(), p_tmp.begin() + res_count, p_tmp.begin(), left->d_columns_float[op_sel1.front()].begin(), d_tmp, is_positive<int>());
                                    thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_float[op_sel1.front()].begin() + offset);
                                }
                                else { //strings
                                    thrust::device_ptr<char> d_tmp((char*)temp);
                                    thrust::fill(d_tmp, d_tmp+res_count*left->char_size[op_sel1.front()],0);
                                    str_gather(thrust::raw_pointer_cast(p_tmp.data()), res_count, (void*)left->d_columns_char[op_sel1.front()],
                                               (void*) thrust::raw_pointer_cast(d_tmp), left->char_size[op_sel1.front()]);
                                    hipMemcpy( (void*)&c->h_columns_char[op_sel1.front()][offset*c->char_size[op_sel1.front()]], (void*) thrust::raw_pointer_cast(d_tmp),
                                                c->char_size[op_sel1.front()] * res_count, hipMemcpyDeviceToHost);
                                };


                                if(op_sel1.front() != colname1)
                                    left->deAllocColumnOnDevice(op_sel1.front());
                            }
                        }
                        else if(std::find(right->columnNames.begin(), right->columnNames.end(), op_sel1.front()) !=  right->columnNames.end()) {

                            //gather
                            if(right->type[op_sel1.front()] == 0) {
                                thrust::device_ptr<int_type> d_tmp((int_type*)temp);
                                thrust::sequence(d_tmp, d_tmp+res_count,0,0);
                                thrust::gather_if(d_res2, d_res2 + res_count, d_res2, right->d_columns_int[op_sel1.front()].begin(), d_tmp, is_positive<int>());
                                thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[op_sel1.front()].begin() + offset);

                            }
                            else if(right->type[op_sel1.front()] == 1) {
                                thrust::device_ptr<float_type> d_tmp((float_type*)temp);
                                thrust::sequence(d_tmp, d_tmp+res_count,0,0);
                                thrust::gather_if(d_res2, d_res2 + res_count, d_res2, right->d_columns_float[op_sel1.front()].begin(), d_tmp, is_positive<int>());
                                thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_float[op_sel1.front()].begin() + offset);
                            }
                            else { //strings
							
                                thrust::device_ptr<char> d_tmp((char*)temp);
                                thrust::fill(d_tmp, d_tmp+res_count*right->char_size[op_sel1.front()],0);
                                str_gather(thrust::raw_pointer_cast(d_res2), res_count, (void*)right->d_columns_char[op_sel1.front()],
                                           (void*) thrust::raw_pointer_cast(d_tmp), right->char_size[op_sel1.front()]);
                                hipMemcpy( (void*)&c->h_columns_char[op_sel1.front()][offset*c->char_size[op_sel1.front()]], (void*) thrust::raw_pointer_cast(d_tmp),
                                            c->char_size[op_sel1.front()] * res_count, hipMemcpyDeviceToHost);
											
                            };
                        }
                        else {
                        };
                        op_sel1.pop();
                    };
                    hipFree(temp);
                };
            };
            std::cout<< endl << "emit_multijoin->seg time " <<  ( ( std::clock() - start2 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        };
    };

    left->deAllocOnDevice();
    right->deAllocOnDevice();
    c->deAllocOnDevice();

    varNames[s] = c;
    c->mRecCount = tot_count;
    c->hostRecCount = tot_count;
    c->name = s;

    if(verbose)
        cout << endl << "tot res " << tot_count << " " << getFreeMem() << endl;

    size_t tot_size = 0;
    for (unsigned int i = 0; i < c->columnNames.size(); i++ ) {
        if(c->type[c->columnNames[i]] <= 1)
            tot_size = tot_size + tot_count*8;
        else
            tot_size = tot_size + tot_count*c->char_size[c->columnNames[i]];
    };

    if ((getFreeMem() - 300000000) > tot_size) {
        c->maxRecs = tot_count;
    }
    else {
        c->segCount = ((tot_size/(getFreeMem() - 300000000)) + 1);
        c->maxRecs = c->hostRecCount - (c->hostRecCount/c->segCount)*(c->segCount-1);
    };


    if(right->tmp_table == 1) {
        right->free();
        varNames.erase(j2);
    }
    else {
        if(stat[j2] == statement_count) {
            right->free();
            varNames.erase(j2);
        };

    };

    if(stat[j1] == statement_count) {
        left->free();
        varNames.erase(j1);
    };
    join_type.pop();

    if(verbose)
        std::cout<< "join time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;


}


///
/**
The order_on_host function is called into action by emit_order when the computation on the records in the column being ordered is to big to fit into gpu. This function
does the ordering on the host device instead.
**/ 
void order_on_host(CudaSet *a, CudaSet* b, queue<string> names, stack<string> exe_type, stack<string> exe_value)
{
    unsigned int tot = 0;
    if(!a->not_compressed) { //compressed
        allocColumns(a, names);

        unsigned int c = 0;
        size_t cnt = 0;
        for(unsigned int i = 0; i < a->segCount; i++) {
            copyColumns(a, names, (a->segCount - i) - 1, cnt);	//uses segment 1 on a host	to copy data from a file to gpu
            if (a->mRecCount) {
                a->CopyToHost((c - tot) - a->mRecCount, a->mRecCount);
                tot = tot + a->mRecCount;
            };
        };
    }
    else
        tot = a->mRecCount;

    b->resize(tot); //resize host arrays
    a->mRecCount = tot;

    unsigned int* permutation = new unsigned int[a->mRecCount];
    thrust::sequence(permutation, permutation + a->mRecCount);

    size_t maxSize =  a->mRecCount;
    char* temp;
    temp = new char[maxSize*max_char(a)];

    // sort on host

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {

        if (a->type[exe_type.top()] == 0)
            update_permutation_host(a->h_columns_int[exe_type.top()].data(), permutation, a->mRecCount, exe_value.top(), (int_type*)temp);
        else if (a->type[exe_type.top()] == 1)
            update_permutation_host(a->h_columns_float[exe_type.top()].data(), permutation, a->mRecCount,exe_value.top(), (float_type*)temp);
        else {
            update_permutation_char_host(a->h_columns_char[exe_type.top()], permutation, a->mRecCount, exe_value.top(), b->h_columns_char[exe_type.top()], a->char_size[exe_type.top()]);
        };
    };

    for (unsigned int i = 0; i < a->mColumnCount; i++) {
        if (a->type[a->columnNames[i]] == 0) {
            apply_permutation_host(a->h_columns_int[a->columnNames[i]].data(), permutation, a->mRecCount, b->h_columns_int[a->columnNames[i]].data());
        }
        else if (a->type[a->columnNames[i]] == 1)
            apply_permutation_host(a->h_columns_float[a->columnNames[i]].data(), permutation, a->mRecCount, b->h_columns_float[a->columnNames[i]].data());
        else {
            apply_permutation_char_host(a->h_columns_char[a->columnNames[i]], permutation, a->mRecCount, b->h_columns_char[a->columnNames[i]], a->char_size[a->columnNames[i]]);
        };
    };

    delete [] temp;
    delete [] permutation;
}
///
/**
 emit_order is called when the keyword 'Order' is parsed. This function has 4 parameters 3 of them are command line variables read by the scanner. This 
function allows columns to be sorted in ascending or descending order by whatever the user specifies. If the computation requires to much  memory it 
will be done on the host device instead.

 **/


void emit_order(char *s, char *f, int e, int ll)
{
    if(ll == 0)
        statement_count++;

    if (scan_state == 0 && ll == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(2, "Order : couldn't find variable " + string(f));
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        return;
    };

    if (scan_state == 0)
        check_used_vars();

    if(varNames.find(f) == varNames.end() ) {
        clean_queues();
        return;
    };

    CudaSet* a = varNames.find(f)->second;


    if (a->mRecCount == 0)	{
        if(varNames.find(s) == varNames.end())
            varNames[s] = new CudaSet(0,1);
        else {
            CudaSet* c = varNames.find(s)->second;
            c->mRecCount = 0;
        };
        return;
    };

    stack<string> exe_type, exe_value;

    if(verbose)
        cout << "order: " << s << " " << f << endl;


    for(int i=0; !op_type.empty(); ++i, op_type.pop(),op_value.pop()) {
        if ((op_type.front()).compare("NAME") == 0) {
            exe_type.push(op_value.front());
            exe_value.push("ASC");
        }
        else {
            exe_type.push(op_type.front());
            exe_value.push(op_value.front());
        };
        if(std::find(a->columnNames.begin(), a->columnNames.end(), exe_type.top()) == a->columnNames.end()) {
            process_error(2, "Couldn't find name " + exe_type.top());
        };

    };

    stack<string> tp(exe_type);
    queue<string> op_vx;
    while (!tp.empty()) {
        op_vx.push(tp.top());
        tp.pop();
    };

    queue<string> names;
    for (unsigned int i = 0; i < a->columnNames.size() ; i++ )
        names.push(a->columnNames[i]);

    CudaSet *b = a->copyDeviceStruct();

    //lets find out if our data set fits into a GPU
    size_t mem_available = getFreeMem();
    size_t rec_size = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        if(a->type[a->columnNames[i]] == 0)
            rec_size = rec_size + int_size;
        else if(a->type[a->columnNames[i]] == 1)
            rec_size = rec_size + float_size;
        else
            rec_size = rec_size + a->char_size[a->columnNames[i]];
    };
    bool fits;
    if (rec_size*a->mRecCount > (mem_available/2)) // doesn't fit into a GPU
        fits = 0;
    else fits = 1;

    if(!fits) {
        order_on_host(a, b, names, exe_type, exe_value);
    }
    else {
        // initialize permutation to [0, 1, 2, ... ,N-1]
        thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(a->mRecCount);
        thrust::sequence(permutation, permutation+(a->mRecCount));

        unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);

        void* temp;
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*max_char(a)));

        if(a->filtered)
            varNames[a->source_name]->hostRecCount = varNames[a->source_name]->mRecCount;
        else
            a->hostRecCount = a->mRecCount;;

        size_t rcount;
        a->mRecCount = load_queue(names, a, 1, op_vx.front(), rcount, 0, a->segCount);

        if(a->filtered)
            varNames[a->source_name]->mRecCount = varNames[a->source_name]->hostRecCount;
        else
            a->mRecCount = a->hostRecCount;;


        for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {
            if (a->type[exe_type.top()] == 0)
                update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, a->mRecCount, exe_value.top(), (int_type*)temp);
            else if (a->type[exe_type.top()] == 1)
                update_permutation(a->d_columns_float[exe_type.top()], raw_ptr, a->mRecCount,exe_value.top(), (float_type*)temp);
            else {
                update_permutation_char(a->d_columns_char[exe_type.top()], raw_ptr, a->mRecCount, exe_value.top(), (char*)temp, a->char_size[exe_type.top()]);
                //update_permutation(a->d_columns_int[int_col_count+str_count], raw_ptr, a->mRecCount, exe_value.top(), (int_type*)temp);
                //str_count++;
            };
        };

        b->resize(a->mRecCount); //resize host arrays
        b->mRecCount = a->mRecCount;
        //str_count = 0;

        for (unsigned int i = 0; i < a->mColumnCount; i++) {
            if (a->type[a->columnNames[i]] == 0)
                apply_permutation(a->d_columns_int[a->columnNames[i]], raw_ptr, a->mRecCount, (int_type*)temp);
            else if (a->type[a->columnNames[i]] == 1)
                apply_permutation(a->d_columns_float[a->columnNames[i]], raw_ptr, a->mRecCount, (float_type*)temp);
            else {
                apply_permutation_char(a->d_columns_char[a->columnNames[i]], raw_ptr, a->mRecCount, (char*)temp, a->char_size[a->columnNames[i]]);
                //str_count++;
            };
        };

        for(unsigned int i = 0; i < a->mColumnCount; i++) {
            switch(a->type[a->columnNames[i]]) {
            case 0 :
                thrust::copy(a->d_columns_int[a->columnNames[i]].begin(), a->d_columns_int[a->columnNames[i]].begin() + a->mRecCount, b->h_columns_int[a->columnNames[i]].begin());
                break;
            case 1 :
                thrust::copy(a->d_columns_float[a->columnNames[i]].begin(), a->d_columns_float[a->columnNames[i]].begin() + a->mRecCount, b->h_columns_float[a->columnNames[i]].begin());
                break;
            default :
                hipMemcpy(b->h_columns_char[a->columnNames[i]], a->d_columns_char[a->columnNames[i]], a->char_size[a->columnNames[i]]*a->mRecCount, hipMemcpyDeviceToHost);
            }
        };

        b->deAllocOnDevice();
        a->deAllocOnDevice();


        thrust::device_free(permutation);
        hipFree(temp);
    };

    varNames[s] = b;
    b->segCount = 1;
    b->not_compressed = 1;

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(f);
    };
}


///
/**
emit_select is the function called when the scanner reads over the keyword 'SELECT'. emit_select has 3 parameters all of whcih are entered into the command line
Function allows user to pick selected data and view as columns  
**/
void emit_select(char *s, char *f, int ll)
{

    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(2, "Select : couldn't find variable " + string(f) );
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        check_used_vars();
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        cout << "Couldn't find1 " << f << endl;
        return;
    };

    queue<string> op_v1(op_value);
    while(op_v1.size() > ll)
        op_v1.pop();


    stack<string> op_v2;
    queue<string> op_v3;

    for(int i=0; i < ll; ++i) {
        op_v2.push(op_v1.front());
        op_v3.push(op_v1.front());
        op_v1.pop();
    };


    CudaSet *a;
    if(varNames.find(f) != varNames.end())
        a = varNames.find(f)->second;
    else {
        process_error(2, "Couldn't find " + string(f) );
    };

    if(a->mRecCount == 0 && !a->filtered) {
        CudaSet *c;
        c = new CudaSet(0,1);
        varNames[s] = c;
        c->name = s;
        clean_queues();
        if(verbose)
            cout << "SELECT " << s << " count : 0,  Mem " << getFreeMem() << endl;
        return;
    };

    if(verbose)
        cout << "SELECT " << s << " " << f << " " << getFreeMem() << endl;
    std::clock_t start1 = std::clock();

    // here we need to determine the column count and composition

    queue<string> op_v(op_value);
    queue<string> op_vx;
    set<string> field_names;
    map<string,string> aliases;
    string tt;

    while(!op_v.empty()) {
        if(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) != a->columnNames.end()) {
            tt = op_v.front();
            op_v.pop();
            if(!op_v.empty()) {
                if(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) == a->columnNames.end()) {
                    if(aliases.count(tt) == 0) {
                        aliases[tt] = op_v.front();
                    };
                }
                else {
                    while(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) == a->columnNames.end() && !op_v.empty()) {
                        op_v.pop();
                    };
                };
            };
        };
        if(!op_v.empty())
            op_v.pop();
    };

    op_v = op_value;
    while(!op_v.empty()) {
        if(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) != a->columnNames.end()) {
            field_names.insert(op_v.front());
        };
        op_v.pop();
    };

    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it)  {
        op_vx.push(*it);
    };

    // find out how many columns a new set will have
    queue<string> op_t(op_type);
    int_type col_count = 0;

    for(int i=0; !op_t.empty(); ++i, op_t.pop())
        if((op_t.front()).compare("emit sel_name") == 0)
            col_count++;

    CudaSet *b, *c;

    if(a->segCount <= 1)
        setSegments(a, op_vx);
    allocColumns(a, op_vx);

    unsigned int cycle_count;
    if(a->filtered)
        cycle_count = varNames[a->source_name]->segCount;
    else
        cycle_count = a->segCount;

    size_t ol_count = a->mRecCount, cnt;
    a->hostRecCount = a->mRecCount;
    b = new CudaSet(0, col_count);
    b->name = "tmp b in select";
    bool b_set = 0, c_set = 0;

    //size_t tmp_size = a->mRecCount;
    //if(a->segCount > 1)
    //    tmp_size = a->maxRecs;

    vector<thrust::device_vector<int_type> > distinct_val; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_hash; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_tmp;

    /* for(unsigned int i = 0; i < distinct_cnt; i++) {
         distinct_tmp.push_back(thrust::device_vector<int_type>(tmp_size));
         distinct_val.push_back(thrust::device_vector<int_type>());
         distinct_hash.push_back(thrust::device_vector<int_type>());
     };
    */

// find out how many string columns we have. Add int_type columns to store string hashes for sort/groupby ops.
    stack<string> op_s = op_v2;

    while(!op_s.empty()) {
        if (a->type[op_s.top()] == 2) {
            a->d_columns_int[op_s.top()] = thrust::device_vector<int_type>(a->maxRecs);
        };
        op_s.pop();
    };

    bool one_liner;


    for(unsigned int i = 0; i < cycle_count; i++) {          // MAIN CYCLE
        if(verbose)
            cout << "segment " << i << " select mem " << getFreeMem() << endl;
        std::clock_t start3 = std::clock();

        cnt = 0;
        copyColumns(a, op_vx, i, cnt);
        //std::cout<< "cpy time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';
        op_s = op_v2;

        while(!op_s.empty() && a->mRecCount != 0 && a->not_compressed) {

            if (a->type[op_s.top()] == 2) {
                a->d_columns_int[op_s.top()].resize(0);
                a->add_hashed_strings(op_s.top(), i);
            };
            op_s.pop();
        };

        if(a->mRecCount) {
            if (ll != 0) {
                order_inplace(a, op_v2, field_names, 1);
                a->GroupBy(op_v2);
            };

            cout << "calling select " << endl;
            select(op_type,op_value,op_nums, op_nums_f,a,b, distinct_tmp, one_liner);

            if(i == 0)
                std::reverse(b->columnNames.begin(), b->columnNames.end());

            if(!b_set) {
                b_set = 1;
                unsigned int old_cnt = b->mRecCount;
                b->mRecCount = 0;
                b->resize(a->maxRecs);
                b->mRecCount = old_cnt;
            };


            if (!c_set && b->mRecCount > 0) {
                c = new CudaSet(0, col_count);
                create_c(c,b);
                c_set = 1;
                c->name = s;
            };

            cout << "emit_select->if to add or copy" << endl;
            if (ll != 0 && cycle_count > 1  && b->mRecCount > 0) {
            	cout << "add" << endl;
                add(c,b,op_v3, aliases, distinct_tmp, distinct_val, distinct_hash, a);
            }
            else {
            	cout << "start copy" << endl;
                //copy b to c
                unsigned int c_offset = c->mRecCount;
                c->resize(b->mRecCount);

                for(unsigned int j=0; j < b->columnNames.size(); j++) {
                    if (b->type[b->columnNames[j]] == 0) {
                        thrust::copy(b->d_columns_int[b->columnNames[j]].begin(), b->d_columns_int[b->columnNames[j]].begin() + b->mRecCount, c->h_columns_int[b->columnNames[j]].begin() + c_offset);
                    }
                    else if (b->type[b->columnNames[j]] == 1) {
                        thrust::copy(b->d_columns_float[b->columnNames[j]].begin(), b->d_columns_float[b->columnNames[j]].begin() + b->mRecCount, c->h_columns_float[b->columnNames[j]].begin() + c_offset);
                    }
                    else {
                        hipMemcpy((void*)(thrust::raw_pointer_cast(c->h_columns_char[b->columnNames[j]] + b->char_size[b->columnNames[j]]*c_offset)), (void*)thrust::raw_pointer_cast(b->d_columns_char[b->columnNames[j]]),
                                   b->char_size[b->columnNames[j]] * b->mRecCount, hipMemcpyDeviceToHost);
                    };
                };
            };
        };
        std::cout<< "cycle sel time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';
    };

    a->mRecCount = ol_count;
    a->mRecCount = a->hostRecCount;
    a->deAllocOnDevice();
    b->deAllocOnDevice();

    if(!c_set) {
        CudaSet *c;
        c = new CudaSet(0,1);
        varNames[s] = c;
        c->name = s;
        clean_queues();
        return;
    };

    if (ll != 0) {
        count_avg(c, distinct_hash);
    }
    else {
        if(one_liner) {
            count_simple(c);
        };
    };

    c->maxRecs = c->mRecCount;
    c->name = s;
    c->keep = 1;
    cout << "select res " << c->mRecCount << endl;


    clean_queues();

    varNames[s] = c;
    b->free();
    varNames[s]->keep = 1;


    if(stat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };


    if(stat[f] == statement_count && a->keep == 0) {
        a->free();
        varNames.erase(f);
    };
    if(verbose)
        std::cout<< "select time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
}


/**
emit_insert is the function called when the keyword 'INSERT' is parsed over. This function calls insert_records(f,s)
**/
//Both the source and destination of the insert operator can be either derived or permanent dataset
//But for now lets see if I can code only permanent to permanent code path and get away with it
void emit_insert(char *f, char* s) {
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(2, "Insert : couldn't find variable " + string(f));
        };
        if (stat.find(s) == stat.end() && data_dict.count(s) == 0) {
            process_error(2, "Insert : couldn't find variable " + string(s) );
        };
        check_used_vars();
        stat[f] = statement_count;
        stat[s] = statement_count;
        clean_queues();
        return;
    };


    if(varNames.find(f) == varNames.end() || varNames.find(s) == varNames.end()) {
        clean_queues();
        return;
    };

    if(verbose)
        cout << "INSERT " << f << " " << s << endl;
    insert_records(f,s);
    clean_queues();


};

void emit_mulite(char *f)
{

};


void emit_delete(char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end()  && data_dict.count(f) == 0) {
            process_error(2, "Delete : couldn't find variable " + string(f));
        };
        stat[f] = statement_count;
        check_used_vars();
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };

    delete_records(f);
    cout << "DELETE " << f <<  endl;
    clean_queues();

}

/**
 emit_case is called when case statments like THEN, ELSE, WHEN, and END are use with CASE
**/ 
void emit_case()
{
    op_case = 1;
    if (scan_state == 1)
        cout << "emit case " << endl;
    //extract releveant values and pass to modified filter
    // get a bool vector back
    /*						while(!op_type.empty())
    						{
    						cout << "CASE type " << op_type.front() << endl;
    						op_type.pop();
    						}
    */
}

/**
emit_display is the function called when the keyword 'DISPLAY' is parsed over. The function checks for variable *f. If variable is found it calls function Display with 
parameters .
 Note:Has improper cout statment might be fixed in code fix
 **/
void emit_display(char *f, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(2, "Filter : couldn't find variable " + string(f) );
        };
        stat[f] = statement_count;
        //check_used_vars();
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };

    CudaSet* a = varNames.find(f)->second;
    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };

    //a->Store("",sep, limit, 0, 1);
    a->Display(limit, 0, 1);
    if(verbose)
        cout << "DISPLAY " << f << endl;

    clean_queues();
    if(stat[f] == statement_count  && a->keep == 0) {
        a->free();
        varNames.erase(f);
    };

}


/** 
When the flex scanner encounters the Keyword(token) 'FILTER' it returns FILTER to Bison and the action is preformed. 
The action for FILTER is a call to the Function emit_filter. This function looks over the user command for recognizable variables
names from the data dictionary. If it finds these variables the requested table and column name is filtered.
**/
void emit_filter(char *s, char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(1, "Filter : couldn't find variable " + string(f));
            //cout << "Filter : couldn't find variable " << f << endl;
            //exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        check_used_vars();
        clean_queues();
        return;
    };


    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;

    if(a->mRecCount == 0 && !a->filtered) {
        b = new CudaSet(0,1);
    }
    else {
        if(verbose)
            cout << "INLINE FILTER " << f << endl;
        b = a->copyDeviceStruct();
        b->name = s;
        b->sorted_fields = a->sorted_fields;
        b->presorted_fields = a->presorted_fields;
        //save the stack
        b->fil_s = s;
        b->fil_f = f;
        b->fil_type = op_type;

        b->fil_value = op_value;
        b->fil_nums = op_nums;
        b->fil_nums_f = op_nums_f;
        b->filtered = 1;
		b->tmp_table = a->tmp_table;
        if(a->filtered) {

            b->source_name = a->source_name;
            b->fil_f = a->fil_f;
            while(!a->fil_value.empty()) {
                b->fil_value.push(a->fil_value.front());
                a->fil_value.pop();
            };

            while(!a->fil_type.empty()) {
                b->fil_type.push(a->fil_type.front());
                a->fil_type.pop();
            };
            b->fil_type.push("AND");

            while(!a->fil_nums.empty()) {
                b->fil_nums.push(a->fil_nums.front());
                a->fil_nums.pop();
            };

            while(!a->fil_nums_f.empty()) {
                b->fil_nums_f.push(a->fil_nums_f.front());
                a->fil_nums_f.pop();
            };
            a->filtered = 0;
            //a->free();
            varNames.erase(f);
        }
        else
            b->source_name = f;
        b->maxRecs = a->maxRecs;
        b->prm_d.resize(a->maxRecs);
    };
    clean_queues();


    if (varNames.count(s) > 0)
        varNames[s]->free();
    varNames[s] = b;

    if(stat[s] == statement_count) {
        b->free();
        varNames.erase(s);
    };
}

/**
emit_store is a function that is called when the keyword 'STORE' is parsed over. emit_store makes sure that the variable the user is using is a pre-existing one.
If varaible exists function calls an instance of Store function for Cudaset class object.
**/
void emit_store(char *s, char *f, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end() && data_dict.count(s) == 0) {
            process_error(2, "Store : couldn't find variable " + string(s) );
        };
        stat[s] = statement_count;
        //check_used_vars();
        clean_queues();
        return;
    };

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;
    if(verbose)
        cout << "STORE: " << s << " " << f << " " << sep << endl;

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };

    a->Store(f,sep, limit, 0);

    if(stat[s] == statement_count  && a->keep == 0) {
        a->free();
        varNames.erase(s);
    };
};


/**
emit_store_binary is a function that is called when the keyword 'STORE' and 'Binary' are parsed over in a command. 
emit_store makes sure that the variable the user is using is a pre-existing one. If varaible exists function calls an instance of Store function 
for Cudaset class object.This function then stores data into binary file
**/
void emit_store_binary(char *s, char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end() && data_dict.count(s) == 0) {
            process_error(2, "Store : couldn't find variable " + string(s));
        };
        stat[s] = statement_count;
        //check_used_vars();
        clean_queues();
        return;
    };

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;

    if(stat[f] == statement_count)
        a->deAllocOnDevice();

    printf("STORE: %s %s \n", s, f);

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };
    total_count = 0;
    total_segments = 0;

    if(fact_file_loaded) {
        a->Store(f,"", limit, 1);
    }
    else {
        FILE* file_p;
        if(a->text_source) {
            file_p = fopen(a->load_file_name.c_str(), "r");
            if (file_p  == NULL) {
                process_error(2, "Could not open file " + a->load_file_name );
            };
        };

        while(!fact_file_loaded) {
            if(verbose)
                cout << "LOADING " << a->load_file_name << " mem: " << getFreeMem() << endl;
            if(a->text_source)
                fact_file_loaded = a->LoadBigFile(file_p);
            a->Store(f,"", limit, 1);
        };
    };
    a->writeSortHeader(f);

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(s);
    };

};

/**
emit_load_binary opens the corrsponding header file to namevars queue value. It then creates new Cudaset class object with
apporaite parameters. Function called by load_vars.
**/

void emit_load_binary(const char *s, const char *f, int d)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };

    if(verbose)
        printf("BINARY LOAD: %s \n", s, f);

    CudaSet *a;
    unsigned int segCount, maxRecs;
    string f1(f);
    f1 += "." + namevars.front() + ".header";

    FILE* ff = fopen(f1.c_str(), "rb");
    if(ff == NULL) {
        process_error(2, "Couldn't open file " + f1);
    };
    size_t totRecs;
    fread((char *)&totRecs, 8, 1, ff);
    fread((char *)&segCount, 4, 1, ff);
    fread((char *)&maxRecs, 4, 1, ff);
    fclose(ff);

    if(verbose)
        cout << "Reading " << totRecs << " records" << endl;

    a = new CudaSet(namevars, typevars, sizevars, cols, totRecs, f, maxRecs);
    a->segCount = segCount;
    a->keep = 1;
    a->name = s;
    varNames[s] = a;
    for(unsigned int i = 0; i < segCount; i++)
        a->orig_segs[f].insert(i);

    if(stat[s] == statement_count )  {
        a->free();
        varNames.erase(s);
    };
}


/** 
emit_load function is called when the keyword 'LOAD' is parsed over. Function used ot create a pointer to Cudaset and intialize its variables. 
**/
void emit_load(char *s, char *f, int d, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };

    printf("LOAD: %s %s %d  %s \n", s, f, d, sep);

    CudaSet *a;

    a = new CudaSet(namevars, typevars, sizevars, cols, process_count, references, references_names);
    a->mRecCount = 0;
    a->resize(process_count);
    a->keep = true;
    a->not_compressed = 1;
    a->load_file_name = f;
    a->separator = sep;
    a->maxRecs = a->mRecCount;
    a->segCount = 0;
    varNames[s] = a;
    fact_file_loaded = 0;

    if(stat[s] == statement_count)  {
        a->free();
        varNames.erase(s);
    };
}

/** 
 emit_show_tables() prints the name of every table in data_dict map structure.
**/
void emit_show_tables()
{
    if (scan_state == 1) {
        for ( map<string, map<string, col_data> >::iterator it=data_dict.begin() ; it != data_dict.end(); ++it ) {
            cout << (*it).first << endl;
        };
    };

    return;
}

/**
This function deletes a table name(table_name ) from the data_dict map structure 
**/
void emit_drop_table(char* table_name)
{
    if (scan_state == 1) {

        map<string, map<string, col_data> >::iterator iter;
        if((iter = data_dict.find(table_name)) != data_dict.end()) {
            map<string, col_data> s = (*iter).second;
            for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {
                int seg = 0;
                string f_name = (*iter).first + "." + (*it).first + "." + int_to_string(seg);
                while(!remove(f_name.c_str())) {
                    seg++;
                    f_name = (*iter).first + "." + (*it).first + "." + int_to_string(seg);
                };
                f_name = (*iter).first + "." + (*it).first + ".header";
                remove(f_name.c_str());
            };
        };
        string s_name = (*iter).first + ".presort";
        remove(s_name.c_str());
        s_name = (*iter).first + ".sort";
        remove(s_name.c_str());

        if(data_dict.find(table_name) != data_dict.end()) {
            data_dict.erase(table_name);
        };
        save_dict = 1;
    };

    return;
}


/**
emit_describe looks through the data dictiory map called data_dict to find parameter table_name. Once table_name is found it goes through each column 
and prints to the screen what type of variables each holds.
**/
void emit_describe_table(char* table_name)
{
    if (scan_state == 1) {
        map<string, map<string, col_data> >::iterator iter;
        if((iter = data_dict.find(table_name)) != data_dict.end()) {
            map<string, col_data> s = (*iter).second;
            for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {
                if ((*it).second.col_type == 0) {
                    cout << (*it).first << " integer" << endl;
                }
                else if ((*it).second.col_type == 1) {
                    cout << (*it).first << " float" << endl;
                }
                else if ((*it).second.col_type == 3) {
                    cout << (*it).first << " decimal" << endl;
                }
                else {
                    cout << (*it).first << " char(" << (*it).second.col_length << ")" << endl;
                };
            };
        };
    };

    return;
}



/** 
yyerror handles errors during the Bison parser phase called by function yyparse()
**/  
void yyerror(char *s, ...)
{
    extern int yylineno;
    extern char *yytext;
    //va_list ap;
    //va_start(ap, s);

    fprintf(stderr, "%d: error: ", yylineno);
    cout << yytext << endl;
    //vfprintf(stderr, s, ap);
    //fprintf(stderr, "\n");


}

/** clean_Queues empties all global queue data structures used during program execution
**/
void clean_queues()
{
    while(!op_type.empty()) op_type.pop();
    while(!op_value.empty()) op_value.pop();
    while(!op_join.empty()) op_join.pop();
    while(!op_nums.empty()) op_nums.pop();
    while(!op_nums_f.empty()) op_nums_f.pop();
    while(!j_col_count.empty()) j_col_count.pop();
    while(!namevars.empty()) namevars.pop();
    while(!typevars.empty()) typevars.pop();
    while(!sizevars.empty()) sizevars.pop();
    while(!cols.empty()) cols.pop();
    while(!op_sort.empty()) op_sort.pop();
    while(!references.empty()) references.pop();
    while(!references_names.empty()) references_names.pop();
    while(!op_presort.empty()) op_presort.pop();
	

    op_case = 0;
    sel_count = 0;
    join_cnt = 0;
    join_col_cnt = 0;
    distinct_cnt = 0;
    join_tab_cnt = 0;
    tab_cnt = 0;
    join_and_cnt.clear();
}

/**
 load_vars() is a function that takes variables used in the Alenka commands entered or read from file/user(used_vars)
 and pushes data onto typevars(datatype of column data) and namevars(column name) queues. The function then calls
 emit_load_binary.load_vars() is called from execute_files.
 **/
void load_vars()
{
    if(used_vars.size() == 0) {
        //cout << "Error, no valid column names have been found " << endl;
        //exit(0);
    }
    else {
        for ( map<string, map<string, bool> >::iterator it=used_vars.begin() ; it != used_vars.end(); ++it ) {

            while(!namevars.empty()) namevars.pop();
            while(!typevars.empty()) typevars.pop();
            while(!sizevars.empty()) sizevars.pop();
            while(!cols.empty()) cols.pop();
            if(stat.count((*it).first) != 0) {
                map<string, bool> c = (*it).second;
                for ( map<string, bool>::iterator sit=c.begin() ; sit != c.end(); ++sit ) {
                    //cout << "name " << (*sit).first << endl;
                    namevars.push((*sit).first);
                    if(data_dict[(*it).first][(*sit).first].col_type == 0)
                        typevars.push("int");
                    else if(data_dict[(*it).first][(*sit).first].col_type == 1)
                        typevars.push("float");
                    else if(data_dict[(*it).first][(*sit).first].col_type == 3)
                        typevars.push("decimal");
                    else typevars.push("char");
                    sizevars.push(data_dict[(*it).first][(*sit).first].col_length);
                    cols.push(0);
                };
                emit_load_binary((*it).first.c_str(), (*it).first.c_str(), 0);
            };
        };
    };
}

/** This Function is considered the main controlling because it control most of what alenka does and is the only function called by the main function.
this function has an if statments for the three different modes of alenka. -l lets you adjust the amount of data processed at once when it is sent over to gpu. -v
is verbose mode and it lets you see more information about the queries and data sent over. -i is Ineractive mode, this lets you apply you on quiries using the 
requested language defined in Bison.y instead of reading from file. Function accepts command arguments and number of arguments.
**/
int execute_file(int ac, char **av)
{
	cout << "execute_file->start" << endl;
bool just_once  = 0;
string script;
string data_dir;

    process_count = 6200000;
    verbose = 0;
    total_buffer_size = 0;

    for (int i = 1; i < ac; i++) {
        if(strcmp(av[i],"-l") == 0) {
            process_count = atoff(av[i+1]);
        }
        else if(strcmp(av[i],"-v") == 0) {
            verbose = 1;
        }
        else if(strcmp(av[i],"-i") == 0) {
            interactive = 1;
            break;
        }
        else if(strcmp(av[i],"-s") == 0) {
            just_once = 1;
			script = av[i+1];
        } else if(strcmp(av[i], "-d") == 0) {
        	data_dir = av[i+1];
        	cout << "Using " << data_dir << " as the data directory" << endl;
        };
    };

    load_col_data(data_dict, "data.dictionary");

	/* If not interactive mode it opens file in read mode */	
    if (!interactive) {
        if((yyin = fopen(av[ac-1], "r")) == NULL) {
            perror(av[ac-1]);
            exit(1);
        };

        if(yyparse()) {
            printf("SQL scan parse failed\n");
            exit(1);
        };

        scan_state = 1;
        std::clock_t start1 = std::clock();

        load_vars();

        statement_count = 0;
        clean_queues();

        yyin = fopen(av[ac-1], "r");
        PROC_FLUSH_BUF ( yyin );
        statement_count = 0;

        extern FILE *yyin;
        context = CreateCudaDevice(0, NULL, verbose);
        hash_seed = 100;

        if(!yyparse()) {
            if(verbose)
                cout << "SQL scan parse worked " << endl;
        }
        else
            cout << "SQL scan parse failed" << endl;

        fclose(yyin);
        for (map<string,CudaSet*>::iterator it=varNames.begin() ; it != varNames.end(); ++it ) {
            (*it).second->free();
        };

        if(verbose) {
            cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        };
    }
    else {
        context = CreateCudaDevice(0, NULL, verbose);
        hash_seed = 100;
        if(!just_once)
            getline(cin, script);

        while (script != "exit" && script != "EXIT") {

            used_vars.clear();
            yy_scan_string(script.c_str());
            scan_state = 0;
            statement_count = 0;
            clean_queues();
            if(yyparse()) {
                printf("SQL scan parse failed \n");
                getline(cin, script);
                continue;
            };

            scan_state = 1;

            load_vars();

            statement_count = 0;
            clean_queues();
            yy_scan_string(script.c_str());
            std::clock_t start1 = std::clock();

            if(!yyparse()) {
                if(verbose)
                    cout << "SQL scan parse worked " <<  endl;
            };
            for (map<string,CudaSet*>::iterator it=varNames.begin() ; it != varNames.end(); ++it ) {
                (*it).second->free();
            };
            varNames.clear();

            if(verbose) {
                cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
            };
            if(!just_once)
                getline(cin, script);
            else
                script = "exit";
        };

        while(!buffer_names.empty()) {
            delete [] buffers[buffer_names.front()];
            buffer_sizes.erase(buffer_names.front());
            buffers.erase(buffer_names.front());
            buffer_names.pop();
        };

    };
    if(save_dict)
        save_col_data(data_dict,"data.dictionary");

    if(alloced_sz) {
        hipFree(alloced_tmp);
        alloced_sz = 0;
    };
    if(raw_decomp_length) {
        hipFree(raw_decomp);
        raw_decomp_length = 0;
    };

    return 0;
}



//external c global to report errors
char alenka_err[4048];
/** 
 One of the lib functions written by Randoplh 
 Main lib function controls program executation calls function yyparse to parse through valid Alenka commands.
 No lib function is ran during the normal flow of Alenka these are add ons
**/

int alenkaExecute(char *s)
{
    YY_BUFFER_STATE bp;

    total_buffer_size = 0;
    scan_state = 0;
    load_col_data(data_dict, "data.dictionary");
    std::clock_t start;

    if(verbose)
        start = std::clock();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    int ret = yyparse();
    //printf("execute: returned [%d]\n", ret);
    if(!ret) {
        if(verbose)
            cout << "SQL scan parse worked" << endl;
    }

    scan_state = 1;
    load_vars();
    statement_count = 0;
    clean_queues();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    if(!yyparse()) {
        if(verbose)
            cout << "SQL scan parse worked " << endl;
    }
    else
        cout << "SQL scan parse failed" << endl;

    yy_delete_buffer(bp);

    // Clear Vars
    for (map<string,CudaSet*>::iterator it=varNames.begin() ; it != varNames.end(); ++it ) {
        (*it).second->free();
    };
    varNames.clear();

    if(verbose)
        cout<< "statement time " <<  ( ( std::clock() - start ) / (double)CLOCKS_PER_SEC ) << endl;
    if(save_dict)
        save_col_data(data_dict,"data.dictionary");
    return ret;
}




void process_error(int severity, string err) {
    switch (severity) {
    case 1:
        err = "(Warning) " + err;
        break;
    case 2:
        err = "(Fatal) " + err;
        break;
    default:
        err = "(Aborting) " + err;
        break;
    }
    error_cb(severity, err.c_str());            // send the error to the c based callback
}

void alenkaInit(char ** av)
{
    process_count = 6200000;
    verbose = 0;
    scan_state = 1;
    statement_count = 0;
    clean_queues();
    context = CreateCudaDevice(0, NULL, true);
    printf("Alenka initialised\n");
}

/** 
Frees up memory on the device that Alenka was useses 
No lib function is ran during the normal flow of Alenka these are add ons
**/
void alenkaClose()
{
    statement_count = 0;
    hash_seed = 100;

    if(alloced_sz)
        hipFree(alloced_tmp);
}

/*
extern "C"
int alenka_JDBC(char *s) {
		//turn on debugging
		verbose = 1;

		YY_BUFFER_STATE bp;

        //Load the data dictionary
        load_col_data(data_dict, "data.dictionary");

        //initialize variables
        hash_seed = 100;
        scan_state = 0;
        statement_count = 0;
        clean_queues();

        //Read the SQL into internal buffers
        bp = yy_scan_string(s);
        yy_switch_to_buffer(bp);

        //Parse the query
        if (yyparse()) {
                cout << "SQL scan parse failed" << endl;
                return 1;
        };

        //Reset variables for execution
        scan_state = 1;
        load_vars();
        statement_count = 0;
        clean_queues();

        bp = yy_scan_string(s);
        yy_switch_to_buffer(bp);
        std::clock_t start1 = std::clock();

        //Excecute Query
        if (yyparse()) {
                cout << "SQL scan parse failed" << endl;
                return 1;
        }

        yy_delete_buffer(bp);

        if (verbose) {
                cout << "cycle time "
                                << ((std::clock() - start1) / (double) CLOCKS_PER_SEC) << endl;
        };

        return 0;
}
*/
extern "C"
int alenka_JDBC(char *s) {
	verbose = 1;
        //Load the data dictionary
        load_col_data(data_dict, "data.dictionary");

        //Read the SQL into internal buffers
        yy_scan_string(s);

        //initialize variables
        hash_seed = 100;
        scan_state = 0;
        statement_count = 0;
        clean_queues();

        //Parse the query
        cout << "first yyparse()" << endl;
        if (yyparse()) {
                printf("SQL scan parse failed \n");
                return 1;
        };

        //Reset variables for execution
        cout << "reset vars and clean_queues" << endl;
        scan_state = 1;
        load_vars();
        statement_count = 0;
        clean_queues();

        yy_scan_string(s);
        std::clock_t start1 = std::clock();

        //Excecute Query
        cout << "second yyparse" << endl;
        if (yyparse()) {
                printf("SQL scan parse failed \n");
                return 1;
        }

        cout << "done" << endl;
        if (verbose) {
                cout << "cycle time "
                                << ((std::clock() - start1) / (double) CLOCKS_PER_SEC) << endl;
        };

        return 0;
}











