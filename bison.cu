#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 17 "bison.y"



#include "lex.yy.c"
#include "cm.h"
#include <iomanip>

    void clean_queues();
    void order_inplace(CudaSet* a, stack<string> exe_type, bool update_int);
    void yyerror(char *s, ...);
    void emit(char *s, ...);
    void emit_mul();
    void emit_add();
    void emit_minus();
    void emit_distinct();
    void emit_div();
    void emit_and();
    void emit_eq();
    void emit_or();
    void emit_cmp(int val);
    void emit_var(char *s, int c, char *f, char* ref, char* ref_name);
	void emit_var_asc(char *s);
    void emit_var_desc(char *s);
    void emit_name(char *name);
    void emit_count();
    void emit_sum();
    void emit_average();
    void emit_min();
    void emit_max();
    void emit_string(char *str);
    void emit_number(int_type val);
    void emit_float(float_type val);
    void emit_decimal(float_type val);
    void emit_sel_name(char* name);
    void emit_limit(int val);
    void emit_union(char *s, char *f1, char *f2);
    void emit_varchar(char *s, int c, char *f, int d, char *ref, char* ref_name);	
    void emit_load(char *s, char *f, int d, char* sep);
    void emit_load_binary(const char *s, const char *f, int d);
    void emit_store(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f);
    void emit_filter(char *s, char *f);
	void emit_delete(char *f);
	void emit_insert(char *f, char* s);
    void emit_order(char *s, char *f, int e, int ll = 0);
    void emit_group(char *s, char *f, int e);
    void emit_select(char *s, char *f, int ll);
    void emit_join(char *s, char *j1, int grp);
    void emit_join_tab(char *s, char tp);
    void emit_distinct();
    void emit_join();
    void emit_sort(char* s, int p);
    void emit_presort(char* s);
	void emit_display(char *s, char* sep);
	void emit_case();
	void emit_show_tables();
	void emit_describe_table(char* table_name);
	void emit_drop_table(char* table_name);
	void process_error(int severity, string err);



/* Line 189 of yacc.c  */
#line 137 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 1
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     OR = 268,
     XOR = 269,
     AND = 270,
     DISTINCT = 271,
     REGEXP = 272,
     LIKE = 273,
     IS = 274,
     IN = 275,
     NOT = 276,
     BETWEEN = 277,
     COMPARISON = 278,
     SHIFT = 279,
     MOD = 280,
     UMINUS = 281,
     LOAD = 282,
     STREAM = 283,
     FILTER = 284,
     BY = 285,
     JOIN = 286,
     STORE = 287,
     INTO = 288,
     GROUP = 289,
     FROM = 290,
     SELECT = 291,
     AS = 292,
     ORDER = 293,
     ASC = 294,
     DESC = 295,
     COUNT = 296,
     USING = 297,
     SUM = 298,
     AVG = 299,
     MIN = 300,
     MAX = 301,
     LIMIT = 302,
     ON = 303,
     BINARY = 304,
     LEFT = 305,
     RIGHT = 306,
     OUTER = 307,
     SORT = 308,
     SEGMENTS = 309,
     PRESORTED = 310,
     PARTITION = 311,
     DELETE = 312,
     INSERT = 313,
     WHERE = 314,
     DISPLAY = 315,
     CASE = 316,
     WHEN = 317,
     THEN = 318,
     ELSE = 319,
     END = 320,
     REFERENCES = 321,
     SHOW = 322,
     TABLES = 323,
     TABLE = 324,
     DESCRIBE = 325,
     DROP = 326
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 80 "bison.y"

    long long int intval;
    double floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 253 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 265 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  20
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   624

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  89
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  15
/* YYNRULES -- Number of rules.  */
#define YYNRULES  84
/* YYNRULES -- Number of states.  */
#define YYNSTATES  227

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   326

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    21,     2,     2,     2,    32,    26,     2,
      82,    83,    30,    28,    88,    29,    84,    31,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    87,    81,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    34,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    85,    25,    86,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    22,    23,    24,    27,
      33,    35,    36,    37,    38,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59,    60,    61,    62,    63,
      64,    65,    66,    67,    68,    69,    70,    71,    72,    73,
      74,    75,    76,    77,    78,    79,    80
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    39,    46,
      55,    65,    73,    78,    86,    94,    97,   100,   104,   106,
     110,   112,   114,   116,   118,   120,   122,   137,   147,   159,
     166,   169,   172,   177,   182,   187,   192,   197,   200,   202,
     206,   210,   214,   218,   222,   226,   230,   234,   238,   242,
     246,   249,   252,   256,   262,   266,   275,   279,   284,   285,
     289,   293,   299,   301,   303,   307,   309,   313,   314,   316,
     319,   322,   327,   333,   339,   345,   351,   358,   365,   372,
     373,   376,   377,   382,   390
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      90,     0,    -1,    91,    81,    -1,    90,    91,    81,    -1,
      92,    -1,     4,    11,    45,    95,    44,     4,    94,    -1,
       4,    11,    36,     3,    51,    82,     3,    83,    46,    82,
      96,    83,    -1,     4,    11,    38,     4,    99,    -1,     4,
      11,    47,     4,    39,    98,    -1,     4,    11,    45,    95,
      44,     4,   101,    94,    -1,    41,     4,    42,     3,    51,
      82,     3,    83,   102,    -1,    41,     4,    42,     3,   102,
      58,   103,    -1,    66,    44,     4,   100,    -1,    67,    42,
       4,    45,    95,    44,     4,    -1,    69,     4,    51,    82,
       3,    83,   102,    -1,    76,    77,    -1,    79,     4,    -1,
      80,    78,     4,    -1,     4,    -1,     4,    84,     4,    -1,
      10,    -1,     5,    -1,     6,    -1,     9,    -1,     7,    -1,
       8,    -1,     4,    85,     6,    86,    87,     4,    82,     6,
      83,    75,     4,    82,     4,    83,    -1,     4,    85,     6,
      86,    87,     4,    82,     6,    83,    -1,     4,    85,     6,
      86,    87,     4,    75,     4,    82,     4,    83,    -1,     4,
      85,     6,    86,    87,     4,    -1,     4,    48,    -1,     4,
      49,    -1,    50,    82,    93,    83,    -1,    52,    82,    93,
      83,    -1,    53,    82,    93,    83,    -1,    54,    82,    93,
      83,    -1,    55,    82,    93,    83,    -1,    16,    93,    -1,
      40,    -1,    93,    28,    93,    -1,    93,    29,    93,    -1,
      93,    30,    93,    -1,    93,    31,    93,    -1,    93,    32,
      93,    -1,    93,    33,    93,    -1,    93,    15,    93,    -1,
      93,    12,    93,    -1,    93,    13,    93,    -1,    93,    14,
      93,    -1,    93,    27,    93,    -1,    22,    93,    -1,    21,
      93,    -1,    93,    24,    93,    -1,    93,    24,    82,    92,
      83,    -1,    82,    93,    83,    -1,    70,    71,    93,    72,
      93,    73,    93,    74,    -1,    93,    19,     8,    -1,    93,
      19,    22,     8,    -1,    -1,    43,    39,    97,    -1,    93,
      46,     4,    -1,    95,    88,    93,    46,     4,    -1,    30,
      -1,    93,    -1,    96,    88,    93,    -1,    93,    -1,    93,
      88,    97,    -1,    -1,    97,    -1,    39,    93,    -1,    68,
      93,    -1,    40,     4,    57,    93,    -1,    59,    40,     4,
      57,    93,    -1,    60,    40,     4,    57,    93,    -1,    61,
      40,     4,    57,    93,    -1,    40,     4,    57,    93,   101,
      -1,    59,    40,     4,    57,    93,   101,    -1,    60,    40,
       4,    57,    93,   101,    -1,    61,    40,     4,    57,    93,
     101,    -1,    -1,    56,     6,    -1,    -1,    62,    63,    39,
       4,    -1,    62,    63,    39,     4,    65,    39,     6,    -1,
      64,    39,     4,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint16 yyrline[] =
{
       0,   173,   173,   174,   178,   181,   183,   185,   187,   189,
     191,   193,   195,   197,   199,   201,   203,   205,   211,   212,
     213,   214,   215,   216,   217,   218,   219,   220,   221,   222,
     223,   224,   225,   226,   227,   228,   229,   230,   231,   235,
     236,   237,   238,   239,   240,   242,   243,   244,   245,   246,
     247,   248,   249,   251,   252,   253,   257,   258,   263,   266,
     270,   271,   272,   276,   277,   281,   282,   285,   287,   290,
     293,   297,   298,   299,   300,   301,   302,   303,   304,   306,
     309,   311,   314,   315,   316
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL", "OR",
  "XOR", "AND", "DISTINCT", "REGEXP", "LIKE", "IS", "IN", "'!'", "NOT",
  "BETWEEN", "COMPARISON", "'|'", "'&'", "SHIFT", "'+'", "'-'", "'*'",
  "'/'", "'%'", "MOD", "'^'", "UMINUS", "LOAD", "STREAM", "FILTER", "BY",
  "JOIN", "STORE", "INTO", "GROUP", "FROM", "SELECT", "AS", "ORDER", "ASC",
  "DESC", "COUNT", "USING", "SUM", "AVG", "MIN", "MAX", "LIMIT", "ON",
  "BINARY", "LEFT", "RIGHT", "OUTER", "SORT", "SEGMENTS", "PRESORTED",
  "PARTITION", "DELETE", "INSERT", "WHERE", "DISPLAY", "CASE", "WHEN",
  "THEN", "ELSE", "END", "REFERENCES", "SHOW", "TABLES", "TABLE",
  "DESCRIBE", "DROP", "';'", "'('", "')'", "'.'", "'{'", "'}'", "':'",
  "','", "$accept", "stmt_list", "stmt", "select_stmt", "expr",
  "opt_group_list", "expr_list", "load_list", "val_list", "opt_val_list",
  "opt_where", "del_where", "join_list", "opt_limit", "sort_def", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,    33,   276,   277,   278,   124,    38,   279,    43,    45,
      42,    47,    37,   280,    94,   281,   282,   283,   284,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
     306,   307,   308,   309,   310,   311,   312,   313,   314,   315,
     316,   317,   318,   319,   320,   321,   322,   323,   324,   325,
     326,    59,    40,    41,    46,   123,   125,    58,    44
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    89,    90,    90,    91,    92,    92,    92,    92,    92,
      92,    92,    92,    92,    92,    92,    92,    92,    93,    93,
      93,    93,    93,    93,    93,    93,    93,    93,    93,    93,
      93,    93,    93,    93,    93,    93,    93,    93,    93,    93,
      93,    93,    93,    93,    93,    93,    93,    93,    93,    93,
      93,    93,    93,    93,    93,    93,    93,    93,    94,    94,
      95,    95,    95,    96,    96,    97,    97,    98,    98,    99,
     100,   101,   101,   101,   101,   101,   101,   101,   101,   102,
     102,   103,   103,   103,   103
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     5,     6,     8,
       9,     7,     4,     7,     7,     2,     2,     3,     1,     3,
       1,     1,     1,     1,     1,     1,    14,     9,    11,     6,
       2,     2,     4,     4,     4,     4,     4,     2,     1,     3,
       3,     3,     3,     3,     3,     3,     3,     3,     3,     3,
       2,     2,     3,     5,     3,     8,     3,     4,     0,     3,
       3,     5,     1,     1,     3,     1,     3,     0,     1,     2,
       2,     4,     5,     5,     5,     5,     6,     6,     6,     0,
       2,     0,     4,     7,     3
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     4,     0,     0,     0,     0,     0,    15,    16,     0,
       1,     0,     2,     0,     0,     0,     0,     0,     0,     0,
       0,    17,     3,     0,     0,    18,    21,    22,    24,    25,
      23,    20,     0,     0,     0,    62,    38,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,    79,     0,    12,
       0,     0,     0,     0,     7,    30,    31,     0,     0,    37,
      51,    50,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,    67,     0,     0,     0,    70,
       0,     0,     0,    69,    19,     0,     0,     0,     0,     0,
       0,     0,    54,    46,    47,    48,    45,    56,     0,     0,
      52,    49,    39,    40,    41,    42,    43,    44,    60,    58,
       0,    65,    68,     8,     0,    80,    81,     0,    79,     0,
       0,    32,    33,    34,    35,    36,     0,    57,    18,     0,
       0,     0,     0,     0,     0,     5,    58,     0,     0,     0,
       0,     0,    11,    13,    14,     0,     0,     0,    53,     0,
       0,     0,     0,     0,     9,    61,    66,    79,     0,     0,
       0,    29,     0,     0,    59,     0,     0,     0,    10,     0,
      84,     0,     0,     0,     0,    71,     0,     0,     0,    82,
      63,     0,     0,     0,    55,    75,    72,    73,    74,     0,
       6,     0,     0,    27,    76,    77,    78,     0,    64,     0,
       0,    83,    28,     0,     0,     0,    26
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,     9,    10,    11,   131,   155,    55,   201,   132,   133,
      64,    59,   156,    98,   162
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -151
static const yytype_int16 yypact[] =
{
     128,     2,     8,   -18,   -15,    26,   -43,    37,   -34,     1,
     -36,  -151,   153,     4,    70,    74,    28,  -151,  -151,    81,
    -151,     5,  -151,    92,    94,   171,    96,    99,    35,    62,
      38,  -151,  -151,    57,    87,   -12,  -151,  -151,  -151,  -151,
    -151,  -151,   223,   223,   223,  -151,  -151,    40,    43,    49,
      58,    59,    80,   223,   526,   -35,   116,   -41,   223,  -151,
     171,   151,    78,   223,  -151,  -151,  -151,   152,   155,   186,
     585,   585,   223,   223,   223,   223,   223,   223,   237,   223,
     223,   223,   223,    -1,   275,   223,   223,   223,   223,   223,
     223,   223,   160,   163,   223,   223,    88,   162,   124,   575,
     -30,    91,   180,   575,  -151,    98,   319,   341,   363,   385,
     407,   480,  -151,   575,    82,   271,   186,  -151,   177,    83,
     279,   591,    86,    86,  -151,  -151,  -151,  -151,  -151,   -37,
     552,   115,  -151,  -151,   185,  -151,   -33,   192,   134,   119,
     112,  -151,  -151,  -151,  -151,  -151,   223,  -151,    -9,   123,
     205,   173,   182,   194,   195,  -151,   193,   216,   223,   154,
     175,   201,  -151,  -151,  -151,   196,   239,   451,  -151,   189,
     223,   243,   244,   250,  -151,  -151,  -151,   134,   218,   251,
     176,   -50,   223,   223,  -151,   202,   203,   214,  -151,   258,
    -151,   223,   268,   281,   429,   502,   223,   223,   223,   209,
     575,   -72,   206,   211,  -151,  -151,   502,   502,   502,   253,
    -151,   223,   285,   238,  -151,  -151,  -151,   308,   575,   233,
     313,  -151,  -151,   236,   315,   240,  -151
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -151,  -151,   312,   207,   -25,   166,   264,  -151,  -150,  -151,
    -151,  -151,  -124,  -134,  -151
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_uint8 yytable[] =
{
      54,    20,    12,   150,   164,     1,   151,   117,   176,    93,
      96,   210,    13,    12,   137,    97,   211,    69,    70,    71,
     184,   118,   152,   153,   154,   192,    14,    15,    78,   160,
      16,   161,   193,    99,    17,    54,    65,    66,   103,    65,
      66,    18,     2,   188,    19,    22,    27,   106,   107,   108,
     109,   110,   111,    94,   113,   114,   115,   116,    94,   120,
     121,   122,   123,   124,   125,   126,   127,     3,     4,   130,
       5,   205,    67,    68,    28,    67,    68,     6,    29,    30,
       7,     8,   214,   215,   216,    31,    32,   148,    36,    37,
      38,    39,    40,    41,    78,    33,    81,    82,    34,    42,
      56,    83,    57,    58,    43,    44,    84,    60,    62,    85,
      86,    87,    88,    89,    90,    91,    88,    89,    90,    91,
      61,   167,    72,    46,     2,    73,    63,    79,    80,    81,
      82,    74,     1,    47,    83,    48,    49,    50,    51,    84,
      75,    76,    85,    86,    87,    88,    89,    90,    91,     3,
       4,    77,     5,    52,   101,    95,   104,   194,   195,     6,
     102,   105,     7,     8,   128,    53,   200,   129,   135,     2,
     134,   206,   207,   208,   138,    35,    36,    37,    38,    39,
      40,    41,   136,   139,   140,   147,   218,    42,   159,    23,
      97,    24,    43,    44,     3,     4,   163,     5,    25,   166,
      26,    45,   165,   158,     6,    83,   168,     7,     8,   169,
      84,    46,   170,    85,    86,    87,    88,    89,    90,    91,
     175,    47,   171,    48,    49,    50,    51,    35,    36,    37,
      38,    39,    40,    41,   172,   173,   151,   177,   178,    42,
     179,    52,   180,   181,    43,    44,   183,   185,   186,    79,
      80,    81,    82,    53,   187,   190,    83,   189,   191,   196,
     197,    84,   199,    46,    85,    86,    87,    88,    89,    90,
      91,   198,   202,    47,   209,    48,    49,    50,    51,    35,
      36,    37,    38,    39,    40,    41,    82,   203,   212,   219,
      83,    42,   217,    52,   213,    84,    43,    44,    85,    86,
      87,    88,    89,    90,    91,    53,    85,    86,    87,    88,
      89,    90,    91,   220,   221,    46,   222,   223,   224,   225,
     112,    21,   174,   226,   100,    47,   149,    48,    49,    50,
      51,    79,    80,    81,    82,     0,     0,     0,    83,     0,
       0,     0,     0,    84,     0,    52,    85,    86,    87,    88,
      89,    90,    91,    79,    80,    81,    82,   119,     0,     0,
      83,     0,     0,     0,     0,    84,     0,     0,    85,    86,
      87,    88,    89,    90,    91,    79,    80,    81,    82,     0,
       0,     0,    83,     0,     0,     0,     0,    84,     0,     0,
      85,    86,    87,    88,    89,    90,    91,    79,    80,    81,
      82,     0,   141,     0,    83,     0,     0,     0,     0,    84,
       0,     0,    85,    86,    87,    88,    89,    90,    91,    79,
      80,    81,    82,     0,   142,     0,    83,     0,     0,     0,
       0,    84,     0,     0,    85,    86,    87,    88,    89,    90,
      91,    79,    80,    81,    82,     0,   143,     0,    83,     0,
       0,     0,     0,    84,     0,     0,    85,    86,    87,    88,
      89,    90,    91,    79,    80,    81,    82,     0,   144,     0,
      83,     0,     0,     0,     0,    84,     0,     0,    85,    86,
      87,    88,    89,    90,    91,     0,     0,     0,     0,     0,
     145,     0,    79,    80,    81,    82,     0,     0,     0,    83,
       0,     0,     0,   204,    84,     0,     0,    85,    86,    87,
      88,    89,    90,    91,    79,    80,    81,    82,     0,     0,
       0,    83,     0,     0,   182,     0,    84,     0,     0,    85,
      86,    87,    88,    89,    90,    91,     0,     0,    79,    80,
      81,    82,   150,     0,     0,    83,     0,     0,     0,     0,
      84,     0,   146,    85,    86,    87,    88,    89,    90,    91,
       0,   152,   153,   154,    79,    80,    81,    82,     0,     0,
       0,    83,    92,     0,     0,     0,    84,     0,     0,    85,
      86,    87,    88,    89,    90,    91,     0,    79,    80,    81,
      82,     0,     0,     0,    83,     0,     0,     0,   157,    84,
       0,     0,    85,    86,    87,    88,    89,    90,    91,    84,
       0,     0,    85,    86,    87,    88,    89,    90,    91,    86,
      87,    88,    89,    90,    91
};

static const yytype_int16 yycheck[] =
{
      25,     0,    11,    40,   138,     4,    43,     8,   158,    44,
      51,    83,     4,    11,    44,    56,    88,    42,    43,    44,
     170,    22,    59,    60,    61,    75,    44,    42,    53,    62,
       4,    64,    82,    58,    77,    60,    48,    49,    63,    48,
      49,     4,    41,   177,    78,    81,    42,    72,    73,    74,
      75,    76,    77,    88,    79,    80,    81,    82,    88,    84,
      85,    86,    87,    88,    89,    90,    91,    66,    67,    94,
      69,   195,    84,    85,     4,    84,    85,    76,     4,    51,
      79,    80,   206,   207,   208,     4,    81,     4,     5,     6,
       7,     8,     9,    10,   119,     3,    14,    15,     4,    16,
       4,    19,     3,    68,    21,    22,    24,    45,    51,    27,
      28,    29,    30,    31,    32,    33,    30,    31,    32,    33,
      82,   146,    82,    40,    41,    82,    39,    12,    13,    14,
      15,    82,     4,    50,    19,    52,    53,    54,    55,    24,
      82,    82,    27,    28,    29,    30,    31,    32,    33,    66,
      67,    71,    69,    70,     3,    39,     4,   182,   183,    76,
      82,     6,    79,    80,     4,    82,   191,     4,     6,    41,
      82,   196,   197,   198,    83,     4,     5,     6,     7,     8,
       9,    10,    58,     3,    86,     8,   211,    16,     3,    36,
      56,    38,    21,    22,    66,    67,     4,    69,    45,    87,
      47,    30,    83,    88,    76,    19,    83,    79,    80,     4,
      24,    40,    39,    27,    28,    29,    30,    31,    32,    33,
       4,    50,    40,    52,    53,    54,    55,     4,     5,     6,
       7,     8,     9,    10,    40,    40,    43,    83,    63,    16,
      39,    70,    46,     4,    21,    22,    57,     4,     4,    12,
      13,    14,    15,    82,     4,     4,    19,    39,    82,    57,
      57,    24,     4,    40,    27,    28,    29,    30,    31,    32,
      33,    57,     4,    50,    65,    52,    53,    54,    55,     4,
       5,     6,     7,     8,     9,    10,    15,     6,    82,     4,
      19,    16,    39,    70,    83,    24,    21,    22,    27,    28,
      29,    30,    31,    32,    33,    82,    27,    28,    29,    30,
      31,    32,    33,    75,     6,    40,    83,     4,    82,     4,
      83,     9,   156,    83,    60,    50,   119,    52,    53,    54,
      55,    12,    13,    14,    15,    -1,    -1,    -1,    19,    -1,
      -1,    -1,    -1,    24,    -1,    70,    27,    28,    29,    30,
      31,    32,    33,    12,    13,    14,    15,    82,    -1,    -1,
      19,    -1,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    12,    13,    14,    15,    -1,
      -1,    -1,    19,    -1,    -1,    -1,    -1,    24,    -1,    -1,
      27,    28,    29,    30,    31,    32,    33,    12,    13,    14,
      15,    -1,    83,    -1,    19,    -1,    -1,    -1,    -1,    24,
      -1,    -1,    27,    28,    29,    30,    31,    32,    33,    12,
      13,    14,    15,    -1,    83,    -1,    19,    -1,    -1,    -1,
      -1,    24,    -1,    -1,    27,    28,    29,    30,    31,    32,
      33,    12,    13,    14,    15,    -1,    83,    -1,    19,    -1,
      -1,    -1,    -1,    24,    -1,    -1,    27,    28,    29,    30,
      31,    32,    33,    12,    13,    14,    15,    -1,    83,    -1,
      19,    -1,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    -1,    -1,    -1,    -1,    -1,
      83,    -1,    12,    13,    14,    15,    -1,    -1,    -1,    19,
      -1,    -1,    -1,    74,    24,    -1,    -1,    27,    28,    29,
      30,    31,    32,    33,    12,    13,    14,    15,    -1,    -1,
      -1,    19,    -1,    -1,    73,    -1,    24,    -1,    -1,    27,
      28,    29,    30,    31,    32,    33,    -1,    -1,    12,    13,
      14,    15,    40,    -1,    -1,    19,    -1,    -1,    -1,    -1,
      24,    -1,    72,    27,    28,    29,    30,    31,    32,    33,
      -1,    59,    60,    61,    12,    13,    14,    15,    -1,    -1,
      -1,    19,    46,    -1,    -1,    -1,    24,    -1,    -1,    27,
      28,    29,    30,    31,    32,    33,    -1,    12,    13,    14,
      15,    -1,    -1,    -1,    19,    -1,    -1,    -1,    46,    24,
      -1,    -1,    27,    28,    29,    30,    31,    32,    33,    24,
      -1,    -1,    27,    28,    29,    30,    31,    32,    33,    28,
      29,    30,    31,    32,    33
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    41,    66,    67,    69,    76,    79,    80,    90,
      91,    92,    11,     4,    44,    42,     4,    77,     4,    78,
       0,    91,    81,    36,    38,    45,    47,    42,     4,     4,
      51,     4,    81,     3,     4,     4,     5,     6,     7,     8,
       9,    10,    16,    21,    22,    30,    40,    50,    52,    53,
      54,    55,    70,    82,    93,    95,     4,     3,    68,   100,
      45,    82,    51,    39,    99,    48,    49,    84,    85,    93,
      93,    93,    82,    82,    82,    82,    82,    71,    93,    12,
      13,    14,    15,    19,    24,    27,    28,    29,    30,    31,
      32,    33,    46,    44,    88,    39,    51,    56,   102,    93,
      95,     3,    82,    93,     4,     6,    93,    93,    93,    93,
      93,    93,    83,    93,    93,    93,    93,     8,    22,    82,
      93,    93,    93,    93,    93,    93,    93,    93,     4,     4,
      93,    93,    97,    98,    82,     6,    58,    44,    83,     3,
      86,    83,    83,    83,    83,    83,    72,     8,     4,    92,
      40,    43,    59,    60,    61,    94,   101,    46,    88,     3,
      62,    64,   103,     4,   102,    83,    87,    93,    83,     4,
      39,    40,    40,    40,    94,     4,    97,    83,    63,    39,
      46,     4,    73,    57,    97,     4,     4,     4,   102,    39,
       4,    82,    75,    82,    93,    93,    57,    57,    57,     4,
      93,    96,     4,     6,    74,   101,    93,    93,    93,    65,
      83,    88,    82,    83,   101,   101,   101,    39,    93,     4,
      75,     6,    83,     4,    82,     4,    83
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 178 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 182 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 186 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval));;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 188 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 190 "bison.y"
    { emit_join((yyvsp[(1) - (8)].strval),(yyvsp[(6) - (8)].strval),(yyvsp[(7) - (8)].intval)); ;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 192 "bison.y"
    { emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 194 "bison.y"
    { emit_store_binary((yyvsp[(2) - (7)].strval),(yyvsp[(4) - (7)].strval)); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 196 "bison.y"
    {  emit_delete((yyvsp[(3) - (4)].strval));;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 198 "bison.y"
    {  emit_insert((yyvsp[(3) - (7)].strval), (yyvsp[(7) - (7)].strval));;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 200 "bison.y"
    {  emit_display((yyvsp[(2) - (7)].strval), (yyvsp[(5) - (7)].strval));;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 202 "bison.y"
    {  emit_show_tables();;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 204 "bison.y"
    {  emit_describe_table((yyvsp[(2) - (2)].strval));;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 206 "bison.y"
    {  emit_drop_table((yyvsp[(3) - (3)].strval));;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 211 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 212 "bison.y"
    { emit("FIELDNAME %s.%s", (yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 213 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 214 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 215 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 216 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 217 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 218 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 219 "bison.y"
    { emit_varchar((yyvsp[(1) - (14)].strval), (yyvsp[(3) - (14)].intval), (yyvsp[(6) - (14)].strval), (yyvsp[(8) - (14)].intval), (yyvsp[(11) - (14)].strval), (yyvsp[(13) - (14)].strval));;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 220 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval), "", "");;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 221 "bison.y"
    { emit_var((yyvsp[(1) - (11)].strval), (yyvsp[(3) - (11)].intval), (yyvsp[(6) - (11)].strval), (yyvsp[(8) - (11)].strval), (yyvsp[(10) - (11)].strval));;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 222 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval), "", "");;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 223 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 224 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 225 "bison.y"
    { emit_count(); ;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 226 "bison.y"
    { emit_sum(); ;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 227 "bison.y"
    { emit_average(); ;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 228 "bison.y"
    { emit_min(); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 229 "bison.y"
    { emit_max(); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 230 "bison.y"
    { emit_distinct(); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 231 "bison.y"
    { emit_join(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 235 "bison.y"
    { emit_add(); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 236 "bison.y"
    { emit_minus(); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 237 "bison.y"
    { emit_mul(); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 238 "bison.y"
    { emit_div(); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 239 "bison.y"
    { emit("MOD"); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 240 "bison.y"
    { emit("MOD"); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 242 "bison.y"
    { emit_and(); ;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 243 "bison.y"
    { emit_eq(); ;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 244 "bison.y"
    { emit_or(); ;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 245 "bison.y"
    { emit("XOR"); ;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 246 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 247 "bison.y"
    { emit("NOT"); ;}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 248 "bison.y"
    { emit("NOT"); ;}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 249 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 251 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 252 "bison.y"
    {emit("EXPR");;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 253 "bison.y"
    { emit_case(); ;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 257 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 57:

/* Line 1455 of yacc.c  */
#line 258 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 58:

/* Line 1455 of yacc.c  */
#line 263 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 59:

/* Line 1455 of yacc.c  */
#line 266 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 270 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 271 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 272 "bison.y"
    { emit_sel_name("*");;}
    break;

  case 63:

/* Line 1455 of yacc.c  */
#line 276 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 64:

/* Line 1455 of yacc.c  */
#line 277 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 65:

/* Line 1455 of yacc.c  */
#line 281 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 66:

/* Line 1455 of yacc.c  */
#line 282 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 67:

/* Line 1455 of yacc.c  */
#line 285 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 69:

/* Line 1455 of yacc.c  */
#line 290 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 70:

/* Line 1455 of yacc.c  */
#line 293 "bison.y"
    { emit("DELETE"); ;}
    break;

  case 71:

/* Line 1455 of yacc.c  */
#line 297 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval), 'I');;}
    break;

  case 72:

/* Line 1455 of yacc.c  */
#line 298 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'L');;}
    break;

  case 73:

/* Line 1455 of yacc.c  */
#line 299 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'R');;}
    break;

  case 74:

/* Line 1455 of yacc.c  */
#line 300 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'O');;}
    break;

  case 75:

/* Line 1455 of yacc.c  */
#line 301 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval), 'I'); ;}
    break;

  case 76:

/* Line 1455 of yacc.c  */
#line 302 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'L'); ;}
    break;

  case 77:

/* Line 1455 of yacc.c  */
#line 303 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'R'); ;}
    break;

  case 78:

/* Line 1455 of yacc.c  */
#line 304 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'O'); ;}
    break;

  case 79:

/* Line 1455 of yacc.c  */
#line 306 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 80:

/* Line 1455 of yacc.c  */
#line 309 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;

  case 81:

/* Line 1455 of yacc.c  */
#line 311 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 82:

/* Line 1455 of yacc.c  */
#line 314 "bison.y"
    { emit_sort((yyvsp[(4) - (4)].strval), 0); ;}
    break;

  case 83:

/* Line 1455 of yacc.c  */
#line 315 "bison.y"
    { emit_sort((yyvsp[(4) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 84:

/* Line 1455 of yacc.c  */
#line 316 "bison.y"
    { emit_presort((yyvsp[(3) - (3)].strval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2300 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 318 "bison.y"


#include "filter.h"
#include "select.h"
#include "merge.h"
#include "zone_map.h"
#include "atof.h"
#include "sstream"
#include "sorts.cu"
#include "callbacks.h"

using namespace mgpu;
using namespace thrust::placeholders;
using namespace std;

size_t int_size = sizeof(int_type);
size_t float_size = sizeof(float_type);

queue<string> namevars;
queue<string> typevars;
queue<int> sizevars;
queue<int> cols;
queue<string> references;
queue<string> references_names;

queue<unsigned int> j_col_count;
unsigned int sel_count = 0;
unsigned int join_cnt = 0;
unsigned int distinct_cnt = 0;
unsigned int join_col_cnt = 0;
unsigned int join_tab_cnt = 0;
unsigned int tab_cnt = 0;
queue<string> op_join;
queue<char> join_type;
unsigned int partition_count;
unsigned int statement_count = 0;
map<string,unsigned int> stat;
map<unsigned int, unsigned int> join_and_cnt;
bool scan_state = 0;
map<string, map<string, bool> > used_vars;
bool save_dict = 0;
ContextPtr context;

void emit_multijoin(string s, string j1, string j2, unsigned int tab, char* res_name);
void filter_op(char *s, char *f, unsigned int segment);

void check_used_vars()
{
	for (map<string, map<string, col_data> >::iterator it=data_dict.begin() ; it != data_dict.end(); ++it ) {
		
		map<string, col_data> s = (*it).second;
		queue<string> vars(op_value);
		while(!vars.empty()) {
			if(s.count(vars.front()) != 0) {
				used_vars[(*it).first][vars.front()] = 1;
			};
			vars.pop();
		}	
	};	
}


void emit_name(char *name)
{
    op_type.push("NAME");
    op_value.push(name);
}

void emit_limit(int val)
{
    op_nums.push(val);
}


void emit_string(char *str)
{   // remove the float_type quotes
    string sss(str,1, strlen(str)-2);
    op_type.push("STRING");
    op_value.push(sss);
}


void emit_number(int_type val)
{
    op_type.push("NUMBER");
    op_nums.push(val);	
}

void emit_float(float_type val)
{
    op_type.push("FLOAT");
    op_nums_f.push(val);
}

void emit_decimal(float_type val)
{
    op_type.push("DECIMAL");
    op_nums_f.push(val);
}



void emit_mul()
{
    op_type.push("MUL");
}

void emit_add()
{
    op_type.push("ADD");
}

void emit_div()
{
    op_type.push("DIV");
}

unsigned int misses = 0;

void emit_and()
{
    op_type.push("AND");
    join_col_cnt++;
}

void emit_eq()
{
    op_type.push("JOIN");
    if(misses == 0) {
        join_and_cnt[tab_cnt] = join_col_cnt;
        misses = join_col_cnt;
        join_col_cnt = 0;
        tab_cnt++;
    }
    else {
        misses--;
    }
}

void emit_distinct()
{
    op_type.push("DISTINCT");
    distinct_cnt++;
}

void emit_join()
{
    
}


void emit_or()
{
    op_type.push("OR");
}


void emit_minus()
{
    op_type.push("MINUS");
}

void emit_cmp(int val)
{
    op_type.push("CMP");
    op_nums.push(val);
}

void emit(char *s, ...)
{


}

void emit_var(char *s, int c, char *f, char* ref, char* ref_name)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(0);
    cols.push(c);
	references.push(ref);
	references_names.push(ref_name);	
}

void emit_var_asc(char *s)
{
    op_type.push(s);
    op_value.push("ASC");
}

void emit_var_desc(char *s)
{
    op_type.push(s);
    op_value.push("DESC");
}

void emit_sort(char *s, int p)
{
    op_sort.push(s);
    partition_count = p;
}

void emit_presort(char *s)
{
    op_presort.push(s);
}


void emit_varchar(char *s, int c, char *f, int d, char *ref, char* ref_name)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(d);
    cols.push(c);
	references.push(ref);
	references_names.push(ref_name);
}

void emit_sel_name(char *s)
{
    op_type.push("emit sel_name");
    op_value.push(s);
    sel_count++;
}

void emit_count()
{
    op_type.push("COUNT");
}

void emit_sum()
{
    op_type.push("SUM");
}


void emit_average()
{
    op_type.push("AVG");
}

void emit_min()
{
    op_type.push("MIN");
}

void emit_max()
{
    op_type.push("MAX");
}

void emit_join_tab(char *s, char tp)
{
    op_join.push(s);
    join_tab_cnt++;
    join_type.push(tp);
};

void order_inplace_host(CudaSet* a, stack<string> exe_type, set<string> field_names, bool update_str)
{
    unsigned int* permutation = new unsigned int[a->mRecCount];
    thrust::sequence(permutation, permutation + a->mRecCount);

    size_t maxSize =  a->mRecCount;
    char* temp;
    temp = new char[maxSize*max_char(a)];
	stack<string> exe_type1(exe_type);
	stack<string> exe_value;
	
    while(!exe_type1.empty()) {
        exe_value.push("ASC");
		exe_type1.pop();		
	};	


    // sort on host

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {

        if (a->type[exe_type.top()] == 0)
            update_permutation_host(a->h_columns_int[exe_type.top()].data(), permutation, a->mRecCount, exe_value.top(), (int_type*)temp);
        else if (a->type[exe_type.top()] == 1)
            update_permutation_host(a->h_columns_float[exe_type.top()].data(), permutation, a->mRecCount,exe_value.top(), (float_type*)temp);
        else {
            update_permutation_char_host(a->h_columns_char[exe_type.top()], permutation, a->mRecCount, exe_value.top(), temp, a->char_size[exe_type.top()]);
        };
    };

    for (unsigned int i = 0; i < a->mColumnCount; i++) {
        if (a->type[a->columnNames[i]] == 0) {
			thrust::gather(permutation, permutation + a->mRecCount, a->h_columns_int[a->columnNames[i]].data(), (int_type*)temp);
			thrust::copy((int_type*)temp, (int_type*)temp + a->mRecCount, a->h_columns_int[a->columnNames[i]].data());
        }
        else if (a->type[a->columnNames[i]] == 1) {
			thrust::gather(permutation, permutation + a->mRecCount, a->h_columns_float[a->columnNames[i]].data(), (float_type*)temp);
			thrust::copy((float_type*)temp, (float_type*)temp + a->mRecCount, a->h_columns_float[a->columnNames[i]].data());
		}	
        else {
            apply_permutation_char_host(a->h_columns_char[a->columnNames[i]], permutation, a->mRecCount, temp, a->char_size[a->columnNames[i]]);
			thrust::copy(temp, temp + a->mRecCount*a->char_size[a->columnNames[i]], a->h_columns_float[a->columnNames[i]].data());
        };
    };

    delete [] temp;
    delete [] permutation;
}


void order_inplace1(CudaSet* a, stack<string> exe_type, set<string> field_names, bool update_str)
{
    unsigned int sz = a->mRecCount;
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(sz);
    thrust::sequence(permutation, permutation+sz,0,1);

    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
    void* temp;	
    CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*max_char(a, field_names)));
	stack<string> exe_type1(exe_type);
	
    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {		
        if (a->type[exe_type.top()] == 0 ) {
			a->d_columns_int[exe_type.top()].resize(sz);
			thrust::copy(a->h_columns_int[exe_type.top()].begin(), a->h_columns_int[exe_type.top()].begin() + sz, a->d_columns_int[exe_type.top()].begin());
            update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, sz, "ASC", (int_type*)temp);			
			a->d_columns_int[exe_type.top()].resize(0);
			a->d_columns_int[exe_type.top()].shrink_to_fit();
		}	
        else if (a->type[exe_type.top()] == 1) {
			a->d_columns_float[exe_type.top()].resize(sz);
			thrust::copy(a->h_columns_float[exe_type.top()].begin(), a->h_columns_float[exe_type.top()].begin() + sz, a->d_columns_float[exe_type.top()].begin());
            update_permutation(a->d_columns_float[exe_type.top()], raw_ptr, sz,"ASC", (float_type*)temp);
			a->d_columns_float[exe_type.top()].resize(0);
			a->d_columns_float[exe_type.top()].shrink_to_fit();			
		}	
        else {
            // use int col int_col_count
			a->d_columns_int[exe_type.top()].resize(sz);
			thrust::copy(a->h_columns_int[exe_type.top()].begin(), a->h_columns_int[exe_type.top()].begin() + sz, a->d_columns_int[exe_type.top()].begin());			
            update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, sz, "ASC", (int_type*)temp);
			a->d_columns_int[exe_type.top()].resize(0);
			a->d_columns_int[exe_type.top()].shrink_to_fit();			
        };		
    };	
	
	
    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        if (a->type[*it] == 0) {				
			a->d_columns_int[*it].resize(sz);
			thrust::copy(a->h_columns_int[*it].begin(), a->h_columns_int[*it].begin() + sz, a->d_columns_int[*it].begin());
            apply_permutation(a->d_columns_int[*it], raw_ptr, sz, (int_type*)temp);						
			thrust::copy(a->d_columns_int[*it].begin(), a->d_columns_int[*it].begin() + sz, a->h_columns_int[*it].begin());
			a->d_columns_int[*it].resize(0);
			a->d_columns_int[*it].shrink_to_fit();			
        }
        else if (a->type[*it] == 1) {
			a->d_columns_float[*it].resize(sz);
			thrust::copy(a->h_columns_float[*it].begin(), a->h_columns_float[*it].begin() + sz, a->d_columns_float[*it].begin());		
            apply_permutation(a->d_columns_float[*it], raw_ptr, sz, (float_type*)temp);
			thrust::copy(a->d_columns_float[*it].begin(), a->d_columns_float[*it].begin() + sz, a->h_columns_float[*it].begin());
			a->d_columns_float[*it].resize(0);
			a->d_columns_float[*it].shrink_to_fit();						
		}	
        else {		
			a->allocColumnOnDevice(*it, sz);
			hipMemcpy( a->d_columns_char[*it], (void *)a->h_columns_char[*it], sz*a->char_size[*it], hipMemcpyHostToDevice);
            apply_permutation_char(a->d_columns_char[*it], raw_ptr, sz, (char*)temp, a->char_size[*it]);
			hipMemcpy( a->h_columns_char[*it], a->d_columns_char[*it], sz*a->char_size[*it], hipMemcpyDeviceToHost);
			a->deAllocColumnOnDevice(*it);
			if(update_str) {
				a->d_columns_int[*it].resize(sz);
				hipMemcpy( a->d_columns_char[*it], (void *)a->h_columns_char[*it], sz*a->char_size[*it], hipMemcpyHostToDevice);
				apply_permutation(a->d_columns_int[*it], raw_ptr, sz, (int_type*)temp);
			}	
			
        };
    };
    hipFree(temp);
    thrust::device_free(permutation);		
	
	for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        if (a->type[*it] == 0) {			
			a->d_columns_int[*it].resize(sz);		
			thrust::copy(a->h_columns_int[*it].begin(), a->h_columns_int[*it].begin() + sz, a->d_columns_int[*it].begin());
        }
        else if (a->type[*it] == 1) {
			a->d_columns_float[*it].resize(sz);
			thrust::copy(a->h_columns_float[*it].begin(), a->h_columns_float[*it].begin() + sz, a->d_columns_float[*it].begin());		
		}	
        else {		
			a->allocColumnOnDevice(*it, sz);
			hipMemcpy( a->d_columns_char[*it], (void *)a->h_columns_char[*it], sz*a->char_size[*it], hipMemcpyHostToDevice);
		};
	
	};
	
}




void order_inplace(CudaSet* a, stack<string> exe_type, set<string> field_names, bool update_str)
{

    unsigned int sz = a->mRecCount;
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(sz);
    thrust::sequence(permutation, permutation+sz,0,1);

    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
    void* temp;	
    CUDA_SAFE_CALL(hipMalloc((void **) &temp, sz*max_char(a, field_names)));
	
    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {		
        if (a->type[exe_type.top()] == 0)
            update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, sz, "ASC", (int_type*)temp);
        else if (a->type[exe_type.top()] == 1)
            update_permutation(a->d_columns_float[exe_type.top()], raw_ptr, sz,"ASC", (float_type*)temp);
        else {
            // use int col int_col_count
            update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, sz, "ASC", (int_type*)temp);
        };				
    };	
	
	
    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        if (a->type[*it] == 0) {				
            apply_permutation(a->d_columns_int[*it], raw_ptr, sz, (int_type*)temp);						
        }
        else if (a->type[*it] == 1)
            apply_permutation(a->d_columns_float[*it], raw_ptr, sz, (float_type*)temp);
        else {		
            apply_permutation_char(a->d_columns_char[*it], raw_ptr, sz, (char*)temp, a->char_size[*it]);
			if(update_str) {
				if(a->d_columns_int[*it].size() > 0) {
					apply_permutation(a->d_columns_int[*it], raw_ptr, sz, (int_type*)temp);
				};	
			};	
        };
    };
    hipFree(temp);
    thrust::device_free(permutation);		
}

bool check_star_join(string j1)
{
    queue<string> op_vals(op_value);

    for(unsigned int i=0; i < sel_count; i++) {
        op_vals.pop();
        op_vals.pop();
    };

    if(join_tab_cnt > 1) {

        while(op_vals.size()) {
			if (std::find(varNames[j1]->columnNames.begin(), varNames[j1]->columnNames.end(), op_vals.front()) != varNames[j1]->columnNames.end()) {
                op_vals.pop();
                op_vals.pop();
            }
            else {
                return 0;
            };
        };
        return 1;
    }
    else
        return 0;
}

std::ostream &operator<<(std::ostream &os, const uint2 &x)
{
    os << x.x << ", " << x.y;
    return os;
}


void emit_join(char *s, char *j1, int grp)
{

    statement_count++;
    if (scan_state == 0) {
        if (stat.find(j1) == stat.end() && data_dict.count(j1) == 0) {
            process_error(2, "Join : couldn't find variable " + string(j1) );
        };
        if (stat.find(op_join.front()) == stat.end() && data_dict.count(op_join.front()) == 0) {
            process_error(2, "Join : couldn't find variable " + op_join.front() );
        };
        stat[s] = statement_count;
        stat[j1] = statement_count;
		check_used_vars();
        while(!op_join.empty()) {
            stat[op_join.front()] = statement_count;
            op_join.pop();
        };		
        return;
    };


    queue<string> op_m(op_value);
	queue<string> op_m1(op_value);

	
	if(join_tab_cnt > 1) {
		string tab_name;
		for(unsigned int i = 1; i <= join_tab_cnt; i++) {

			if(i == join_tab_cnt)
				tab_name = s;
			else
				tab_name = s + int_to_string(i);

			string j, j2;
			if(i == 1) {
				j2 = op_join.front();
				op_join.pop();
				j = op_join.front();
				op_join.pop();
			}
			else {
				if(!op_join.empty()) {
					j = op_join.front();
					op_join.pop();
				}
				else
					j = j1;
				j2 = s + int_to_string(i-1);
			};
			emit_multijoin(tab_name, j, j2, i, s);
			op_value = op_m;
		};
	}
	else {
		string j2 = op_join.front();
		op_join.pop();
		emit_multijoin(s, j1, j2, 1, s);
	};

	
    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_m.front());
        op_m.pop();
        op_sel_as.push(op_m.front());
        op_m.pop();
    };
	while(!op_sel_as.empty()) {
		//cout << "alias " << op_sel.front() << " : " << op_sel_as.front() << endl;
		if(op_sel.front() != op_sel_as.front()) {
			if(varNames[s]->type[op_sel.front()] == 0) {
				varNames[s]->h_columns_int[op_sel_as.front()] = varNames[s]->h_columns_int[op_sel.front()];
				varNames[s]->h_columns_int.erase(op_sel.front());	
				varNames[s]->d_columns_int[op_sel_as.front()] = varNames[s]->d_columns_int[op_sel.front()];
				varNames[s]->d_columns_int.erase(op_sel.front());	
				varNames[s]->type[op_sel_as.front()] = 0;	
				varNames[s]->type.erase(op_sel.front());
			}
			else if(varNames[s]->type[op_sel.front()] == 1) {
				varNames[s]->h_columns_float[op_sel_as.front()] = varNames[s]->h_columns_float[op_sel.front()];
				varNames[s]->h_columns_float.erase(op_sel.front());	
				varNames[s]->d_columns_float[op_sel_as.front()] = varNames[s]->d_columns_float[op_sel.front()];
				varNames[s]->d_columns_float.erase(op_sel.front());					
				varNames[s]->type[op_sel_as.front()] = 1;	
				varNames[s]->type.erase(op_sel.front());					
				varNames[s]->decimal.erase(op_sel.front());
			}
			else {
				varNames[s]->h_columns_char[op_sel_as.front()] = varNames[s]->h_columns_char[op_sel.front()];
				varNames[s]->h_columns_char.erase(op_sel.front());	
				varNames[s]->d_columns_char[op_sel_as.front()] = varNames[s]->d_columns_char[op_sel.front()];
				varNames[s]->d_columns_char.erase(op_sel.front());					
				varNames[s]->type[op_sel_as.front()] = 2;	
				varNames[s]->type.erase(op_sel.front());	
				varNames[s]->char_size[op_sel_as.front()] = varNames[s]->char_size[op_sel.front()];
				varNames[s]->char_size.erase(op_sel.front());								
			};
			varNames[s]->decimal[op_sel_as.front()] = varNames[s]->decimal[op_sel.front()];
			std::vector<string>::iterator it;
			it = std::find(varNames[s]->columnNames.begin(), varNames[s]->columnNames.end(), op_sel.front());
			*it = op_sel_as.front();
		};	
		op_sel_as.pop();
		op_sel.pop();
	};
	
	

    clean_queues();

    if(stat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };

    if(op_join.size()) {
        if(stat[op_join.front()] == statement_count && op_join.front().compare(j1) != 0) {
            varNames[op_join.front()]->free();
            varNames.erase(op_join.front());
        };
    };

}



void emit_multijoin(string s, string j1, string j2, unsigned int tab, char* res_name)
{

    if(varNames.find(j1) == varNames.end() || varNames.find(j2) == varNames.end()) {
        clean_queues();
        if(varNames.find(j1) == varNames.end())
            cout << "Couldn't find j1 " << j1 << endl;
        if(varNames.find(j2) == varNames.end())
            cout << "Couldn't find j2 " << j2 << endl;

        return;
    };

    CudaSet* left = varNames.find(j1)->second;
    CudaSet* right = varNames.find(j2)->second;

    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_value.front());
        op_value.pop();
        op_sel_as.push(op_value.front());
        op_value.pop();
    };

    queue<string> op_sel_s(op_sel);
    queue<string> op_sel_s_as(op_sel_as);
    queue<string> op_g(op_value);

    if(tab > 0) {
        for(unsigned int z = 0; z < join_tab_cnt - tab; z++) {
            for(unsigned int j = 0; j < join_and_cnt[z]*2 + 2; j++) {
                op_sel_s.push(op_g.front());
                op_sel_s_as.push(op_g.front());
                op_g.pop();
            };
        };
    };
	
    string f1 = op_g.front();
    op_g.pop();
    string f2 = op_g.front();
    op_g.pop();

	if (verbose)
		cout << "JOIN " << s <<  " " <<  f1 << " " << f2 << " " << getFreeMem() <<  endl;

    std::clock_t start1 = std::clock();
    CudaSet* c = new CudaSet(right, left, op_sel_s, op_sel_s_as);

    if ((left->mRecCount == 0 && !left->filtered) || (right->mRecCount == 0 && !right->filtered)) {
        c = new CudaSet(left, right, op_sel_s, op_sel_s_as);
        varNames[res_name] = c;
        clean_queues();
        return;
    };
	
    if(join_tab_cnt > 1 && tab < join_tab_cnt)
        c->tmp_table = 1;
    else
        c->tmp_table = 0;	

	string colname1, colname2;
    string tmpstr;
    if (std::find(left->columnNames.begin(), left->columnNames.end(), f1) != left->columnNames.end()) {
		colname1 = f1;
        if (std::find(right->columnNames.begin(), right->columnNames.end(), f2) != right->columnNames.end()) {
			colname2 = f2;
        }
        else {
            process_error(2, "Couldn't find column " + f2 );
        };
    }
    else if (std::find(right->columnNames.begin(), right->columnNames.end(), f1) != right->columnNames.end()) {
		colname2 = f1;
        tmpstr = f1;
        f1 = f2;
        if (std::find(left->columnNames.begin(), left->columnNames.end(), f2) != left->columnNames.end()) {
			colname1 = f2;
            f2 = tmpstr;
        }
        else {
            process_error(2, "Couldn't find column " +f2 );
        };
    }
    else {
        process_error(2, "Couldn't find column " + f1);
    };


    if (!((left->type[colname1] == 0 && right->type[colname2]  == 0) || (left->type[colname1] == 2 && right->type[colname2]  == 2)
            || (left->type[colname1] == 1 && right->type[colname2]  == 1 && left->decimal[colname1] && right->decimal[colname2]))) {
        process_error(2, "Joins on floats are not supported ");
    };


    bool decimal_join = 0;
    if (left->type[colname1] == 1 && right->type[colname2]  == 1)
        decimal_join = 1;

    queue<string> op_vd(op_g);
	queue<string> op_g1(op_g);
    queue<string> op_alt(op_sel);
    unsigned int jc = join_and_cnt[join_tab_cnt - tab];
    while(jc) {
        jc--;
        op_vd.pop();
        op_alt.push(op_vd.front());
        op_vd.pop();
    };

    bool str_join = 0;    
    size_t rcount = 0, cnt_r;
	unsigned int r_parts = calc_right_partition(left, right, op_sel);
	unsigned int start_part = 0;
    queue<string> cc;
	
    if (left->type[colname1]  == 2) {
        left->d_columns_int[colname1] = thrust::device_vector<int_type>();
    }
    else {
        cc.push(f1);
        allocColumns(left, cc);
    };	

    left->hostRecCount = left->mRecCount;

    size_t cnt_l, res_count, tot_count = 0, offset = 0, k = 0;
    queue<string> lc(cc);
    thrust::device_vector<int> p_tmp;
    thrust::device_vector<unsigned int> v_l(left->maxRecs);
    MGPU_MEM(int) aIndicesDevice, bIndicesDevice;
	std::vector<int> j_data;
	
	stack<string> exe_type;
	set<string> field_names;
	exe_type.push(f2);
	for (unsigned int i = 0; i < right->columnNames.size() ; i++ ) {
		if (std::find(c->columnNames.begin(), c->columnNames.end(), right->columnNames[i]) != c->columnNames.end() || right->columnNames[i] == f2) {
			field_names.insert(right->columnNames[i]);
		};	
	};	
	
	
	while(start_part < right->segCount) {

	    bool rsz = 1;
		right->deAllocOnDevice();		

		//cout << "ordering " << endl;
		//if(right->not_compressed)
			//order_inplace_host(right, exe_type, field_names, 0);						
		//cout << "ordered " << endl;
		
		if(start_part + r_parts >= right->segCount) {
			cnt_r = load_right(right, colname2, f2, op_g1, op_sel, op_alt, decimal_join, str_join, rcount, start_part, right->segCount, rsz);
			start_part = right->segCount;
		}
		else {
			cnt_r = load_right(right, colname2, f2, op_g1, op_sel, op_alt, decimal_join, str_join, rcount, start_part, start_part+r_parts, rsz);
			start_part = start_part+r_parts;			
		};			
		
		if(right->not_compressed && getFreeMem() < right->mRecCount*max_char(right)*2) {
			right->CopyToHost(0, right->mRecCount);
			right->deAllocOnDevice();
			if (left->type[colname1]  != 2)
				order_inplace1(right, exe_type, field_names, 0);					
			else	
				order_inplace1(right, exe_type, field_names, 1);					
		}
		else {
			if (left->type[colname1]  != 2)
				order_inplace(right, exe_type, field_names, 0);					
			else {	
				order_inplace(right, exe_type, field_names, 1);					
			};	
		};
		

		for (unsigned int i = 0; i < left->segCount; i++) {
			
			if(verbose)
				cout << "segment " << i <<  '\xd';	
			j_data.clear();		
			std::clock_t start2 = std::clock();		
			
			//for (set<unsigned int>::iterator it = left->ref_joins[colInd1][i].begin(); it != left->ref_joins[colInd1][i].end(); it++) {
			//	cout << "seg match " << *it << endl;
			//};
			
			//for (set<unsigned int>::iterator it = right->orig_segs[left->ref_sets[colInd1]].begin(); it != right->orig_segs[left->ref_sets[colInd1]].end(); it++) {
			//	cout << "right segs " << *it << endl;
			//};
			
			
			if(left->ref_joins[colname1][i].size() && right->orig_segs[left->ref_sets[colname1]].size()) {
				set_intersection(left->ref_joins[colname1][i].begin(),left->ref_joins[colname1][i].end(),
								 right->orig_segs[left->ref_sets[colname1]].begin(), right->orig_segs[left->ref_sets[colname1]].end(),
								 std::back_inserter(j_data));
				if(j_data.empty()) {
					continue;
				};	
				
			};	
			
					
			cnt_l = 0;
			if (left->type[colname1]  != 2) {
				copyColumns(left, lc, i, cnt_l);			
			}
			else {
				left->add_hashed_strings(f1, i);
			};
			
	
			if(!left->filtered) {
				if (left->type[colname1]  != 2)
					cnt_l = left->mRecCount;
				else
					cnt_l = left->d_columns_int[colname1].size();
			}
			else {
				cnt_l = left->mRecCount;
			};
			
						
			if (cnt_l) {

				// sort the left index column, save the permutation vector, it might be needed later

				thrust::device_ptr<int_type> d_col((int_type*)thrust::raw_pointer_cast(left->d_columns_int[colname1].data()));
				thrust::sequence(v_l.begin(), v_l.begin() + cnt_l,0,1);

				bool do_sort = 1;
				if(!left->sorted_fields.empty()) {
					if(left->sorted_fields.front() == f1) {
						do_sort = 0;
					};
				}
				else if(!left->presorted_fields.empty()) {
						if(left->presorted_fields.front() == f1) {
							do_sort = 0;
						};
				};
				
				if(do_sort)
					thrust::sort_by_key(d_col, d_col + cnt_l, v_l.begin());
				else
					if(verbose)
						cout << "No need of sorting " << endl;
					
				//cout << "join " << cnt_l << ":" << cnt_r << " " << join_type.front() << endl;
				//cout << "SZ " << left->d_columns_int[colname1].size() << endl;
					
				
				if (left->d_columns_int[colname1][0] > right->d_columns_int[colname2][cnt_r-1] ||
					left->d_columns_int[colname1][cnt_l-1] < right->d_columns_int[colname2][0]) {
					continue;
				};	
				
								
				char join_kind = join_type.front();
				
				if (left->type[colname1] == 2) {
					thrust::device_ptr<int_type> d_col_r((int_type*)thrust::raw_pointer_cast(right->d_columns_int[colname2].data()));		

					//for(int z = 0; z < cnt_r ; z++)
					//	cout << " R " << right->d_columns_int[colname2][z] << endl;
						
					//for(int z = 0; z < cnt_l ; z++)
					//	cout << " L " << left->d_columns_int[colname1][z] << endl;
	
					
					res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(d_col), cnt_l,
								thrust::raw_pointer_cast(d_col_r), cnt_r,
								&aIndicesDevice, &bIndicesDevice,
								mgpu::less<unsigned long long int>(), *context);

				}
				else {

					if (join_kind == 'I')
						res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
									thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<int_type>(), *context);
					else if(join_kind == 'L')
						res_count = RelationalJoin<MgpuJoinKindLeft>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
									thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<int_type>(), *context);
					else if(join_kind == 'R')
						res_count = RelationalJoin<MgpuJoinKindRight>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
									thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<int_type>(), *context);
					else if(join_kind == 'O')
						res_count = RelationalJoin<MgpuJoinKindOuter>(thrust::raw_pointer_cast(left->d_columns_int[colname1].data()), cnt_l,
									thrust::raw_pointer_cast(right->d_columns_int[colname2].data()), cnt_r,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<int_type>(), *context);
				};
				
				//cout << "RES " << res_count << " seg " << i << endl;
				
				int* r1 = aIndicesDevice->get();
				thrust::device_ptr<int> d_res1((int*)r1);
				int* r2 = bIndicesDevice->get();
				thrust::device_ptr<int> d_res2((int*)r2);
				
				
				if(res_count) {
					p_tmp.resize(res_count);
					thrust::sequence(p_tmp.begin(), p_tmp.end(),-1);
					thrust::gather_if(d_res1, d_res1+res_count, d_res1, v_l.begin(), p_tmp.begin(), is_positive<int>());
				};					


				// check if the join is a multicolumn join
				unsigned int mul_cnt = join_and_cnt[join_tab_cnt - tab];

				while(mul_cnt) {

					mul_cnt--;
					string f3 = op_g.front();
					op_g.pop();
					string f4 = op_g.front();
					op_g.pop();

					//cout << "ADDITIONAL COL JOIN " << f3 << " " << f4 << " " << getFreeMem() << endl;

					queue<string> rc;
					rc.push(f3);

					allocColumns(left, rc);
					left->hostRecCount = left->mRecCount;
					size_t offset = 0;
					copyColumns(left, rc, i, offset, 0, 0);
					rc.pop();

					void* temp;
					CUDA_SAFE_CALL(hipMalloc((void **) &temp, res_count*float_size));
					void* temp1;
					CUDA_SAFE_CALL(hipMalloc((void **) &temp1, res_count*float_size));
					hipMemset(temp,0,res_count*float_size);
					hipMemset(temp1,0,res_count*float_size);

				

					if (res_count) {
						thrust::device_ptr<bool> d_add = thrust::device_malloc<bool>(res_count);

						if (left->type[f3] == 1 && right->type[f4]  == 1) {

							if(right->d_columns_float[f4].size() == 0)
								load_queue(rc, right, 0, f4, rcount, 0, right->segCount, 0, 0);

							thrust::device_ptr<float_type> d_tmp((float_type*)temp);
							thrust::device_ptr<float_type> d_tmp1((float_type*)temp1);
							thrust::gather_if(p_tmp.begin(), p_tmp.end(), p_tmp.begin(), left->d_columns_float[f3].begin(), d_tmp, is_positive<int>());
							thrust::gather_if(d_res2, d_res2+res_count, d_res2, right->d_columns_float[f4].begin(), d_tmp1, is_positive<int>());
							thrust::transform(d_tmp, d_tmp+res_count, d_tmp1, d_add, float_equal_to());
						}
						else {
							if(right->d_columns_int[f4].size() == 0) {
								load_queue(rc, right, 0, f4, rcount, 0, right->segCount, 0, 0);
							};
							thrust::device_ptr<int_type> d_tmp((int_type*)temp);
							thrust::device_ptr<int_type> d_tmp1((int_type*)temp1);
							thrust::gather_if(p_tmp.begin(), p_tmp.end(), p_tmp.begin(), left->d_columns_int[f3].begin(), d_tmp, is_positive<int>());
							thrust::gather_if(d_res2, d_res2+res_count, d_res2, right->d_columns_int[f4].begin(), d_tmp1, is_positive<int>());
							thrust::transform(d_tmp, d_tmp+res_count, d_tmp1, d_add, thrust::equal_to<int_type>());							
						};

						if (join_kind == 'I') {  // result count changes only in case of an inner join
							unsigned int new_cnt = thrust::count(d_add, d_add+res_count, 1);
							thrust::stable_partition(d_res2, d_res2 + res_count, d_add, thrust::identity<unsigned int>());
							thrust::stable_partition(p_tmp.begin(), p_tmp.end(), d_add, thrust::identity<unsigned int>());
							thrust::device_free(d_add);
							res_count = new_cnt;
						}
						else { //otherwise we consider it a valid left join result with non-nulls on the left side and nulls on the right side
							thrust::transform(d_res2, d_res2 + res_count, d_add , d_res2, set_minus());
						};
					};
					hipFree(temp);
					hipFree(temp1);
				};

				//string sss = s;
				//cout << " " << tot_count << " " << res_count << endl;								
				//if(sss != "OLC") {
				//	cout << "RUN " << tot_count << " " << res_count << endl;				

				tot_count = tot_count + res_count;				

				if(res_count) {

					for (map<string, set<unsigned int> >::iterator itr = left->orig_segs.begin(); itr != left->orig_segs.end(); itr++) {
						for (set<unsigned int>::iterator it = itr->second.begin(); it != itr->second.end(); it++) {
							//cout << "LEFT SEGS " << itr->first << " : " << *it << endl;
							c->orig_segs[itr->first].insert(*it);
						};						
					};	

					for (map<string, set<unsigned int> >::iterator itr = right->orig_segs.begin(); itr != right->orig_segs.end(); itr++) {
						for (set<unsigned int>::iterator it = itr->second.begin(); it != itr->second.end(); it++) {
							//cout << "RIGHT SEGS " << itr->first << " : " << *it << endl;
							c->orig_segs[itr->first].insert(*it);
						};						
					};	

					offset = c->mRecCount;				
					if(i == 0 && left->segCount != 1) {
						c->reserve(res_count*(left->segCount+1));
					};
					
					
					queue<string> op_sel1(op_sel_s);					
					c->resize_join(res_count);					
					void* temp;
					CUDA_SAFE_CALL(hipMalloc((void **) &temp, res_count*max_char(c)));

					bool copied = 0;	
					thrust::host_vector<unsigned int> prm_vh;
					std::map<string,bool> processed;				
					bool cmp_type;			

					
					//std::clock_t start1 = std::clock();	
					while(!op_sel1.empty()) {
					
						if (processed.find(op_sel1.front()) != processed.end()) {
							op_sel1.pop();
							continue;
						}	
						else	
							processed[op_sel1.front()] = 1;
							
						while(!cc.empty())
							cc.pop();

						cc.push(op_sel1.front());
												
						if(std::find(left->columnNames.begin(), left->columnNames.end(), op_sel1.front()) !=  left->columnNames.end()) {
							// copy field's segment to device, gather it and copy to the host
							
							if(left->filtered)
								cmp_type = varNames[left->source_name]->compTypes[op_sel1.front()];
							else
								cmp_type = left->compTypes[op_sel1.front()];
							
							if ((((left->type[op_sel1.front()] == 0) || ((left->type[op_sel1.front()] == 1) && 
								   left->decimal[op_sel1.front()])) && cmp_type == 0) && (op_sel1.front() != colname1) && left->not_compressed == 0) { // do the processing on host												
								
								void* h;	
								unsigned int cnt, lower_val, bits;		

								//if(verbose)
								//	cout << "processing " << op_sel1.front() << " " << i << " " << cmp_type << endl;
								
								if(!copied) {								
									if(left->filtered) {
										thrust::device_vector<unsigned int> prm_v(res_count);
										thrust::gather(p_tmp.begin(), p_tmp.begin() + res_count, left->prm_d.begin(), prm_v.begin());
										prm_vh = prm_v;									
									}	
									else {
										prm_vh = p_tmp;
									};
									copied = 1;
								};								
								
								
								CudaSet *t;
								if(left->filtered)
									t = varNames[left->source_name];
								else
									t = left;
									
								t->readSegmentsFromFile(i, op_sel1.front(), 0);
								
								if(t->type[op_sel1.front()] == 0) {
									h = t->h_columns_int[op_sel1.front()].data();								
								}
								else {
									h = t->h_columns_float[op_sel1.front()].data();
								};	
								
								cnt = ((unsigned int*)h)[0];
								lower_val = ((unsigned int*)h)[1];
								bits = ((unsigned int*)((char*)h + cnt))[8];	
								//cout << cnt << " " << lower_val << " " << bits << endl;																
			
								if(bits == 8) {
									if(left->type[op_sel1.front()] == 0) {	
										thrust::gather(prm_vh.begin(), prm_vh.end(), (char*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
									}	
									else {	
										int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
										thrust::gather(prm_vh.begin(), prm_vh.end(), (char*)((unsigned int*)h + 6), ptr + offset);
									};										
								}
								else if(bits == 16) {
									if(left->type[op_sel1.front()] == 0) {	
										thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned short int*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
									}	
									else {	
										int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
										thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned short int*)((unsigned int*)h + 6), ptr + offset);
									};
								}
								else if(bits == 32) {
									if(left->type[op_sel1.front()] == 0) {	
										thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned int*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
									}	
									else {	
										int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
										thrust::gather(prm_vh.begin(), prm_vh.end(), (unsigned int*)((unsigned int*)h + 6), ptr + offset);
									};	
								}
								else if(bits == 64) {
									if(left->type[op_sel1.front()] == 0) {	
										thrust::gather(prm_vh.begin(), prm_vh.end(),  (int_type*)((unsigned int*)h + 6), c->h_columns_int[op_sel1.front()].begin() + offset);
									}	
									else {	
										int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
										thrust::gather(prm_vh.begin(), prm_vh.end(), (int_type*)((unsigned int*)h + 6), ptr + offset);
									};
								};
								
								if(left->type[op_sel1.front()] == 0) {	
									thrust::transform(c->h_columns_int[op_sel1.front()].begin() + offset, c->h_columns_int[op_sel1.front()].begin() + offset + res_count, 
													  thrust::make_constant_iterator(lower_val), c->h_columns_int[op_sel1.front()].begin() + offset, thrust::plus<int_type>()); 																	
								}
								else {
									int_type* ptr = (int_type*)c->h_columns_float[op_sel1.front()].data();
									thrust::transform(ptr + offset, ptr + offset + res_count, 
													  thrust::make_constant_iterator(lower_val), ptr + offset, thrust::plus<int_type>()); 																						
									thrust::transform(ptr + offset, ptr + offset + res_count, c->h_columns_float[op_sel1.front()].begin() + offset, long_to_float());													
								};							
								
							}
							else {						
							
								allocColumns(left, cc);				
								copyColumns(left, cc, i, k, 0, 0);
							
								//gather
								if(left->type[op_sel1.front()] == 0) {
									thrust::device_ptr<int_type> d_tmp((int_type*)temp);
									thrust::sequence(d_tmp, d_tmp+res_count,0,0);		
									thrust::gather_if(p_tmp.begin(), p_tmp.begin() + res_count, p_tmp.begin(), left->d_columns_int[op_sel1.front()].begin(), d_tmp, is_positive<int>());								
									thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[op_sel1.front()].begin() + offset);
								}
								else if(left->type[op_sel1.front()] == 1) {
									thrust::device_ptr<float_type> d_tmp((float_type*)temp);
									thrust::sequence(d_tmp, d_tmp+res_count,0,0);
									thrust::gather_if(p_tmp.begin(), p_tmp.begin() + res_count, p_tmp.begin(), left->d_columns_float[op_sel1.front()].begin(), d_tmp, is_positive<int>());
									thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_float[op_sel1.front()].begin() + offset);
								}
								else { //strings
									thrust::device_ptr<char> d_tmp((char*)temp);
									thrust::fill(d_tmp, d_tmp+res_count*left->char_size[op_sel1.front()],0);
									
								
									str_gather(thrust::raw_pointer_cast(p_tmp.data()), res_count, (void*)left->d_columns_char[op_sel1.front()],
											(void*) thrust::raw_pointer_cast(d_tmp), left->char_size[op_sel1.front()]);
									hipMemcpy( (void*)&c->h_columns_char[op_sel1.front()][offset*c->char_size[op_sel1.front()]], (void*) thrust::raw_pointer_cast(d_tmp),
												c->char_size[op_sel1.front()] * res_count, hipMemcpyDeviceToHost);
								};
								
							
								if(op_sel1.front() != colname1)
									left->deAllocColumnOnDevice(op_sel1.front());							
							}
						}
						else if(std::find(right->columnNames.begin(), right->columnNames.end(), op_sel1.front()) !=  right->columnNames.end()) {
						
							//gather
							if(right->type[op_sel1.front()] == 0) {
								thrust::device_ptr<int_type> d_tmp((int_type*)temp);
								thrust::sequence(d_tmp, d_tmp+res_count,0,0);
								thrust::gather_if(d_res2, d_res2 + res_count, d_res2, right->d_columns_int[op_sel1.front()].begin(), d_tmp, is_positive<int>());							
								thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[op_sel1.front()].begin() + offset);
							}
							else if(right->type[op_sel1.front()] == 1) {
								thrust::device_ptr<float_type> d_tmp((float_type*)temp);
								thrust::sequence(d_tmp, d_tmp+res_count,0,0);
								thrust::gather_if(d_res2, d_res2 + res_count, d_res2, right->d_columns_float[op_sel1.front()].begin(), d_tmp, is_positive<int>());
								thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_float[op_sel1.front()].begin() + offset);
							}
							else { //strings

								thrust::device_ptr<char> d_tmp((char*)temp);
								thrust::sequence(d_tmp, d_tmp+res_count*right->char_size[op_sel1.front()],0,0);							
								str_gather(thrust::raw_pointer_cast(d_res2), res_count, (void*)right->d_columns_char[op_sel1.front()],
										   (void*) thrust::raw_pointer_cast(d_tmp), right->char_size[op_sel1.front()]);									   
								hipMemcpy( (void*)&c->h_columns_char[op_sel1.front()][offset*c->char_size[op_sel1.front()]], (void*) thrust::raw_pointer_cast(d_tmp),
											c->char_size[op_sel1.front()] * res_count, hipMemcpyDeviceToHost);
							};	
						}
						else {
						};
						op_sel1.pop();					
					};					
					hipFree(temp);
				};
			};
			//std::cout<< endl << "seg time " <<  ( ( std::clock() - start2 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
		};
	};
	
		

    left->deAllocOnDevice();
    right->deAllocOnDevice();
    c->deAllocOnDevice();
	

    varNames[s] = c;
    c->mRecCount = tot_count;    
	c->hostRecCount = tot_count;
	c->name = s;
	
	if(verbose)
		cout << endl << "tot res " << tot_count << endl;
	
	unsigned int tot_size = 0;	    
    for (unsigned int i = 0; i < c->columnNames.size(); i++ ) {
		if(c->type[c->columnNames[i]] <= 1) 
			tot_size = tot_size + tot_count*8;
		else	
			tot_size = tot_size + tot_count*c->char_size[c->columnNames[i]];
    };
	if ((getFreeMem() - 300000000) > tot_size) {
		c->maxRecs = tot_count;
	}
	else {	 
		c->segCount = ((tot_size/(getFreeMem() - 300000000)) + 1);		
		c->maxRecs = c->hostRecCount - (c->hostRecCount/c->segCount)*(c->segCount-1);
	};	
	

    if(right->tmp_table == 1) {
        right->free();
        varNames.erase(j2);
    }
    else {
        if(stat[j2] == statement_count) {
            right->free();
            varNames.erase(j2);
        };

    };

    if(stat[j1] == statement_count) {
        left->free();
        varNames.erase(j1);
    };
	join_type.pop();	
	
	if(verbose)
		std::cout<< "join time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;					
		
	
}


void order_on_host(CudaSet *a, CudaSet* b, queue<string> names, stack<string> exe_type, stack<string> exe_value)
{
    unsigned int tot = 0;
    if(!a->not_compressed) { //compressed
        allocColumns(a, names);

        unsigned int c = 0;
        size_t cnt = 0;
        for(unsigned int i = 0; i < a->segCount; i++) {
            copyColumns(a, names, (a->segCount - i) - 1, cnt);	//uses segment 1 on a host	to copy data from a file to gpu
            if (a->mRecCount) {
                a->CopyToHost((c - tot) - a->mRecCount, a->mRecCount);
                tot = tot + a->mRecCount;
            };
        };
    }
    else
        tot = a->mRecCount;

    b->resize(tot); //resize host arrays
    a->mRecCount = tot;

    unsigned int* permutation = new unsigned int[a->mRecCount];
    thrust::sequence(permutation, permutation + a->mRecCount);

    size_t maxSize =  a->mRecCount;
    char* temp;
    temp = new char[maxSize*max_char(a)];

    // sort on host

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {

        if (a->type[exe_type.top()] == 0)
            update_permutation_host(a->h_columns_int[exe_type.top()].data(), permutation, a->mRecCount, exe_value.top(), (int_type*)temp);
        else if (a->type[exe_type.top()] == 1)
            update_permutation_host(a->h_columns_float[exe_type.top()].data(), permutation, a->mRecCount,exe_value.top(), (float_type*)temp);
        else {
            update_permutation_char_host(a->h_columns_char[exe_type.top()], permutation, a->mRecCount, exe_value.top(), b->h_columns_char[exe_type.top()], a->char_size[exe_type.top()]);
        };
    };

    for (unsigned int i = 0; i < a->mColumnCount; i++) {
        if (a->type[a->columnNames[i]] == 0) {
            apply_permutation_host(a->h_columns_int[a->columnNames[i]].data(), permutation, a->mRecCount, b->h_columns_int[a->columnNames[i]].data());
        }
        else if (a->type[a->columnNames[i]] == 1)
            apply_permutation_host(a->h_columns_float[a->columnNames[i]].data(), permutation, a->mRecCount, b->h_columns_float[a->columnNames[i]].data());
        else {
            apply_permutation_char_host(a->h_columns_char[a->columnNames[i]], permutation, a->mRecCount, b->h_columns_char[a->columnNames[i]], a->char_size[a->columnNames[i]]);
        };
    };

    delete [] temp;
    delete [] permutation;
}



void emit_order(char *s, char *f, int e, int ll)
{
    if(ll == 0)
        statement_count++;

    if (scan_state == 0 && ll == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(2, "Order : couldn't find variable " + string(f));
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        return;
    };
	
	if (scan_state == 0)
		check_used_vars();

    if(varNames.find(f) == varNames.end() ) {
        clean_queues();
        return;
    };

    CudaSet* a = varNames.find(f)->second;


    if (a->mRecCount == 0)	{
        if(varNames.find(s) == varNames.end())
            varNames[s] = new CudaSet(0,1);
        else {
            CudaSet* c = varNames.find(s)->second;
            c->mRecCount = 0;
        };
        return;
    };

    stack<string> exe_type, exe_value;

	if(verbose)
		cout << "order: " << s << " " << f << endl;


    for(int i=0; !op_type.empty(); ++i, op_type.pop(),op_value.pop()) {
        if ((op_type.front()).compare("NAME") == 0) {
            exe_type.push(op_value.front());
            exe_value.push("ASC");
        }
        else {
            exe_type.push(op_type.front());
            exe_value.push(op_value.front());
        };
		if(std::find(a->columnNames.begin(), a->columnNames.end(), exe_type.top()) == a->columnNames.end()) {		
			process_error(2, "Couldn't find name " + exe_type.top());
		};

    };

    stack<string> tp(exe_type);
    queue<string> op_vx;
    while (!tp.empty()) {
        op_vx.push(tp.top());
        tp.pop();
    };

    queue<string> names;
    for (unsigned int i = 0; i < a->columnNames.size() ; i++ )
        names.push(a->columnNames[i]);

    CudaSet *b = a->copyDeviceStruct();

    //lets find out if our data set fits into a GPU
    size_t mem_available = getFreeMem();
    size_t rec_size = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        if(a->type[a->columnNames[i]] == 0)
            rec_size = rec_size + int_size;
        else if(a->type[a->columnNames[i]] == 1)
            rec_size = rec_size + float_size;
        else
            rec_size = rec_size + a->char_size[a->columnNames[i]];
    };
    bool fits;
    if (rec_size*a->mRecCount > (mem_available/2)) // doesn't fit into a GPU
        fits = 0;
    else fits = 1;

    if(!fits) {
        order_on_host(a, b, names, exe_type, exe_value);
    }
    else {
        // initialize permutation to [0, 1, 2, ... ,N-1]
        thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(a->mRecCount);
        thrust::sequence(permutation, permutation+(a->mRecCount));

        unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);

        void* temp;        
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, a->mRecCount*max_char(a)));
        
		if(a->filtered)
			varNames[a->source_name]->hostRecCount = varNames[a->source_name]->mRecCount;
		else
			a->hostRecCount = a->mRecCount;; 

        size_t rcount;
        a->mRecCount = load_queue(names, a, 1, op_vx.front(), rcount, 0, a->segCount);
		
		if(a->filtered)
			varNames[a->source_name]->mRecCount = varNames[a->source_name]->hostRecCount;
		else
			a->mRecCount = a->hostRecCount;; 

		
        for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {
            if (a->type[exe_type.top()] == 0)
                update_permutation(a->d_columns_int[exe_type.top()], raw_ptr, a->mRecCount, exe_value.top(), (int_type*)temp);
            else if (a->type[exe_type.top()] == 1)
                update_permutation(a->d_columns_float[exe_type.top()], raw_ptr, a->mRecCount,exe_value.top(), (float_type*)temp);
            else {
                update_permutation_char(a->d_columns_char[exe_type.top()], raw_ptr, a->mRecCount, exe_value.top(), (char*)temp, a->char_size[exe_type.top()]);
                //update_permutation(a->d_columns_int[int_col_count+str_count], raw_ptr, a->mRecCount, exe_value.top(), (int_type*)temp);
                //str_count++;
            };
        };

        b->resize(a->mRecCount); //resize host arrays
        b->mRecCount = a->mRecCount;
        //str_count = 0;

        for (unsigned int i = 0; i < a->mColumnCount; i++) {
            if (a->type[a->columnNames[i]] == 0)
                apply_permutation(a->d_columns_int[a->columnNames[i]], raw_ptr, a->mRecCount, (int_type*)temp);
            else if (a->type[a->columnNames[i]] == 1)
                apply_permutation(a->d_columns_float[a->columnNames[i]], raw_ptr, a->mRecCount, (float_type*)temp);
            else {
                apply_permutation_char(a->d_columns_char[a->columnNames[i]], raw_ptr, a->mRecCount, (char*)temp, a->char_size[a->columnNames[i]]);
                //str_count++;
            };
        };

        for(unsigned int i = 0; i < a->mColumnCount; i++) {
            switch(a->type[a->columnNames[i]]) {
            case 0 :
                thrust::copy(a->d_columns_int[a->columnNames[i]].begin(), a->d_columns_int[a->columnNames[i]].begin() + a->mRecCount, b->h_columns_int[a->columnNames[i]].begin());
                break;
            case 1 :
                thrust::copy(a->d_columns_float[a->columnNames[i]].begin(), a->d_columns_float[a->columnNames[i]].begin() + a->mRecCount, b->h_columns_float[a->columnNames[i]].begin());
                break;
            default :
                hipMemcpy(b->h_columns_char[a->columnNames[i]], a->d_columns_char[a->columnNames[i]], a->char_size[a->columnNames[i]]*a->mRecCount, hipMemcpyDeviceToHost);
            }
        };

        b->deAllocOnDevice();
        a->deAllocOnDevice();


        thrust::device_free(permutation);
        hipFree(temp);
    };

    varNames[s] = b;
    b->segCount = 1;
    b->not_compressed = 1;

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(f);
    };
}


void emit_select(char *s, char *f, int ll)
{

    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(2, "Select : couldn't find variable " + string(f) );
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
		check_used_vars();
	    clean_queues();
        return;
    };
	
	if(varNames.find(f) == varNames.end()) {
        clean_queues();
        cout << "Couldn't find1 " << f << endl;
        return;
    };
	
	queue<string> op_v1(op_value);
    while(op_v1.size() > ll)
        op_v1.pop();


    stack<string> op_v2;
    queue<string> op_v3;

    for(int i=0; i < ll; ++i) {
        op_v2.push(op_v1.front());
        op_v3.push(op_v1.front());
        op_v1.pop();
    };
	
		
    CudaSet *a;
    if(varNames.find(f) != varNames.end())
        a = varNames.find(f)->second;
    else {
        process_error(2, "Couldn't find " + string(f) );
    };

    if(a->mRecCount == 0 && !a->filtered) {
        CudaSet *c;
        c = new CudaSet(0,1);
        varNames[s] = c;
		c->name = s;
        clean_queues();
		if(verbose)
			cout << "SELECT " << s << " count : 0,  Mem " << getFreeMem() << endl;
        return;
    };

	if(verbose)
		cout << "SELECT " << s << " " << f << " " << getFreeMem() << endl;
    std::clock_t start1 = std::clock();

    // here we need to determine the column count and composition

    queue<string> op_v(op_value);
    queue<string> op_vx;
    set<string> field_names;
    map<string,string> aliases;
    string tt;
	
    while(!op_v.empty()) {
        if(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) != a->columnNames.end()) {
            tt = op_v.front();
            if(!op_v.empty()) {
                op_v.pop();
                if(!op_v.empty()) {
                    if(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) == a->columnNames.end()) {
                        if(aliases.count(tt) == 0) {
                            aliases[tt] = op_v.front();
                        };
                    }
                    else {
                        if (!op_v.empty()) {
                            while(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) == a->columnNames.end())
                                op_v.pop();
                        };
                    };
                };
            };
        };
        if(!op_v.empty())
            op_v.pop();
    };

    op_v = op_value;
    while(!op_v.empty()) {
        if(std::find(a->columnNames.begin(), a->columnNames.end(), op_v.front()) != a->columnNames.end()) {
            field_names.insert(op_v.front());
        };
        op_v.pop();
    };

    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it)  {
        op_vx.push(*it);
    };

    // find out how many columns a new set will have
    queue<string> op_t(op_type);
    int_type col_count = 0;

    for(int i=0; !op_t.empty(); ++i, op_t.pop())
        if((op_t.front()).compare("emit sel_name") == 0)
            col_count++;

    CudaSet *b, *c;

    if(a->segCount <= 1)
        setSegments(a, op_vx);
    allocColumns(a, op_vx);
	
    unsigned int cycle_count;
    if(a->filtered)
        cycle_count = varNames[a->source_name]->segCount;
    else
        cycle_count = a->segCount;

    size_t ol_count = a->mRecCount, cnt;
    a->hostRecCount = a->mRecCount;
    b = new CudaSet(0, col_count);
	b->name = "tmp b in select";
    bool b_set = 0, c_set = 0;

    //size_t tmp_size = a->mRecCount;
    //if(a->segCount > 1)
    //    tmp_size = a->maxRecs;

    vector<thrust::device_vector<int_type> > distinct_val; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_hash; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_tmp;

   /* for(unsigned int i = 0; i < distinct_cnt; i++) {
        distinct_tmp.push_back(thrust::device_vector<int_type>(tmp_size));
        distinct_val.push_back(thrust::device_vector<int_type>());
        distinct_hash.push_back(thrust::device_vector<int_type>());
    };
	*/

// find out how many string columns we have. Add int_type columns to store string hashes for sort/groupby ops.
    stack<string> op_s = op_v2;

    while(!op_s.empty()) {
        if (a->type[op_s.top()] == 2) {
            a->d_columns_int[op_s.top()] = thrust::device_vector<int_type>(a->maxRecs);
        };
        op_s.pop();
    };

    bool one_liner;
	
	
    for(unsigned int i = 0; i < cycle_count; i++) {          // MAIN CYCLE
		if(verbose)
			cout << "segment " << i << " select mem " << getFreeMem() << endl;
		std::clock_t start3 = std::clock();		
				
        cnt = 0;		
        copyColumns(a, op_vx, i, cnt);
		//std::cout<< "cpy time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';			
        op_s = op_v2;

        while(!op_s.empty() && a->mRecCount != 0 && a->not_compressed) {

			if (a->type[op_s.top()] == 2) {
                a->d_columns_int[op_s.top()].resize(0);			
                a->add_hashed_strings(op_s.top(), i);	
            };
            op_s.pop();
        };
		

        if(a->mRecCount) {
            if (ll != 0) {
				start3 = std::clock();		
                order_inplace(a, op_v2, field_names, 1);
				//std::cout<< "order time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';	
				//start3 = std::clock();		
                a->GroupBy(op_v2);				
				//std::cout<< "grp time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';	
				//start3 = std::clock();		
            };
			
			
			//cout << "select time " << endl;
            select(op_type,op_value,op_nums, op_nums_f,a,b, distinct_tmp, one_liner);
			//std::cout<< "sel time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';	
			
			
					
			if(i == 0)
				std::reverse(b->columnNames.begin(), b->columnNames.end());
			
            if(!b_set) {
                b_set = 1;
                unsigned int old_cnt = b->mRecCount;
                b->mRecCount = 0;
                b->resize(a->maxRecs);
                b->mRecCount = old_cnt;
            };		
			

            if (!c_set && b->mRecCount > 0) {
                c = new CudaSet(0, col_count);
                create_c(c,b);				
                c_set = 1;
				c->name = s;
            };
			
            if (ll != 0 && cycle_count > 1  && b->mRecCount > 0) {
                add(c,b,op_v3, aliases, distinct_tmp, distinct_val, distinct_hash, a);						
            }
            else {
                //copy b to c
                unsigned int c_offset = c->mRecCount;
                c->resize(b->mRecCount);				
				
                for(unsigned int j=0; j < b->columnNames.size(); j++) {
                    if (b->type[b->columnNames[j]] == 0) {
                        thrust::copy(b->d_columns_int[b->columnNames[j]].begin(), b->d_columns_int[b->columnNames[j]].begin() + b->mRecCount, c->h_columns_int[b->columnNames[j]].begin() + c_offset);
                    }
                    else if (b->type[b->columnNames[j]] == 1) {
                        thrust::copy(b->d_columns_float[b->columnNames[j]].begin(), b->d_columns_float[b->columnNames[j]].begin() + b->mRecCount, c->h_columns_float[b->columnNames[j]].begin() + c_offset);
                    }
                    else {
                        hipMemcpy((void*)(thrust::raw_pointer_cast(c->h_columns_char[b->columnNames[j]] + b->char_size[b->columnNames[j]]*c_offset)), (void*)thrust::raw_pointer_cast(b->d_columns_char[b->columnNames[j]]),
                                   b->char_size[b->columnNames[j]] * b->mRecCount, hipMemcpyDeviceToHost);
                    };
                };				
            };
        };		
		//std::cout<< "cycle sel time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';	
    };
	
    a->mRecCount = ol_count;
    a->mRecCount = a->hostRecCount;
    a->deAllocOnDevice();
    b->deAllocOnDevice();
	
    if(!c_set) {
        CudaSet *c;
        c = new CudaSet(0,1);
        varNames[s] = c;
		c->name = s;
        clean_queues();
        return;
    };	
	
    if (ll != 0) {
        count_avg(c, distinct_hash);
    }
    else {
        if(one_liner) {
            count_simple(c);
        };
    };
	
    c->maxRecs = c->mRecCount;
    c->name = s;
    c->keep = 1;
	//cout << "select res " << c->mRecCount << endl;


    clean_queues();
	
    varNames[s] = c;
    b->free();
    varNames[s]->keep = 1;
	

    if(stat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };
	

    if(stat[f] == statement_count && a->keep == 0) {
        a->free();
        varNames.erase(f);		
    };
	if(verbose)
		std::cout<< "select time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
}


void emit_insert(char *f, char* s) {
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(2, "Delete : couldn't find variable " + string(f));
        };
        if (stat.find(s) == stat.end() && data_dict.count(s) == 0) {
            process_error(2, "Delete : couldn't find variable " + string(s) );
        };		
		check_used_vars();			
        stat[f] = statement_count;
		stat[s] = statement_count;
        clean_queues();
        return;
    };

	
    if(varNames.find(f) == varNames.end() || varNames.find(s) == varNames.end()) {
        clean_queues();
        return;
    };
	
	if(verbose)
		cout << "INSERT " << f << " " << s << endl;	
	insert_records(f,s);	
    clean_queues();


};

void emit_delete(char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end()  && data_dict.count(f) == 0) {
            process_error(2, "Delete : couldn't find variable " + string(f));
        };
        stat[f] = statement_count;
		check_used_vars();
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };

	delete_records(f);	
    cout << "DELETE " << f <<  endl;	
	clean_queues();
	
}	

void emit_case()
{
	op_case = 1;
	if (scan_state == 1)
		cout << "emit case " << endl;
	//extract releveant values and pass to modified filter	
	// get a bool vector back
/*						while(!op_type.empty())
						{
						cout << "CASE type " << op_type.front() << endl;
						op_type.pop();
						}				
*/						
}

void emit_display(char *f, char* sep)
{
   statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(2, "Filter : couldn't find variable " + string(f) );
        };
        stat[f] = statement_count;
		//check_used_vars();
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };

    CudaSet* a = varNames.find(f)->second;
    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };
	
    //a->Store("",sep, limit, 0, 1);
    a->Display(limit, 0, 1);
	if(verbose)
		cout << "DISPLAY " << f << endl;

	clean_queues();
    if(stat[f] == statement_count  && a->keep == 0) {
        a->free();
        varNames.erase(f);
    };
	
}


void emit_filter(char *s, char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end() && data_dict.count(f) == 0) {
            process_error(1, "Filter : couldn't find variable " + string(f));
            //cout << "Filter : couldn't find variable " << f << endl;
            //exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;		
		check_used_vars();
        clean_queues();
        return;
    };


    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;
	
    if(a->mRecCount == 0 && !a->filtered) {
        b = new CudaSet(0,1);
    }
    else {
		if(verbose)
			cout << "INLINE FILTER " << f << endl;
        b = a->copyDeviceStruct();
        b->name = s;
        b->sorted_fields = a->sorted_fields;
        b->presorted_fields = a->presorted_fields;
        //save the stack
        b->fil_s = s;
        b->fil_f = f;
        b->fil_type = op_type;
		
        b->fil_value = op_value;
        b->fil_nums = op_nums;
        b->fil_nums_f = op_nums_f;
        b->filtered = 1;
		if(a->filtered) {
		
			b->source_name = a->source_name;
			b->fil_f = a->fil_f;
			while(!a->fil_value.empty()) {
				b->fil_value.push(a->fil_value.front());
				a->fil_value.pop();
			};
			
			while(!a->fil_type.empty()) {
				b->fil_type.push(a->fil_type.front());
				a->fil_type.pop();
			};
			b->fil_type.push("AND");			
			
			while(!a->fil_nums.empty()) {
				b->fil_nums.push(a->fil_nums.front());
				a->fil_nums.pop();
			};

			while(!a->fil_nums_f.empty()) {
				b->fil_nums_f.push(a->fil_nums_f.front());
				a->fil_nums_f.pop();
			};
			a->filtered = 0;
			//a->free();	
			varNames.erase(f);
		}	
		else	
			b->source_name = f;
		b->maxRecs = a->maxRecs;
        b->prm_d.resize(a->maxRecs);		
    };
    clean_queues();
	

    if (varNames.count(s) > 0)
        varNames[s]->free();
    varNames[s] = b;

    if(stat[s] == statement_count) {
        b->free();
        varNames.erase(s);
    };
}

void emit_store(char *s, char *f, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end() && data_dict.count(s) == 0) {
            process_error(2, "Store : couldn't find variable " + string(s) );
        };
        stat[s] = statement_count;
		//check_used_vars();
		clean_queues();
        return;
    };

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;
	if(verbose)
		cout << "STORE: " << s << " " << f << " " << sep << endl;

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };

    a->Store(f,sep, limit, 0);

    if(stat[s] == statement_count  && a->keep == 0) {
        a->free();
        varNames.erase(s);
    };
};


void emit_store_binary(char *s, char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end() && data_dict.count(s) == 0) {
            process_error(2, "Store : couldn't find variable " + string(s));
        };
        stat[s] = statement_count;
		//check_used_vars();
		clean_queues();
        return;
    };

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;

    if(stat[f] == statement_count)
        a->deAllocOnDevice();

    printf("STORE: %s %s \n", s, f);

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };
    total_count = 0;
    total_segments = 0;
	
    if(fact_file_loaded) {
        a->Store(f,"", limit, 1);
    }
    else {
		FILE* file_p;
		if(a->text_source) {
			file_p = fopen(a->load_file_name.c_str(), "r");
		    if (file_p  == NULL) {
				process_error(2, "Could not open file " + a->load_file_name );
			};
		};

        while(!fact_file_loaded) {
			if(verbose)
				cout << "LOADING " << a->load_file_name << " mem: " << getFreeMem() << endl;
            if(a->text_source)
                fact_file_loaded = a->LoadBigFile(file_p);
            a->Store(f,"", limit, 1);
        };
    };
	a->writeSortHeader(f);

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(s);
    };

};


void emit_load_binary(const char *s, const char *f, int d)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };

	if(verbose)
		printf("BINARY LOAD: %s \n", s, f);

    CudaSet *a;
    unsigned int segCount, maxRecs;
    string f1(f);
    f1 += "." + namevars.front() + ".header";

    FILE* ff = fopen(f1.c_str(), "rb");
    if(ff == NULL) {
         process_error(2, "Couldn't open file " + f1);
    };
	size_t totRecs;
    fread((char *)&totRecs, 8, 1, ff);
    fread((char *)&segCount, 4, 1, ff);
    fread((char *)&maxRecs, 4, 1, ff);
    fclose(ff);

	if(verbose)
		cout << "Reading " << totRecs << " records" << endl;

	a = new CudaSet(namevars, typevars, sizevars, cols, totRecs, f, maxRecs);
    a->segCount = segCount;    
    a->keep = 1;
	a->name = s;
    varNames[s] = a;
	for(unsigned int i = 0; i < segCount; i++)
		a->orig_segs[f].insert(i); 

    if(stat[s] == statement_count )  {
        a->free();
        varNames.erase(s);
    };
}


void emit_load(char *s, char *f, int d, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };

    printf("LOAD: %s %s %d  %s \n", s, f, d, sep);

    CudaSet *a;

    a = new CudaSet(namevars, typevars, sizevars, cols, process_count, references, references_names);
    a->mRecCount = 0;
    a->resize(process_count);
    a->keep = true;
    a->not_compressed = 1;
    a->load_file_name = f;
	a->separator = sep;
    a->maxRecs = a->mRecCount;
    a->segCount = 0;
    varNames[s] = a;
    fact_file_loaded = 0;

    if(stat[s] == statement_count)  {
        a->free();
        varNames.erase(s);
    };
}

void emit_show_tables()
{
	if (scan_state == 1) {
		for ( map<string, map<string, col_data> >::iterator it=data_dict.begin() ; it != data_dict.end(); ++it ) {		
			cout << (*it).first << endl;
		};
	};
		
    return;
}

void emit_drop_table(char* table_name)
{
	if (scan_state == 1) {
	
		map<string, map<string, col_data> >::iterator iter;
		if((iter = data_dict.find(table_name)) != data_dict.end()) {	
			map<string, col_data> s = (*iter).second;			
			for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {	
				int seg = 0;
				string f_name = (*iter).first + "." + (*it).first + "." + int_to_string(seg);				
				while(!remove(f_name.c_str())) {
					seg++;					
					f_name = (*iter).first + "." + (*it).first + "." + int_to_string(seg);														
				};	
				f_name = (*iter).first + "." + (*it).first + ".header";	
				remove(f_name.c_str());				
			};						
		};	
		string s_name = (*iter).first + ".presort";				
		remove(s_name.c_str());
		s_name = (*iter).first + ".sort";				
		remove(s_name.c_str());
	
		if(data_dict.find(table_name) != data_dict.end()) {	
			data_dict.erase(table_name);
		};
		save_dict = 1;
	};
		
    return;
}


void emit_describe_table(char* table_name)
{
	if (scan_state == 1) {
		map<string, map<string, col_data> >::iterator iter;
		if((iter = data_dict.find(table_name)) != data_dict.end()) {	
			map<string, col_data> s = (*iter).second;
			for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {	
				if ((*it).second.col_type == 0) {
					cout << (*it).first << " integer" << endl;
				}	
				else if ((*it).second.col_type == 1) {
					cout << (*it).first << " float" << endl;
				}
				else if ((*it).second.col_type == 3) {
					cout << (*it).first << " decimal" << endl;
				}				
				else {
					cout << (*it).first << " char(" << (*it).second.col_length << ")" << endl;
				};	
			};
		};
	};
		
    return;
}



void yyerror(char *s, ...)
{
    extern int yylineno;
    va_list ap;
    va_start(ap, s);

    fprintf(stderr, "%d: error: ", yylineno);
    vfprintf(stderr, s, ap);
    fprintf(stderr, "\n");
}

void clean_queues()
{
    while(!op_type.empty()) op_type.pop();
    while(!op_value.empty()) op_value.pop();
    while(!op_join.empty()) op_join.pop();
    while(!op_nums.empty()) op_nums.pop();
    while(!op_nums_f.empty()) op_nums_f.pop();
    while(!j_col_count.empty()) j_col_count.pop();
    while(!namevars.empty()) namevars.pop();
    while(!typevars.empty()) typevars.pop();
    while(!sizevars.empty()) sizevars.pop();
    while(!cols.empty()) cols.pop();
    while(!op_sort.empty()) op_sort.pop();
    while(!references.empty()) references.pop();
    while(!references_names.empty()) references_names.pop();
    while(!op_presort.empty()) op_presort.pop();

	op_case = 0;
    sel_count = 0;
    join_cnt = 0;
    join_col_cnt = 0;
    distinct_cnt = 0;
    join_tab_cnt = 0;
    tab_cnt = 0;
    join_and_cnt.clear();
}

void load_vars()
{
	if(used_vars.size() == 0) {
		//cout << "Error, no valid column names have been found " << endl;
		//exit(0);
	}
	else {
		for ( map<string, map<string, bool> >::iterator it=used_vars.begin() ; it != used_vars.end(); ++it ) {
			
			while(!namevars.empty()) namevars.pop();
			while(!typevars.empty()) typevars.pop();
			while(!sizevars.empty()) sizevars.pop();
			while(!cols.empty()) cols.pop();
			if(stat.count((*it).first) != 0) {
				map<string, bool> c = (*it).second;
				for ( map<string, bool>::iterator sit=c.begin() ; sit != c.end(); ++sit ) {
					//cout << "name " << (*sit).first << endl;
					namevars.push((*sit).first);
					if(data_dict[(*it).first][(*sit).first].col_type == 0)
						typevars.push("int");
					else if(data_dict[(*it).first][(*sit).first].col_type == 1)
						typevars.push("float");
					else if(data_dict[(*it).first][(*sit).first].col_type == 3)
						typevars.push("decimal");	
					else typevars.push("char");	
					sizevars.push(data_dict[(*it).first][(*sit).first].col_length);
					cols.push(0);				
				};			
				emit_load_binary((*it).first.c_str(), (*it).first.c_str(), 0);
			};		
		};
	};	
}

int execute_file(int ac, char **av)
{
	cout << "execute_file->start" << endl;
bool just_once  = 0;
string script;

    process_count = 6200000;
    verbose = 0;
	total_buffer_size = 0;
	
    for (int i = 1; i < ac; i++) {
        if(strcmp(av[i],"-l") == 0) {
            process_count = atoff(av[i+1]);
        }
        else if(strcmp(av[i],"-v") == 0) {
            verbose = 1;
        }
        else if(strcmp(av[i],"-i") == 0) {
            interactive = 1;
			break;
        }
        else if(strcmp(av[i],"-s") == 0) {
            just_once = 1;
			interactive = 1;
			script = av[i+1];
        };		
    };

	load_col_data(data_dict, "data.dictionary");
		
    if (!interactive) {
        if((yyin = fopen(av[ac-1], "r")) == NULL) {
            perror(av[ac-1]);
            exit(1);
        };

        if(yyparse()) {
            printf("SQL scan parse failed\n");
            exit(1);
        };
		
		//exit(0);

        scan_state = 1;
        std::clock_t start1 = std::clock();
		
		load_vars();
		
        statement_count = 0;
        clean_queues();

        yyin = fopen(av[ac-1], "r");
        PROC_FLUSH_BUF ( yyin );
        statement_count = 0;

        extern FILE *yyin;		
        context = CreateCudaDevice(0, av, verbose);	
        hash_seed = 100;

        if(!yyparse()) {
            if(verbose)
            cout << "SQL scan parse worked " << endl;
        }
        else
            cout << "SQL scan parse failed" << endl;

        fclose(yyin);
        for (map<string,CudaSet*>::iterator it=varNames.begin() ; it != varNames.end(); ++it ) {
            (*it).second->free();
        };

        if(verbose) {
            cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        };
    }
    else {        
        context = CreateCudaDevice(0, av, verbose);
        hash_seed = 100;
		if(!just_once)
			getline(cin, script);		

        while (script != "exit" && script != "EXIT") {

			used_vars.clear();
            yy_scan_string(script.c_str());
            scan_state = 0;
            statement_count = 0;
            clean_queues();
            if(yyparse()) {
                printf("SQL scan parse failed \n");
                getline(cin, script);
                continue;
            };

            scan_state = 1;
			
			load_vars();
			
            statement_count = 0;
            clean_queues();
            yy_scan_string(script.c_str());
            std::clock_t start1 = std::clock();
        
            if(!yyparse()) {
                if(verbose)
                    cout << "SQL scan parse worked " <<  endl;
            };
            for (map<string,CudaSet*>::iterator it=varNames.begin() ; it != varNames.end(); ++it ) {
                (*it).second->free();
            };
            varNames.clear();

            if(verbose) {
                cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
            };
			if(!just_once)
				getline(cin, script);
			else
				script = "exit";
        };
		
		while(!buffer_names.empty()) { 
			delete [] buffers[buffer_names.front()];
			buffer_sizes.erase(buffer_names.front());
			buffers.erase(buffer_names.front());
			buffer_names.pop();
		};
	
    };
	if(save_dict)
		save_col_data(data_dict,"data.dictionary");
		
    if(alloced_sz) {
        hipFree(alloced_tmp);
        alloced_sz = 0;
    };
    if(raw_decomp_length) {
        hipFree(raw_decomp);
        raw_decomp_length = 0;
    };	
		
    return 0;
}



//external c global to report errors
//char alenka_err[4048];

int alenkaExecute(char *s)
{
YY_BUFFER_STATE bp;

    total_buffer_size = 0;
    scan_state = 0;
    load_col_data(data_dict, "data.dictionary");
    std::clock_t start;

        if(verbose)
            start = std::clock();
        bp = yy_scan_string(s);
        yy_switch_to_buffer(bp);
        int ret = yyparse();
        //printf("execute: returned [%d]\n", ret);
        if(!ret) {
                if(verbose)
                        cout << "SQL scan parse worked" << endl;
        }

        scan_state = 1;
        load_vars();
        statement_count = 0;
        clean_queues();
        bp = yy_scan_string(s);
        yy_switch_to_buffer(bp);
        if(!yyparse()) {
            if(verbose)
            cout << "SQL scan parse worked " << endl;
        }
        else
            cout << "SQL scan parse failed" << endl;

        yy_delete_buffer(bp);

	// Clear Vars
        for (map<string,CudaSet*>::iterator it=varNames.begin() ; it != varNames.end(); ++it ) {
            (*it).second->free();
        };
        varNames.clear();

        if(verbose)
                cout<< "statement time " <<  ( ( std::clock() - start ) / (double)CLOCKS_PER_SEC ) << endl;
	if(save_dict)
		save_col_data(data_dict,"data.dictionary");
        return ret;
}




void process_error(int severity, string err) {
    switch (severity) {
        case 1: err = "(Warning) " + err;
           break;
        case 2: err = "(Fatal) " + err;
           break;
        default:
             err = "(Aborting) " + err;
           break;
    }
    error_cb(severity, err.c_str());            // send the error to the c based callback
}

void alenkaInit(char ** av)
{
        process_count = 6200000;
        verbose = 0;
        scan_state = 1;
        statement_count = 0;
        clean_queues();
        context = CreateCudaDevice(0, av, true);
        printf("Alenka initialised\n");
}

void alenkaClose()
{
        statement_count = 0;
        hash_seed = 100;

        if(alloced_sz)
                hipFree(alloced_tmp);
}

extern "C"
int alenka_JDBC(char *s) {
		//turn on debugging
		verbose = 1;

        //Load the data dictionary

        load_col_data(data_dict, "data.dictionary");

        //Read the SQL into internal buffers
        yy_scan_string(s);

        //initialize variables
        hash_seed = 100;
        scan_state = 0;
        statement_count = 0;
        clean_queues();

        //Parse the query
        if (yyparse()) {
                printf("SQL scan parse failed \n");
                return 1;
        };

        //Reset variables for execution
        scan_state = 1;
        load_vars();
        statement_count = 0;
        clean_queues();

        yy_scan_string(s);
        std::clock_t start1 = std::clock();

        //Excecute Query
        if (yyparse()) {
                printf("SQL scan parse failed \n");
                return 1;
        }

        if (verbose) {
                cout << "cycle time "
                                << ((std::clock() - start1) / (double) CLOCKS_PER_SEC) << endl;
        };

        return 0;
}

/*
 * Is use to initialize the resultset and prepare what we need to iterate through the results.
 * Passed in string is the name of the variable that we need the result set for.  Probably
 * not the best way but should work for now.
*/
extern "C" CudaSet* initializeResultSet_JDBC(char *f) {
        //Can only get the resultset if we are in scan_state 1
        if (scan_state == 0) {
                process_error(1, "Unable to get resultset in current scan_state");
                return NULL;
        }

        if(varNames.find(f) == varNames.end()) {
                process_error(1, "Unable to find CudaSet");
                clean_queues();
                return NULL;
        }

        CudaSet* cs = varNames.find(f)->second;
        cout << "Found CudaSet for variable " << f << endl;
        return cs;
}

extern "C" void resultSetClose_JDBC() {
        //Clean Up variables
        for (map<string, CudaSet*>::iterator it = varNames.begin();
                        it != varNames.end(); ++it) {
                (*it).second->free();
        };
        varNames.clear();
}











