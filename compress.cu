#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


// PFOR and PFOR-DELTA Compression and decompression routines

#include <stdio.h>
#include <iomanip>
#include <thrust/extrema.h>
#include "cm.h"

using namespace std;

unsigned long long int* raw_decomp = NULL;
unsigned int raw_decomp_length = 0;

std::map<string, unsigned int> cnt_counts;
string curr_file;

struct int64_to_char
{
   __host__ __device__
    char operator()(const int_type x)
    {
        return (char)x;
    }
};

struct char_to_int64
{
   __host__ __device__
    int_type operator()(const char x)
    {
        return (int_type)x;
    }
};


struct int64_to_int16
{
   __host__ __device__
    unsigned short int operator()(const int_type x)
    {
        return (unsigned short int)x;
    }
};

struct int16_to_int64
{
   __host__ __device__
    int_type operator()(const unsigned short int x)
    {
        return (int_type)x;
    }
};


struct int64_to_int32
{
   __host__ __device__
    unsigned int operator()(const int_type x)
    {
        return (unsigned int)x;
    }
};

struct int32_to_int64
{
   __host__ __device__
    int_type operator()(const unsigned int x)
    {
        return (int_type)x;
    }
};



struct bool_to_int
{
    __host__ __device__
    unsigned int operator()(const bool x)
    {
        return (unsigned int)x;
    }
};

struct ui_to_ll
{
    __host__ __device__
    long long int operator()(const unsigned int x)
    {
        return (long long int)x;
    }
};


struct compress_functor_int
{

    const int_type * source;
    unsigned long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;


    compress_functor_int(const int_type * _source, unsigned long long int  * _dest,
                         const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        long long int val = source[i] - start_val[0];
        unsigned int shifted = vals[2] - vals[0] - (i%vals[1])*vals[0];
        dest[i] = val << shifted;
    }
};

struct compress_functor_float
{

    const long long int * source;
    unsigned long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;


    compress_functor_float(const long long int * _source, unsigned long long int  * _dest,
                           const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        long long int val;

        unsigned int bits = vals[0];
        unsigned int fit_count = vals[1];
        unsigned int int_sz = vals[2];

        val = source[i] - start_val[0];
        unsigned int z = i%fit_count;

        unsigned int shifted = int_sz - bits - z*bits;
        dest[i] = val << shifted;
    }
};



struct decompress_functor_int
{

    const unsigned long long int * source;
    int_type * dest;
    const long long int * start_val;
    const unsigned int * vals;


    decompress_functor_int(const unsigned long long int * _source, int_type * _dest,
                           const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned long long int tmp = source[i/vals[1]]  >> (vals[2] - vals[0] - (i%vals[1])*vals[0]);
        // set  the rest of bits to 0
        tmp	= tmp << (vals[2] - vals[0]);
        tmp	= tmp >> (vals[2] - vals[0]);
        dest[i] = tmp + start_val[0];

    }
};




struct decompress_functor_str
{

    const unsigned long long  * source;
    unsigned int * dest;
    const unsigned int * vals;


    decompress_functor_str(const unsigned long long int * _source, unsigned int * _dest,
                           const unsigned int * _vals):
        source(_source), dest(_dest), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned int bits = vals[0];
        unsigned int fit_count = vals[1];
        unsigned int int_sz = 64;

        //find the source index
        unsigned int src_idx = i/fit_count;
        // find the exact location
        unsigned int src_loc = i%fit_count;
        //right shift the values
        unsigned int shifted = ((fit_count-src_loc)-1)*bits;
        unsigned long long int tmp = source[src_idx]  >> shifted;
        // set  the rest of bits to 0
        tmp	= tmp << (int_sz - bits);
        tmp	= tmp >> (int_sz - bits);
        dest[i] = tmp;

    }
};




size_t pfor_decompress(void* destination, void* host, void* d_v, void* s_v)
{
    unsigned int bits, cnt, fit_count, orig_recCount;
    long long int  orig_lower_val;
    unsigned int bit_count = 64;
    unsigned int comp_type;
    long long int start_val;

    cnt = ((unsigned int*)host)[0];
    orig_recCount = ((unsigned int*)((char*)host + cnt))[7];
    bits = ((unsigned int*)((char*)host + cnt))[8];
    orig_lower_val = ((long long int*)((unsigned int*)((char*)host + cnt) + 9))[0];
    fit_count = ((unsigned int*)((char*)host + cnt))[11];
    start_val = ((long long int*)((unsigned int*)((char*)host + cnt) + 12))[0];
    comp_type = ((unsigned int*)((char*)host + cnt))[14];
	comp_type = ((unsigned int*)host)[5];

    //cout << "Decomp Header " <<  orig_recCount << " " << bits << " " << orig_lower_val << " " << cnt << " " << fit_count << " " << comp_type << endl;

    if(raw_decomp_length < cnt) {
        if(raw_decomp != NULL) {
            hipFree(raw_decomp);
        };
        hipMalloc((void **) &raw_decomp, cnt);
        raw_decomp_length = cnt;
    };

    hipMemcpy( (void*)raw_decomp, (void*)((unsigned int*)host + 6), cnt, hipMemcpyHostToDevice);
	thrust::device_ptr<int_type> d_int((int_type*)destination);
  
	if(comp_type == 1) {
		thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
		thrust::device_ptr<long long int> dd_sv((long long int*)s_v);

		dd_sv[0] = orig_lower_val;
		dd_v[0] = bits;
		dd_v[1] = fit_count;
		dd_v[2] = bit_count;

		thrust::counting_iterator<unsigned int> begin(0);
		decompress_functor_int ff1(raw_decomp,(int_type*)destination, (long long int*)s_v, (unsigned int*)d_v);
		thrust::for_each(begin, begin + orig_recCount, ff1);
		
		d_int[0] = start_val;
        thrust::inclusive_scan(d_int, d_int + orig_recCount, d_int);
	}
	else {
		if(bits == 8) {
			thrust::device_ptr<char> src((char*)raw_decomp);
			thrust::transform(src, src+orig_recCount, d_int, char_to_int64());
		}	
		else if(bits == 16) {
			thrust::device_ptr<unsigned short int> src((unsigned short int*)raw_decomp);
			thrust::transform(src, src+orig_recCount, d_int, int16_to_int64());
		}
		else if(bits == 32) {
			thrust::device_ptr<unsigned int> src((unsigned int*)raw_decomp);
			thrust::transform(src, src+orig_recCount, d_int, int32_to_int64());
		}
		else {
			thrust::device_ptr<int_type> src((int_type*)raw_decomp);
			thrust::copy(src, src+orig_recCount, d_int);
		};
		thrust::constant_iterator<int_type> iter(orig_lower_val);
		thrust::transform(d_int, d_int+orig_recCount, iter, d_int, thrust::plus<int_type>());
		
	};	

    return orig_recCount;
}


template< typename T>
void pfor_delta_compress(void* source, size_t source_len, string file_name, thrust::host_vector<T, pinned_allocator<T> >& host, bool tp)
{
    long long int orig_lower_val, orig_upper_val, start_val, real_lower, real_upper;
    unsigned int  bits, recCount;
    unsigned int bit_count = 8*8;
    unsigned int fit_count;
    unsigned int comp_type = 1; // FOR-DELTA

    if(tp == 0)
        recCount = source_len/int_size;
    else
        recCount = source_len/float_size;

    void* ss;
    CUDA_SAFE_CALL(hipMalloc((void **) &ss, recCount*float_size));

    if (tp == 0) {
        thrust::device_ptr<int_type> s((int_type*)source);
        thrust::device_ptr<int_type> d_ss((int_type*)ss);
        thrust::adjacent_difference(s, s+recCount, d_ss);
		
        start_val = d_ss[0];
        if(recCount > 1)
            d_ss[0] = d_ss[1];

        orig_lower_val = *(thrust::min_element(d_ss, d_ss + recCount));
        orig_upper_val = *(thrust::max_element(d_ss, d_ss + recCount));

        real_lower = s[0];
        real_upper = s[recCount-1];
        //cout << "orig " << orig_upper_val << " " <<  orig_lower_val << endl;
        //cout << "We need for delta " << (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1))) << " bits to encode " <<  orig_upper_val-orig_lower_val << " values " << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1)));
        if (bits == 0)
            bits = 1;

    }
    else {
        thrust::device_ptr<long long int> s((long long int*)source);
        thrust::device_ptr<long long int> d_ss((long long int*)ss);
        thrust::adjacent_difference(s, s+recCount, d_ss);		
        start_val = d_ss[0];
        if(recCount > 1)
            d_ss[0] = d_ss[1];

        orig_lower_val = *(thrust::min_element(d_ss, d_ss + recCount));
        orig_upper_val = *(thrust::max_element(d_ss, d_ss + recCount));
        real_lower = s[0];
        real_upper = s[recCount-1];

        //cout << "orig " << orig_upper_val << " " <<  orig_lower_val << endl;
        //cout << "We need for delta " << (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1))) << " bits to encode " << orig_upper_val-orig_lower_val << " values" << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1)));
        if (bits == 0)
            bits = 1;
    };

    thrust::counting_iterator<unsigned int> begin(0);

    fit_count = bit_count/bits;
    void* d_v1;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_v1, 12));
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v1);

    void* s_v1;
    CUDA_SAFE_CALL(hipMalloc((void **) &s_v1, 8));
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v1);

    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;


    //void* d;
    //CUDA_SAFE_CALL(hipMalloc((void **) &d, recCount*float_size));

    thrust::device_ptr<char> dd((char*)source);
    thrust::fill(dd, dd+source_len,0);

    //cout << "FF " << orig_lower_val << " " << bits << " " << fit_count << " " << bit_count << endl;

    if (tp == 0) {
        compress_functor_int ff((int_type*)ss,(unsigned long long int*)source, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    }
    else {
        compress_functor_float ff((long long int*)ss,(unsigned long long int*)source, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    };


    thrust::device_ptr<unsigned long long int> s_copy1((unsigned long long int*)source);

    // make an addition  sequence

    thrust::device_ptr<unsigned long long int> add_seq((unsigned long long int*)ss);
    thrust::constant_iterator<unsigned long long int> iter(fit_count);
    thrust::sequence(add_seq, add_seq + recCount, 0, 1);
    thrust::transform(add_seq, add_seq + recCount, iter, add_seq, thrust::divides<unsigned long long int>());

    unsigned int cnt = (recCount)/fit_count;
    if (recCount%fit_count > 0)
        cnt++;

    thrust::device_ptr<unsigned long long int> fin_seq = thrust::device_malloc<unsigned long long int>(cnt);

    thrust::reduce_by_key(add_seq, add_seq+recCount,s_copy1,thrust::make_discard_iterator(),
                          fin_seq);

    // copy fin_seq to host
    unsigned long long int * raw_src = thrust::raw_pointer_cast(fin_seq);
	
	//cout << file_name << " CNT  " << cnt << " " << recCount << endl;
	cnt = cnt*8;

    hipMemcpy( host.data(), (void *)raw_src, cnt, hipMemcpyDeviceToHost);
    fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&cnt, 4);
    binary_file.write((char *)&real_lower, 8);
    binary_file.write((char *)&real_upper, 8);
	binary_file.write((char *)&comp_type, 4);
    binary_file.write((char *)host.data(),cnt);    
    binary_file.write((char *)&cnt, 4);
    binary_file.write((char *)&recCount, 4);
    binary_file.write((char *)&bits, 4);
    binary_file.write((char *)&orig_lower_val, 8);
    binary_file.write((char *)&fit_count, 4);
    binary_file.write((char *)&start_val, 8);
    binary_file.close();
    if(cnt_counts[curr_file] < cnt)
        cnt_counts[curr_file] = cnt;	
	
    thrust::device_free(fin_seq);
    hipFree(ss);
    hipFree(d_v1);
    hipFree(s_v1);
}


// non sorted compressed fields should have 1,2,4 or 8 byte values for direct operations on compressed values
template< typename T>
void pfor_compress(void* source, size_t source_len, string file_name, thrust::host_vector<T, pinned_allocator<T> >& host,  bool tp)
{
    unsigned int recCount;
    long long int orig_lower_val;
    long long int orig_upper_val;
    unsigned int  bits;
    unsigned int fit_count = 0;
    unsigned int comp_type = 0; // FOR
    long long int start_val = 0;
    bool sorted = 0;

    // check if sorted

    if (tp == 0) {
        recCount = source_len/int_size;
        thrust::device_ptr<int_type> s((int_type*)source);
        sorted = thrust::is_sorted(s, s+recCount);
    }
    else {
        recCount = source_len/float_size;
        thrust::device_ptr<long long int> s((long long int*)source);
        sorted = thrust::is_sorted(s, s+recCount);		
    };
    //cout << "file " << file_name << " is sorted " << sorted << endl;

    if(sorted) {
        pfor_delta_compress(source, source_len, file_name, host, tp);
		return;
	};
	

    if (tp == 0) {
        thrust::device_ptr<int_type> s((int_type*)source);

        orig_lower_val = *(thrust::min_element(s, s + recCount));
        orig_upper_val = *(thrust::max_element(s, s + recCount));

		//cout << "orig " << orig_upper_val << " " <<  orig_lower_val << endl;
        //cout << "We need " << (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1))) << " bits to encode original range of " << orig_lower_val << " to " << orig_upper_val << endl;		
        bits = (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1)));
    }
    else {

        thrust::device_ptr<long long int> s((long long int*)source);

        orig_lower_val = *(thrust::min_element(s, s + recCount));
        orig_upper_val = *(thrust::max_element(s, s + recCount));

        //cout << "We need " << (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1))) << " bits to encode original range of " << orig_lower_val << " to " << orig_upper_val << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1)));
    };
	
	if (bits != 8 && bits != 16 && bits != 32 && bits != 64) {
		if(bits < 8)
			bits = 8;
		else if(bits < 16)
			bits = 16;
		else if(bits < 32)
			bits = 32;
		else if(bits < 64)
			bits = 64;	
	};	
	//cout << "We will really need " << bits << endl;
	
	unsigned int cnt;
	thrust::device_ptr<int_type> s((int_type*)source);
	thrust::constant_iterator<int_type> iter(orig_lower_val);
	thrust::transform(s, s+recCount, iter, s, thrust::minus<int_type>());	
		
	thrust::device_vector<int8_type> d_columns_int8;	
	thrust::device_vector<int16_type> d_columns_int16;	
	thrust::device_vector<int32_type> d_columns_int32;	
	if(bits == 8) {
		d_columns_int8.resize(recCount);
		thrust::transform(s, s+recCount, d_columns_int8.begin(), int64_to_char());	
		hipMemcpy( host.data(), thrust::raw_pointer_cast(d_columns_int8.data()), recCount, hipMemcpyDeviceToHost);
		cnt = recCount;
	}
	else if(bits == 16) {
		d_columns_int16.resize(recCount);
		thrust::transform(s, s+recCount, d_columns_int16.begin(), int64_to_int16());	
		hipMemcpy( host.data(), thrust::raw_pointer_cast(d_columns_int16.data()), recCount*2, hipMemcpyDeviceToHost);
		cnt = recCount*2;
	}
	else if(bits == 32) {
		d_columns_int32.resize(recCount);
		thrust::transform(s, s+recCount, d_columns_int32.begin(), int64_to_int32());	
		hipMemcpy( host.data(), thrust::raw_pointer_cast(d_columns_int32.data()), recCount*4, hipMemcpyDeviceToHost);
		cnt = recCount*4;
	}
	else {
		hipMemcpy( host.data(), (void*)source, recCount*8, hipMemcpyDeviceToHost);
		cnt = recCount*8;
	};

    /*thrust::counting_iterator<unsigned int> begin(0);

    fit_count = bit_count/bits;
    void* d_v1;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_v1, 12));
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v1);

    void* s_v1;
    CUDA_SAFE_CALL(hipMalloc((void **) &s_v1, 8));
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v1);

    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;

    void* d;
    CUDA_SAFE_CALL(hipMalloc((void **) &d, recCount*float_size));
    thrust::device_ptr<char> dd((char*)d);
    thrust::fill(dd, dd+source_len,0);

    if (tp == 0) {
        compress_functor_int ff((int_type*)source,(unsigned long long int*)d, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    }
    else {
        compress_functor_float ff((long long int*)source,(unsigned long long int*)d, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    };


    thrust::device_ptr<unsigned long long int> s_copy1((unsigned long long int*)d);

    // make an addition  sequence
    thrust::device_ptr<unsigned int> add_seq = thrust::device_malloc<unsigned int>(recCount);
    thrust::constant_iterator<unsigned int> iter(fit_count);
    thrust::sequence(add_seq, add_seq + recCount, 0, 1);
    thrust::transform(add_seq, add_seq + recCount, iter, add_seq, thrust::divides<unsigned int>());

    unsigned int cnt = (recCount)/fit_count;
    if(cnt == 0)
        cnt = 1; // need at least 1

    if (recCount%fit_count > 0)
        cnt++;

    //thrust::device_ptr<unsigned long long int> fin_seq = thrust::device_malloc<unsigned long long int>(cnt);
    thrust::device_ptr<unsigned long long int> fin_seq((unsigned long long int*)source);

    thrust::reduce_by_key(add_seq, add_seq+recCount,s_copy1,thrust::make_discard_iterator(),
                          fin_seq);

    // copy fin_seq to host
    unsigned long long int * raw_src = thrust::raw_pointer_cast(fin_seq);

    //cout << file_name << " CNT  " << cnt << " " << recCount << endl;
*/
 
	//cout << "comp Header " <<  recCount << " " << bits << " " << orig_lower_val << " " << cnt << " " << fit_count << " " << comp_type << endl; 
    fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&cnt, 4);
    binary_file.write((char *)&orig_lower_val, 8);
    binary_file.write((char *)&orig_upper_val, 8);
	binary_file.write((char *)&comp_type, 4);
    binary_file.write((char *)host.data(),cnt);    
    binary_file.write((char *)&cnt, 4);
    binary_file.write((char *)&recCount, 4);
    binary_file.write((char *)&bits, 4);
    binary_file.write((char *)&orig_lower_val, 8);
    binary_file.write((char *)&fit_count, 4);
    binary_file.write((char *)&start_val, 8);
    binary_file.close();
    if(cnt_counts[curr_file] < cnt)
        cnt_counts[curr_file] = cnt;
    /*thrust::device_free(add_seq);
    hipFree(d);
    hipFree(d_v1);
    hipFree(s_v1);
	*/
}