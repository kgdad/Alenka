#include "hip/hip_runtime.h"
/*
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */



#include <cctype>
#include <algorithm>
#include <functional>
#include <numeric>
#include "cm.h"
#include "atof.h"
#include "compress.cu"
#include "sorts.cu"
#include "filter.h"
#include "callbacks.h"


#ifdef _WIN64
#define atoll(S) _atoi64(S)
#include <windows.h>
#else
#include <unistd.h>
#endif


using namespace std;
using namespace thrust::placeholders;

size_t total_count = 0, total_max;
clock_t tot;
unsigned int total_segments = 0;
unsigned int process_count;
size_t alloced_sz = 0;
bool fact_file_loaded = 1;
bool verbose;
bool interactive;
void* d_v = NULL;
void* s_v = NULL;
queue<string> op_sort;
queue<string> op_presort;
queue<string> op_type;
bool op_case = 0;
queue<string> op_value;
queue<int_type> op_nums;
queue<float_type> op_nums_f;
queue<string> col_aliases;
map<string, map<string, col_data> > data_dict;
string data_dir;

map<string, char*> buffers;
map<string, size_t> buffer_sizes;
size_t total_buffer_size;
queue<string> buffer_names;

void* alloced_tmp;
bool alloced_switch = 0;

map<string,CudaSet*> varNames; //  STL map to manage CudaSet variables
//map<string,string> setMap; //map to keep track of column names and set names

struct is_match
{
    __host__ __device__
    bool operator()(unsigned int x)
    {
        return x != 4294967295;
    }
};


struct f_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct f_less
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((y-x) > EPSILON);
    }
};

struct f_greater
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON);
    }
};

struct f_greater_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_less_equal
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_not_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON) || ((x-y) < -EPSILON);
    }
};


struct long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (float_type)x;
    }
};


struct l_to_ui
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (unsigned int)x;
    }
};


struct to_zero
{
    __host__ __device__
    bool operator()(const int_type x)
    {
        if(x == -1)
            return 0;
        else
            return 1;
    }
};



struct div_long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x, const float_type y)
    {
        return (float_type)x/y;
    }
};



// trim from start
static inline std::string &ltrim(std::string &s) {
    s.erase(s.begin(), std::find_if(s.begin(), s.end(), std::not1(std::ptr_fun<int, int>(std::isspace))));
    return s;
}

// trim from end
static inline std::string &rtrim(std::string &s) {
    s.erase(std::find_if(s.rbegin(), s.rend(), std::not1(std::ptr_fun<int, int>(std::isspace))).base(), s.end());
    return s;
}

// trim from both ends
static inline std::string &trim(std::string &s) {
    return ltrim(rtrim(s));
}

char *mystrtok(char **m,char *s,const char c)
{
    char *p=s?s:*m;
    if( !*p )
        return 0;
    *m=strchr(p,c);
    if( *m )
        *(*m)++=0;
    else
        *m=p+strlen(p);
    return p;
}


void allocColumns(CudaSet* a, queue<string> fields);
void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt);
void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void write_compressed_char(string file_name, unsigned int index, size_t mCount);
size_t max_tmp(CudaSet* a);
size_t getFreeMem();
char zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a, unsigned int segment);
void filter_op(char *s, char *f, unsigned int segment);
size_t getTotalSystemMemory();
void process_error(int severity, string err);

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, queue<string> &references, queue<string> &references_names)
    : mColumnCount(0), mRecCount(0)
{
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs, references, references_names);
    keep = false;
    source = 1;
    text_source = 1;
    grp = NULL;
    fil_f = NULL;
    fil_s = NULL;
};

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name, unsigned int max)
    : mColumnCount(0),  mRecCount(0)
{
    maxRecs = max;
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs, file_name);
    keep = false;
    source = 1;
    text_source = 0;
    grp = NULL;
    fil_f = NULL;
    fil_s = NULL;
};

CudaSet::CudaSet(size_t RecordCount, unsigned int ColumnCount)
{
    initialize(RecordCount, ColumnCount);
    keep = false;
    source = 0;
    text_source = 0;
    grp = NULL;
    fil_f = NULL;
    fil_s = NULL;
};


CudaSet::CudaSet(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
    initialize(a,b, op_sel, op_sel_as);
    keep = false;
    source = 0;
    text_source = 0;
    grp = NULL;
    fil_f = NULL;
    fil_s = NULL;
};


CudaSet::~CudaSet()
{
    free();
};


void CudaSet::allocColumnOnDevice(string colname, size_t RecordCount)
{
    if (type[colname] == 0) {
        d_columns_int[colname].resize(RecordCount);
    }
    else if (type[colname] == 1)
        d_columns_float[colname].resize(RecordCount);
    else {
        void* d;
        size_t sz = RecordCount*char_size[colname];
        hipError_t cudaStatus = hipMalloc(&d, sz);
        if(cudaStatus != hipSuccess) {
            char buf[1024];
            sprintf( buf, "Could not allocate %llu bytes of GPU memory for %d records ", sz, RecordCount);
            process_error(3, string(buf));
        };
        d_columns_char[colname] = (char*)d;
    };
};


void CudaSet::decompress_char_hash(string colname, unsigned int segment)
{
    unsigned int bits_encoded, fit_count, sz, vals_count, real_count;
    size_t old_count;
    const unsigned int len = char_size[colname];

    string f1 = load_file_name + "." + colname + "." + int_to_string(segment);

    FILE* f;
    cout << "decompress_char_hash->Reading file: " << f1.c_str() << endl;
    f = fopen (f1.c_str() , "rb" );
    fread(&sz, 4, 1, f);
    char* d_array = new char[sz*len];
    fread((void*)d_array, sz*len, 1, f);

    unsigned long long int* hashes  = new unsigned long long int[sz];

    for(unsigned int i = 0; i < sz ; i++) {
        hashes[i] = MurmurHash64A(&d_array[i*len], len, hash_seed)/2;
    };

    void* d;
    hipMalloc((void **) &d, sz*int_size);
    hipMemcpy( d, (void *) hashes, sz*8, hipMemcpyHostToDevice);

    thrust::device_ptr<unsigned long long int> dd_int((unsigned long long int*)d);

    delete[] d_array;
    delete[] hashes;

    fread(&fit_count, 4, 1, f);
    fread(&bits_encoded, 4, 1, f);
    fread(&vals_count, 4, 1, f);
    fread(&real_count, 4, 1, f);

    unsigned long long int* int_array = new unsigned long long int[vals_count];
    fread((void*)int_array, 1, vals_count*8, f);
    fclose(f);

    void* d_val;
    hipMalloc((void **) &d_val, vals_count*8);
    hipMemcpy(d_val, (void *) int_array, vals_count*8, hipMemcpyHostToDevice);

    delete[] int_array;
    void* d_int;
    hipMalloc((void **) &d_int, real_count*4);

    // convert bits to ints and then do gather

    void* d_v1;
    hipMalloc((void **) &d_v1, 8);
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v1);

    dd_v[1] = fit_count;
    dd_v[0] = bits_encoded;

    thrust::counting_iterator<unsigned int> begin(0);
    decompress_functor_str ff((unsigned long long int*)d_val,(unsigned int*)d_int, (unsigned int*)d_v1);
    thrust::for_each(begin, begin + real_count, ff);

    thrust::device_ptr<unsigned int> dd_val((unsigned int*)d_int);

    if(filtered) {
        if(prm_index == 'R') {
            thrust::device_ptr<int_type> d_tmp = thrust::device_malloc<int_type>(real_count);
            thrust::gather(dd_val, dd_val + real_count, dd_int, d_tmp);
            old_count = d_columns_int[colname].size();
            d_columns_int[colname].resize(old_count + mRecCount);
            thrust::gather(prm_d.begin(), prm_d.begin() + mRecCount, d_tmp, d_columns_int[colname].begin() + old_count);
            thrust::device_free(d_tmp);

        }
        else if(prm_index == 'A') {
            old_count = d_columns_int[colname].size();
            d_columns_int[colname].resize(old_count + real_count);
            thrust::gather(dd_val, dd_val + real_count, dd_int, d_columns_int[colname].begin() + old_count);
        }
    }
    else {
        old_count = d_columns_int[colname].size();
        d_columns_int[colname].resize(old_count + real_count);
        thrust::gather(dd_val, dd_val + real_count, dd_int, d_columns_int[colname].begin() + old_count);
    };


    hipFree(d);
    hipFree(d_val);
    hipFree(d_v1);
    hipFree(d_int);
};




// takes a char column , hashes strings, copies them to a gpu
void CudaSet::add_hashed_strings(string field, unsigned int segment)
{
    CudaSet *t;
    if(filtered)
        t = varNames[source_name];
    else
        t = this;

    if(not_compressed) { // decompressed strings on a host

        size_t old_count;
        unsigned long long int* hashes  = new unsigned long long int[t->mRecCount];

        for(unsigned int i = 0; i < t->mRecCount ; i++) {
            hashes[i] = MurmurHash64A(t->h_columns_char[field] + i*t->char_size[field] + segment*t->maxRecs*t->char_size[field], t->char_size[field], hash_seed)/2;
        };

        if(filtered) {

            if(prm_index == 'R') {
                thrust::device_ptr<unsigned long long int> d_tmp = thrust::device_malloc<unsigned long long int>(t->mRecCount);
                thrust::copy(hashes, hashes+mRecCount, d_tmp);
                old_count = d_columns_int[field].size();
                d_columns_int[field].resize(old_count + mRecCount);
                thrust::gather(prm_d.begin(), prm_d.begin() + mRecCount, d_tmp, d_columns_int[field].begin() + old_count);
                thrust::device_free(d_tmp);
            }
            else if(prm_index == 'A') {
                old_count = d_columns_int[field].size();
                d_columns_int[field].resize(old_count + mRecCount);
                thrust::copy(hashes, hashes + mRecCount, d_columns_int[field].begin() + old_count);
            }
        }
        else {
            old_count = d_columns_int[field].size();
            d_columns_int[field].resize(old_count + mRecCount);
            thrust::copy(hashes, hashes + mRecCount, d_columns_int[field].begin() + old_count);
        }
        delete [] hashes;
    }
    else { // hash the dictionary
        decompress_char_hash(field, segment);
    };
};




void CudaSet::resize_join(size_t addRecs)
{
    mRecCount = mRecCount + addRecs;
    bool prealloc = 0;
    for(unsigned int i=0; i < columnNames.size(); i++) {
        if(type[columnNames[i]] == 0) {
            h_columns_int[columnNames[i]].resize(mRecCount);
        }
        else if(type[columnNames[i]] == 1) {
            h_columns_float[columnNames[i]].resize(mRecCount);
        }
        else {
            if (h_columns_char.find(columnNames[i]) != h_columns_char.end()) {
                if (mRecCount > prealloc_char_size) {
                    h_columns_char[columnNames[i]] = (char*)realloc(h_columns_char[columnNames[i]], mRecCount*char_size[columnNames[i]]);
                    prealloc = 1;
                };
            }
            else {
                h_columns_char[columnNames[i]] = new char[mRecCount*char_size[columnNames[i]]];
            };
        };
    };
    if(prealloc)
        prealloc_char_size = mRecCount;
};


void CudaSet::resize(size_t addRecs)
{
    mRecCount = mRecCount + addRecs;
    for(unsigned int i=0; i < columnNames.size(); i++) {
        if(type[columnNames[i]] == 0) {
            h_columns_int[columnNames[i]].resize(mRecCount);
        }
        else if(type[columnNames[i]] == 1) {
            h_columns_float[columnNames[i]].resize(mRecCount);
        }
        else {
            if (h_columns_char[columnNames[i]]) {
                h_columns_char[columnNames[i]] = (char*)realloc(h_columns_char[columnNames[i]], mRecCount*char_size[columnNames[i]]);
            }
            else {
                h_columns_char[columnNames[i]] = new char[mRecCount*char_size[columnNames[i]]];
                memset(h_columns_char[columnNames[i]], 0, mRecCount*char_size[columnNames[i]]);
            };
        };

    };
};

void CudaSet::reserve(size_t Recs)
{

	cout << "reserve->Recs size" << Recs << endl;
    for(unsigned int i=0; i < columnNames.size(); i++) {
        if(type[columnNames[i]] == 0)
            h_columns_int[columnNames[i]].reserve(Recs);
        else if(type[columnNames[i]] == 1)
            h_columns_float[columnNames[i]].reserve(Recs);
        else {
            h_columns_char[columnNames[i]] = new char[Recs*char_size[columnNames[i]]];
            if(h_columns_char[columnNames[i]] == NULL) {
                char buf[1024];
                sprintf(buf, "(Alenka) Could not allocate on a host %d records of size %llu", Recs, char_size[columnNames[i]]);
                process_error(3, string(buf));
            };
            prealloc_char_size = Recs;
        };

    };
};


void CudaSet::deAllocColumnOnDevice(string colname)
{
    if (type[colname] == 0 && !d_columns_int.empty()) {
        if(d_columns_int[colname].size() > 0) {
            d_columns_int[colname].resize(0);
            d_columns_int[colname].shrink_to_fit();
        };
    }
    else if (type[colname] == 1 && !d_columns_float.empty()) {
        if (d_columns_float[colname].size() > 0) {
            d_columns_float[colname].resize(0);
            d_columns_float[colname].shrink_to_fit();
        };
    }
    else if (type[colname] == 2 && d_columns_char[colname] != NULL) {
        hipFree(d_columns_char[colname]);
        d_columns_char[colname] = NULL;
    };
};

void CudaSet::allocOnDevice(size_t RecordCount)
{
    for(unsigned int i=0; i < columnNames.size(); i++)
        allocColumnOnDevice(columnNames[i], RecordCount);
};

void CudaSet::deAllocOnDevice()
{
    for(unsigned int i=0; i < columnNames.size(); i++)
        deAllocColumnOnDevice(columnNames[i]);

    for ( map<string, thrust::device_vector<int_type > >::iterator it=d_columns_int.begin(); it != d_columns_int.end(); ++it ) {
        if(it->second.size() > 0) {
            it->second.resize(0);
            it->second.shrink_to_fit();
        };
    };

    for ( map<string, thrust::device_vector<float_type > >::iterator it=d_columns_float.begin(); it != d_columns_float.end(); ++it ) {
        if(it->second.size() > 0) {
            it->second.resize(0);
            it->second.shrink_to_fit();
        };
    };

    if(grp) {
        hipFree(grp);
        grp = NULL;
    };

    if(filtered) { // free the sources
        if(varNames.find(source_name) != varNames.end()) {
            varNames[source_name]->deAllocOnDevice();
        };
    };
};

void CudaSet::resizeDeviceColumn(size_t RecCount, string colname)
{
    if (type[colname] == 0) {
        d_columns_int[colname].resize(RecCount);
    }
    else if (type[colname] == 1)
        d_columns_float[colname].resize(RecCount);
    else {
        void *d;
        hipMalloc((void **) &d, RecCount*char_size[colname]);        
        if (d_columns_char[colname] != NULL) {
			hipMemcpy( d, (void*)d_columns_char[colname], char_size[colname] * mRecCount, hipMemcpyDeviceToDevice);			
            hipFree(d_columns_char[colname]);		
		};	
		d_columns_char[colname] = (char*)d;		
    };
};



void CudaSet::resizeDevice(size_t RecCount)
{
    for(unsigned int i=0; i < columnNames.size(); i++) {
        resizeDeviceColumn(RecCount, columnNames[i]);
    };
};

bool CudaSet::onDevice(string colname)
{

    if (type[colname] == 0) {
        if (d_columns_int.empty())
            return 0;
        if (d_columns_int[colname].size() == 0)
            return 0;
    }
    else if (type[colname] == 1) {
        if (d_columns_float.empty())
            return 0;
        if(d_columns_float[colname].size() == 0)
            return 0;
    }
    else if  (type[colname] == 2) {
        if(d_columns_char.empty())
            return 0;
        if(d_columns_char[colname] == NULL)
            return 0;
    };
    return 1;
}



CudaSet* CudaSet::copyDeviceStruct()
{

    CudaSet* a = new CudaSet(mRecCount, mColumnCount);
    a->not_compressed = not_compressed;
    a->segCount = segCount;
    a->maxRecs = maxRecs;
    a->ref_joins = ref_joins;
    a->ref_sets = ref_sets;
    a->ref_cols = ref_cols;
    a->columnNames = columnNames;
    a->cols = cols;
    a->type = type;
    a->char_size = char_size;
    a->decimal = decimal;

    for(unsigned int i=0; i < columnNames.size(); i++) {

        if(a->type[columnNames[i]] == 0) {
            a->d_columns_int[columnNames[i]] = thrust::device_vector<int_type>();
            a->h_columns_int[columnNames[i]] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >();
        }
        else if(a->type[columnNames[i]] == 1) {
            a->d_columns_float[columnNames[i]] = thrust::device_vector<float_type>();
            a->h_columns_float[columnNames[i]] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >();
        }
        else {
            a->h_columns_char[columnNames[i]] = NULL;
            a->d_columns_char[columnNames[i]] = NULL;
        };
    };
    a->load_file_name = load_file_name;

    a->mRecCount = 0;
    return a;
}


void CudaSet::readSegmentsFromFile(unsigned int segNum, string colname, size_t offset)
{
    string f1 = load_file_name + "." + colname + "." + int_to_string(segNum);;
		cout << "decompress_char_hash->Reading file: " << f1.c_str() << endl;
    if(interactive) { //check if data are in buffers
        if(buffers.find(f1) == buffers.end()) { // add data to buffers
            FILE* f = fopen(f1.c_str(), "rb" );
            if(f == NULL) {
                process_error(3, "Error opening " + string(f1) +" file " );
            };
            fseek(f, 0, SEEK_END);
            long fileSize = ftell(f);
            while(total_buffer_size + fileSize > getTotalSystemMemory() && !buffer_names.empty()) { //free some buffers
                delete [] buffers[buffer_names.front()];
                total_buffer_size = total_buffer_size - buffer_sizes[buffer_names.front()];
                buffer_sizes.erase(buffer_names.front());
                buffers.erase(buffer_names.front());
                buffer_names.pop();
            };
            fseek(f, 0, SEEK_SET);
            char* buff = new char[fileSize];
            fread(buff, fileSize, 1, f);
            fclose(f);
            buffers[f1] = buff;
            buffer_sizes[f1] = fileSize;
            buffer_names.push(f1);
            total_buffer_size = total_buffer_size + fileSize;
            buffer_names.push(f1);
            cout << "added buffer " << f1 << " " << fileSize << endl;
        };
        // get data from buffers
        if(type[colname] == 0) {
            unsigned int cnt = ((unsigned int*)buffers[f1])[0];
            if(cnt > h_columns_int[colname].size()/8 + 10)
                h_columns_int[colname].resize(cnt/8 + 10);
            memcpy(h_columns_int[colname].data(), buffers[f1], cnt+56);
        }
        else if(type[colname] == 1) {
            unsigned int cnt = ((unsigned int*)buffers[f1])[0];
            if(cnt > h_columns_float[colname].size()/8 + 10)
                h_columns_float[colname].resize(cnt/8 + 10);
            memcpy(h_columns_float[colname].data(), buffers[f1], cnt+56);
        }
        else {
            decompress_char(NULL, colname, segNum, offset, buffers[f1]);
        };
    }
    else {

        FILE* f = fopen(f1.c_str(), "rb" );
        if(f == NULL) {
            cout << "Error opening " << f1 << " file " << endl;
            exit(0);
        };

        if(type[colname] == 0) {
            if(1 > h_columns_int[colname].size())
                h_columns_int[colname].resize(1);
            fread(h_columns_int[colname].data(), 4, 1, f);
            unsigned int cnt = ((unsigned int*)(h_columns_int[colname].data()))[0];
            if(cnt > h_columns_int[colname].size()/8 + 10)
                h_columns_int[colname].resize(cnt/8 + 10);
            size_t rr = fread((unsigned int*)(h_columns_int[colname].data()) + 1, 1, cnt+52, f);
            if(rr != cnt+52) {
                char buf[1024];
                sprintf(buf, "Couldn't read %d bytes from %s ,read only", cnt+52, f1.c_str());
                process_error(3, string(buf));
            };
        }
        else if(type[colname] == 1) {
            if(1 > h_columns_float[colname].size())
                h_columns_float[colname].resize(1);
            fread(h_columns_float[colname].data(), 4, 1, f);
            unsigned int cnt = ((unsigned int*)(h_columns_float[colname].data()))[0];
            if(cnt > h_columns_float[colname].size()/8 + 10)
                h_columns_float[colname].resize(cnt/8 + 10);
            size_t rr = fread((unsigned int*)(h_columns_float[colname].data()) + 1, 1, cnt+52, f);
            if(rr != cnt+52) {
                char buf[1024];
                sprintf(buf, "Couldn't read %d bytes from %s ,read only", cnt+52, f1.c_str());
                process_error(3, string(buf));
            };
        }
        else {
            decompress_char(f, colname, segNum, offset, NULL);
        };
        fclose(f);
    };
};


void CudaSet::decompress_char(FILE* f, string colname, unsigned int segNum, size_t offset, char* mem)
{
    unsigned int bits_encoded, fit_count, sz, vals_count, real_count;
    const unsigned int len = char_size[colname];

    if(mem == NULL)
        fread(&sz, 4, 1, f);
    else
        sz = ((unsigned int*)mem)[0];

    size_t a_sz = (size_t)sz*(size_t)len;
    char* d_array = new char[a_sz];
    if(mem == NULL)
        fread((void*)d_array, a_sz, 1, f);
    else
        memcpy(d_array, ((unsigned int*)mem + 1), a_sz);
	
    void* d;
    hipMalloc((void **) &d, a_sz);

    hipMemcpy( d, (void *) d_array, a_sz, hipMemcpyHostToDevice);
    delete[] d_array;

    if(mem == NULL) {
        fread(&fit_count, 4, 1, f);
        fread(&bits_encoded, 4, 1, f);
        fread(&vals_count, 4, 1, f);
        fread(&real_count, 4, 1, f);
    }
    else {
        fit_count = ((unsigned int*)(&mem[4+a_sz]))[0];
        bits_encoded = ((unsigned int*)(&mem[4+a_sz]))[1];
        vals_count = ((unsigned int*)(&mem[4+a_sz]))[2];
        real_count = ((unsigned int*)(&mem[4+a_sz]))[3];
    };
	
	//cout << "DECOMP " << colname << " " << fit_count << " " << bits_encoded << " " << vals_count << " " << real_count << endl;

    thrust::device_ptr<unsigned int> param = thrust::device_malloc<unsigned int>(2);
    param[1] = fit_count;
    param[0] = bits_encoded;

    unsigned long long int* int_array = new unsigned long long int[vals_count];
    if(mem == NULL) {
        fread((void*)int_array, 1, vals_count*8, f);
    }
    else {
        memcpy(int_array, &mem[4+sz*len+16], vals_count*8);
    };
	

    void* d_val;
    hipMalloc((void **) &d_val, vals_count*8);
    hipMemcpy(d_val, (void *) int_array, vals_count*8, hipMemcpyHostToDevice);
    delete[] int_array;

    void* d_int;
    hipMalloc((void **) &d_int, real_count*4);

    thrust::counting_iterator<unsigned int> begin(0);
    decompress_functor_str ff((unsigned long long int*)d_val,(unsigned int*)d_int, (unsigned int*)thrust::raw_pointer_cast(param));
    thrust::for_each(begin, begin + real_count, ff);

    thrust::device_ptr<unsigned int> d_int2((unsigned int*)d_int);
    d_columns_int[colname].resize(real_count);
    thrust::copy(d_int2, d_int2+real_count, d_columns_int[colname].begin());

    if(!alloced_switch)
        str_gather(d_int, real_count, d, d_columns_char[colname] + offset*len, len);
    else
        str_gather(d_int, real_count, d, alloced_tmp, len);

    mRecCount = real_count;

    hipFree(d);
    hipFree(d_val);
    thrust::device_free(param);
    hipFree(d_int);
}



void CudaSet::CopyColumnToGpu(string colname,  unsigned int segment, size_t offset)
{

    if(not_compressed) 	{
        // calculate how many records we need to copy
        if(segment < segCount-1) {
            mRecCount = maxRecs;
        }
        else {
            mRecCount = hostRecCount - maxRecs*(segCount-1);
        };

        switch(type[colname]) {
        case 0 :
            if(!alloced_switch)
                thrust::copy(h_columns_int[colname].begin() + maxRecs*segment, h_columns_int[colname].begin() + maxRecs*segment + mRecCount, d_columns_int[colname].begin() + offset);
            else {
                thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
                thrust::copy(h_columns_int[colname].begin() + maxRecs*segment, h_columns_int[colname].begin() + maxRecs*segment + mRecCount, d_col);
            };
            break;
        case 1 :
            if(!alloced_switch) {
                thrust::copy(h_columns_float[colname].begin() + maxRecs*segment, h_columns_float[colname].begin() + maxRecs*segment + mRecCount, d_columns_float[colname].begin() + offset);
            }
            else {
                thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
                thrust::copy(h_columns_float[colname].begin() + maxRecs*segment, h_columns_float[colname].begin() + maxRecs*segment + mRecCount, d_col);
            };
            break;
        default :
            if(!alloced_switch) {
                hipMemcpy(d_columns_char[colname] + char_size[colname]*offset, h_columns_char[colname] + maxRecs*segment*char_size[colname], char_size[colname]*mRecCount, hipMemcpyHostToDevice);
            }
            else
                hipMemcpy(alloced_tmp , h_columns_char[colname] + maxRecs*segment*char_size[colname], char_size[colname]*mRecCount, hipMemcpyHostToDevice);
        };
    }
    else {

        readSegmentsFromFile(segment,colname, offset);

        if(type[colname] != 2) {
            if(d_v == NULL)
                CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
            if(s_v == NULL)
                CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));
        };


        if(type[colname] == 0) {
            if(!alloced_switch) {
                mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + offset), h_columns_int[colname].data(), d_v, s_v);
            }
            else {
                mRecCount = pfor_decompress(alloced_tmp, h_columns_int[colname].data(), d_v, s_v);
            };
        }
        else if(type[colname] == 1) {
            if(decimal[colname]) {
                if(!alloced_switch) {
                    mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + offset) , h_columns_float[colname].data(), d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[colname].data() + offset));
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[colname].begin(), long_to_float());
                }
                else {
                    mRecCount = pfor_decompress(alloced_tmp, h_columns_float[colname].data(), d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)alloced_tmp);
                    thrust::device_ptr<float_type> d_col_float((float_type*)alloced_tmp);
                    thrust::transform(d_col_int,d_col_int+mRecCount, d_col_float, long_to_float());
                };
            }
            //else // uncompressed float
            //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
            // will have to fix it later so uncompressed data will be written by segments too
        }

    };
}



void CudaSet::CopyColumnToGpu(string colname) // copy all segments
{
    if(not_compressed) {
        switch(type[colname]) {
        case 0 :
            thrust::copy(h_columns_int[colname].begin(), h_columns_int[colname].begin() + mRecCount, d_columns_int[colname].begin());
            break;
        case 1 :
            thrust::copy(h_columns_float[colname].begin(), h_columns_float[colname].begin() + mRecCount, d_columns_float[colname].begin());
            break;
        default :
            hipMemcpy(d_columns_char[colname], h_columns_char[colname], char_size[colname]*mRecCount, hipMemcpyHostToDevice);
        };
    }
    else {
        size_t totals = 0;
        if(d_v == NULL)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
        if(s_v == NULL)
            CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

        size_t cnt = 0;
        for(unsigned int i = 0; i < segCount; i++) {

        	cout << "CopyColumnToGpu->Processing compressed segments" << endl;
            readSegmentsFromFile(i,colname, cnt);

            if(type[colname] == 0) {
                mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + totals), h_columns_int[colname].data(), d_v, s_v);
            }
            else if(type[colname] == 1) {
                if(decimal[colname]) {
                    mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[colname].data() + totals) , h_columns_float[colname].data(), d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[colname].data() + totals));
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[colname].begin() + totals, long_to_float());
                }
                // else  uncompressed float
                //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
                // will have to fix it later so uncompressed data will be written by segments too
            };
            cnt = cnt + mRecCount;

            //totalRecs = totals + mRecCount;
        };

        mRecCount = totals;
    };
}

void CudaSet::CopyColumnToHost(string colname, size_t offset, size_t RecCount)
{

    switch(type[colname]) {
    case 0 :
        thrust::copy(d_columns_int[colname].begin(), d_columns_int[colname].begin() + RecCount, h_columns_int[colname].begin() + offset);
        break;
    case 1 :
        thrust::copy(d_columns_float[colname].begin(), d_columns_float[colname].begin() + RecCount, h_columns_float[colname].begin() + offset);
        break;
    default :
        hipMemcpy(h_columns_char[colname] + offset*char_size[colname], d_columns_char[colname], char_size[colname]*RecCount, hipMemcpyDeviceToHost);
    }
}


void CudaSet::CopyColumnToHost(string colname)
{
    CopyColumnToHost(colname, 0, mRecCount);
}

void CudaSet::CopyToHost(size_t offset, size_t count)
{
    for(unsigned int i = 0; i < columnNames.size(); i++) {
        CopyColumnToHost(columnNames[i], offset, count);
    };
}

float_type* CudaSet::get_float_type_by_name(string name)
{
    return thrust::raw_pointer_cast(d_columns_float[name].data());
}

int_type* CudaSet::get_int_by_name(string name)
{
    return thrust::raw_pointer_cast(d_columns_int[name].data());
}

float_type* CudaSet::get_host_float_by_name(string name)
{
    return thrust::raw_pointer_cast(h_columns_float[name].data());
}

int_type* CudaSet::get_host_int_by_name(string name)
{
    return thrust::raw_pointer_cast(h_columns_int[name].data());
}



void CudaSet::GroupBy(stack<string> columnRef)
{
    if(grp)
        hipFree(grp);

    CUDA_SAFE_CALL(hipMalloc((void **) &grp, mRecCount * sizeof(bool)));
    thrust::device_ptr<bool> d_grp(grp);

    thrust::sequence(d_grp, d_grp+mRecCount, 0, 0);

    thrust::device_ptr<bool> d_group = thrust::device_malloc<bool>(mRecCount);

    d_group[mRecCount-1] = 1;

    for(int i = 0; i < columnRef.size(); columnRef.pop()) {

        columnGroups.push(columnRef.top()); // save for future references

        if (type[columnRef.top()] == 0) {  // int_type
            thrust::transform(d_columns_int[columnRef.top()].begin(), d_columns_int[columnRef.top()].begin() + mRecCount - 1,
                              d_columns_int[columnRef.top()].begin()+1, d_group, thrust::not_equal_to<int_type>());
        }
        else if (type[columnRef.top()] == 1) {  // float_type
            thrust::transform(d_columns_float[columnRef.top()].begin(), d_columns_float[columnRef.top()].begin() + mRecCount - 1,
                              d_columns_float[columnRef.top()].begin()+1, d_group, f_not_equal_to());
        }
        else  {  // Char
            //str_grp(d_columns_char[type_index[colIndex]], mRecCount, d_group, char_size[type_index[colIndex]]);
            //use int_type

            thrust::transform(d_columns_int[columnRef.top()].begin(), d_columns_int[columnRef.top()].begin() + mRecCount - 1,
                              d_columns_int[columnRef.top()].begin()+1, d_group, thrust::not_equal_to<int_type>());

        };
        thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<bool>());

    };

    thrust::device_free(d_group);
    grp_count = thrust::count(d_grp, d_grp+mRecCount,1);
};



void CudaSet::addDeviceColumn(int_type* col, string colname, size_t recCount)
{
    if (std::find(columnNames.begin(), columnNames.end(), colname) == columnNames.end()) {
        columnNames.push_back(colname);
        type[colname] = 0;
        d_columns_int[colname] = thrust::device_vector<int_type>(recCount);
        h_columns_int[colname] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >();
    }
    else {  // already exists, my need to resize it
        if(d_columns_int[colname].size() < recCount) {
            d_columns_int[colname].resize(recCount);
        };
    };
    // copy data to d columns
    thrust::device_ptr<int_type> d_col((int_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_int[colname].begin());
};

void CudaSet::addDeviceColumn(float_type* col, string colname, size_t recCount, bool is_decimal)
{
    if (std::find(columnNames.begin(), columnNames.end(), colname) == columnNames.end()) {
        columnNames.push_back(colname);
        type[colname] = 1;
        d_columns_float[colname] = thrust::device_vector<float_type>(recCount);
        h_columns_float[colname] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >();
    }
    else {  // already exists, my need to resize it
        if(d_columns_float[colname].size() < recCount)
            d_columns_float[colname].resize(recCount);
    };

    decimal[colname] = is_decimal;
    thrust::device_ptr<float_type> d_col((float_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_float[colname].begin());
};

void CudaSet::compress(string file_name, size_t offset, unsigned int check_type, unsigned int check_val, size_t mCount)
{
    string str(file_name);
    thrust::device_vector<unsigned int> permutation;

    void* d;
    CUDA_SAFE_CALL(hipMalloc((void **) &d, mCount*float_size));

    total_count = total_count + mCount;
    if (mCount > total_max && op_sort.empty()) {
        total_max = mCount;
    };

    if(!op_sort.empty()) { //sort the segment
        //copy the key columns to device
        queue<string> sf(op_sort);

        permutation.resize(mRecCount);
        thrust::sequence(permutation.begin(), permutation.begin() + mRecCount,0,1);
        unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation.data());
        void* temp;

        CUDA_SAFE_CALL(hipMalloc((void **) &temp, mRecCount*max_char(this, sf)));

        string sort_type = "ASC";

        while(!sf.empty()) {

            allocColumnOnDevice(sf.front(), maxRecs);
            CopyColumnToGpu(sf.front());

            if (type[sf.front()] == 0)
                update_permutation(d_columns_int[sf.front()], raw_ptr, mRecCount, sort_type, (int_type*)temp);
            else if (type[sf.front()] == 1)
                update_permutation(d_columns_float[sf.front()], raw_ptr, mRecCount, sort_type, (float_type*)temp);
            else {
                update_permutation_char(d_columns_char[sf.front()], raw_ptr, mRecCount, sort_type, (char*)temp, char_size[sf.front()]);
            };
            deAllocColumnOnDevice(sf.front());
            sf.pop();
        };
        hipFree(temp);
    };

    // here we need to check for partitions and if partition_count > 0 -> create partitions
    if(mCount < partition_count || partition_count == 0)
        partition_count = 1;
    unsigned int partition_recs = mCount/partition_count;

    if(!op_sort.empty()) {
        if(total_max < partition_recs)
            total_max = partition_recs;
    };

    total_segments++;
    unsigned int old_segments = total_segments;
    size_t new_offset;
    for(unsigned int i = 0; i < columnNames.size(); i++) {

        string colname = columnNames[i];

        str = file_name + "." + colname;
        curr_file = str;
        str += "." + int_to_string(total_segments-1);
        new_offset = 0;

        if(!op_sort.empty()) {
            allocColumnOnDevice(colname, maxRecs);
            CopyColumnToGpu(colname);
        };

        if(type[colname] == 0) {
            thrust::device_ptr<int_type> d_col((int_type*)d);
            if(!op_sort.empty()) {
                thrust::gather(permutation.begin(), permutation.end(), d_columns_int[colname].begin(), d_col);

                for(unsigned int p = 0; p < partition_count; p++) {
                    str = file_name + "." + colname;
                    curr_file = str;
                    str += "." + int_to_string(total_segments-1);
                    if (p < partition_count - 1) {
                        pfor_compress( (int_type*)d + new_offset, partition_recs*int_size, str, h_columns_int[colname], 0);
                    }
                    else {
                        pfor_compress( (int_type*)d + new_offset, (mCount - partition_recs*p)*int_size, str, h_columns_int[colname], 0);
                    };
                    new_offset = new_offset + partition_recs;
                    total_segments++;
                };
            }
            else {
                thrust::copy(h_columns_int[colname].begin() + offset, h_columns_int[colname].begin() + offset + mCount, d_col);
                pfor_compress( d, mCount*int_size, str, h_columns_int[colname], 0);
            };
        }
        else if(type[colname] == 1) {
            if(decimal[colname]) {
                thrust::device_ptr<float_type> d_col((float_type*)d);
                if(!op_sort.empty()) {
                    thrust::gather(permutation.begin(), permutation.end(), d_columns_float[colname].begin(), d_col);
                    thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                    thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());

                    for(unsigned int p = 0; p < partition_count; p++) {
                        str = file_name + "." + colname;
                        curr_file = str;
                        str += "." + int_to_string(total_segments-1);
                        if (p < partition_count - 1)
                            pfor_compress( (int_type*)d + new_offset, partition_recs*float_size, str, h_columns_float[colname], 1);
                        else
                            pfor_compress( (int_type*)d + new_offset, (mCount - partition_recs*p)*float_size, str, h_columns_float[colname], 1);
                        new_offset = new_offset + partition_recs;
                        total_segments++;
                    };
                }
                else {
                    thrust::copy(h_columns_float[colname].begin() + offset, h_columns_float[colname].begin() + offset + mCount, d_col);
                    thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                    thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
                    pfor_compress( d, mCount*float_size, str, h_columns_float[colname], 1);
                };
            }
            else { // do not compress -- float
                thrust::device_ptr<float_type> d_col((float_type*)d);
                if(!op_sort.empty()) {
                    thrust::gather(permutation.begin(), permutation.end(), d_columns_float[colname].begin(), d_col);
                    thrust::copy(d_col, d_col+mRecCount, h_columns_float[colname].begin());
                    for(unsigned int p = 0; p < partition_count; p++) {
                        str = file_name + "." + colname;
                        curr_file = str;
                        str += "." + int_to_string(total_segments-1);
                        unsigned int curr_cnt;
                        if (p < partition_count - 1)
                            curr_cnt = partition_recs;
                        else
                            curr_cnt = mCount - partition_recs*p;

                        fstream binary_file(str.c_str(),ios::out|ios::binary|fstream::app);
                        binary_file.write((char *)&curr_cnt, 4);
                        binary_file.write((char *)(h_columns_float[colname].data() + new_offset),curr_cnt*float_size);
                        new_offset = new_offset + partition_recs;
                        unsigned int comp_type = 3;
                        binary_file.write((char *)&comp_type, 4);
                        binary_file.close();
                    };
                }
                else {
                    fstream binary_file(str.c_str(),ios::out|ios::binary|fstream::app);
                    binary_file.write((char *)&mCount, 4);
                    binary_file.write((char *)(h_columns_float[colname].data() + offset),mCount*float_size);
                    unsigned int comp_type = 3;
                    binary_file.write((char *)&comp_type, 4);
                    binary_file.close();
                };
            };
        }
        else { //char
            if(!op_sort.empty()) {
                unsigned int*  h_permutation = new unsigned int[mRecCount];
                thrust::copy(permutation.begin(), permutation.end(), h_permutation);
                char* t = new char[char_size[colname]*mRecCount];
                apply_permutation_char_host(h_columns_char[colname], h_permutation, mRecCount, t, char_size[colname]);
                delete [] h_permutation;
                thrust::copy(t, t+ char_size[colname]*mRecCount, h_columns_char[colname]);
                delete [] t;
                for(unsigned int p = 0; p < partition_count; p++) {
                    str = file_name + "." + colname;
                    curr_file = str;
                    str += "." + int_to_string(total_segments-1);

                    if (p < partition_count - 1)
                        compress_char(str, colname, partition_recs, new_offset);
                    else
                        compress_char(str, colname, mCount - partition_recs*p, new_offset);
                    new_offset = new_offset + partition_recs;
                    total_segments++;
                };
            }
            else {
                compress_char(str, colname, mCount, offset);
            };
        };


        if((check_type == 1 && fact_file_loaded) || (check_type == 1 && check_val == 0)) {
            if(!op_sort.empty())
                writeHeader(file_name, colname, total_segments-1);
            else {
                writeHeader(file_name, colname, total_segments);
            };
        };

        total_segments = old_segments;
    };
    hipFree(d);

    if(!op_sort.empty()) {
        total_segments = (old_segments-1)+partition_count;
    };
    permutation.resize(0);
    permutation.shrink_to_fit();
}


void CudaSet::writeHeader(string file_name, string colname, unsigned int tot_segs) {
    string str = file_name + "." + colname;
    string ff = str;
    str += ".header";

    fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&total_count, 8);
    binary_file.write((char *)&tot_segs, 4);
    binary_file.write((char *)&total_max, 4);
    binary_file.write((char *)&cnt_counts[ff], 4);
    binary_file.close();
};

void CudaSet::reWriteHeader(string file_name, string colname, unsigned int tot_segs, size_t newRecs, size_t maxRecs1) {
    string str = file_name + "." + colname;
    string ff = str;
    str += ".header";
    fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&newRecs, 8);
    binary_file.write((char *)&tot_segs, 4);
    binary_file.write((char *)&maxRecs1, 4);
    binary_file.close();
};



void CudaSet::writeSortHeader(string file_name)
{
    string str(file_name);
    unsigned int idx;

    if(!op_sort.empty()) {
        str += ".sort";
        fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
        idx = (unsigned int)op_sort.size();
        binary_file.write((char *)&idx, 4);
        queue<string> os(op_sort);
        while(!os.empty()) {
            //idx = cols[columnNames[os.front()]];
            if(verbose)
                cout << "sorted on " << idx << endl;
            idx = os.front().size();
            binary_file.write((char *)&idx, 4);
            binary_file.write(os.front().data(), idx);
            os.pop();
        };
        binary_file.close();
    }
    else if(!op_presort.empty()) {
        str += ".presort";
        fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
        idx = (unsigned int)op_presort.size();
        binary_file.write((char *)&idx, 4);
        queue<string> os(op_presort);
        while(!os.empty()) {
            //idx = cols[columnNames[os.front()]];
            idx = os.front().size();
            binary_file.write((char *)&idx, 4);
            binary_file.write(os.front().data(), idx);
            os.pop();
        };
        binary_file.close();
    };
}

using namespace mgpu;

void CudaSet::Display(unsigned int limit, bool binary, bool term)
{
#define MAXCOLS 128
#define MAXFIELDSIZE 128

    //-- This should/will be converted to an array holding pointers of malloced sized structures--
    char    bigbuf[MAXCOLS * MAXFIELDSIZE];
    char    *fields[MAXCOLS];
    const   char *dcolumns[MAXCOLS];
    size_t  mCount;         // num records in play
    bool    print_all = 0;
    string  ss;
    int rows = 0;

    if(limit != 0 && limit < mRecCount)
        mCount = limit;
    else {
        mCount = mRecCount;
        print_all = 1;
    };

    cout << "mRecCount=" << mRecCount << " mcount = " << mCount << " term " << term <<  " limit=" << limit << " print_all=" << print_all << endl;

    //map<unsigned int, string> ordered_columnNames;
    //for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
    //        ordered_columnNames[it->second] = it->first;

    unsigned int cc =0;
    //for (map<unsigned int, string>::iterator it=ordered_columnNames.begin() ; it != ordered_columnNames.end(); ++it )
    for(unsigned int i = 0; i < columnNames.size(); i++)
    {
        fields[cc] = &(bigbuf[cc*MAXFIELDSIZE]);                        // a hack to avoid malloc overheads     - refine later
        //dcolumns[cc++] = it->second.c_str();
        dcolumns[cc++] = columnNames[i].c_str();
    }

    // The goal here is to loop fast and avoid any double handling of outgoing data - pointers are good.
    if(not_compressed && prm_d.size() == 0) {
        for(unsigned int i=0; i < mCount; i++) {                            // for each record
            for(unsigned int j=0; j < columnNames.size(); j++) {                // for each col
                if (type[columnNames[j]] == 0)
                    sprintf(fields[j], "%lld", (h_columns_int[columnNames[j]])[i] );
                else if (type[columnNames[j]] == 1)
                    sprintf(fields[j], "%.2f", (h_columns_float[columnNames[j]])[i] );
                else {
                    strncpy(fields[j], h_columns_char[columnNames[j]] + (i*char_size[columnNames[j]]), char_size[columnNames[j]]);
                    //ss.assign(h_columns_char[type_index[j]] + (i*char_size[type_index[j]]), char_size[type_index[j]]);
                    //fields[j] = (char *) ss.c_str();
                };
            };
            row_cb(mColumnCount, (char **)fields, (char **)dcolumns);
            rows++;
        };
    }
    else {
        queue<string> op_vx;
        //for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
        for(unsigned int i = 0; i < columnNames.size(); i++)
            op_vx.push(columnNames[i]);


        if(prm_d.size() || source) {
            allocColumns(this, op_vx);
        };
        unsigned int curr_seg = 0;
        size_t cnt = 0;
        size_t curr_count, sum_printed = 0;
        resize(maxRecs);
        while(sum_printed < mCount || print_all) {

            if(prm_d.size() || source)  {                            // if host arrays are empty
                copyColumns(this, op_vx, curr_seg, cnt);
                size_t olRecs = mRecCount;
                mRecCount = olRecs;
                CopyToHost(0,mRecCount);
                if(sum_printed + mRecCount <= mCount || print_all)
                    curr_count = mRecCount;
                else
                    curr_count = mCount - sum_printed;
            }
            else
                curr_count = mCount;

            sum_printed = sum_printed + mRecCount;
            for(unsigned int i=0; i < curr_count; i++) {
                for(unsigned int j=0; j < columnNames.size(); j++) {
                    if (type[columnNames[j]] == 0)
                        sprintf(fields[j], "%lld", (h_columns_int[columnNames[j]])[i] );
                    else if (type[columnNames[j]] == 1)
                        sprintf(fields[j], "%.2f", (h_columns_float[columnNames[j]])[i] );
                    else {
                        ss.assign(h_columns_char[columnNames[j]] + (i*char_size[columnNames[j]]), char_size[columnNames[j]]);
                        fields[j] = (char *) ss.c_str();
                    };
                };
                row_cb(mColumnCount, (char **)fields, (char**)dcolumns);
                rows++;
            };
            curr_seg++;
            if(curr_seg == segCount)
                print_all = 0;
        };
    };      // end else
}

void CudaSet::Store(string file_name, char* sep, unsigned int limit, bool binary, bool term)
{
    if (mRecCount == 0 && binary == 1 && !term) { // write tails
        for(unsigned int j=0; j < columnNames.size(); j++) {
            writeHeader(file_name, columnNames[j], total_segments);
        };
        return;
    };

    size_t mCount;
    bool print_all = 0;

    if(limit != 0 && limit < mRecCount)
        mCount = limit;
    else {
        mCount = mRecCount;
        print_all = 1;
    };
    //cout << "mCount " << mCount << " " << mRecCount << endl;

    if(binary == 0) {

        FILE *file_pr;
        if(!term) {
            file_pr = fopen(file_name.c_str(), "w");
            if (file_pr  == NULL)
                cout << "Could not open file " << file_name << endl;
        }
        else
            file_pr = stdout;

        string ss;

        if(not_compressed && prm_d.size() == 0) {
            for(unsigned int i=0; i < mCount; i++) {
                for(unsigned int j=0; j < columnNames.size(); j++) {
                    if (type[columnNames[j]] == 0) {
                        fprintf(file_pr, "%lld", (h_columns_int[columnNames[j]])[i]);
                        fputs(sep, file_pr);
                    }
                    else if (type[columnNames[j]] == 1) {
                        fprintf(file_pr, "%.2f", (h_columns_float[columnNames[j]])[i]);
                        fputs(sep, file_pr);
                    }
                    else {
                        ss.assign(h_columns_char[columnNames[j]] + (i*char_size[columnNames[j]]), char_size[columnNames[j]]);
                        fputs(ss.c_str(), file_pr);
                        fputs(sep, file_pr);
                    };
                };
                if (i != mCount -1 )
                    fputs("\n",file_pr);
            };
            if(!term)
                fclose(file_pr);
        }
        else {

            queue<string> op_vx;
            for(unsigned int j=0; j < columnNames.size(); j++)
                op_vx.push(columnNames[j]);

            if(prm_d.size() || source) {
                allocColumns(this, op_vx);
            };

            unsigned int curr_seg = 0;
            size_t cnt = 0;
            size_t curr_count, sum_printed = 0;
            mRecCount = 0;
            resize(maxRecs);

            while(sum_printed < mCount || print_all) {

                if(prm_d.size() || source)  {
                    copyColumns(this, op_vx, curr_seg, cnt);
                    if(curr_seg == 0) {
                        if(limit != 0 && limit < mRecCount) {
                            mCount = limit;
                            print_all = 0;
                        }
                        else {
                            mCount = mRecCount;
                            print_all = 1;
                        };

                    };

                    // if host arrays are empty
                    size_t olRecs = mRecCount;
                    mRecCount = olRecs;
                    CopyToHost(0,mRecCount);
                    //cout << "start " << sum_printed << " " <<  mRecCount << " " <<  mCount << endl;
                    if(sum_printed + mRecCount <= mCount || print_all) {
                        curr_count = mRecCount;
                    }
                    else {
                        curr_count = mCount - sum_printed;
                    };
                }
                else {
                    curr_count = mCount;
                };

                sum_printed = sum_printed + mRecCount;
                //cout << "sum printed " << sum_printed << " " << curr_count << " " << curr_seg << endl;

                for(unsigned int i=0; i < curr_count; i++) {
                    for(unsigned int j=0; j < columnNames.size(); j++) {
                        if (type[columnNames[j]] == 0) {
                            fprintf(file_pr, "%lld", (h_columns_int[columnNames[j]])[i]);
                            fputs(sep, file_pr);
                        }
                        else if (type[columnNames[j]] == 1) {
                            fprintf(file_pr, "%.2f", (h_columns_float[columnNames[j]])[i]);
                            fputs(sep, file_pr);
                        }
                        else {
                            ss.assign(h_columns_char[columnNames[j]] + (i*char_size[columnNames[j]]), char_size[columnNames[j]]);
                            trim(ss);
                            fputs(ss.c_str(), file_pr);
                            fputs(sep, file_pr);
                        };
                    };
                    if (i != mCount -1 && (curr_seg != segCount || i < curr_count))
                        fputs("\n",file_pr);
                };
                curr_seg++;
                if(curr_seg == segCount)
                    print_all = 0;
            };
            if(!term) {
                fclose(file_pr);
            };
        };
    }
    else {

        //lets update the data dictionary
        for(unsigned int j=0; j < columnNames.size(); j++) {

            if(decimal[columnNames[j]] == 1)
                data_dict[file_name][columnNames[j]].col_type = 3;
            else
                data_dict[file_name][columnNames[j]].col_type = type[columnNames[j]];
            if(type[columnNames[j]] != 2)
                data_dict[file_name][columnNames[j]].col_length = 0;
            else
                data_dict[file_name][columnNames[j]].col_length = char_size[columnNames[j]];
        };
        save_dict = 1;


        if(text_source) {  //writing a binary file using a text file as a source

            // time to perform join checks on REFERENCES dataset segments
            //for(unsigned int i = 0; i< mColumnCount; i++) {

            for(unsigned int i=0; i < columnNames.size(); i++) {

                if(ref_sets.find(columnNames[i]) != ref_sets.end()) {

                    string f1 = file_name + "." + columnNames[i] + ".refs";
                    fstream f_file;
                    if(total_segments == 0) {
                        f_file.open(f1.c_str(), ios::out|ios::trunc|ios::binary);
                        unsigned int len = ref_sets[columnNames[i]].size();
                        f_file.write((char *)&len, 4);
                        f_file.write(ref_sets[columnNames[i]].c_str(), len);
                        len = ref_cols[columnNames[i]].size();
                        f_file.write((char *)&len, 4);
                        f_file.write(ref_cols[columnNames[i]].c_str(), len);
                    }
                    else {
                        f_file.open(f1.c_str(), ios::out|ios::app|ios::binary);
                    };

                    f1 = ref_sets[columnNames[i]] + "." + ref_cols[columnNames[i]] + ".header";
                    FILE* ff = fopen(f1.c_str(), "rb");
                    if(ff == NULL) {
                        process_error(3, "Couldn't open file " + string(f1));
                    };
                    unsigned int ref_segCount, ref_maxRecs;
                    fread((char *)&ref_segCount, 4, 1, ff);
                    fread((char *)&ref_segCount, 4, 1, ff);
                    fread((char *)&ref_segCount, 4, 1, ff);
                    fread((char *)&ref_maxRecs, 4, 1, ff);
                    fclose(ff);
                    //cout << "CALC " << i << " " << columnNames[i] << " " << ref_sets[columnNames[i]] << " " << ref_cols[columnNames[i]] << " " << ref_segCount << " " << ref_maxRecs << endl;

                    CudaSet* a = new CudaSet(maxRecs, 1);
                    a->h_columns_int[ref_cols[columnNames[i]]] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
                    a->d_columns_int[ref_cols[columnNames[i]]] = thrust::device_vector<int_type>(ref_maxRecs);
                    a->type[ref_cols[columnNames[i]]] = 0;
                    a->not_compressed = 0;
                    a->load_file_name = ref_sets[columnNames[i]];
                    a->cols[ref_cols[columnNames[i]]] = 1;
                    a->columnNames.push_back(ref_cols[columnNames[i]]);
                    MGPU_MEM(int) aIndicesDevice, bIndicesDevice;
                    size_t res_count;

                    if(!onDevice(columnNames[i])) {
                        allocColumnOnDevice(columnNames[i], maxRecs);
                    };
                    CopyColumnToGpu(columnNames[i]);
                    thrust::sort(d_columns_int[columnNames[i]].begin(), d_columns_int[columnNames[i]].begin() + mRecCount);

                    f_file.write((char *)&total_segments, 4);
                    f_file.write((char *)&ref_segCount, 4);
                    for(unsigned int z = 0; z < ref_segCount; z++) {

                        a->CopyColumnToGpu(ref_cols[columnNames[i]], z, 0);
                        thrust::sort(a->d_columns_int[ref_cols[columnNames[i]]].begin(), a->d_columns_int[ref_cols[columnNames[i]]].begin() + a->mRecCount);
                        // check if there is a join result
                        //cout << "join " << mRecCount << " " << a->mRecCount << " " << getFreeMem() << endl;
                        //cout << d_columns_int[columnNames[i]][0] << " " <<  d_columns_int[columnNames[i]][mRecCount-1] << " " << a->d_columns_int[ref_cols[columnNames[i]]][a->mRecCount-1]	<< " " <<  a->d_columns_int[ref_cols[columnNames[i]]][0] << endl;
                        if(d_columns_int[columnNames[i]][0] > a->d_columns_int[ref_cols[columnNames[i]]][a->mRecCount-1]	||
                                d_columns_int[columnNames[i]][mRecCount-1] < a->d_columns_int[ref_cols[columnNames[i]]][0]) {
                            res_count = 0;
                        }
                        else {
                            res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()), mRecCount,
                                        thrust::raw_pointer_cast(a->d_columns_int[ref_cols[columnNames[i]]].data()), a->mRecCount,
                                        &aIndicesDevice, &bIndicesDevice,
                                        mgpu::less<int_type>(), *context);
                        };
                        //cout << "RES " << i << " " << total_segments << ":" << z << " " << res_count << endl;
                        f_file.write((char *)&z, 4);
                        f_file.write((char *)&res_count, 8);
                    };
                    f_file.close();
                    a->deAllocColumnOnDevice(ref_cols[columnNames[i]]);
                    a->free();
                };
            };
            compress(file_name, 0, 1, 0, mCount);
            for(unsigned int i = 0; i< columnNames.size(); i++)
                if(type[columnNames[i]] == 2)
                    deAllocColumnOnDevice(columnNames[i]);
        }
        else { //writing a binary file using a binary file as a source
            fact_file_loaded = 1;
            size_t offset = 0;

            if(!not_compressed) { // records are compressed, for example after filter op.
                //decompress to host
                queue<string> op_vx;
                for(unsigned int i = 0; i< columnNames.size(); i++) {
                    op_vx.push(columnNames[i]);
                };

                allocColumns(this, op_vx);
                size_t oldCnt = mRecCount;
                mRecCount = 0;
                resize(oldCnt);
                mRecCount = oldCnt;
                for(unsigned int i = 0; i < segCount; i++) {
                    size_t cnt = 0;
                    copyColumns(this, op_vx, i, cnt);
                    CopyToHost(0, mRecCount);
                    offset = offset + mRecCount;
                    compress(file_name, 0, 0, i - (segCount-1), mRecCount);
                };
            }
            else {
                // now we have decompressed records on the host
                //call setSegments and compress columns in every segment

                segCount = (mRecCount/process_count + 1);
                offset = 0;

                for(unsigned int z = 0; z < segCount; z++) {

                    if(z < segCount-1) {
                        if(mRecCount < process_count) {
                            mCount = mRecCount;
                        }
                        else {
                            mCount = process_count;
                        }
                    }
                    else {
                        mCount = mRecCount - (segCount-1)*process_count;
                    };
                    compress(file_name, offset, 0, z - (segCount-1), mCount);
                    offset = offset + mCount;
                };
            };
        };
    };
}


void CudaSet::compress_char(string file_name, string colname, size_t mCount, size_t offset)
{
    std::map<string,unsigned int> dict;
    std::vector<string> dict_ordered;
    std::vector<unsigned int> dict_val;
    map<string,unsigned int>::iterator iter;
    unsigned int bits_encoded, ss;
    unsigned int len = char_size[colname];


    for (unsigned int i = 0 ; i < mCount; i++) {

        string f(h_columns_char[colname] + (i+offset)*len, len);
        if((iter = dict.find(f)) != dict.end()) {
            dict_val.push_back(iter->second);
        }
        else {
            ss = (unsigned int)dict.size();
            dict[f] = ss;
            dict_val.push_back(ss);
            dict_ordered.push_back(f);
        };
    };

    bits_encoded = (unsigned int)ceil(log2(double(dict.size()+1)));
	//cout << "bits " << bits_encoded << endl;

    char *cc = new char[len+1];
    cc[len] = 0;
    unsigned int sz = (unsigned int)dict_ordered.size();
    // write to a file
    fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&sz, 4);	
    for(unsigned int i = 0; i < sz; i++) {
        memset(&cc[0], 0, len);
        strcpy(cc,dict_ordered[i].c_str());
        binary_file.write(cc, len);
    };

    delete [] cc;
    unsigned int fit_count = 64/bits_encoded;
    unsigned long long int val = 0;
    binary_file.write((char *)&fit_count, 4);
    binary_file.write((char *)&bits_encoded, 4);
    unsigned int curr_cnt = 1;
    unsigned int vals_count = (unsigned int)dict_val.size()/fit_count;
    if(!vals_count || dict_val.size()%fit_count)
        vals_count++;
    binary_file.write((char *)&vals_count, 4);
    unsigned int real_count = (unsigned int)dict_val.size();
    binary_file.write((char *)&real_count, 4);

    for(unsigned int i = 0; i < dict_val.size(); i++) {

        val = val | dict_val[i];

        if(curr_cnt < fit_count)
            val = val << bits_encoded;

        if( (curr_cnt == fit_count) || (i == (dict_val.size() - 1)) ) {
            if (curr_cnt < fit_count) {
                val = val << ((fit_count-curr_cnt)-1)*bits_encoded;
            };
            curr_cnt = 1;
            binary_file.write((char *)&val, 8);
            val = 0;
        }
        else
            curr_cnt = curr_cnt + 1;
    };
    binary_file.close();
};



bool CudaSet::LoadBigFile(FILE* file_p)
{
    char line[1000];
    unsigned int current_column, count = 0;
    string colname;
    char *p,*t;
    const char* sep = separator.c_str();
	cout << "LoadBigFile" << endl;


    unsigned int maxx = 0;
    for(unsigned int i = 0; i < mColumnCount; i++) {
        if(cols[columnNames[i]] > maxx)
            maxx = cols[columnNames[i]];
    };

    bool *check_col = new bool[maxx+1];
    vector<string> names(maxx+1);

    for(unsigned int i = 0; i <= maxx; i++) {
        check_col[i] = 0;
    };

    for(unsigned int i = 0; i < mColumnCount; i++) {
        names[cols[columnNames[i]]] = columnNames[i];
        check_col[cols[columnNames[i]]] = 1;
    };
	
	//clear the varchars
	for(auto it=columnNames.begin(); it!=columnNames.end();it++) {
		if(type[*it] == 2) {			
			memset(h_columns_char[*it], 0, maxRecs*char_size[*it]);
		};
	};


    //while (count < process_count && fgets(line, 1000, file_p) != NULL) {
    while (count < process_count && fgets(line, 1000, file_p) != NULL) {
        strtok(line, "\n");
        current_column = 0;

        for(t=mystrtok(&p,line,*sep); t && current_column < maxx; t=mystrtok(&p,0,*sep)) {
            current_column++;
            if(!check_col[current_column]) {
                //cout << "Didn't find " << current_column << endl;
                continue;
            };
            //cout << "curr " << current_column << " " << names[current_column] << endl;

            if (type[names[current_column]] == 0) {
                if (strchr(t,'-') == NULL) {
                    (h_columns_int[names[current_column]])[count] = atoll(t);
                }
                else {   // handling possible dates
                    strncpy(t+4,t+5,2);
                    strncpy(t+6,t+8,2);
                    t[8] = '\0';
                    (h_columns_int[names[current_column]])[count] = atoll(t);
                };
            }
            else if (type[names[current_column]] == 1) {
                (h_columns_float[names[current_column]])[count] = atoff(t);
            }
            else  {//char
                strcpy(h_columns_char[names[current_column]] + count*char_size[names[current_column]], t);
            }
        };
        count++;
    };

    delete [] check_col;
    mRecCount = count;

    if(count < process_count)  {
        fclose(file_p);
        return 1;
    }
    else
        return 0;
};


void CudaSet::free()  {

    for(unsigned int i = 0; i < columnNames.size(); i++ ) {
        if(type[columnNames[i]] == 2 && h_columns_char[columnNames[i]]) {
            delete [] h_columns_char[columnNames[i]];
            h_columns_char[columnNames[i]] = NULL;
        }
        else {
            if(type[columnNames[i]] == 0 ) {
                h_columns_int[columnNames[i]].resize(0);
                h_columns_int[columnNames[i]].shrink_to_fit();
            }
            else if(type[columnNames[i]] == 1) {
                h_columns_float[columnNames[i]].resize(0);
                h_columns_float[columnNames[i]].shrink_to_fit();
            };
        }
    };

    prm_d.resize(0);
    prm_d.shrink_to_fit();
    deAllocOnDevice();

    if(fil_s)
        delete fil_s;
    if(fil_f)
        delete fil_f;

};


bool* CudaSet::logical_and(bool* column1, bool* column2)
{
    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_and<bool>());

    thrust::device_free(dev_ptr2);
    return column1;
}


bool* CudaSet::logical_or(bool* column1, bool* column2)
{

    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_or<bool>());
    thrust::device_free(dev_ptr2);
    return column1;
}



bool* CudaSet::compare(int_type s, int_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if(d>s) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if(d<s) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if(d>=s) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if(d<=s) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if(d==s) res = 1;
        else res = 0;
    else // !=
        if(d!=s) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
};


bool* CudaSet::compare(float_type s, float_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if ((d-s) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if ((s-d) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if (((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if (((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if (((d-s) < EPSILON) && ((d-s) > -EPSILON)) res = 1;
        else res = 0;
    else // !=
        if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
}


bool* CudaSet::compare(int_type* column1, int_type d, int_type op_type)
{
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);


    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);

}

bool* CudaSet::compare(float_type* column1, float_type d, int_type op_type)
{
    thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr(column1);

    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_equal_to());
    else  // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_not_equal_to());

    return thrust::raw_pointer_cast(res);
}


bool* CudaSet::compare(int_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);
}

bool* CudaSet::compare(float_type* column1, float_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    return thrust::raw_pointer_cast(temp);

}


bool* CudaSet::compare(float_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr(column2);
    thrust::device_ptr<float_type> dev_ptr2 = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr2, long_to_float_type());

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    thrust::device_free(dev_ptr2);
    return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(int_type* column1, float_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, temp, long_to_float_type()); // in-place transformation

    thrust::device_ptr<float_type> dev_ptr1(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };

    return thrust::raw_pointer_cast(temp);

}




int_type* CudaSet::op(int_type* column1, int_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<int_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    }

    return thrust::raw_pointer_cast(temp);

}

float_type* CudaSet::op(float_type* column1, float_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    };
    return thrust::raw_pointer_cast(temp);
}

int_type* CudaSet::op(int_type* column1, int_type d, string op_type, int reverse)
{
    thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr1(column1);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<int_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    };
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(int_type* column1, float_type d, string op_type, int reverse)
{
    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr(column1);
    thrust::device_ptr<float_type> dev_ptr1 = thrust::device_malloc<float_type>(mRecCount);
    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr1, long_to_float_type());

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };
    thrust::device_free(dev_ptr1);
    return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(float_type* column1, float_type d, string op_type,int reverse)
{
    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr1(column1);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::divides<float_type>());
    }
    else	{
        if (op_type.compare("MUL") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };

    return thrust::raw_pointer_cast(temp);

}





void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name) // compressed data for DIM tables
{
    mColumnCount = (unsigned int)nameRef.size();
    FILE* f;
    string f1;
    unsigned int cnt;
    char buffer[4000];
    string str;

    prealloc_char_size = 0;
    not_compressed = 0;
    mRecCount = Recs;
    hostRecCount = Recs;
    totalRecs = Recs;
    load_file_name = file_name;

    f1 = file_name + ".sort";
    cout << "initialize->Reading file: " << f1.c_str() << endl;
    f = fopen (f1.c_str() , "rb" );
    if(f != NULL) {
        unsigned int sz, idx;
        fread((char *)&sz, 4, 1, f);
        for(unsigned int j = 0; j < sz; j++) {
            fread((char *)&idx, 4, 1, f);
            fread(buffer, idx, 1, f);
            str.assign(buffer, idx);
            sorted_fields.push(str);
            if(verbose)
                cout << "segment sorted on " << idx << endl;
        };
        fclose(f);
    };

    f1 = file_name + ".presort";
    cout << "initialize->Reading file: " << f1.c_str() << endl;
    f = fopen (f1.c_str() , "rb" );
    if(f != NULL) {
        unsigned int sz, idx;
        fread((char *)&sz, 4, 1, f);
        for(unsigned int j = 0; j < sz; j++) {
            fread((char *)&idx, 4, 1, f);
            fread(buffer, idx, 1, f);
            str.assign(buffer, idx);
            presorted_fields.push(str);
            if(verbose)
                cout << "presorted on " << str << endl;
        };
        fclose(f);
    };
	
    tmp_table = 0;
    filtered = 0;

    for(unsigned int i=0; i < mColumnCount; i++) {

        //f1 = file_name + "." + nameRef.front() + ".0";
        //f = fopen (f1.c_str() , "rb" );
        //fread((char *)&bytes, 4, 1, f); //need to read metadata such as type and length
        //fclose(f);

        columnNames.push_back(nameRef.front());
        cols[nameRef.front()] = colsRef.front();

        if (((typeRef.front()).compare("decimal") == 0) || ((typeRef.front()).compare("int") == 0)) {
            f1 = file_name + "." + nameRef.front() + ".0";
            f = fopen (f1.c_str() , "rb" );
			if(f == NULL) {
				cout << "Couldn't find field " << nameRef.front() << endl;
				exit(0);
			};
            for(unsigned int j = 0; j < 6; j++)
                fread((char *)&cnt, 4, 1, f);
            fclose(f);
            compTypes[nameRef.front()] = cnt;
        };

        //check the references
        f1 = file_name + "." + nameRef.front() + ".refs";
        f = fopen (f1.c_str() , "rb" );
        if(f != NULL) {
            unsigned int len;
            fread(&len, 4, 1, f);
            char* array = new char[len];
            fread((void*)array, len, 1, f);
            ref_sets[nameRef.front()] = array;
            delete [] array;
            unsigned int segs, seg_num, curr_seg;
            size_t res_count;
            fread(&len, 4, 1, f);
            char* array1 = new char[len];
            fread((void*)array1, len, 1, f);
            ref_cols[nameRef.front()] = array1;
            delete [] array1;

            unsigned int bytes_read = fread((void*)&curr_seg, 4, 1, f);

            while(bytes_read == 1) {
                fread((void*)&segs, 4, 1, f); //ref seg count
                //cout << "for " << i << " read " << array << " and " << z << " " << segs << endl;

                for(unsigned int j = 0; j < segs; j++) {
                    fread((void*)&seg_num, 4, 1, f);
                    fread((void*)&res_count, 8, 1, f);
                    //cout << "curr_seg " << curr_seg << " " << seg_num << " " << res_count << endl;
                    if(res_count)
                        ref_joins[columnNames[i]][curr_seg].insert(seg_num);
                    else
                        ref_joins[columnNames[i]][curr_seg].insert(std::numeric_limits<unsigned int>::max());
                };
                bytes_read = fread((void*)&curr_seg, 4, 1, f);
            };
            fclose(f);
        };
		
        if ((typeRef.front()).compare("int") == 0) {
            type[nameRef.front()] = 0;
            decimal[nameRef.front()] = 0;
            h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 0;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type >();
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 1;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type>();
        }
        else {
            type[nameRef.front()] = 2;
            decimal[nameRef.front()] = 0;
            h_columns_char[nameRef.front()] = NULL;
            d_columns_char[nameRef.front()] = NULL;
            char_size[nameRef.front()] = sizeRef.front();
        };
		
        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
    };
	
};



void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, queue<string> &references, queue<string> &references_names)
{
    mColumnCount = (unsigned int)nameRef.size();
    prealloc_char_size = 0;
    tmp_table = 0;
    filtered = 0;
    mRecCount = Recs;
    hostRecCount = Recs;
    segCount = 1;

    for(unsigned int i=0; i < mColumnCount; i++) {

        columnNames.push_back(nameRef.front());
        cols[nameRef.front()] = colsRef.front();

        if ((typeRef.front()).compare("int") == 0) {
            type[nameRef.front()] = 0;
            decimal[nameRef.front()] = 0;
            h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 0;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type>();
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 1;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type>();
        }

        else {
            type[nameRef.front()] = 2;
            decimal[nameRef.front()] = 0;
            h_columns_char[nameRef.front()] = NULL;
            d_columns_char[nameRef.front()] = NULL;
            char_size[nameRef.front()] = sizeRef.front();
        };

        if(!references.front().empty()) {
            ref_sets[nameRef.front()] = references.front();
            ref_cols[nameRef.front()] = references_names.front();
        };
        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
        references.pop();
        references_names.pop();
    };
};

void CudaSet::initialize(size_t RecordCount, unsigned int ColumnCount)
{
    mRecCount = RecordCount;
    hostRecCount = RecordCount;
    mColumnCount = ColumnCount;
    prealloc_char_size = 0;
    filtered = 0;
};




void CudaSet::initialize(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
    mRecCount = 0;
    mColumnCount = 0;
    queue<string> q_cnt(op_sel);
    unsigned int i = 0;
    set<string> field_names;
    while(!q_cnt.empty()) {
        if( std::find(a->columnNames.begin(), a->columnNames.end(), q_cnt.front()) !=  a->columnNames.end() ||
                std::find(b->columnNames.begin(), b->columnNames.end(), q_cnt.front()) !=  b->columnNames.end())  {
            field_names.insert(q_cnt.front());
        };
        q_cnt.pop();
    }
    mColumnCount = (unsigned int)field_names.size();
    maxRecs = b->maxRecs;
    map<string,unsigned int>::iterator it;

    segCount = 1;
    filtered = 0;
    not_compressed = 1;

    col_aliases = op_sel_as;
    prealloc_char_size = 0;

    i = 0;
    while(!op_sel.empty() && (std::find(columnNames.begin(), columnNames.end(), op_sel.front()) ==  columnNames.end())) {

        if(std::find(a->columnNames.begin(), a->columnNames.end(), op_sel.front()) !=  a->columnNames.end()) {
            cols[op_sel.front()] = i;
            decimal[op_sel.front()] = a->decimal[op_sel.front()];
            columnNames.push_back(op_sel.front());
            type[op_sel.front()] = a->type[op_sel.front()];

            if (a->type[op_sel.front()] == 0)  {
                d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
                h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            }
            else if (a->type[op_sel.front()] == 1) {
                d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
                h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            }
            else {
                h_columns_char[op_sel.front()] = NULL;
                d_columns_char[op_sel.front()] = NULL;
                char_size[op_sel.front()] = a->char_size[op_sel.front()];
            };
            i++;
        }
        else if(std::find(b->columnNames.begin(), b->columnNames.end(), op_sel.front()) !=  b->columnNames.end()) {
            columnNames.push_back(op_sel.front());
            cols[op_sel.front()] = i;
            decimal[op_sel.front()] = b->decimal[op_sel.front()];
            type[op_sel.front()] = b->type[op_sel.front()];

            if (b->type[op_sel.front()] == 0) {
                d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
                h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            }
            else if (b->type[op_sel.front()] == 1) {
                d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
                h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            }
            else {
                h_columns_char[op_sel.front()] = NULL;
                d_columns_char[op_sel.front()] = NULL;
                char_size[op_sel.front()] = b->char_size[op_sel.front()];
            };
            i++;
        }
        op_sel.pop();
    };
};



int_type reverse_op(int_type op_type)
{
    if (op_type == 2) // >
        return 5;
    else if (op_type == 1)  // <
        return 6;
    else if (op_type == 6) // >=
        return 1;
    else if (op_type == 5)  // <=
        return 2;
    else return op_type;
}


size_t getFreeMem()
{
    size_t available, total;
    hipMemGetInfo(&available, &total);
    return available;
} ;



void allocColumns(CudaSet* a, queue<string> fields)
{
    if(a->filtered) {
        size_t max_sz = max_tmp(a);
        CudaSet* t;
        if(a->filtered)
            t = varNames[a->source_name];
        else
            t = a;

        if(max_sz*t->maxRecs > alloced_sz) {
            if(alloced_sz) {
                hipFree(alloced_tmp);
            };
            hipMalloc((void **) &alloced_tmp, max_sz*t->maxRecs);
            alloced_sz = max_sz*t->maxRecs;
        }
    }
    else {

        while(!fields.empty()) {
            if(var_exists(a, fields.front())) {

                bool onDevice = 0;

                if(a->type[fields.front()] == 0) {
                    if(a->d_columns_int[fields.front()].size() > 0) {
                        onDevice = 1;
                    }
                }
                else if(a->type[fields.front()] == 1) {
                    if(a->d_columns_float[fields.front()].size() > 0) {
                        onDevice = 1;
                    };
                }
                else {
                    if((a->d_columns_char[fields.front()]) != NULL) {
                        onDevice = 1;
                    };
                };

                if (!onDevice) {
                    a->allocColumnOnDevice(fields.front(), a->maxRecs);
                }
            }
            fields.pop();
        };
    };
}



void gatherColumns(CudaSet* a, CudaSet* t, string field, unsigned int segment, size_t& count)
{
    if(!a->onDevice(field)) {
        a->allocColumnOnDevice(field, a->maxRecs);
    };

    if(a->prm_index == 'R') {
        mygather(field, a, t, count, a->mRecCount);
    }
    else {
        mycopy(field, a, t, count, t->mRecCount);
        a->mRecCount = t->mRecCount;
    };
}


size_t getSegmentRecCount(CudaSet* a, unsigned int segment) {
    if (segment == a->segCount-1) {
        return a->hostRecCount - a->maxRecs*segment;
    }
    else
        return 	a->maxRecs;
}



void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt)
{
    set<string> uniques;

    if(a->filtered) { //filter the segment
        if(flt) {
            filter_op(a->fil_s, a->fil_f, segment);
		};
        if(rsz && a->mRecCount) {
            queue<string> fields1(fields);			
            while(!fields1.empty()) {
                a->resizeDeviceColumn(a->devRecCount + a->mRecCount, fields1.front());			
                fields1.pop();
            };
            a->devRecCount = a->devRecCount + a->mRecCount;
        };
    };
	
    while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()))	{
            if(a->filtered) {
                if(a->mRecCount) {		
                    CudaSet *t = varNames[a->source_name];
                    alloced_switch = 1;
                    t->CopyColumnToGpu(fields.front(), segment);
                    gatherColumns(a, t, fields.front(), segment, count);
                    alloced_switch = 0;
                    a->orig_segs[t->load_file_name].insert(segment);
                };
            }
            else {
                if(a->mRecCount) {
                    a->CopyColumnToGpu(fields.front(), segment, count);
                };
            };
            uniques.insert(fields.front());
        };
        fields.pop();
    };
}



void setPrm(CudaSet* a, CudaSet* b, char val, unsigned int segment) {

    b->prm_index = val;
    if (val == 'A') {
        b->mRecCount = getSegmentRecCount(a,segment);
    }
    else if (val == 'N') {
        b->mRecCount = 0;
    }
}



void mygather(string colname, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
    if(t->type[colname] == 0) {
        if(!alloced_switch) {
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_int[colname].begin(), a->d_columns_int[colname].begin() + offset);
        }
        else {
            thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           d_col, a->d_columns_int[colname].begin() + offset);
        };
    }
    else if(t->type[colname] == 1) {
        if(!alloced_switch) {
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_float[colname].begin(), a->d_columns_float[colname].begin() + offset);
        }
        else {
            thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           d_col, a->d_columns_float[colname].begin() + offset);
        };
    }
    else {
        if(!alloced_switch) {
            str_gather((void*)thrust::raw_pointer_cast(a->prm_d.data()), g_size,
                       (void*)t->d_columns_char[colname], (void*)(a->d_columns_char[colname] + offset*a->char_size[colname]), (unsigned int)a->char_size[colname] );
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_int[colname].begin(), a->d_columns_int[colname].begin() + offset);

        }
        else {
            str_gather((void*)thrust::raw_pointer_cast(a->prm_d.data()), g_size,
                       alloced_tmp, (void*)(a->d_columns_char[colname] + offset*a->char_size[colname]), (unsigned int)a->char_size[colname] );			   
        };
        if(a->d_columns_int.find(colname) != a->d_columns_int.end())
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_int[colname].begin(), a->d_columns_int[colname].begin() + offset);
    }
};

void mycopy(string colname, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
    if(t->type[colname] == 0) {
        if(!alloced_switch) {
            thrust::copy(t->d_columns_int[colname].begin(), t->d_columns_int[colname].begin() + g_size,
                         a->d_columns_int[colname].begin() + offset);
        }
        else {
            thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
            thrust::copy(d_col, d_col + g_size, a->d_columns_int[colname].begin() + offset);

        };
    }
    else if(t->type[colname] == 1) {
        if(!alloced_switch) {
            thrust::copy(t->d_columns_float[colname].begin(), t->d_columns_float[colname].begin() + g_size,
                         a->d_columns_float[colname].begin() + offset);
        }
        else {
            thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
            thrust::copy(d_col, d_col + g_size,	a->d_columns_float[colname].begin() + offset);
        };
    }
    else {
        if(!alloced_switch) {
            hipMemcpy((void**)(a->d_columns_char[colname] + offset*a->char_size[colname]), (void**)t->d_columns_char[colname],
                       g_size*t->char_size[colname], hipMemcpyDeviceToDevice);
            thrust::copy(t->d_columns_int[colname].begin(), t->d_columns_int[colname].begin() + g_size,
                         a->d_columns_int[colname].begin() + offset);
        }
        else {
            hipMemcpy((void**)(a->d_columns_char[colname] + offset*a->char_size[colname]), alloced_tmp,
                       g_size*t->char_size[colname], hipMemcpyDeviceToDevice);
        };
        if(a->d_columns_int.find(colname) != a->d_columns_int.end())
            thrust::copy(t->d_columns_int[colname].begin(), t->d_columns_int[colname].begin() + g_size,
                         a->d_columns_int[colname].begin() + offset);

    };
};



size_t load_queue(queue<string> c1, CudaSet* right, bool str_join, string f2, size_t &rcount,
                  unsigned int start_segment, unsigned int end_segment, bool rsz, bool flt)
{
    queue<string> cc;
    while(!c1.empty()) {
        if(std::find(right->columnNames.begin(), right->columnNames.end(), c1.front()) !=  right->columnNames.end()) {
            if(f2 != c1.front() ) {
                cc.push(c1.front());
            };
        };
        c1.pop();
    };
    if(std::find(right->columnNames.begin(), right->columnNames.end(), f2) !=  right->columnNames.end()) {
        cc.push(f2);
    };

    if(right->filtered) {
        allocColumns(right, cc);
    };

    rcount = right->maxRecs;
    queue<string> ct(cc);

    while(!ct.empty()) {
        if(right->filtered && rsz) {
            right->mRecCount = 0;
        }
        else {
            right->allocColumnOnDevice(ct.front(), rcount);
        };
        ct.pop();
    };


    size_t cnt_r = 0;
    right->devRecCount = 0;
    for(unsigned int i = start_segment; i < end_segment; i++) {
        if(!right->filtered)
            copyColumns(right, cc, i, cnt_r, rsz, 0);
        else
            copyColumns(right, cc, i, cnt_r, rsz, flt);
        cnt_r = cnt_r + right->mRecCount;
    };
    right->mRecCount = cnt_r;
    return cnt_r;

}

size_t max_char(CudaSet* a)
{
    size_t max_char1 = 8;
    for(unsigned int i = 0; i < a->columnNames.size(); i++) {
        if(a->type[a->columnNames[i]] == 2) {
            if (a->char_size[a->columnNames[i]] > max_char1)
                max_char1 = a->char_size[a->columnNames[i]];
        };
    };
    return max_char1;
};

size_t max_char(CudaSet* a, set<string> field_names)
{
    size_t max_char1 = 8;
    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        if (a->type[*it] == 2) {
            if (a->char_size[*it] > max_char1)
                max_char1 = a->char_size[*it];
        };
    };
    return max_char1;
};

size_t max_char(CudaSet* a, queue<string> field_names)
{
    size_t max_char = 8;
    while (!field_names.empty()) {
        if (a->type[field_names.front()] == 2) {
            if (a->char_size[field_names.front()] > max_char)
                max_char = a->char_size[field_names.front()];
        };
        field_names.pop();
    };
    return max_char;
};



size_t max_tmp(CudaSet* a)
{
    size_t max_sz = 0;
    for(unsigned int i = 0; i < a->columnNames.size(); i++) {
        if(a->type[a->columnNames[i]] == 0) {
            if(int_size > max_sz)
                max_sz = int_size;
        }
        else if(a->type[a->columnNames[i]] == 1) {
            if(float_size > max_sz)
                max_sz = float_size;
        };
    };
    size_t m_char = max_char(a);
    if(m_char > max_sz)
        return m_char;
    else
        return max_sz;

};


void setSegments(CudaSet* a, queue<string> cols)
{
    size_t mem_available = getFreeMem();
    size_t tot_sz = 0;
    while(!cols.empty()) {
        if(a->type[cols.front()] != 2)
            tot_sz = tot_sz + int_size;
        else
            tot_sz = tot_sz + a->char_size[cols.front()];
        cols.pop();
    };
    if(a->mRecCount*tot_sz > mem_available/3) { //default is 3
        a->segCount = (a->mRecCount*tot_sz)/(mem_available/5) + 1;
        a->maxRecs = (a->mRecCount/a->segCount)+1;
    };

};

void update_permutation_char(char* key, unsigned int* permutation, size_t RecCount, string SortType, char* tmp, unsigned int len)
{

    str_gather((void*)permutation, RecCount, (void*)key, (void*)tmp, len);

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
        str_sort(tmp, RecCount, permutation, 1, len);
    else
        str_sort(tmp, RecCount, permutation, 0, len);
}

void update_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, string SortType, char* tmp, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)tmp, len);

    if (SortType.compare("DESC") == 0 )
        str_sort_host(tmp, RecCount, permutation, 1, len);
    else
        str_sort_host(tmp, RecCount, permutation, 0, len);

}



void apply_permutation_char(char* key, unsigned int* permutation, size_t RecCount, char* tmp, unsigned int len)
{
    // copy keys to temporary vector
    hipMemcpy( (void*)tmp, (void*) key, RecCount*len, hipMemcpyDeviceToDevice);
    // permute the keys
    str_gather((void*)permutation, RecCount, (void*)tmp, (void*)key, len);
}


void apply_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, char* res, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)res, len);
}



void filter_op(char *s, char *f, unsigned int segment)
{
    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;
    //std::clock_t start1 = std::clock();

    if(a->mRecCount == 0 && !a->filtered) {
        b = new CudaSet(0,1);
    }
    else {
        if(verbose)
            cout << "FILTER " << s << " " << f << " " << getFreeMem() << '\xd';


        b = varNames[s];
        b->name = s;
        size_t cnt = 0;
        allocColumns(a, b->fil_value);

        if (b->prm_d.size() == 0)
            b->prm_d.resize(a->maxRecs);

        //cout << endl << "MAP CHECK start " << segment <<  endl;
        char map_check = zone_map_check(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, a, segment);
        //cout << endl << "MAP CHECK segment " << segment << " " << map_check <<  endl;

        if(map_check == 'R') {
            copyColumns(a, b->fil_value, segment, cnt);
            bool* res = filter(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, a, segment);
            thrust::device_ptr<bool> bp((bool*)res);
            b->prm_index = 'R';
            b->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 1);
            thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
                            bp, b->prm_d.begin(), thrust::identity<bool>());
            hipFree(res);
        }
        else  {
            setPrm(a,b,map_check,segment);
        };
        if(segment == a->segCount-1)
            a->deAllocOnDevice();
    }
    if(verbose)
        cout << endl << "filter res " << b->mRecCount << endl;
    //std::cout<< "filter time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';
}




size_t load_right(CudaSet* right, string colname, string f2, queue<string> op_g, queue<string> op_sel,
                  queue<string> op_alt, bool decimal_join, bool& str_join,
                  size_t& rcount, unsigned int start_seg, unsigned int end_seg, bool rsz) {

    size_t cnt_r = 0;
    //if join is on strings then add integer columns to left and right tables and modify colInd1 and colInd2

    // need to allocate all right columns
    if(right->not_compressed) {
        queue<string> op_alt1;
        op_alt1.push(f2);
        cnt_r = load_queue(op_alt1, right, str_join, "", rcount, start_seg, end_seg, rsz, 1);
    }
    else {
        cnt_r = load_queue(op_alt, right, str_join, f2, rcount, start_seg, end_seg, rsz, 1);
    };


    if (right->type[colname]  == 2) {
        str_join = 1;
        right->d_columns_int[f2] = thrust::device_vector<int_type>();
        for(unsigned int i = start_seg; i < end_seg; i++) {
            right->add_hashed_strings(f2, i);
        };
        cnt_r = right->d_columns_int[f2].size();
    };


    if(right->not_compressed) {
        queue<string> op_alt1;
        while(!op_alt.empty()) {
            if(f2.compare(op_alt.front())) {
                if (std::find(right->columnNames.begin(), right->columnNames.end(), op_alt.front()) != right->columnNames.end()) {
                    op_alt1.push(op_alt.front());
                };
            };
            op_alt.pop();
        };
        if(!op_alt1.empty())
            cnt_r = load_queue(op_alt1, right, str_join, "", rcount, start_seg, end_seg, 0, 0);
    };
    return cnt_r;
};

unsigned int calc_right_partition(CudaSet* left, CudaSet* right, queue<string> op_sel) {
    size_t tot_size = left->maxRecs*8;

    while(!op_sel.empty()) {
        if (std::find(right->columnNames.begin(), right->columnNames.end(), op_sel.front()) != right->columnNames.end()) {

            if(right->type[op_sel.front()] <= 1) {
                tot_size = tot_size + right->maxRecs*8*right->segCount;
            }
            else {
                tot_size = tot_size + right->maxRecs*
                           right->char_size[op_sel.front()]*
                           right->segCount;
            };
        };
        op_sel.pop();
    };

    //cout << "tot size " << tot_size << " " << right->maxRecs << " " << right->mRecCount << endl;

    if(tot_size + 300000000 < getFreeMem()) //00
        return right->segCount;
    else {
        if(right->segCount == 1) { //need to partition it. Not compressed.
            right->segCount = ((tot_size*2 )/getFreeMem())+1;
            //right->segCount = 8;
            cout << "seg count " << right->segCount << endl;
            right->maxRecs = (right->mRecCount/right->segCount)+1;
            cout << "max recs " << right->maxRecs << endl;
            return 1;
        }
        else { //compressed
            return right->segCount / ((tot_size+300000000)/getFreeMem());
        };
    };

};


string int_to_string(int number) {
    string number_string = "";
    char ones_char;
    int ones = 0;
    while(true) {
        ones = number % 10;
        switch(ones) {
        case 0:
            ones_char = '0';
            break;
        case 1:
            ones_char = '1';
            break;
        case 2:
            ones_char = '2';
            break;
        case 3:
            ones_char = '3';
            break;
        case 4:
            ones_char = '4';
            break;
        case 5:
            ones_char = '5';
            break;
        case 6:
            ones_char = '6';
            break;
        case 7:
            ones_char = '7';
            break;
        case 8:
            ones_char = '8';
            break;
        case 9:
            ones_char = '9';
            break;
        default :
            cout << ("Trouble converting number to string.");
        }
        number -= ones;
        number_string = ones_char + number_string;
        if(number == 0) {
            break;
        }
        number = number/10;
    }
    return number_string;
}


void insert_records(char* f, char* s) {
    char buf[4096];
    size_t size, maxRecs, cnt = 0;
    string str_s, str_d;

    if(varNames.find(s) == varNames.end()) {
        process_error(3, "couldn't find " + string(s) );
    };
    CudaSet *a;
    a = varNames.find(s)->second;
    a->name = s;

    if(varNames.find(f) == varNames.end()) {
        process_error(3, "couldn't find " + string(f) );
    };

    CudaSet *b;
    b = varNames.find(f)->second;
    b->name = f;

    // if both source and destination are on disk
    cout << "SOURCES " << a->source << ":" << b->source << endl;
    if(a->source && b->source) {
        for(unsigned int i = 0; i < a->segCount; i++) {
            for(unsigned int z = 0; z < a->columnNames.size(); z++) {
                str_s = a->load_file_name + "." + a->columnNames[z] + "." + int_to_string(i);
                str_d = b->load_file_name + "." + a->columnNames[z] + "." + int_to_string(b->segCount + i);
                cout << str_s << " " << str_d << endl;
                FILE* source = fopen(str_s.c_str(), "rb");
                FILE* dest = fopen(str_d.c_str(), "wb");
                while (size = fread(buf, 1, BUFSIZ, source)) {
                    fwrite(buf, 1, size, dest);
                }
                fclose(source);
                fclose(dest);
            };
        };

        if(a->maxRecs > b->maxRecs)
            maxRecs = a->maxRecs;
        else
            maxRecs = b->maxRecs;

        for(unsigned int i = 0; i < b->columnNames.size(); i++) {
            b->reWriteHeader(b->load_file_name, b->columnNames[i], a->segCount + b->segCount, a->totalRecs + b->totalRecs, maxRecs);
        };
    }
    else if(!a->source && !b->source) { //if both source and destination are in memory
        size_t oldCount = b->mRecCount;
        b->resize(a->mRecCount);
        for(unsigned int z = 0; z< b->mColumnCount; z++) {
            if(b->type[a->columnNames[z]] == 0) {
                thrust::copy(a->h_columns_int[a->columnNames[z]].begin(), a->h_columns_int[a->columnNames[z]].begin() + a->mRecCount, b->h_columns_int[b->columnNames[z]].begin() + oldCount);
            }
            else if(b->type[a->columnNames[z]] == 1) {
                thrust::copy(a->h_columns_float[a->columnNames[z]].begin(), a->h_columns_float[a->columnNames[z]].begin() + a->mRecCount, b->h_columns_float[b->columnNames[z]].begin() + oldCount);
            }
            else {
                hipMemcpy(b->h_columns_char[b->columnNames[z]] + b->char_size[b->columnNames[z]]*oldCount, a->h_columns_char[a->columnNames[z]], a->char_size[a->columnNames[z]]*a->mRecCount, hipMemcpyHostToHost);
            };
        };
    }
    else if(!a->source && b->source) {


        total_segments = b->segCount;
        total_count = b->mRecCount;
        total_max = b->maxRecs;;

        queue<string> op_vx;
        for(unsigned int i=0; i < a->columnNames.size(); i++)
            op_vx.push(a->columnNames[i]);

        allocColumns(a, op_vx);
        a->resize(a->maxRecs);
        for(unsigned int i = 0; i < a->segCount; i++) {

            if (a->filtered) {
                copyColumns(a, op_vx, i, cnt);
                a->CopyToHost(0, a->mRecCount);
            };

            a->compress(b->load_file_name, 0, 1, i - (a->segCount-1), a->mRecCount);
        };
        //update headers
        //total_count = a->mRecCount + b->mRecCount;
        //cout << "and now lets write " << total_segments << " " <<  total_count << " " << total_max << endl;
        for(unsigned int i = 0; i < b->columnNames.size(); i++) {
            b->writeHeader(b->load_file_name, b->columnNames[i], total_segments);
        };
    };
};



void delete_records(char* f) {

    CudaSet *a;
    a = varNames.find(f)->second;
    a->name = f;
    size_t totalRemoved = 0;
    size_t maxRecs = 0;

    if(!a->keep) { // temporary variable
        process_error(2, "Delete operator is only applicable to disk based sets\nfor deleting records from derived sets please use filter operator ");
    }
    else {  // read matching segments, delete, compress and write on a disk replacing the original segments

        string str, str_old;
        queue<string> op_vx;
        size_t cnt;
        map<string, col_data> s = data_dict[a->load_file_name];
        for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {
            op_vx.push((*it).first);
            if (std::find(a->columnNames.begin(), a->columnNames.end(), (*it).first) == a->columnNames.end()) {

                if ((*it).second.col_type == 0) {
                    a->type[(*it).first] = 0;
                    a->decimal[(*it).first] = 0;
                    a->h_columns_int[(*it).first] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
                    a->d_columns_int[(*it).first] = thrust::device_vector<int_type>();
                }
                else if((*it).second.col_type == 1) {
                    a->type[(*it).first] = 1;
                    a->decimal[(*it).first] = 0;
                    a->h_columns_float[(*it).first] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                    a->d_columns_float[(*it).first] = thrust::device_vector<float_type>();
                }
                else if ((*it).second.col_type == 3) {
                    a->type[(*it).first] = 1;
                    a->decimal[(*it).first] = 1;
                    a->h_columns_float[(*it).first] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                    a->d_columns_float[(*it).first] = thrust::device_vector<float_type>();
                }
                else {
                    a->type[(*it).first] = 2;
                    a->decimal[(*it).first] = 0;
                    a->h_columns_char[(*it).first] = NULL;
                    a->d_columns_char[(*it).first] = NULL;
                    a->char_size[(*it).first] = (*it).second.col_length;
                };
                a->columnNames.push_back((*it).first);
            }
        };

        allocColumns(a, op_vx);
        a->resize(a->maxRecs);
        a->prm_d.resize(a->maxRecs);
        size_t cc = a->mRecCount;
        size_t tmp;

        void* d;
        CUDA_SAFE_CALL(hipMalloc((void **) &d, a->maxRecs*float_size));
        unsigned int new_seg_count = 0;
        char map_check;

        for(unsigned int i = 0; i < a->segCount; i++) {

            map_check = zone_map_check(op_type,op_value,op_nums, op_nums_f, a, i);
            if(verbose)
                cout << "MAP CHECK segment " << i << " " << map_check <<  endl;
            if(map_check != 'N') {

                cnt = 0;
                copyColumns(a, op_vx, i, cnt);
                tmp = a->mRecCount;

                if(a->mRecCount) {
                    bool* res = filter(op_type,op_value,op_nums, op_nums_f, a, i);
                    thrust::device_ptr<bool> bp((bool*)res);
                    thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
                                    bp, a->prm_d.begin(), not_identity<bool>());

                    a->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 0);
                    hipFree(res);

//					cout << "Remained recs count " << a->mRecCount << endl;
                    if(a->mRecCount > maxRecs)
                        maxRecs = a->mRecCount;

                    if (a->mRecCount) {

                        totalRemoved = totalRemoved + (tmp - a->mRecCount);
                        if (a->mRecCount == tmp) { //none deleted
                            if(new_seg_count != i) {
                                map<string, col_data> s = data_dict[a->load_file_name];
                                for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {
                                    string colname = (*it).first;


                                    str_old = a->load_file_name + "." + colname;
                                    str_old += "." + int_to_string(i);
                                    str = a->load_file_name + "." + colname;
                                    str += "." + int_to_string(new_seg_count);

                                    remove(str.c_str());
                                    rename(str_old.c_str(), str.c_str());
                                };
                            };
                            new_seg_count++;

                        }
                        else { //some deleted
                            //cout << "writing segment " << new_seg_count << endl;

                            map<string, col_data> s = data_dict[a->load_file_name];
                            for ( map<string, col_data>::iterator it=s.begin() ; it != s.end(); ++it ) {
                                string colname = (*it).first;
                                str = a->load_file_name + "." + colname;
                                str += "." + int_to_string(new_seg_count);

                                if(a->type[colname] == 0) {
                                    thrust::device_ptr<int_type> d_col((int_type*)d);
                                    thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_int[colname].begin(), d_col);
                                    pfor_compress( d, a->mRecCount*int_size, str, a->h_columns_int[colname], 0);
                                }
                                else if(a->type[colname] == 1) {
                                    thrust::device_ptr<float_type> d_col((float_type*)d);
                                    if(a->decimal[colname]) {
                                        thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[colname].begin(), d_col);
                                        thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                                        thrust::transform(d_col,d_col+a->mRecCount, d_col_dec, float_to_long());
                                        pfor_compress( d, a->mRecCount*float_size, str, a->h_columns_float[colname], 1);
                                    }
                                    else {
                                        thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[colname].begin(), d_col);
                                        thrust::copy(d_col, d_col + a->mRecCount, a->h_columns_float[colname].begin());
                                        fstream binary_file(str.c_str(),ios::out|ios::binary);
                                        binary_file.write((char *)&a->mRecCount, 4);
                                        binary_file.write((char *)(a->h_columns_float[colname].data()),a->mRecCount*float_size);
                                        unsigned int comp_type = 3;
                                        binary_file.write((char *)&comp_type, 4);
                                        binary_file.close();

                                    };
                                }
                                else {
                                    void* t;
                                    CUDA_SAFE_CALL(hipMalloc((void **) &t, tmp*a->char_size[colname]));
                                    apply_permutation_char(a->d_columns_char[colname], (unsigned int*)thrust::raw_pointer_cast(a->prm_d.data()), tmp, (char*)t, a->char_size[colname]);
                                    hipMemcpy(a->h_columns_char[colname], a->d_columns_char[colname], a->char_size[colname]*a->mRecCount, hipMemcpyDeviceToHost);
                                    hipFree(t);
                                    a->compress_char(str, colname, a->mRecCount, 0);
                                };
                            };
                            new_seg_count++;
                        };
                    }
                    else {
                        totalRemoved = totalRemoved + tmp;
                    };
                }
            }
            else {
                if(new_seg_count != i) {
                    //cout << "rename " << i << " to " << new_seg_count << endl;
                    //for(unsigned int z = 0; z< a->mColumnCount; z++) {
                    for(unsigned int z = 0; z < a->columnNames.size(); z++) {

                        str_old = a->load_file_name + "." + a->columnNames[z];
                        str_old += "." + int_to_string(i);
                        str = a->load_file_name + "." + a->columnNames[z];
                        str += "." + int_to_string(new_seg_count);

                        remove(str.c_str());
                        rename(str_old.c_str(), str.c_str());
                    };
                };
                new_seg_count++;
                maxRecs	= a->maxRecs;
            };
            //cout << "TOTAL REM " << totalRemoved << endl;
        };

        if (new_seg_count < a->segCount) {
            for(unsigned int i = new_seg_count; i < a->segCount; i++) {
                //cout << "delete segment " << i << endl;
                for(unsigned int z = 0; z < a->columnNames.size(); z++) {
                    str = a->load_file_name + "." + a->columnNames[z];
                    str += "." + int_to_string(i);
                    remove(str.c_str());
                };
            };
        };

        for(unsigned int i = new_seg_count; i < a->segCount; i++) {
            a->reWriteHeader(a->load_file_name, a->columnNames[i], new_seg_count, a->totalRecs-totalRemoved, maxRecs);
        };


        a->mRecCount = cc;
        a->prm_d.resize(0);
        a->segCount = new_seg_count;
        a->deAllocOnDevice();
        hipFree(d);
    };


};


void save_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
    size_t str_len;
    fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
    size_t len = data_dict.size();
    binary_file.write((char *)&len, 8);
    for ( map<string, map<string, col_data> >::iterator it=data_dict.begin() ; it != data_dict.end(); ++it ) {
        str_len = (*it).first.size();
        binary_file.write((char *)&str_len, 8);
        binary_file.write((char *)(*it).first.data(), str_len);
        map<string, col_data> s = (*it).second;
        size_t len1 = s.size();
        binary_file.write((char *)&len1, 8);

        for ( map<string, col_data>::iterator sit=s.begin() ; sit != s.end(); ++sit ) {
            str_len = (*sit).first.size();
            binary_file.write((char *)&str_len, 8);
            binary_file.write((char *)(*sit).first.data(), str_len);
            binary_file.write((char *)&(*sit).second.col_type, 4);
            binary_file.write((char *)&(*sit).second.col_length, 4);
        };
    };
    binary_file.close();
}

void load_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
    size_t str_len, recs, len1;
    string str1, str2;
    char buffer[4000];
    unsigned int col_type, col_length;
    fstream binary_file;
    binary_file.open(file_name.c_str(),ios::in|ios::binary);
    if(binary_file.is_open()) {
        binary_file.read((char*)&recs, 8);
        for(unsigned int i = 0; i < recs; i++) {
            binary_file.read((char*)&str_len, 8);
            binary_file.read(buffer, str_len);
            str1.assign(buffer, str_len);
            binary_file.read((char*)&len1, 8);

            for(unsigned int j = 0; j < len1; j++) {
                binary_file.read((char*)&str_len, 8);
                binary_file.read(buffer, str_len);
                str2.assign(buffer, str_len);
                binary_file.read((char*)&col_type, 4);
                binary_file.read((char*)&col_length, 4);
                data_dict[str1][str2].col_type = col_type;
                data_dict[str1][str2].col_length = col_length;
                //cout << "data DICT " << str1 << " " << str2 << " " << col_type << " " << col_length << endl;
            };
        };
        binary_file.close();
    }
    else {
        cout << "Coudn't open data dictionary" << endl;
    };
	cout << "load_col_data->end" << endl;
}

bool var_exists(CudaSet* a, string name) {

    if(std::find(a->columnNames.begin(), a->columnNames.end(), name) !=  a->columnNames.end())
        return 1;
    else

        return 0;
}


#ifdef _WIN64
size_t getTotalSystemMemory()
{
    MEMORYSTATUSEX status;
    status.dwLength = sizeof(status);
    GlobalMemoryStatusEx(&status);
    return status.ullTotalPhys;
}
#else
size_t getTotalSystemMemory()
{
    long pages = sysconf(_SC_PHYS_PAGES);
    long page_size = sysconf(_SC_PAGE_SIZE);
    return pages * page_size;
}
#endif

