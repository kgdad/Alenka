/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "cm.h"


using namespace mgpu;

template<typename T>
struct distinct : public binary_function<T,T,T>
{
    __host__ __device__ T operator()(const T &lhs, const T &rhs) const {
        return lhs != rhs;
    }
};


void select(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums, queue<float_type> op_nums_f, CudaSet* a,
            CudaSet* b, vector<thrust::device_vector<int_type> >& distinct_tmp, bool& one_liner)
{

    stack<string> exe_type;
    stack<string> exe_value;
    stack<int_type*> exe_vectors;
    stack<int_type> exe_nums;
    string  s1, s2, s1_val, s2_val;
    int_type n1, n2, res;
    unsigned int colCount = 0;
    stack<int> col_type;
    string grp_type;
    stack<string> grp_type1;
    stack<string> col_val;
    size_t res_size = 0;

    stack<string> exe_value1;
    stack<int_type*> exe_vectors1;
    stack<float_type*> exe_vectors1_d;
    stack<int_type> exe_nums1;

    stack<float_type*> exe_vectors_f;
    stack<float_type> exe_nums_f;
    float_type n1_f, n2_f, res_f;
    bool one_line;
    unsigned int dist_processed = 0;
    bool prep = 0;
    one_line = 0;

    thrust::device_ptr<bool> d_di(a->grp);

    std::auto_ptr<ReduceByKeyPreprocessData> ppData;

    if (!a->columnGroups.empty() && (a->mRecCount != 0))
        res_size = a->grp_count;

    for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

        string ss = op_type.front();


        if(ss.compare("emit sel_name") != 0) {
            grp_type = "NULL";

            if (ss.compare("COUNT") == 0  || ss.compare("SUM") == 0  || ss.compare("AVG") == 0 || ss.compare("MIN") == 0 || ss.compare("MAX") == 0 || ss.compare("DISTINCT") == 0) {

                if(!prep && !a->columnGroups.empty()) {

                    mgpu::ReduceByKeyPreprocess<float_type>((int)a->mRecCount, thrust::raw_pointer_cast(d_di),
                                                            (bool*)0, head_flag_predicate<bool>(), (int*)0, (int*)0,
                                                            &ppData, *context);
                    prep = 1;
                };


                if(a->columnGroups.empty())
                    one_line = 1;

                if (ss.compare("DISTINCT") == 0) {
                    s1_val = exe_value.top();
                    exe_type.pop();
                    exe_value.pop();


                    if(a->type[s1_val] == 0) {

                        thrust::copy(a->d_columns_int[s1_val].begin(), a->d_columns_int[s1_val].begin() + a->mRecCount,
                                     distinct_tmp[dist_processed].begin());
                        dist_processed++;
                        thrust::device_ptr<int_type> res = thrust::device_malloc<int_type>(res_size);
                        exe_vectors.push(thrust::raw_pointer_cast(res));
                        exe_type.push("VECTOR");
                    }
                    else if(a->type[s1_val] == 2) {
                        //will add a DISTINCT on strings if anyone needs it
                        cout << "DISTINCT on strings is not supported yet" << endl;
                        exit(0);
                    }
                    else {
                        cout << "DISTINCT on float is not supported" << endl;
                        exit(0);
                    };
                }

                else if (ss.compare("COUNT") == 0) {

                    s1 = exe_type.top();
                    if(s1.compare("VECTOR") != 0) {  // non distinct

                        grp_type = "COUNT";
                        exe_type.pop();
                        s1_val = exe_value.top();
                        exe_value.pop();


                        if (!a->columnGroups.empty()) {
                            thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                            thrust::device_ptr<int_type> const_seq = thrust::device_malloc<int_type>(a->mRecCount);
                            thrust::fill(const_seq, const_seq+a->mRecCount, (int_type)1);
                            ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(const_seq), (int_type)0,
                                             mgpu::plus<int_type>(), thrust::raw_pointer_cast(count_diff), *context);

                            thrust::device_free(const_seq);
                            //thrust::reduce_by_key(d_di, d_di+(a->mRecCount), thrust::constant_iterator<int_type>(1),
                            //                      thrust::make_discard_iterator(), count_diff,
                            //                      head_flag_predicate<bool>(),thrust::plus<int_type>());

                            exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR");
                        }
                        else {
                            thrust::device_ptr<int_type> dest  = thrust::device_malloc<int_type>(1);
                            dest[0] = a->mRecCount;
                            exe_vectors.push(thrust::raw_pointer_cast(dest));
                            exe_type.push("VECTOR");
                        }
                    }
                    else
                        grp_type = "COUNTD";

                }
                else if (ss.compare("SUM") == 0) {

                    /*if(op_case) {
                    	cout << "found case " << endl;
                    	op_case = 0;
                    	while(!exe_type.empty())
                    	{
                    	cout << "CASE type " << exe_type.top() << endl;
                    	exe_type.pop();
                    	exit(0);
                    	}

                    };
                    */


                    grp_type = "SUM";
                    s1 = exe_type.top();
                    exe_type.pop();

                    if (s1.compare("VECTOR F") == 0) {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();

                        if (!a->columnGroups.empty()) {
                            thrust::device_ptr<float_type> source((float_type*)(s3));
                            //thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                            thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(a->mRecCount);

                            ReduceByKeyApply(*ppData, s3, (float_type)0,
                                             mgpu::plus<float_type>(), thrust::raw_pointer_cast(count_diff), *context);

                            //thrust::reduce_by_key(d_di, d_di + a->mRecCount, source,
                            //                      thrust::make_discard_iterator(), count_diff,
                            //                      head_flag_predicate<bool>(),thrust::plus<float_type>());

                            exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR F");
                        }
                        else {
                            thrust::device_ptr<float_type> source((float_type*)(s3));
                            thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(1);
                            count_diff[0] = mgpu::Reduce(thrust::raw_pointer_cast(source), a->mRecCount, *context);
                            exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR F");
                        };
                        hipFree(s3);
                    }
                    if (s1.compare("VECTOR") == 0) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();

                        if (!a->columnGroups.empty()) {
                            thrust::device_ptr<int_type> source((int_type*)(s3));
                            thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                            ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(source), (int_type)0,
                                             mgpu::plus<int_type>(), thrust::raw_pointer_cast(count_diff), *context);
                            exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR");
                        }
                        else {
                            thrust::device_ptr<int_type> source((int_type*)(s3));
                            thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(1);
                            count_diff[0] = mgpu::Reduce(thrust::raw_pointer_cast(source), a->mRecCount, *context);
                            exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                            exe_type.push("VECTOR");
                        };
                        hipFree(s3);
                    }

                    else if (s1.compare("NAME") == 0) {
                        s1_val = exe_value.top();
                        exe_value.pop();

                        if (!a->columnGroups.empty()) {

                            if(a->type[s1_val] == 0) {
                                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                                ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), (int_type)0,
                                                 mgpu::plus<int_type>(), thrust::raw_pointer_cast(count_diff), *context);
                                exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                                exe_type.push("VECTOR");
                            }
                            else if(a->type[s1_val] == 1) {
                                thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                                ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), (float_type)0,
                                                 mgpu::plus<float_type>(), thrust::raw_pointer_cast(count_diff), *context);
                                exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                                exe_type.push("VECTOR F");
                            }
                        }
                        else {
                            if(a->type[s1_val] == 0) {
                                thrust::device_ptr<int_type> dest;
                                int_type cc = mgpu::Reduce(thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), a->mRecCount, *context);
                                if (one_line) {
                                    dest = thrust::device_malloc<int_type>(1);
                                    dest[0] = cc;
                                }
                                else {
                                    dest = thrust::device_malloc<int_type>(a->mRecCount);
                                    thrust::sequence(dest, dest+(a->mRecCount), cc, (int_type)0);
                                };
                                exe_vectors.push(thrust::raw_pointer_cast(dest));
                                exe_type.push("VECTOR");
                            }
                            else if(a->type[s1_val] == 1) {
                                thrust::device_ptr<float_type> dest;
                                float_type cc = mgpu::Reduce(thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), a->mRecCount, *context);

                                if (one_line) {
                                    dest = thrust::device_malloc<float_type>(1);
                                    dest[0] = cc;
                                }
                                else {
                                    dest = thrust::device_malloc<float_type>(a->mRecCount);
                                    thrust::sequence(dest, dest+a->mRecCount, cc, (float_type)0);
                                };
                                exe_vectors_f.push(thrust::raw_pointer_cast(dest));
                                exe_type.push("VECTOR F");
                            };
                        };
                    }
                }
                else if (ss.compare("MIN") == 0) {

                    grp_type = "MIN";
                    s1 = exe_type.top();
                    exe_type.pop();

                    s1_val = exe_value.top();
                    exe_value.pop();

                    if(a->type[s1_val] == 0) {

                        thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                        ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), (int_type)0,
                                         mgpu::minimum<int_type>(), thrust::raw_pointer_cast(count_diff), *context);
                        exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR");

                    }
                    else if(a->type[s1_val] == 1) {

                        thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                        //ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), (float_type)0,
                        //						mgpu::minimum<float_type>(), thrust::raw_pointer_cast(count_diff), *context);


                        thrust::reduce_by_key(d_di, d_di+(a->mRecCount), a->d_columns_float[s1_val].begin(),
                                              thrust::make_discard_iterator(), count_diff,
                                              head_flag_predicate<bool>(), thrust::minimum<float_type>());

                        exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR F");
                    }
                }
                else if (ss.compare("MAX") == 0) {

                    grp_type = "MAX";
                    s1 = exe_type.top();
                    exe_type.pop();

                    s1_val = exe_value.top();
                    exe_value.pop();

                    if(a->type[s1_val] == 0) {

                        thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                        ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), (int_type)0,
                                         mgpu::maximum<int_type>(), thrust::raw_pointer_cast(count_diff), *context);
						cout << "reducing " << endl;
						//thrust::reduce_by_key(d_di, d_di+(a->mRecCount), a->d_columns_int[s1_val].begin(),
                        //                      thrust::make_discard_iterator(), count_diff,
                        //                      head_flag_predicate<bool>(), thrust::maximum<int_type>());
						cout << "reducing1 " << endl;
                        exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR");

                    }
                    else if(a->type[s1_val] == 1) {

                        thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                        ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), (float_type)0,
                        						mgpu::maximum<float_type>(), thrust::raw_pointer_cast(count_diff), *context);


                        //thrust::reduce_by_key(d_di, d_di+(a->mRecCount), a->d_columns_float[s1_val].begin(),
                        //                      thrust::make_discard_iterator(), count_diff,
                        //                      head_flag_predicate<bool>(), thrust::maximum<float_type>());

                        exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR F");
                    }
                }				

                else if (ss.compare("AVG") == 0) {
                    grp_type = "AVG";
                    s1 = exe_type.top();
                    exe_type.pop();

                    s1_val = exe_value.top();
                    exe_value.pop();

                    if(a->type[s1_val] == 0) {

                        thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                        ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_int[s1_val].data()), (int_type)0,
                                         mgpu::plus<int_type>(), thrust::raw_pointer_cast(count_diff), *context);

                        exe_vectors.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR");
                    }
                    else if(a->type[s1_val] == 1) {

                        thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                        ReduceByKeyApply(*ppData, thrust::raw_pointer_cast(a->d_columns_float[s1_val].data()), (float_type)0,
                                         mgpu::plus<float_type>(), thrust::raw_pointer_cast(count_diff), *context);
                        exe_vectors_f.push(thrust::raw_pointer_cast(count_diff));
                        exe_type.push("VECTOR F");
                    }
                };
            };

            if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("VECTOR") == 0 || ss.compare("VECTOR F") == 0) {

                exe_type.push(ss);
                if (ss.compare("NUMBER") == 0) {
                    exe_nums.push(op_nums.front());
                    op_nums.pop();
                }
                else if (ss.compare("NAME") == 0) {
                    exe_value.push(op_value.front());
                    op_value.pop();
                }
            }
            else {
                if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
                    // get 2 values from the stack
                    s1 = exe_type.top();
                    exe_type.pop();
                    s2 = exe_type.top();
                    exe_type.pop();


                    if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                        n1 = exe_nums.top();
                        exe_nums.pop();
                        n2 = exe_nums.top();
                        exe_nums.pop();

                        if (ss.compare("ADD") == 0 )
                            res = n1+n2;
                        else if (ss.compare("MUL") == 0 )
                            res = n1*n2;
                        else if (ss.compare("DIV") == 0 )
                            res = n1/n2;
                        else
                            res = n1-n2;

                        thrust::device_ptr<int_type> p = thrust::device_malloc<int_type>(a->mRecCount);
                        thrust::sequence(p, p+(a->mRecCount),res,(int_type)0);

                        exe_type.push("VECTOR");
                        exe_vectors.push(thrust::raw_pointer_cast(p));
                    }
                    else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                        n1_f = exe_nums_f.top();
                        exe_nums_f.pop();
                        n2_f = exe_nums_f.top();
                        exe_nums_f.pop();

                        if (ss.compare("ADD") == 0 )
                            res_f = n1_f+n2_f;
                        else if (ss.compare("MUL") == 0 )
                            res_f = n1_f*n2_f;
                        else if (ss.compare("DIV") == 0 )
                            res_f = n1_f/n2_f;
                        else
                            res_f = n1_f-n2_f;

                        thrust::device_ptr<float_type> p = thrust::device_malloc<float_type>(a->mRecCount);
                        thrust::sequence(p, p+(a->mRecCount),res_f,(float_type)0);

                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(thrust::raw_pointer_cast(p));

                    }

                    else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                        s1_val = exe_value.top();
                        exe_value.pop();
                        n1_f = exe_nums_f.top();
                        exe_nums_f.pop();

                        exe_type.push("VECTOR F");

                        if (a->type[s1_val] == 1) {
                            float_type* t = a->get_float_type_by_name(s1_val);
                            exe_vectors_f.push(a->op(t,n1_f,ss,1));
                        }
                        else {
                            int_type* t = a->get_int_by_name(s1_val);
                            exe_vectors_f.push(a->op(t,n1_f,ss,1));
                        };

                    }
                    else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                        n1_f = exe_nums_f.top();
                        exe_nums_f.pop();
                        s2_val = exe_value.top();
                        exe_value.pop();

                        exe_type.push("VECTOR F");

                        if (a->type[s2_val] == 1) {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            exe_vectors_f.push(a->op(t,n1_f,ss,0));
                        }
                        else {
                            int_type* t = a->get_int_by_name(s2_val);
                            exe_vectors_f.push(a->op(t,n1_f,ss,0));
                        };
                    }
                    else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {

                        s1_val = exe_value.top();
                        exe_value.pop();
                        n1 = exe_nums.top();
                        exe_nums.pop();

                        if (a->type[s1_val] == 1) {
                            float_type* t = a->get_float_type_by_name(s1_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,(float_type)n1,ss,1));
                        }
                        else {
                            int_type* t = a->get_int_by_name(s1_val);
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,n1,ss,1));
                        };
                    }
                    else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                        n1 = exe_nums.top();
                        exe_nums.pop();
                        s2_val = exe_value.top();
                        exe_value.pop();

                        if (a->type[s2_val] == 1) {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,(float_type)n1,ss,0));
                        }
                        else {
                            int_type* t = a->get_int_by_name(s2_val);
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,n1,ss,0));
                        };
                    }
                    else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                        s1_val = exe_value.top();
                        exe_value.pop();
                        s2_val = exe_value.top();
                        exe_value.pop();

                        if (a->type[s1_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s1_val);
                            if (a->type[s2_val] == 0) {
                                int_type* t = a->get_int_by_name(s2_val);
                                exe_type.push("VECTOR");
                                exe_vectors.push(a->op(t1,t,ss,0));
                            }
                            else {
                                float_type* t = a->get_float_type_by_name(s2_val);
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t1,t,ss,0));
                            };
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s1_val);

                            if (a->type[s2_val] == 0) {
                                int_type* t1 = a->get_int_by_name(s2_val);
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t1,t,ss,0));
                            }
                            else {
                                float_type* t1 = a->get_float_type_by_name(s2_val);
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t1,t,ss,0));
                            };
                        }
                    }
                    else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0 ) && s2.compare("NAME") == 0) {

                        s2_val = exe_value.top();
                        exe_value.pop();

                        if (a->type[s2_val] == 0) {
                            int_type* t = a->get_int_by_name(s2_val);

                            if (s1.compare("VECTOR") == 0 ) {
                                int_type* s3 = exe_vectors.top();
                                exe_vectors.pop();
                                exe_type.push("VECTOR");
                                exe_vectors.push(a->op(t,s3,ss,0));
                                //free s3
                                hipFree(s3);

                            }
                            else {
                                float_type* s3 = exe_vectors_f.top();
                                exe_vectors_f.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t,s3,ss,0));
                                hipFree(s3);
                            }
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            if (s1.compare("VECTOR") == 0 ) {
                                int_type* s3 = exe_vectors.top();
                                exe_vectors.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(s3,t, ss,0));
                                hipFree(s3);
                            }
                            else {
                                float_type* s3 = exe_vectors_f.top();
                                exe_vectors_f.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t,s3,ss,0));
                                hipFree(s3);
                            }
                        };
                    }
                    else if ((s2.compare("VECTOR") == 0 || s2.compare("VECTOR F") == 0 ) && s1.compare("NAME") == 0) {

                        s1_val = exe_value.top();
                        exe_value.pop();

                        if (a->type[s1_val] == 0) {
                            int_type* t = a->get_int_by_name(s1_val);

                            if (s2.compare("VECTOR") == 0 ) {
                                int_type* s3 = exe_vectors.top();
                                exe_vectors.pop();
                                exe_type.push("VECTOR");
                                exe_vectors.push(a->op(t,s3,ss,1));
                                hipFree(s3);
                            }
                            else {
                                float_type* s3 = exe_vectors_f.top();
                                exe_vectors_f.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t,s3,ss,1));
                                hipFree(s3);
                            }
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s1_val);
                            if (s2.compare("VECTOR") == 0 ) {
                                int_type* s3 = exe_vectors.top();
                                exe_vectors.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(s3,t,ss,1));
                                hipFree(s3);
                            }
                            else {
                                float_type* s3 = exe_vectors_f.top();
                                exe_vectors_f.pop();
                                exe_type.push("VECTOR F");
                                exe_vectors_f.push(a->op(t,s3,ss,1));
                                hipFree(s3);
                            }
                        };
                    }
                    else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("NUMBER") == 0) {
                        n1 = exe_nums.top();
                        exe_nums.pop();

                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(s3,n1, ss,1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,(float_type)n1, ss,1));
                            hipFree(s3);
                        }
                    }
                    else if (s1.compare("NUMBER") == 0 && (s2.compare("VECTOR") || s2.compare("VECTOR F") == 0)) {
                        n1 = exe_nums.top();
                        exe_nums.pop();

                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(s3,n1, ss,0));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,(float_type)n1, ss,0));
                            hipFree(s3);
                        }
                    }

                    else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("FLOAT") == 0) {
                        n1_f = exe_nums_f.top();
                        exe_nums_f.pop();

                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,n1_f, ss,1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,n1_f, ss,1));
                            hipFree(s3);
                        }
                    }
                    else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                        n1_f = exe_nums_f.top();
                        exe_nums.pop();

                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,n1_f, ss,0));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,n1_f, ss,0));
                            hipFree(s3);
                        }
                    }

                    else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        int_type* s4 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(s3, s4,ss,0));
                        hipFree(s3);
                        hipFree(s4);
                    }
                    else if(s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        float_type* s4 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3, s4,ss,1));
                        hipFree(s3);
                        hipFree(s4);
                    }
                    else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        float_type* s4 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3, s4,ss,0));
                        hipFree(s3);
                        hipFree(s4);
                    }
                    else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        float_type* s4 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3, s4,ss,1));
                        hipFree(s3);
                        hipFree(s4);
                    }
                }
            }

        } //
        else {
            // here we need to save what is where

            col_val.push(op_value.front());
            op_value.pop();

            grp_type1.push(grp_type);

            if(!exe_nums.empty()) {  //number
                col_type.push(0);
                exe_nums1.push(exe_nums.top());
                exe_nums.pop();
            };
            if(!exe_value.empty()) {  //field name
                col_type.push(1);
                exe_value1.push(exe_value.top());
                exe_value.pop();
            };
            if(!exe_vectors.empty()) {  //vector int
                exe_vectors1.push(exe_vectors.top());
                exe_vectors.pop();
                col_type.push(2);
            };
            if(!exe_vectors_f.empty()) {  //vector float
                exe_vectors1_d.push(exe_vectors_f.top());
                exe_vectors_f.pop();
                col_type.push(3);
            };

            colCount++;
        };
    };



    for(unsigned int j=0; j < colCount; j++) {

        if ((grp_type1.top()).compare("COUNT") == 0 )
            b->grp_type[col_val.top()] = 0;
        else if ((grp_type1.top()).compare("AVG") == 0 )
            b->grp_type[col_val.top()] = 1;
        else if ((grp_type1.top()).compare("SUM") == 0 )
            b->grp_type[col_val.top()] = 2;
        else if ((grp_type1.top()).compare("NULL") == 0 )
            b->grp_type[col_val.top()] = 3;
        else if ((grp_type1.top()).compare("MIN") == 0 )
            b->grp_type[col_val.top()] = 4;
        else if ((grp_type1.top()).compare("MAX") == 0 )
            b->grp_type[col_val.top()] = 5;
        else if ((grp_type1.top()).compare("COUNTD") == 0 ) {
            b->grp_type[col_val.top()] = 6;
        };


        if(col_type.top() == 0) {

            // create a vector

            thrust::device_ptr<int_type> s = thrust::device_malloc<int_type>(a->mRecCount);
            thrust::sequence(s, s+(a->mRecCount), (int)exe_nums1.top(), 0);
            if (!a->columnGroups.empty()) {
                thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                thrust::device_ptr<bool> d_grp(a->grp);
                thrust::copy_if(s,s+(a->mRecCount), d_grp, count_diff, thrust::identity<bool>());
                b->addDeviceColumn(thrust::raw_pointer_cast(count_diff) , col_val.top(), res_size);
                thrust::device_free(count_diff);
            }
            else
                b->addDeviceColumn(thrust::raw_pointer_cast(s), col_val.top(), a->mRecCount);
            exe_nums1.pop();
        };
        if(col_type.top() == 1) {

            if(a->type[exe_value1.top()] == 0) {

                //modify what we push there in case of a grouping
                if (!a->columnGroups.empty()) {
                    thrust::device_ptr<int_type> count_diff = thrust::device_malloc<int_type>(res_size);
                    thrust::device_ptr<bool> d_grp(a->grp);

                    thrust::copy_if(a->d_columns_int[exe_value1.top()].begin(),a->d_columns_int[exe_value1.top()].begin() + a->mRecCount,
                                    d_grp, count_diff, thrust::identity<bool>());
                    b->addDeviceColumn(thrust::raw_pointer_cast(count_diff) ,  col_val.top(), res_size);
                    thrust::device_free(count_diff);
                }
                else
                    b->addDeviceColumn(thrust::raw_pointer_cast(a->d_columns_int[exe_value1.top()].data()) , col_val.top(), a->mRecCount);
            }
            else if(a->type[exe_value1.top()] == 1) {

                //modify what we push there in case of a grouping
                if (!a->columnGroups.empty()) {
                    thrust::device_ptr<float_type> count_diff = thrust::device_malloc<float_type>(res_size);
                    thrust::device_ptr<bool> d_grp(a->grp);

                    thrust::copy_if(a->d_columns_float[exe_value1.top()].begin(), a->d_columns_float[exe_value1.top()].begin() + a->mRecCount,
                                    d_grp, count_diff, thrust::identity<bool>());
                    b->addDeviceColumn(thrust::raw_pointer_cast(count_diff) , col_val.top(), res_size, a->decimal[exe_value1.top()]);
                    thrust::device_free(count_diff);
                }
                else
                    b->addDeviceColumn(thrust::raw_pointer_cast(a->d_columns_float[exe_value1.top()].data()), col_val.top(), a->mRecCount, a->decimal[exe_value1.top()]);
            }
            else if(a->type[exe_value1.top()] == 2) { //varchar

                if (a->columnGroups.empty())
                    res_size = a->mRecCount;

                if (std::find(b->columnNames.begin(), b->columnNames.end(), col_val.top()) == b->columnNames.end()) {
                    void *d;
                    hipMalloc((void **) &d, res_size*a->char_size[exe_value1.top()]);
                    b->d_columns_char[col_val.top()] = (char*)d;
                    b->h_columns_char[col_val.top()] = NULL;
                    b->char_size[col_val.top()] = a->char_size[exe_value1.top()];
                    b->columnNames.push_back(col_val.top());
                    b->type[col_val.top()] = 2;
                }
                else {  // already exists, my need to resize it
                    if(b->mRecCount < res_size)
                        b->resizeDeviceColumn(res_size, col_val.top());
                };

                if (!a->columnGroups.empty()) {
                    thrust::device_ptr<bool> d_grp(a->grp);
                    str_copy_if(a->d_columns_char[exe_value1.top()], a->mRecCount, b->d_columns_char[col_val.top()], d_grp, a->char_size[exe_value1.top()]);
                }
                else {
                    hipMemcpy((void*)(thrust::raw_pointer_cast(b->d_columns_char[col_val.top()])), (void*)thrust::raw_pointer_cast(a->d_columns_char[exe_value1.top()]),
                               a->mRecCount*a->char_size[exe_value1.top()], hipMemcpyDeviceToDevice);
                }
            }
            exe_value1.pop();

        };

        if(col_type.top() == 2) {	    // int

            if (!a->columnGroups.empty())
                b->addDeviceColumn(exe_vectors1.top() , col_val.top(), res_size);
            else {
                if(!one_line)
                    b->addDeviceColumn(exe_vectors1.top() , col_val.top(), a->mRecCount);
                else
                    b->addDeviceColumn(exe_vectors1.top() , col_val.top(), 1);
            };

            hipFree(exe_vectors1.top());
            exe_vectors1.pop();

        }
        if(col_type.top() == 3) {        //float

            if (!a->columnGroups.empty()) {
                b->addDeviceColumn(exe_vectors1_d.top() , col_val.top(), res_size, 1);
            }
            else {
                if(!one_line) {
                    b->addDeviceColumn(exe_vectors1_d.top() , col_val.top(), a->mRecCount, 1);
                }
                else {
                    b->addDeviceColumn(exe_vectors1_d.top() , col_val.top(), 1, 1);
                };
            };
            hipFree(exe_vectors1_d.top());
            exe_vectors1_d.pop();
        };
        col_type.pop();
        col_val.pop();
        grp_type1.pop();
    };

    if (a->columnGroups.empty()) {
        if(!one_line)
            b->mRecCount = a->mRecCount;
        else
            b->mRecCount = 1;
        one_liner = one_line;
    }
    else {
        b->mRecCount = res_size;
        one_liner = 0;
    };
}


