#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "zone_map.h"

void process_error(int severity, string err);   // this should probably live in a utils header file


bool fh_equal_to(const float_type x, const float_type y)
{
    return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
}


bool fh_less(const float_type x, const float_type y)
{
    return ((y-x) > EPSILON);
}


bool fh_greater(const float_type x, const float_type y)
{
    return ((x-y) > EPSILON);
}


bool fh_greater_equal_to(const float_type x, const float_type y)
{
    return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
}


bool fh_less_equal_to(const float_type x, const float_type y)
{
    return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
}

char host_logical_and(char column1, char column2)
{
    //cout << "AND " << column1 << " " << column2 << endl;
    if (column1 == 'A' && column2 == 'A')
        return 'A';
    else if (column1 == 'N' || column2 == 'N') {
        return 'N';
    }
    else
        return 'R';

}


char host_logical_or(char column1, char column2)
{
    //cout << "OR " << column1 << " " << column2 << endl;
    if (column1 == 'A' && column2 == 'A')
        return 'A';
    else if (column1 == 'N' && column2 == 'N')
        return 'N';
    else
        return 'R';

}



char host_compare(int_type s, int_type d, int_type op_type)
{
    char res = 'N';

    if (op_type == 2 && d>s ) // >
        res = 'A';
    else if (op_type == 1 && d<s)  // <
        res = 'A';
    else if (op_type == 6 && d>=s) // >=
        res = 'A';
    else if (op_type == 5 && d<=s)  // <=
        res = 'A';
    else if (op_type == 4 && d==s)// =
        res = 'A';
    else // !=
        if(d!=s) res = 'A';

    return res;
}


char host_compare(float_type s, float_type d, int_type op_type)
{
    char res = 'N';

    if (op_type == 2 && (d-s) > EPSILON) // >
        res = 'A';
    else if (op_type == 1 && (s-d) > EPSILON)  // <
        res = 'A';
    else if (op_type == 6 && ((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) // >=
        res = 'A';
    else if (op_type == 5 && ((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON)))  // <=
        res = 'A';
    else if (op_type == 4 && ((d-s) < EPSILON) && ((d-s) > -EPSILON))// =
        res = 'A';
    else // !=
        if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 'A';

    return res;
}


char host_compare(int_type* column1, int_type d, int_type op_type)
{
    char res = 'R';
    //cout << "CMP " << column1[0] << " " << column1[1] << " with " << d << endl;

    if (op_type == 2) {   // >
        if (column1[1] <= d)
            res = 'N';
        else if (column1[0] > d)
            res = 'A';
    }
    else if (op_type == 1) { // <
        if (column1[0] >= d)
            res = 'N';
        else if (column1[1] < d)
            res = 'A';
    }
    else if (op_type == 6) {  // >=
        if (column1[1] < d)
            res = 'N';
        else if (column1[0] >= d)
            res = 'A';
    }
    else if (op_type == 5) { // <=
        if (column1[0] > d)
            res = 'N';
        else if (column1[1] <= d)
            res = 'A';
    }
    else if (op_type == 4 && column1[0] == d && column1[1] == d) { // =
        res = 'A';
    };
    //cout << "res " << res << endl;

    return res;
}

char host_compare(float_type* column1, float_type d, int_type op_type)
{
    char res = 'R';
    //cout << "CMP " << column1[0] << " " << column1[1] << " with " << d << endl;

    if (op_type == 2) { // >
        if(fh_less_equal_to(column1[1],d)) {
            res = 'N';
        }
        else if(fh_greater(column1[0],d)) {
            res = 'A';
        };
    }
    else if (op_type == 1) { // <
        if(fh_less(column1[1],d)) {
            res = 'A';
        }
        else if(fh_greater_equal_to(column1[0],d)) {
            res = 'N';
        };
    }
    else if (op_type == 6) { // >=
        if(fh_greater_equal_to(column1[0],d)) {
            res = 'A';
        }
        else if(fh_less(column1[1],d)) {
            res = 'N';
        };
    }
    else if (op_type == 5) { // <=
        if(fh_less_equal_to(column1[1],d)) {
            res = 'A';
        }
        else if(fh_greater(column1[0],d)) {
            res = 'N';
        };
    }
    else if (op_type == 4 && fh_equal_to(column1[0],d) && fh_equal_to(column1[1],d)) // =
        res = 'A';

    //cout << "res " << res << endl;
    return res;

}


char host_compare(int_type* column1, int_type* column2, int_type op_type)
{
    char res = 'R';

    if (op_type == 2) { // >
        if(column1[0] > column2[1])
            res = 'A';
        else if(column1[1] <= column2[0])
            res = 'N';
    }
    else if (op_type == 1) { // <
        if(column1[1] < column2[0])
            res = 'A';
        else if(column1[0] >= column2[1])
            res = 'N';
    }
    else if (op_type == 6) { // >=
        if(column1[0] >= column2[1])
            res = 'A';
        else if(column1[1] < column2[0])
            res = 'N';
    }
    else if (op_type == 5) { // <=
        if(column1[1] <= column2[0])
            res = 'A';
        else if(column1[0] > column2[1])
            res = 'N';
    }
    else if (op_type == 4  && column1[0] == column2[1] && column1[1] == column2[0]) // =
        res = 'A';

    return res;


}

char host_compare(float_type* column1, float_type* column2, int_type op_type)
{
    char res = 'R';

    if (op_type == 2) { // >
        if(fh_greater(column1[0],column2[1]))
            res = 'A';
        else if(fh_less_equal_to(column1[1],column2[0]))
            res = 'N';
    }
    else if (op_type == 1) { // <
        if(fh_less(column1[1],column2[0]))
            res = 'A';
        else if(fh_greater_equal_to(column1[0],column2[1]))
            res = 'N';
    }
    else if (op_type == 6) { // >=
        if(fh_greater_equal_to(column1[1],column2[0]))
            res = 'A';
        else if(fh_less(column1[1],column2[0]))
            res = 'N';
    }
    else if (op_type == 5) { // <=
        if(fh_less_equal_to(column1[1],column2[0]))
            res = 'A';
        else if(fh_greater(column1[0],column2[1]))
            res = 'N';
    }
    else if (op_type == 4  && fh_equal_to(column1[0], column2[1]) && fh_equal_to(column1[1],column2[0])) // =
        res = 'A';

    return res;
}


char host_compare(float_type* column1, int_type* column2, int_type op_type)
{
    char res = 'R';

    if (op_type == 2) { // >
        if(fh_greater(column1[0],(float_type)column2[1]))
            res = 'A';
        else if(fh_less_equal_to(column1[1],(float_type)column2[0]))
            res = 'N';
    }
    else if (op_type == 1) { // <
        if(fh_less(column1[1],(float_type)column2[0]))
            res = 'A';
        else if(fh_greater_equal_to(column1[0],(float_type)column2[1]))
            res = 'N';
    }
    else if (op_type == 6) { // >=
        if(fh_greater_equal_to(column1[1],(float_type)column2[0]))
            res = 'A';
        else if(fh_less(column1[1],(float_type)column2[0]))
            res = 'N';
    }
    else if (op_type == 5) { // <=
        if(fh_less_equal_to(column1[1],(float_type)column2[0]))
            res = 'A';
        else if(fh_greater(column1[0],(float_type)column2[1]))
            res = 'N';
    }
    else if (op_type == 4  && fh_equal_to(column1[0],(float_type) column2[1]) && fh_equal_to(column1[1],(float_type)column2[0])) // =
        res = 'A';

    return res;
}



float_type* host_op(int_type* column1, float_type* column2, string op_type, int reverse)
{

    float_type* temp = (float_type*)malloc(2*float_size);
    temp[0] = (float_type)column1[0];
    temp[1] = (float_type)column1[1];

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = temp[0] * column2[0];
            temp[1] = temp[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = temp[0] + column2[0];
            temp[1] = temp[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column2[0] - temp[0];
            temp[1] = column2[1] - temp[1];
        }
        else {
            temp[0] = column2[0] / temp[0];
            temp[1] = column2[1] / temp[1];
        }
    }
    else {
        if (op_type.compare("MUL") == 0) {
            temp[0] = temp[0] * column2[0];
            temp[1] = temp[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = temp[0] + column2[0];
            temp[1] = temp[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = temp[0] - column2[0];
            temp[1] = temp[1] - column2[1];
        }
        else {
            temp[0] = temp[0] / column2[0];
            temp[1] = temp[1] / column2[1];
        }
    };

    return temp;
}




int_type* host_op(int_type* column1, int_type* column2, string op_type, int reverse)
{
    int_type* temp = (int_type*)malloc(2*int_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * column2[0];
            temp[1] = column1[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + column2[0];
            temp[1] = column1[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column1[0] - column2[0];
            temp[1] = column1[1] - column2[1];
        }
        else {
            temp[0] = column1[0] / column2[0];
            temp[1] = column1[1] / column2[1];
        }
    }
    else  {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * column2[0];
            temp[1] = column1[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + column2[0];
            temp[1] = column1[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column2[0] - column1[0];
            temp[1] = column2[1] - column1[1];
        }
        else {
            temp[0] = column2[0] / column1[0];
            temp[1] = column2[1] / column1[1];
        }
    }

    return temp;

}

float_type* host_op(float_type* column1, float_type* column2, string op_type, int reverse)
{
    float_type* temp = (float_type*)malloc(2*float_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * column2[0];
            temp[1] = column1[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + column2[0];
            temp[1] = column1[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column1[0] - column2[0];
            temp[1] = column1[1] - column2[1];
        }
        else {
            temp[0] = column1[0] / column2[0];
            temp[1] = column1[1] / column2[1];
        }
    }
    else  {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * column2[0];
            temp[1] = column1[1] * column2[1];
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + column2[0];
            temp[1] = column1[1] + column2[1];
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column2[0] - column1[0];
            temp[1] = column2[1] - column1[1];
        }
        else {
            temp[0] = column2[0] / column1[0];
            temp[1] = column2[1] / column1[1];
        }
    }

    return temp;

}

int_type* host_op(int_type* column1, int_type d, string op_type, int reverse)
{
    int_type* temp = (int_type*)malloc(2*int_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * d;
            temp[1] = column1[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + d;
            temp[1] = column1[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column1[0] - d;
            temp[1] = column1[1] - d;
        }
        else {
            temp[0] = column1[0] / d;
            temp[1] = column1[1] / d;
        }
    }
    else {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * d;
            temp[1] = column1[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + d;
            temp[1] = column1[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = d - column1[0];
            temp[1] = d - column1[1];
        }
        else {
            temp[0] = d / column1[0];
            temp[1] = d / column1[1];
        }

    };
    return temp;

}

float_type* host_op(int_type* column1, float_type d, string op_type, int reverse)
{
    float_type* temp = (float_type*)malloc(2*float_size);
    temp[0] = (float_type)column1[0];
    temp[1] = (float_type)column1[1];

    float_type* temp1 = (float_type*)malloc(2*float_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp1[0] = temp[0] * d;
            temp1[1] = temp[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp1[0] = temp[0] + d;
            temp1[1] = temp[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp1[0] = temp[0] - d;
            temp1[1] = temp[1] - d;
        }
        else {
            temp1[0] = temp[0] / d;
            temp1[1] = temp[1] / d;
        }
    }
    else  {
        if (op_type.compare("MUL") == 0) {
            temp1[0] = temp[0] * d;
            temp1[1] = temp[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp1[0] = temp[0] + d;
            temp1[1] = temp[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp1[0] = d - temp[0];
            temp1[1] = d - temp[1];
        }
        else {
            temp1[0] = d / temp[0];
            temp1[1] = d / temp[1];
        }
    };

    free(temp);
    return temp1;

}



float_type* host_op(float_type* column1, float_type d, string op_type,int reverse)
{
    float_type* temp = (float_type*)malloc(2*float_size);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * d;
            temp[1] = column1[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + d;
            temp[1] = column1[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = column1[0] - d;
            temp[1] = column1[1] - d;
        }
        else {
            temp[0] = column1[0] / d;
            temp[1] = column1[1] / d;
        }
    }
    else {
        if (op_type.compare("MUL") == 0) {
            temp[0] = column1[0] * d;
            temp[1] = column1[1] * d;
        }
        else if (op_type.compare("ADD") == 0) {
            temp[0] = column1[0] + d;
            temp[1] = column1[1] + d;
        }
        else if (op_type.compare("MINUS") == 0) {
            temp[0] = d - column1[0];
            temp[1] = d - column1[1];
        }
        else {
            temp[0] = d / column1[0];
            temp[1] = d / column1[1];
        }
    };

    return temp;

}



//CudaSet a contains two records - with all minimum and maximum values of the segment
//We need to determine if this segment needs to be processed
//The check takes place in host's memory

char zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a, unsigned int segment)
{

    stack<string> exe_type;
    stack<string> exe_value;
    stack<int_type*> exe_vectors;
    stack<float_type*> exe_vectors_f;
    stack<int_type> exe_nums;
    stack<char> bool_vectors;
    stack<float_type> exe_nums_f;
    string  s1, s2, s1_val, s2_val;
    int_type n1, n2, res;
    float_type n1_f, n2_f, res_f;

    if(a->not_compressed)
        return 'R';

    //first we need to set all host arrays [0] and [1] of t to min and max values of appropriate files
    set<string> uniques;
    queue<string> fields(op_value);
    CudaSet *t;
    FILE* f;
    unsigned int cnt;
    string f1;
	

    while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && var_exists(a, fields.front()))	{	

			if(a->filtered)
				t = varNames[a->source_name];
			else
				t = a;
			
            // copy t min and max values to a only if int, decimal or float
            if(t->type[fields.front()] <= 1) {

                f1 = t->load_file_name + "." + fields.front() + "." + int_to_string(segment);
                f = fopen (f1.c_str() , "rb" );
				if(f == NULL) {
					cout << "Error opening " << f1 << " file " << endl;
					exit(0);
				};

                fread((char *)&cnt, 4, 1, f);
                if (t->type[fields.front()] == 0) {
                    a->h_columns_int[fields.front()].resize(2);
                    fread((char *)&a->h_columns_int[fields.front()][0], 8, 1, f);
                    fread((char *)&a->h_columns_int[fields.front()][1], 8, 1, f);
                    //cout << "file " << f1 << " " << segment << " " << a->h_columns_int[fields.front()][0] << ":" << a->h_columns_int[fields.front()][1] << endl;
                }
                else  {
                    long long int t;
                    a->h_columns_float[fields.front()].resize(2);
                    fread((char *)&t, 8, 1, f);
                    a->h_columns_float[fields.front()][0] = (float_type)t/100.0;
                    fread((char *)&t, 8, 1, f);
                    a->h_columns_float[fields.front()][1] = (float_type)t/100.0;
                    //cout << "file " << f1 << " " << segment << " " << a->h_columns_float[a->type_index[colIndex]][0] << ":" << a->h_columns_float[a->type_index[colIndex]][1] << endl;
                };
                fclose(f);
            };			
        };
        uniques.insert(fields.front());
        fields.pop();
    };
	

    for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

        string ss = op_type.front();

        if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("VECTOR") == 0 || ss.compare("FLOAT") == 0
                || ss.compare("STRING") == 0) {

            exe_type.push(ss);
            if (ss.compare("NUMBER") == 0) {
                exe_nums.push(op_nums.front());
                op_nums.pop();
            }
            else if (ss.compare("NAME") == 0)  {
				if(var_exists(a, op_value.front())) {
					exe_value.push(op_value.front());
					op_value.pop();
				}
				else {
					process_error(1, "Couldn't find column " + op_value.front());
					//cout << "Couldn't find column " << op_value.front() << endl;
					//exit(0);
				};
            }
            else if (ss.compare("STRING") == 0) {
                exe_value.push(op_value.front());
                op_value.pop();
            }			
            if (ss.compare("FLOAT") == 0) {
                exe_nums_f.push(op_nums_f.front());
                op_nums_f.pop();
            }

        }
        else {
            if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
                // get 2 values from the stack
                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();


                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();

                    if (ss.compare("ADD") == 0 )
                        res = n1+n2;
                    else if (ss.compare("MUL") == 0 )
                        res = n1*n2;
                    else if (ss.compare("DIV") == 0 )
                        res = n1/n2;
                    else
                        res = n1-n2;


                    exe_type.push("NUMBER");
                    exe_nums.push(res);
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (ss.compare("ADD") == 0 )
                        res_f = n1_f+n2_f;
                    else if (ss.compare("MUL") == 0 )
                        res_f = n1_f*n2_f;
                    else if (ss.compare("DIV") == 0 )
                        res_f = n1_f/n2_f;
                    else
                        res_f = n1_f-n2_f;

                    exe_type.push("FLOAT");
                    exe_nums_f.push(res_f);

                }
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    exe_type.push("VECTOR F");

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_host_float_by_name(s1_val);
                        exe_vectors_f.push(host_op(t,n1_f,ss,1));
                    }
                    else {
                        int_type* t = a->get_host_int_by_name(s1_val);
                        exe_vectors_f.push(host_op(t,n1_f,ss,1));
                    };

                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    exe_type.push("VECTOR F");

                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        exe_vectors_f.push(host_op(t,n1_f,ss,0));
                    }
                    else {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        exe_vectors_f.push(host_op(t,n1_f,ss,0));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_host_float_by_name(s1_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(t,(float_type)n1,ss,1));

                    }
                    else {
                        int_type* t = a->get_host_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        exe_vectors.push(host_op(t,n1,ss,1));
                    };
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();


                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(t,(float_type)n1,ss,0));
                    }
                    else {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        exe_vectors.push(host_op(t,n1,ss,0));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {

                    return 'R';
                    /*s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t1 = a->get_host_int_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t = a->get_host_int_by_name(s2_val);
                            exe_type.push("VECTOR");
                            exe_vectors.push(host_op(t,t1,ss,0));
                        }
                        else {
                            float_type* t = a->get_host_float_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t1,t,ss,0));
                        };
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_host_int_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t1,t,ss,0));
                        }
                        else {
                            float_type* t1 = a->get_host_float_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t,t1,ss,0));
                        };
                    }
                    */
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0 ) && s2.compare("NAME") == 0) {

                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);

                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
                            exe_vectors.push(host_op(t,s3,ss,0));
                            //free s3
                            hipFree(s3);

                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t,s3,ss,0));
                            hipFree(s3);
                        }
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(s3,t, ss,0));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t,s3,ss,0));
                            hipFree(s3);
                        }
                    };
                }
                else if ((s2.compare("VECTOR") == 0 || s2.compare("VECTOR F") == 0 ) && s1.compare("NAME") == 0) {

                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s1_val);

                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
                            exe_vectors.push(host_op(t,s3,ss,1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t,s3,ss,1));
                            hipFree(s3);
                        }
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s1_val);
                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(s3,t,ss,1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(host_op(t,s3,ss,1));
                            hipFree(s3);
                        }
                    };
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
                        exe_vectors.push(host_op(s3,n1, ss,1));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,(float_type)n1, ss,1));
                        hipFree(s3);
                    }
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR") || s2.compare("VECTOR F") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
                        exe_vectors.push(host_op(s3,n1, ss,0));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,(float_type)n1, ss,0));
                        hipFree(s3);
                    }
                }

                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,n1_f, ss,1));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,n1_f, ss,1));
                        hipFree(s3);
                    }
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    n1_f = (float_type)exe_nums.top();
                    exe_nums.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,n1_f, ss,0));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(host_op(s3,n1_f, ss,0));
                        hipFree(s3);
                    }
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s4 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    exe_vectors.push(host_op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(host_op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(host_op(s3, s4,ss,0));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(host_op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
            }

            else if (ss.compare("CMP") == 0) {


                int_type cmp_type = op_nums.front();
                op_nums.pop();

                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();



                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(n1,n2,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(n1_f,n2_f,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NUMBER") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(n1_f,float_type(n2),cmp_type));
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(n1_f,float_type(n2),cmp_type));
                }

                else if (s1.compare("STRING") == 0 && s2.compare("NAME") == 0) {


                    exe_type.push("VECTOR");
                    bool_vectors.push('R'); // later I plan to change implementation of char type so I will leave indexing of char off for now
                }
                else if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
                    exe_type.push("VECTOR");
                    bool_vectors.push('R');
                }


                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,n1,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,(float_type)n1,cmp_type));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,n1,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,(float_type)n1,cmp_type));
                    };
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,n1_f,cmp_type));
                    }
                    else {
						
                        float_type* t = a->get_host_float_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,n1_f,cmp_type));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,(int_type)n1_f,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(host_compare(t,n1_f,cmp_type));
                    };
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,(float_type)n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,n1,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR F") == 0) {

                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,(float_type)n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR") == 0) {

                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,n1_f,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("VECTOR") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,(int_type)n1_f,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,n1_f,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,(int_type)n1_f,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NAME") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        bool_vectors.push(host_compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }


                else if (s1.compare("VECTOR") == 0 && s2.compare("NAME") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR F") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        bool_vectors.push(host_compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    }
                    else {
                        float_type* t = a->get_host_float_by_name(s2_val);
                        bool_vectors.push(host_compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s2,s3,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s2 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s2,s3,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,s2,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(host_compare(s3,s2,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }


                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    return 'R';
                    /*
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_host_int_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_host_int_by_name(s2_val);
                            bool_vectors.push(host_compare(t1,t,cmp_type));
                        }
                        else {
                            float_type* t1 = a->get_host_float_by_name(s2_val);
                            bool_vectors.push(host_compare(t1,t,cmp_type));
                        };
                    }
                    else {
                        cmp_type = reverse_op(cmp_type);
                        float_type* t = a->get_host_float_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_host_int_by_name(s2_val);
                            bool_vectors.push(host_compare(t,t1,cmp_type));
                        }
                        else {
                            float_type* t1 = a->get_host_float_by_name(s2_val);
                            bool_vectors.push(host_compare(t,t1,cmp_type));
                        };
                    }
                    */
                }
            }

            else if (ss.compare("AND") == 0) {
                char s3 = bool_vectors.top();
                bool_vectors.pop();
                char s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(host_logical_and(s2,s3));
            }
            else if (ss.compare("OR") == 0) {
                char s3 = bool_vectors.top();
                bool_vectors.pop();
                char s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(host_logical_or(s2,s3));
            }
            else {
				if(ss.compare("JOIN") == 0)
					process_error(2, "operation = is not valid");
					//cout << "operation = is not valid" << endl;
				else 	
					 process_error(2, "operation " + string(ss)+ " is not valid");
					//cout << "operation " << ss << " is not valid" << endl;
				exit(0);	// never gets here
            }
        };
    };


    return bool_vectors.top();

}

