/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "strings.h"

template <typename KeyType>
void update_permutation(thrust::device_vector<KeyType>& key, unsigned int* permutation, unsigned long long int RecCount, string SortType, KeyType* tmp)
{
	cout << "update_permutation->declare device_ptr " << getFreeMem() << endl;
    thrust::device_ptr<unsigned int> dev_per(permutation);
    // temporary storage for keys
    thrust::device_ptr<KeyType> temp(tmp);
    // permute the keys with the current reordering
    thrust::gather(dev_per, dev_per+RecCount, key.begin(), temp);

    // stable_sort the permuted keys and update the permutation
    cout << "update_permutation->stable_sort_by_key: " << getFreeMem() << endl;
    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per, thrust::greater<KeyType>());
    else
        thrust::stable_sort_by_key(temp, temp+RecCount, dev_per);
}

template <typename KeyType>
void update_permutation_host(KeyType* key, unsigned int* permutation, unsigned long long int RecCount, string SortType, KeyType* tmp)
{
    thrust::gather(permutation, permutation+RecCount, key, tmp);

    if (SortType.compare("DESC") == 0 )
        thrust::stable_sort_by_key(tmp, tmp+RecCount, permutation, thrust::greater<KeyType>());
    else
        thrust::stable_sort_by_key(tmp, tmp+RecCount, permutation);
}



template <typename KeyType>
void apply_permutation(thrust::device_vector<KeyType>& key, unsigned int* permutation, unsigned long long int RecCount, KeyType* tmp)
{
    thrust::device_ptr<unsigned int> dev_per(permutation);
    thrust::device_ptr<KeyType> temp(tmp);
    // copy keys to temporary vector
    thrust::copy(key.begin(), key.begin() + RecCount, temp);
    // permute the keys
    thrust::gather(dev_per, dev_per+RecCount, temp, key.begin());
}

template <typename KeyType>
void apply_permutation_host(KeyType* key, unsigned int* permutation, unsigned long long int RecCount, KeyType* res)
{
    thrust::gather(permutation, permutation + RecCount, key, res);
}







