#include "hip/hip_runtime.h"
/*
 	Example main routine for Alenka
*/

#include <map>
#include <string>
#include <iostream>
#include <ctime>
#include <vector>


using namespace std;
#include "alenka.h"

int main(int ac, char **av)
{
    cout << "main->start" << endl;
    std::clock_t start;
    int x;

    // test QPS via alenkaExecute	-- this section is the only C++ dependency
    if (string(av[1]) == "--QPS-test") {
        alenkaInit(NULL);
        start = std::clock();
        for (x=0; x< 1000; x++)  {
            alenkaExecute("A1 := SELECT  count(n_name) AS col1 FROM nation;\n DISPLAY A1 USING ('|');");
        }
        cout<< "Ave QPS is : " <<  ( 1000/ (( std::clock() - start ) / (double)CLOCKS_PER_SEC )) << endl;
        alenkaClose();
    }
    else {				// ordinary alenka file mode
        if (ac < 2) {
            cout << "Usage : alenka [--QPS-test] | [ [-l process_count] | [-d data_directory] | [-v] script.sql ]" << endl;
            exit(1);
        }
        else {
        	return execute_file( ac, av) ;
        }

    }
}
