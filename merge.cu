#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "merge.h"
#include "zone_map.h"

using namespace std;

void process_error(int severity, string err);	// this should probably live in a utils header file


#if defined(_MSC_VER)
#define BIG_CONSTANT(x) (x)
// Other compilers
#else   // defined(_MSC_VER)
#define BIG_CONSTANT(x) (x##LLU)
#endif // !defined(_MSC_VER)


uint64_t MurmurHash64A ( const void * key, int len, uint64_t seed )
{
  const uint64_t m = BIG_CONSTANT(0xc6a4a7935bd1e995);
  const int r = 47;

  uint64_t h = seed ^ (len * m);

  const uint64_t * data = (const uint64_t *)key;
  const uint64_t * end = data + (len/8);

  while(data != end)
  {
    uint64_t k = *data++;

    k *= m;
    k ^= k >> r;
    k *= m;
   
    h ^= k;
    h *= m;
  }

  const unsigned char * data2 = (const unsigned char*)data;

  switch(len & 7)
  {
  case 7: h ^= uint64_t(data2[6]) << 48;
  case 6: h ^= uint64_t(data2[5]) << 40;
  case 5: h ^= uint64_t(data2[4]) << 32;
  case 4: h ^= uint64_t(data2[3]) << 24;
  case 3: h ^= uint64_t(data2[2]) << 16;
  case 2: h ^= uint64_t(data2[1]) << 8;
  case 1: h ^= uint64_t(data2[0]);
          h *= m;
  };
 
  h ^= h >> r;
  h *= m;
  h ^= h >> r;

  return h;
} 


struct float_avg
{
    __host__  float_type operator()(const float_type &lhs, const int_type &rhs) const {
        return lhs/rhs;
    }
};

struct float_avg1
{
    __host__  float_type operator()(const int_type &lhs, const int_type &rhs) const {
        return ((float_type)lhs)/rhs;
    }
};


/*struct float_avg  : public binary_function<float_type,int_type,float_type>
{
  __host__ __device__ float_type operator()(const float_type &lhs, const int_type &rhs) const {return lhs/(float_type)rhs;}
}; // end not_equal_to
*/





//typedef thrust::device_vector<int_type>::iterator    IntIterator;
//typedef thrust::tuple<IntIterator,IntIterator> IteratorTuple;
//typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
unsigned int hash_seed = 100;
thrust::host_vector<unsigned long long int> h_merge;

using namespace std;
using namespace thrust::placeholders;


void create_c(CudaSet* c, CudaSet* b)
{
    map<string,unsigned int>::iterator it;
    c->not_compressed = 1;
    c->segCount = 1;

	c->columnNames = b->columnNames;    
	h_merge.clear();
	c->cols = b->cols;
	c->type = b->type;
	c->decimal = b->decimal;
    c->grp_type = b->grp_type;

    for(unsigned int i=0; i < b->columnNames.size(); i++) {
	
        if (b->type[b->columnNames[i]] == 0) {
            c->h_columns_int[b->columnNames[i]] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >();
            c->d_columns_int[b->columnNames[i]] = thrust::device_vector<int_type>();
        }
        else if (b->type[b->columnNames[i]] == 1) {
            c->h_columns_float[b->columnNames[i]] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >();
            c->d_columns_float[b->columnNames[i]] = thrust::device_vector<float_type>();			
        }
        else {
            c->h_columns_char[b->columnNames[i]] = NULL;
            c->d_columns_char[b->columnNames[i]] = NULL;
            c->char_size[b->columnNames[i]] = b->char_size[b->columnNames[i]];
        };
    };	
}

void add(CudaSet* c, CudaSet* b, queue<string> op_v3, map<string,string> aliases,
         vector<thrust::device_vector<int_type> >& distinct_tmp, vector<thrust::device_vector<int_type> >& distinct_val,
         vector<thrust::device_vector<int_type> >& distinct_hash, CudaSet* a)
{		
    if (c->columnNames.empty()) {
        // create d_columns and h_columns
        create_c(c,b);
    }

    size_t cycle_sz = op_v3.size();	
	
    vector<string> opv;
    queue<string> ss;
    for(unsigned int z = 0; z < cycle_sz; z++) {
        opv.push_back(aliases[op_v3.front()]);
        ss.push(aliases[op_v3.front()]);
        op_v3.pop();
    };

    // create hashes of groupby columns
    unsigned long long int* hashes = new unsigned long long int[b->mRecCount];
    unsigned long long int* sum = new unsigned long long int[cycle_sz*b->mRecCount];
        

	b->CopyToHost(0, b->mRecCount);					
	
    for(unsigned int z = 0; z < cycle_sz; z++) {
        
        if(b->type[opv[z]] == 0) {  //int
			for(int i = 0; i < b->mRecCount; i++) {
				sum[i*cycle_sz + z] = MurmurHash64A(&b->h_columns_int[opv[z]][i], 8, hash_seed);				
			};
        }
        else if(b->type[opv[z]] == 2) {  //string			
			for(int i = 0; i < b->mRecCount; i++) {
				sum[i*cycle_sz + z] = MurmurHash64A(&b->h_columns_char[opv[z]][i*b->char_size[opv[z]]], b->char_size[opv[z]], hash_seed);				
			};			
			
        }
        else {  //float
            process_error(2, "No group by on float/decimal columns ");
            //cout << "No group by on float/decimal columns " << endl;
            //exit(0);
        };
    };
	
	for(int i = 0; i < b->mRecCount; i++) {
		hashes[i] = MurmurHash64A(&sum[i*cycle_sz], 8*cycle_sz, hash_seed);				
	};			


	// sort the results by hash
    unsigned int* v = new unsigned int[b->mRecCount];
    thrust::sequence(v, v + b->mRecCount, 0, 1);
    thrust::sort_by_key(hashes, hashes + b->mRecCount, v);	

    for(unsigned int i = 0; i < b->columnNames.size(); i++) {

        if(b->type[b->columnNames[i]] == 0) {
            int_type* d_tmp = new int_type[b->mRecCount];
            thrust::gather(v, v+b->mRecCount, b->h_columns_int[b->columnNames[i]].begin(), d_tmp);
            thrust::copy(d_tmp, d_tmp + b->mRecCount, b->h_columns_int[b->columnNames[i]].begin());
			delete [] d_tmp;
        }
        else if(b->type[b->columnNames[i]] == 1) {
            float_type* d_tmp = new float_type[b->mRecCount];
            thrust::gather(v, v+b->mRecCount, b->h_columns_float[b->columnNames[i]].begin(), d_tmp);
            thrust::copy(d_tmp, d_tmp + b->mRecCount, b->h_columns_float[b->columnNames[i]].begin());
			delete [] d_tmp;
        }
        else {
            char* d_tmp = new char[b->mRecCount*b->char_size[b->columnNames[i]]];
            str_gather_host(v, b->mRecCount, b->h_columns_char[b->columnNames[i]], d_tmp, b->char_size[b->columnNames[i]]);
			memcpy(b->h_columns_char[b->columnNames[i]], d_tmp, b->mRecCount*b->char_size[b->columnNames[i]]);	
			delete [] d_tmp;
        };
    };

    delete [] v;	
	
    thrust::host_vector<unsigned long long int> hh(b->mRecCount);
	thrust::copy(hashes, hashes+b->mRecCount, hh.begin());
	;
    char* tmp = new char[max_char(b)*(c->mRecCount + b->mRecCount)];
    c->resize(b->mRecCount);
	
    //lets merge every column
	
    for(unsigned int i = 0; i < b->columnNames.size(); i++) {
	
        if(b->type[b->columnNames[i]] == 0) {
		
            thrust::merge_by_key(h_merge.begin(), h_merge.end(),
                                 hh.begin(), hh.end(),
                                 c->h_columns_int[c->columnNames[i]].begin(), b->h_columns_int[b->columnNames[i]].begin(),
                                 thrust::make_discard_iterator(), (int_type*)tmp);
            thrust::copy((int_type*)tmp, (int_type*)tmp + h_merge.size() + b->mRecCount, c->h_columns_int[c->columnNames[i]].begin());
        }
        else if(b->type[b->columnNames[i]] == 1) {
            thrust::merge_by_key(h_merge.begin(), h_merge.end(),
                                 hh.begin(), hh.end(),
                                 c->h_columns_float[c->columnNames[i]].begin(), b->h_columns_float[b->columnNames[i]].begin(),
                                 thrust::make_discard_iterator(), (float_type*)tmp);										 
            thrust::copy((float_type*)tmp, (float_type*)tmp + h_merge.size() + b->mRecCount, c->h_columns_float[c->columnNames[i]].begin());			
			
        }
        else {						
            str_merge_by_key(h_merge, hh, c->h_columns_char[c->columnNames[i]], b->h_columns_char[b->columnNames[i]], b->char_size[b->columnNames[i]], tmp);
            thrust::copy(tmp, tmp + (h_merge.size() + b->mRecCount)*b->char_size[b->columnNames[i]], c->h_columns_char[c->columnNames[i]]);							
        };
    };
	
    //merge the keys
    thrust::merge(h_merge.begin(), h_merge.end(),
                  hh.begin(), hh.end(), (unsigned long long int*)tmp);

    size_t cpy_sz = h_merge.size() + b->mRecCount;
    h_merge.resize(h_merge.size() + b->mRecCount);
    thrust::copy((unsigned long long int*)tmp, (unsigned long long int*)tmp + cpy_sz, h_merge.begin());
    delete [] tmp;
	
	
	
    //cout << endl << "end b and c " << b->mRecCount << " " << c->mRecCount << endl;
    //for(int i = 0; i < h_merge.size();i++)
    //cout << "H " << h_merge[i] << endl;

    /*   bool dis_exists = 0;
        for(unsigned int j=0; j < c->mColumnCount; j++) {
            if (c->grp_type[j] == 6)
                dis_exists = 1;
        };

        if (dis_exists) {
            bool grp_scanned = 0;
            thrust::device_ptr<bool> d_di(a->grp);
            thrust::device_ptr<unsigned int> d_dii = thrust::device_malloc<unsigned int>(a->mRecCount);
            thrust::identity<bool> op;
            thrust::transform(d_di, d_di+a->mRecCount, d_dii, op);

            thrust::device_ptr<int_type> tmp = thrust::device_malloc<int_type>(a->mRecCount);

            unsigned int dist_count = 0;

            for(unsigned int j=0; j < c->mColumnCount; j++) {

                if (c->grp_type[j] == 6) {

                    if(!grp_scanned) {

                        d_dii[a->mRecCount-1] = 0;
                        thrust::inclusive_scan(d_dii, d_dii + a->mRecCount, d_dii);
                        thrust::gather(d_dii, d_dii + a->mRecCount, hashes.begin(), tmp);	// now hashes are in tmp
                        grp_scanned = 1;
                    };
                    unsigned int offset = distinct_val[dist_count].size();

                    distinct_val[dist_count].resize(distinct_val[dist_count].size() + a->mRecCount);
                    distinct_hash[dist_count].resize(distinct_hash[dist_count].size() + a->mRecCount);

                    thrust::copy(distinct_tmp[dist_count].begin(), distinct_tmp[dist_count].begin() + a->mRecCount, distinct_val[dist_count].begin() + offset);
                    thrust::copy(tmp, tmp + a->mRecCount, distinct_hash[dist_count].begin() + offset);

                    thrust::stable_sort_by_key(distinct_val[dist_count].begin(), distinct_val[dist_count].end(), distinct_hash[dist_count].begin());
                    thrust::stable_sort_by_key(distinct_hash[dist_count].begin(), distinct_hash[dist_count].end(), distinct_val[dist_count].begin());

                    ZipIterator new_last = thrust::unique(thrust::make_zip_iterator(thrust::make_tuple(distinct_hash[dist_count].begin(), distinct_val[dist_count].begin())),
                                                          thrust::make_zip_iterator(thrust::make_tuple(distinct_hash[dist_count].end(), distinct_val[dist_count].end())));

                    IteratorTuple t = new_last.get_iterator_tuple();
                    distinct_val[dist_count].resize(thrust::get<0>(t) - distinct_hash[dist_count].begin());
                    distinct_hash[dist_count].resize(thrust::get<0>(t) - distinct_hash[dist_count].begin());

                    dist_count++;

                };
            };
            thrust::device_free(tmp);
            thrust::device_free(d_dii);
        };
    	*/


}


void count_simple(CudaSet* c)
{
    int_type count;

    for(unsigned int i = 0; i < c->columnNames.size(); i++) {
        if(c->grp_type[c->columnNames[i]] == 0) { // COUNT
            count = thrust::reduce(c->h_columns_int[c->columnNames[i]].begin(), c->h_columns_int[c->columnNames[i]].begin() + c->mRecCount);
            c->h_columns_int[c->columnNames[i]][0] = count;
        };
    };


    if (c->mRecCount != 0) {

        for(unsigned int k = 0; k < c->columnNames.size(); k++) {
            if(c->grp_type[c->columnNames[k]] == 1) {   // AVG
                if(c->type[c->columnNames[k]] == 0) {
                    int_type sum  = thrust::reduce(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].begin() + c->mRecCount);
                    c->h_columns_int[c->columnNames[k]][0] = sum/count;
                }
                if(c->type[c->columnNames[k]] == 1) {
                    float_type sum  = thrust::reduce(c->h_columns_float[c->columnNames[k]].begin(), c->h_columns_float[c->columnNames[k]].begin() + c->mRecCount);
                    c->h_columns_float[c->columnNames[k]][0] = sum/count;
                };
            }
            else if(c->grp_type[c->columnNames[k]] == 2) {   // SUM
                if(c->type[c->columnNames[k]] == 0) {
                    int_type sum  = thrust::reduce(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].begin() + c->mRecCount);
                    c->h_columns_int[c->columnNames[k]][0] = sum;
                }
                if(c->type[c->columnNames[k]] == 1) {
                    float_type sum  = thrust::reduce(c->h_columns_float[c->columnNames[k]].begin(), c->h_columns_float[c->columnNames[k]].begin() + c->mRecCount);
                    c->h_columns_float[c->columnNames[k]][0] = sum;
                };

            }
        };
    }
    c->mRecCount = 1;
};


void count_avg(CudaSet* c,  vector<thrust::device_vector<int_type> >& distinct_hash)
{
    string countstr;
	
    for(unsigned int i = 0; i < c->columnNames.size(); i++) {
        if(c->grp_type[c->columnNames[i]] == 0) { // COUNT
            countstr = c->columnNames[i];
            break;
        };
    };
	

    thrust::host_vector<bool> grp;
    size_t res_count;

    if(h_merge.size()) {
        grp.resize(h_merge.size());
        thrust::adjacent_difference(h_merge.begin(), h_merge.end(), grp.begin());
        res_count = h_merge.size() - thrust::count(grp.begin(), grp.end(), 0);
    };
	

    if (c->mRecCount != 0) {

        //unsigned int dis_count = 0;
        if (h_merge.size()) {
            for(unsigned int k = 0; k < c->columnNames.size(); k++)	{

                if(c->grp_type[c->columnNames[k]] <= 2) { //sum || avg || count
                    if (c->type[c->columnNames[k]] == 0 ) { // int

                        int_type* tmp =  new int_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_int[c->columnNames[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
                        delete [] tmp;
                    }
                    else if (c->type[c->columnNames[k]] == 1 ) { // float
                        float_type* tmp =  new float_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_float[c->columnNames[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_float[c->columnNames[k]].begin());
                        delete [] tmp;
                    };
                }
                if(c->grp_type[c->columnNames[k]] == 4) { //min
                    if (c->type[c->columnNames[k]] == 0 ) { // int
                        int_type* tmp =  new int_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_int[c->columnNames[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
                        delete [] tmp;
                    }
                    else if (c->type[c->columnNames[k]] == 1 ) { // float
                        float_type* tmp =  new float_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_float[c->columnNames[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_float[c->columnNames[k]].begin());
                        delete [] tmp;
                    };
                }
                if(c->grp_type[c->columnNames[k]] == 5) { //max
                    if (c->type[c->columnNames[k]] == 0 ) { // int
                        int_type* tmp =  new int_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_int[c->columnNames[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
                        delete [] tmp;
                    }
                    else if (c->type[c->columnNames[k]] == 1 ) { // float
                        float_type* tmp =  new float_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_float[c->columnNames[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_float[c->columnNames[k]].begin());
                        delete [] tmp;
                    };
                }
                else if(c->grp_type[c->columnNames[k]] == 3) { //no group function
                    if (c->type[c->columnNames[k]] == 0 ) { // int
                        int_type* tmp =  new int_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->columnNames[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_int[c->columnNames[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->columnNames[k]].begin());
                        delete [] tmp;
                    }
                    else if (c->type[c->columnNames[k]] == 1 ) { // float
                        float_type* tmp =  new float_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->columnNames[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_float[c->columnNames[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_float[c->columnNames[k]].begin());
                        delete [] tmp;
                    }
                    else { //char
                        char* tmp = new char[res_count*c->char_size[c->columnNames[k]]];
                        str_copy_if_host(c->h_columns_char[c->columnNames[k]], c->mRecCount, tmp, grp, c->char_size[c->columnNames[k]]);
                        thrust::copy(tmp, tmp + c->char_size[c->columnNames[k]]*res_count, c->h_columns_char[c->columnNames[k]]);
                        delete [] tmp;
                    };
                };
            };
		
            c->mRecCount = res_count;
        };

        for(unsigned int k = 0; k < c->columnNames.size(); k++)	{
            if(c->grp_type[c->columnNames[k]] == 1) {   // AVG

                if (c->type[c->columnNames[k]] == 0 ) { // int
                    //create a float column k
                    c->h_columns_float[c->columnNames[k]] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >(c->mRecCount);
					c->d_columns_float[c->columnNames[k]] = thrust::device_vector<float_type>();

                    thrust::transform(c->h_columns_int[c->columnNames[k]].begin(), c->h_columns_int[c->columnNames[k]].begin() + c->mRecCount,
                                      c->h_columns_int[countstr].begin(), c->h_columns_float[c->columnNames[k]].begin(), float_avg1());
                    c->type[c->columnNames[k]] = 1;
                    c->h_columns_int[c->columnNames[k]].resize(0);
                    c->h_columns_int[c->columnNames[k]].shrink_to_fit();
                    c->grp_type[c->columnNames[k]] = 3;
                }
                else {              // float
                    thrust::transform(c->h_columns_float[c->columnNames[k]].begin(), c->h_columns_float[c->columnNames[k]].begin() + c->mRecCount,
                                      c->h_columns_int[countstr].begin(), c->h_columns_float[c->columnNames[k]].begin(), float_avg());
                };
            }
            else if(c->grp_type[c->columnNames[k]] == 6) {
                /*   unsigned int res_count = 0;

                   thrust::host_vector<int_type> h_hash = distinct_hash[dis_count];
                   int_type curr_val = h_hash[0];
                   unsigned int cycle_sz = h_hash.size();

                   for(unsigned int i = 0; i < cycle_sz; i++) {
                       if (h_hash[i] == curr_val) {
                           res_count++;
                           if(i == cycle_sz-1) {
                               c->h_columns_int[c->columnNames[k]][mymap[h_hash[i]]] = res_count;
                           };
                       }
                       else {
                           unsigned int idx = mymap[h_hash[i-1]];
                           c->h_columns_int[c->columnNames[k]][idx] = res_count;
                           curr_val = h_hash[i];
                           res_count = 1;
                       };
                   };
                   dis_count++;*/
            }
            else if(c->grp_type[c->columnNames[k]] == 2) {

            };
        };

    };

    c->segCount = 1;
    c->maxRecs = c->mRecCount;
};

