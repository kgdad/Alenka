/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "cm.h"

struct cmp_functor_str
{
    const char  * source;
    const char *str;
    bool * dest;
    const unsigned int * len;

    cmp_functor_str(const char * _source, const char * _str, bool * _dest,
                    const unsigned int * _len):
        source(_source), str(_str), dest(_dest), len(_len) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned int length = len[0];
        unsigned int start = i*length;

        for(unsigned int z = 0; z < length ; z++) {
            if(source[start+z] != str[z]) {
                dest[i] = 0;
                return;
            };

        };
        dest[i] = 1;

    }
};


struct cmp_functor_str_like_left
{
    const char  * source;
    const char *str;
    bool * dest;
    const unsigned int * len;

    cmp_functor_str_like_left(const char * _source, const char * _str, bool * _dest,
                              const unsigned int * _len):
        source(_source), str(_str), dest(_dest), len(_len) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned int length = len[0];
        unsigned int str_length = len[1];
        unsigned int start = i*length;

        for(unsigned int z = 0; z < str_length ; z++) {
            if(source[start+z] != str[z]) {
                dest[i] = 0;
                return;
            };

        };
        dest[i] = 1;

    }
};


struct cmp_functor_str_like_right
{
    const char  * source;
    const char *str;
    bool * dest;
    const unsigned int * len;

    cmp_functor_str_like_right(const char * _source, const char * _str, bool * _dest,
                               const unsigned int * _len):
        source(_source), str(_str), dest(_dest), len(_len) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned int length = len[0];
        unsigned int str_length = len[1];
        unsigned int start = i*length;

        unsigned int j = 1;
        while(source[(start+length)-j] == 0)
            j++;

        for(unsigned int z = 0; z < str_length ; z++) {
            if(source[((start+length)-j)-z] != str[str_length-z-1]) {
                dest[i] = 0;
                return;
            };

        };
        dest[i] = 1;

    }
};




bool* filter(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a,
             unsigned int segment)
{

    stack<string> exe_type;
    stack<string> exe_value;
    stack<int_type*> exe_vectors;
    stack<float_type*> exe_vectors_f;
    stack<int_type> exe_nums;
    stack<bool*> bool_vectors;
    stack<float_type> exe_nums_f;
    string  s1, s2, s1_val, s2_val;
    int_type n1, n2, res;
    float_type n1_f, n2_f, res_f;

	
    for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

        string ss = op_type.front();

        if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("VECTOR") == 0 || ss.compare("FLOAT") == 0
                || ss.compare("STRING") == 0) {

            exe_type.push(ss);
            if (ss.compare("NUMBER") == 0) {
                exe_nums.push(op_nums.front());
                op_nums.pop();
            }
            else if (ss.compare("NAME") == 0 || ss.compare("STRING") == 0) {
                exe_value.push(op_value.front());
                op_value.pop();
            }
            if (ss.compare("FLOAT") == 0) {
                exe_nums_f.push(op_nums_f.front());
                op_nums_f.pop();
            }

        }
        else {
            if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
                // get 2 values from the stack
                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();


                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();

                    if (ss.compare("ADD") == 0 )
                        res = n1+n2;
                    else if (ss.compare("MUL") == 0 )
                        res = n1*n2;
                    else if (ss.compare("DIV") == 0 )
                        res = n1/n2;
                    else
                        res = n1-n2;

                    thrust::device_ptr<int_type> p = thrust::device_malloc<int_type>(a->mRecCount);
                    thrust::sequence(p, p+(a->mRecCount),res,(int_type)0);

                    exe_type.push("VECTOR");
                    exe_vectors.push(thrust::raw_pointer_cast(p));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (ss.compare("ADD") == 0 )
                        res_f = n1_f+n2_f;
                    else if (ss.compare("MUL") == 0 )
                        res_f = n1_f*n2_f;
                    else if (ss.compare("DIV") == 0 )
                        res_f = n1_f/n2_f;
                    else
                        res_f = n1_f-n2_f;

                    thrust::device_ptr<float_type> p = thrust::device_malloc<float_type>(a->mRecCount);
                    thrust::sequence(p, p+(a->mRecCount),res_f,(float_type)0);

                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(thrust::raw_pointer_cast(p));

                }
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
					printf("CMPF1 %lld \n" , n1);

                    exe_type.push("VECTOR F");

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,1));
                    }
                    else {
                        int_type* t = a->get_int_by_name(s1_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,1));
                    };

                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					printf("CMPF %lld \n" , n1);

                    exe_type.push("VECTOR F");

                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,0));
                    }
                    else {
                        int_type* t = a->get_int_by_name(s2_val);
                        exe_vectors_f.push(a->op(t,n1_f,ss,0));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
					
					printf("CMP1 %lld \n" , n1);

                    if (a->type[s1_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(t,(float_type)n1,ss,1));

                    }
                    else {
                        int_type* t = a->get_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(t,n1,ss,1));
                    };
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					printf("CMP %lld \n" , n1);


                    if (a->type[s2_val] == 1) {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(t,(float_type)n1,ss,0));
                    }
                    else {
                        int_type* t = a->get_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(t,n1,ss,0));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t1 = a->get_int_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t = a->get_int_by_name(s2_val);
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,t1,ss,0));
                        }
                        else {
                            float_type* t = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t1,t,ss,0));
                        };
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t1,t,ss,0));
                        }
                        else {
                            float_type* t1 = a->get_float_type_by_name(s2_val);
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,t1,ss,0));
                        };
                    }
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0 ) && s2.compare("NAME") == 0) {

                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);

                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,s3,ss,0));
                            //free s3
                            hipFree(s3);

                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,0));
                            hipFree(s3);
                        }
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        if (s1.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,t, ss,0));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,0));
                            hipFree(s3);
                        }
                    };
                }
                else if ((s2.compare("VECTOR") == 0 || s2.compare("VECTOR F") == 0 ) && s1.compare("NAME") == 0) {

                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_int_by_name(s1_val);

                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR");
                            exe_vectors.push(a->op(t,s3,ss,1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,1));
                            hipFree(s3);
                        }
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        if (s2.compare("VECTOR") == 0 ) {
                            int_type* s3 = exe_vectors.top();
                            exe_vectors.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(s3,t,ss,1));
                            hipFree(s3);
                        }
                        else {
                            float_type* s3 = exe_vectors_f.top();
                            exe_vectors_f.pop();
                            exe_type.push("VECTOR F");
                            exe_vectors_f.push(a->op(t,s3,ss,1));
                            hipFree(s3);
                        }
                    };
                }
                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(s3,n1, ss,1));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,(float_type)n1, ss,1));
                        hipFree(s3);
                    }
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR") || s2.compare("VECTOR F") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR");
                        exe_vectors.push(a->op(s3,n1, ss,0));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,(float_type)n1, ss,0));
                        hipFree(s3);
                    }
                }

                else if ((s1.compare("VECTOR") == 0 || s1.compare("VECTOR F") == 0)  && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();

                    if (s1.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,1));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,1));
                        hipFree(s3);
                    }
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums.pop();

                    if (s2.compare("VECTOR") == 0 ) {
                        int_type* s3 = exe_vectors.top();
                        exe_vectors.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,0));
                        hipFree(s3);
                    }
                    else {
                        float_type* s3 = exe_vectors_f.top();
                        exe_vectors_f.pop();
                        exe_type.push("VECTOR F");
                        exe_vectors_f.push(a->op(s3,n1_f, ss,0));
                        hipFree(s3);
                    }
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s4 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    exe_vectors.push(a->op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(a->op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(a->op(s3, s4,ss,0));
                    hipFree(s3);
                    hipFree(s4);
                }
                else if(s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s4 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR F");
                    exe_vectors_f.push(a->op(s3, s4,ss,1));
                    hipFree(s3);
                    hipFree(s4);
                }
            }

            else if (ss.compare("CMP") == 0) {


                int_type cmp_type = op_nums.front();
                op_nums.pop();

                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();
				
                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1,n2,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1_f,n2_f,cmp_type));
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("NUMBER") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1_f,float_type(n2),cmp_type));
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("FLOAT") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(n1_f,float_type(n2),cmp_type));
                }

                else if (s1.compare("STRING") == 0 && s2.compare("NAME") == 0) {

                    s1_val = exe_value.top();
                    bool like_start = 0;

                    if(s1_val[0] == '%') {
                        like_start = 1;
                        s1_val.erase(0,1);
                    };

                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    void* d_v;
                    hipMalloc((void **) &d_v, 8);
                    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
                    dd_v[0] = a->char_size[s2_val];
                    dd_v[1] = (unsigned int)s1_val.length();
                    void* d_res;
                    hipMalloc((void **) &d_res, a->mRecCount);

                    void* d_str;

                    thrust::counting_iterator<unsigned int> begin(0);
                    if(!like_start) {
                        hipMalloc((void **) &d_str, a->char_size[s2_val]);
                        hipMemset(d_str,0,a->char_size[s2_val]);
                        hipMemcpy( d_str, (void *) s1_val.c_str(), s1_val.length(), hipMemcpyHostToDevice);

                        cmp_functor_str ff(a->d_columns_char[s2_val], (char*)d_str, (bool*)d_res, (unsigned int*)d_v);
                        thrust::for_each(begin, begin + a->mRecCount, ff);
                    }
                    else {
                        hipMalloc((void **) &d_str, s1_val.length());
                        hipMemcpy( d_str, (void *) s1_val.c_str(), s1_val.length(), hipMemcpyHostToDevice);

                        cmp_functor_str_like_right ff(a->d_columns_char[s2_val], (char*)d_str, (bool*)d_res, (unsigned int*)d_v);
                        thrust::for_each(begin, begin + a->mRecCount, ff);
                    };


                    exe_type.push("VECTOR");
                    bool_vectors.push((bool*)d_res);
                    hipFree(d_v);
                    hipFree(d_str);

                }
                else if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    bool like_start = 0;
                    if(s1_val[0] == '%') {
                        like_start = 1;
                        s1_val.erase(0,1);
                    };

                    void* d_v;
                    hipMalloc((void **) &d_v, 4);
                    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
                    dd_v[0] = a->char_size[s1_val];

                    void* d_res;
                    hipMalloc((void **) &d_res, a->mRecCount);

                    void* d_str;
                    hipMalloc((void **) &d_str, a->char_size[s1_val]);
                    hipMemset(d_str,0,a->char_size[s1_val]);
                    hipMemcpy( d_str, (void *) s1_val.c_str(), s1_val.length(), hipMemcpyHostToDevice);

                    thrust::counting_iterator<unsigned int> begin(0);
                    if(!like_start) {
                        cmp_functor_str ff(a->d_columns_char[s1_val], (char*)d_str, (bool*)d_res, (unsigned int*)d_v);
                        thrust::for_each(begin, begin + a->mRecCount, ff);
                    }
                    else {
                        cmp_functor_str_like_right ff(a->d_columns_char[s1_val], (char*)d_str, (bool*)d_res, (unsigned int*)d_v);
                        thrust::for_each(begin, begin + a->mRecCount, ff);
                    };

                    exe_type.push("VECTOR");
                    bool_vectors.push((bool*)d_res);
                    hipFree(d_v);
                    hipFree(d_str);
                }


                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,n1,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,(float_type)n1,cmp_type));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,n1,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,(float_type)n1,cmp_type));
                    };
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("NAME") == 0) {
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();					

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_int_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,(int_type)n1_f,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s1_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,n1_f,cmp_type));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,(int_type)n1_f,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        exe_type.push("VECTOR");
                        bool_vectors.push(a->compare(t,n1_f,cmp_type));
                    };
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(float_type)n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR F") == 0) {

                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(float_type)n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("NUMBER") == 0 && s2.compare("VECTOR") == 0) {

                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1_f,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("VECTOR") == 0 && s2.compare("FLOAT") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(int_type)n1_f,cmp_type));
                    hipFree(s3);
                }
                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,n1_f,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("FLOAT") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    n1_f = exe_nums_f.top();
                    exe_nums_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,(int_type)n1_f,cmp_type));
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("NAME") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }


                else if (s1.compare("VECTOR") == 0 && s2.compare("NAME") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR F") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(s3,t,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("NAME") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s2_val] == 0) {
                        int_type* t = a->get_int_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    }
                    else {
                        float_type* t = a->get_float_type_by_name(s2_val);
                        bool_vectors.push(a->compare(t,s3,cmp_type));
                    };
                    hipFree(s3);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR") == 0) {
                    int_type* s3 = exe_vectors.top();
                    exe_vectors.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s2,s3,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    float_type* s2 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s2,s3,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR F") == 0 && s2.compare("VECTOR") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,s2,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }

                else if (s1.compare("VECTOR") == 0 && s2.compare("VECTOR F") == 0) {
                    float_type* s3 = exe_vectors_f.top();
                    exe_vectors_f.pop();
                    int_type* s2 = exe_vectors.top();
                    exe_vectors.pop();
                    exe_type.push("VECTOR");
                    bool_vectors.push(a->compare(s3,s2,cmp_type));
                    hipFree(s3);
                    hipFree(s2);
                }


                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("VECTOR");

                    if (a->type[s1_val] == 0) {
                        int_type* t = a->get_int_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s2_val);
                            bool_vectors.push(a->compare(t1,t,cmp_type));
                        }
                        else {
                            float_type* t1 = a->get_float_type_by_name(s2_val);
                            bool_vectors.push(a->compare(t1,t,cmp_type));
                        };
                    }
                    else {
                        cmp_type = reverse_op(cmp_type);
                        float_type* t = a->get_float_type_by_name(s1_val);
                        if (a->type[s2_val] == 0) {
                            int_type* t1 = a->get_int_by_name(s2_val);
                            bool_vectors.push(a->compare(t,t1,cmp_type));
                        }
                        else {
                            float_type* t1 = a->get_float_type_by_name(s2_val);
                            bool_vectors.push(a->compare(t,t1,cmp_type));
                        };
                    }
                }
            }

            else if (ss.compare("AND") == 0) {
                bool* s3 = bool_vectors.top();
                bool_vectors.pop();
                bool* s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(a->logical_and(s2,s3));
            }
            else if (ss.compare("OR") == 0) {
                bool* s3 = bool_vectors.top();
                bool_vectors.pop();
                bool* s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("VECTOR");
                bool_vectors.push(a->logical_or(s2,s3));
            }
            else {
                cout << "found nothing " << endl;
            }
        };
    };
	
	return bool_vectors.top();
}